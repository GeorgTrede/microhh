/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "field3d.h"
#include "grid.h"
#include "master.h"
#include "tools.h"

#ifdef USECUDA
Field3d::~Field3d()
{
    cudaSafeCall(hipHostFree(data));
    cudaSafeCall(hipHostFree(databot));
    cudaSafeCall(hipHostFree(datatop));
    cudaSafeCall(hipHostFree(datagradbot));
    cudaSafeCall(hipHostFree(datagradtop));
    cudaSafeCall(hipHostFree(datafluxbot));
    cudaSafeCall(hipHostFree(datafluxtop));
    cudaSafeCall(hipHostFree(datamean));
}

int Field3d::init()
{
    const int ijksize = grid->ncells *sizeof(double);
    const int ijsize  = grid->ijcells*sizeof(double);
    const int ksize   = grid->kcells *sizeof(double);

    // Allocate the 3d field.
    cudaSafeCall(hipHostMalloc(&data, ijksize));

    // Allocate the boundary cells.
    cudaSafeCall(hipHostMalloc(&databot, ijsize));
    cudaSafeCall(hipHostMalloc(&datatop, ijsize));
    cudaSafeCall(hipHostMalloc(&datagradbot, ijsize));
    cudaSafeCall(hipHostMalloc(&datagradtop, ijsize));
    cudaSafeCall(hipHostMalloc(&datafluxbot, ijsize));
    cudaSafeCall(hipHostMalloc(&datafluxtop, ijsize));
    cudaSafeCall(hipHostMalloc(&datamean, ksize));

    // Set all values to zero
    for (int n=0; n<grid->ncells; n++)
        data[n] = 0.;

    for (int n=0; n<grid->kcells; n++)
        datamean[n] = 0.;

    for (int n=0; n<grid->icells*grid->jcells; n++)
    {
        databot    [n] = 0.;
        datatop    [n] = 0.;
        datagradbot[n] = 0.;
        datagradtop[n] = 0.;
        datafluxbot[n] = 0.;
        datafluxtop[n] = 0.;
    }

    return 0;
}
#endif

void Field3d::init_device()
{
    const int nmemsize   = grid->ncellsp*sizeof(double);
    const int nmemsize1d = grid->kcells *sizeof(double);
    const int nmemsize2d = (grid->ijcellsp+grid->memoffset)*sizeof(double);

    cudaSafeCall(hipMalloc(&data_g,        nmemsize  ));
    cudaSafeCall(hipMalloc(&databot_g,     nmemsize2d));
    cudaSafeCall(hipMalloc(&datatop_g,     nmemsize2d));
    cudaSafeCall(hipMalloc(&datagradbot_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&datagradtop_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&datafluxbot_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&datafluxtop_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&datamean_g,    nmemsize1d));
}

void Field3d::clear_device()
{
    cudaSafeCall(hipFree(data_g));
    cudaSafeCall(hipFree(databot_g));
    cudaSafeCall(hipFree(datatop_g));
    cudaSafeCall(hipFree(datagradbot_g));
    cudaSafeCall(hipFree(datagradtop_g));
    cudaSafeCall(hipFree(datafluxbot_g));
    cudaSafeCall(hipFree(datafluxtop_g));
    cudaSafeCall(hipFree(datamean_g));
}
