#include "field3d.h"
#include "grid.h"
#include "master.h"

#ifdef USECUDA
cfield3d::~cfield3d()
{
  hipHostFree(data);
  hipHostFree(databot);
  hipHostFree(datatop);
  hipHostFree(datagradbot);
  hipHostFree(datagradtop);
  hipHostFree(datafluxbot);
  hipHostFree(datafluxtop);
  hipHostFree(datamean);
}

int cfield3d::init()
{
  // allocate the memory
  master->printMessage("Allocating %d bytes of memory for %s\n", grid->ncells*(int)sizeof(double), name.c_str());

  const int ijksize = grid->ncells *sizeof(double);
  const int ijsize  = grid->ijcells*sizeof(double);
  const int ksize   = grid->kcells *sizeof(double);

  // Allocate the 3d field.
  hipHostMalloc(&data, ijksize);

  // Allocate the boundary cells.
  hipHostMalloc(&databot, ijsize);
  hipHostMalloc(&datatop, ijsize);
  hipHostMalloc(&datagradbot, ijsize);
  hipHostMalloc(&datagradtop, ijsize);
  hipHostMalloc(&datafluxbot, ijsize);
  hipHostMalloc(&datafluxtop, ijsize);
  hipHostMalloc(&datamean, ksize);

  // Set all values to zero
  for(int n=0; n<grid->ncells; n++)
    data[n] = 0.;

  for(int n=0; n<grid->kcells; n++)
    datamean[n] = 0.;

  for(int n=0; n<grid->icells*grid->jcells; n++)
  {
    databot    [n] = 0.;
    datatop    [n] = 0.;
    datagradbot[n] = 0.;
    datagradtop[n] = 0.;
    datafluxbot[n] = 0.;
    datafluxtop[n] = 0.;
  }

  return 0;
}
#endif
