#include "field3d.h"
#include "grid.h"
#include "master.h"
#include "tools.h"

#ifdef USECUDA
Field3d::~Field3d()
{
  cudaSafeCall(hipHostFree(data));
  cudaSafeCall(hipHostFree(databot));
  cudaSafeCall(hipHostFree(datatop));
  cudaSafeCall(hipHostFree(datagradbot));
  cudaSafeCall(hipHostFree(datagradtop));
  cudaSafeCall(hipHostFree(datafluxbot));
  cudaSafeCall(hipHostFree(datafluxtop));
  cudaSafeCall(hipHostFree(datamean));
}

int Field3d::init()
{
  // allocate the memory
  master->printMessage("Allocating %d bytes of memory for %s\n", grid->ncells*(int)sizeof(double), name.c_str());

  const int ijksize = grid->ncells *sizeof(double);
  const int ijsize  = grid->ijcells*sizeof(double);
  const int ksize   = grid->kcells *sizeof(double);

  // Allocate the 3d field.
  cudaSafeCall(hipHostMalloc(&data, ijksize));

  // Allocate the boundary cells.
  cudaSafeCall(hipHostMalloc(&databot, ijsize));
  cudaSafeCall(hipHostMalloc(&datatop, ijsize));
  cudaSafeCall(hipHostMalloc(&datagradbot, ijsize));
  cudaSafeCall(hipHostMalloc(&datagradtop, ijsize));
  cudaSafeCall(hipHostMalloc(&datafluxbot, ijsize));
  cudaSafeCall(hipHostMalloc(&datafluxtop, ijsize));
  cudaSafeCall(hipHostMalloc(&datamean, ksize));

  // Set all values to zero
  for(int n=0; n<grid->ncells; n++)
    data[n] = 0.;

  for(int n=0; n<grid->kcells; n++)
    datamean[n] = 0.;

  for(int n=0; n<grid->icells*grid->jcells; n++)
  {
    databot    [n] = 0.;
    datatop    [n] = 0.;
    datagradbot[n] = 0.;
    datagradtop[n] = 0.;
    datafluxbot[n] = 0.;
    datafluxtop[n] = 0.;
  }

  return 0;
}
#endif

void Field3d::initDevice()
{
  const int nmemsize   = grid->ncellsp*sizeof(double);
  const int nmemsize1d = grid->kcells *sizeof(double);
  const int nmemsize2d = (grid->ijcellsp+grid->memoffset)*sizeof(double);

  cudaSafeCall(hipMalloc(&data_g,        nmemsize  ));
  cudaSafeCall(hipMalloc(&databot_g,     nmemsize2d));
  cudaSafeCall(hipMalloc(&datatop_g,     nmemsize2d));
  cudaSafeCall(hipMalloc(&datagradbot_g, nmemsize2d));
  cudaSafeCall(hipMalloc(&datagradtop_g, nmemsize2d));
  cudaSafeCall(hipMalloc(&datafluxbot_g, nmemsize2d));
  cudaSafeCall(hipMalloc(&datafluxtop_g, nmemsize2d));
  cudaSafeCall(hipMalloc(&datamean_g,    nmemsize1d));
}
