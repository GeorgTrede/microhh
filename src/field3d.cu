#include "field3d.h"
#include "grid.h"
#include "master.h"

#ifdef USECUDA
cfield3d::~cfield3d()
{
  delete[] data;
  delete[] databot;
  delete[] datatop;
  delete[] datamean;
  delete[] datagradbot;
  delete[] datagradtop;
  delete[] datafluxbot;
  delete[] datafluxtop;
}

int cfield3d::init()
{
  // allocate the memory
  master->printMessage("Allocating %d bytes of memory for %s\n", grid->ncells*(int)sizeof(double), name.c_str());

  const int ijksize = grid->ncells *sizeof(double);
  const int ijsize  = grid->ijcells*sizeof(double);
  const int ksize   = grid->kcells *sizeof(double);

  // Allocate the 3d field.
  hipHostMalloc(&data, ijksize);

  // Allocate the boundary cells.
  hipHostMalloc(&databot, ijsize);
  hipHostMalloc(&datatop, ijsize);
  hipHostMalloc(&datagradbot, ijsize);
  hipHostMalloc(&datagradtop, ijsize);
  hipHostMalloc(&datafluxbot, ijsize);
  hipHostMalloc(&datafluxtop, ijsize);
  hipHostMalloc(&datamean, ksize);

  // Set all values to zero
  for(int n=0; n<grid->ncells; n++)
    data[n] = 0.;

  for(int n=0; n<grid->kcells; n++)
    datamean[n] = 0.;

  for(int n=0; n<grid->icells*grid->jcells; n++)
  {
    databot    [n] = 0.;
    datatop    [n] = 0.;
    datagradbot[n] = 0.;
    datagradtop[n] = 0.;
    datafluxbot[n] = 0.;
    datafluxtop[n] = 0.;
  }

  return 0;
}
#endif
