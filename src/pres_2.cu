#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2013 Chiel van Heerwaarden
 * Copyright (c) 2011-2013 Thijs Heus
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <fftw3.h>
#include <hipfft/hipfft.h>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "pres_2.h"
#include "defines.h"
#include "model.h"
#include "tools.h"

__global__ void pres_2_presin(double * __restrict__ p,
                              double * __restrict__ u ,  double * __restrict__ v , double * __restrict__ w ,
                              double * __restrict__ ut,  double * __restrict__ vt, double * __restrict__ wt,
                              double * __restrict__ dzi, double dxi, double dyi, double dti,
                              const int jj, const int kk,
                              const int jjp, const int kkp,
                              const int imax, const int jmax, const int kmax,
                              const int igc, const int jgc, const int kgc)
{
  const int ii = 1;
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const int j = blockIdx.y*blockDim.y + threadIdx.y;
  const int k = blockIdx.z;

  if(i < imax && j < jmax && k < kmax)
  {
    const int ijkp = i + j*jjp + k*kkp;
    const int ijk  = i+igc + (j+jgc)*jj + (k+kgc)*kk;
    p[ijkp] = ( (ut[ijk+ii] + u[ijk+ii] * dti) - (ut[ijk] + u[ijk] * dti) ) * dxi
            + ( (vt[ijk+jj] + v[ijk+jj] * dti) - (vt[ijk] + v[ijk] * dti) ) * dyi
            + ( (wt[ijk+kk] + w[ijk+kk] * dti) - (wt[ijk] + w[ijk] * dti) ) * dzi[k+kgc];
  }
}

__global__ void pres_2_presout(double * __restrict__ ut, double * __restrict__ vt, double * __restrict__ wt,
                               double * __restrict__ p,
                               double * __restrict__ dzhi, const double dxi, const double dyi,
                               const int jj, const int kk,
                               const int istart, const int jstart, const int kstart,
                               const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  const int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] -= (p[ijk] - p[ijk-ii]) * dxi;
    vt[ijk] -= (p[ijk] - p[ijk-jj]) * dyi;
    wt[ijk] -= (p[ijk] - p[ijk-kk]) * dzhi[k];
  }
}

__global__ void pres_2_solveout(double * __restrict__ p, double * __restrict__ work3d,
                                const int jj, const int kk,
                                const int jjp, const int kkp,
                                const int istart, const int jstart, const int kstart,
                                const int imax, const int jmax, const int kmax)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const int j = blockIdx.y*blockDim.y + threadIdx.y;
  const int k = blockIdx.z;
  const int ijk  = i + j*jj + k*kk;
  const int ijkp = i+istart + (j+jstart)*jjp + (k+kstart)*kkp;

  if(i < imax && j < jmax && k < kmax)
  {
    p[ijkp] = work3d[ijk];

    if(k == 0)
      p[ijkp-kkp] = p[ijkp];
  }
}

__global__ void pres_2_solvein(double * __restrict__ p,
                               double * __restrict__ work3d, double * __restrict__ b,
                               double * __restrict__ a, double * __restrict__ c,
                               double * __restrict__ dz, double * __restrict__ bmati, double * __restrict__ bmatj,
                               const int jj, const int kk, 
                               const int imax, const int jmax, const int kmax,
                               const int kstart)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z;

  if(i < imax && j < jmax && k < kmax)
  {
    int ijk = i + j*jj + k*kk;

    // CvH this needs to be taken into account in case of an MPI run
    // iindex = mpi->mpicoordy * iblock + i;
    // jindex = mpi->mpicoordx * jblock + j;
    // b[ijk] = dz[k+kgc]*dz[k+kgc] * (bmati[iindex]+bmatj[jindex]) - (a[k]+c[k]);
    //  if(iindex == 0 && jindex == 0)

    b[ijk] = dz[k+kstart]*dz[k+kstart] * (bmati[i]+bmatj[j]) - (a[k]+c[k]);
    p[ijk] = dz[k+kstart]*dz[k+kstart] * p[ijk];

    if(k == 0)
    {
      // substitute BC's
      // ijk = i + j*jj;
      b[ijk] += a[0];
    }
    else if(k == kmax-1)
    {
      // for wave number 0, which contains average, set pressure at top to zero
      if(i == 0 && j == 0)
        b[ijk] -= c[k];
      // set dp/dz at top to zero
      else
        b[ijk] += c[k];
    }
  }
}

__global__ void pres_2_tdma(double * __restrict__ a, double * __restrict__ b, double * __restrict__ c, 
                            double * __restrict__ p, double * __restrict__ work3d,
                            const int jj, const int kk,
                            const int imax, const int jmax, const int kmax)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const int j = blockIdx.y*blockDim.y + threadIdx.y;

  if(i < imax && j < jmax)
  {
    const int ij = i + j*jj;
    int k,ijk;

    double work2d = b[ij];
    p[ij] /= work2d;

    for(k=1; k<kmax; k++)
    {
      ijk = ij + k*kk;
      work3d[ijk] = c[k-1] / work2d;
      work2d = b[ijk] - a[k]*work3d[ijk];
      p[ijk] -= a[k]*p[ijk-kk];
      p[ijk] /= work2d;
    }

    for(k=kmax-2; k>=0; k--)
    {
      ijk = ij + k*kk;
      p[ijk] -= work3d[ijk+kk]*p[ijk+kk];
    }
  }
}

__global__ void pres_2_complex_double_x(hipfftDoubleComplex * __restrict__ cdata, double * __restrict__ ddata, const unsigned int itot, const unsigned int jtot, bool forward)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int ij = i + j * itot;
  int ij2 = (itot-i) + j*itot;
  int imax = itot/2+1;

  if((j < jtot) && (i < imax))
  {
    if(forward) // complex -> double
    {
      ddata[ij]  = cdata[ij].x;
      if(i>0 && i<imax-1) 
        ddata[ij2] = cdata[ij].y;
    }
    else // double -> complex
    {
      cdata[ij].x = ddata[ij];
      if(i>0 && i<imax-1) 
        cdata[ij].y = ddata[ij2];
    }
  }
} 

__global__ void pres_2_complex_double_y(hipfftDoubleComplex * __restrict__ cdata, double * __restrict__ ddata, const unsigned int itot, const unsigned int jtot, bool forward)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int ij = i + j * itot;
  int ij2 = i + (jtot-j)*itot;
  int jmax = jtot/2+1; 

  if((i < itot) && (j < jmax))
  {
    if(forward) // complex -> double
    {
      ddata[ij] = cdata[ij].x;
      if(j>0 && j<jmax-1) 
        ddata[ij2] = cdata[ij].y;
    }
    else // double -> complex
    {
      cdata[ij].x = ddata[ij];
      if(j>0 && j<jmax-1) 
        cdata[ij].y = ddata[ij2];
    }
  }
}

 __global__ void pres_2_normalize(double * __restrict__ data, const unsigned int itot, const unsigned int jtot, const double in)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int ij = i + j * itot;
  if((i < itot) && (j < jtot))
    data[ij] = data[ij] * in;
} 

int cpres_2::prepareGPU()
{
  const int kmemsize = grid->kmax*sizeof(double);
  const int imemsize = grid->itot*sizeof(double);
  const int jmemsize = grid->jtot*sizeof(double);

  const int ijmemsize = grid->imax*grid->jmax*sizeof(double);

  hipMalloc((void**)&bmati_g, imemsize);
  hipMalloc((void**)&bmatj_g, jmemsize);
  hipMalloc((void**)&a_g, kmemsize);
  hipMalloc((void**)&c_g, kmemsize);
  hipMalloc((void**)&work2d_g, ijmemsize);

  hipMemcpy(bmati_g, bmati, imemsize, hipMemcpyHostToDevice);
  hipMemcpy(bmatj_g, bmatj, jmemsize, hipMemcpyHostToDevice);
  hipMemcpy(a_g, a, kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(c_g, c, kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(work2d_g, work2d, ijmemsize, hipMemcpyHostToDevice);

  // cuFFT
  hipMalloc((void **)&ffti_complex_g, sizeof(hipfftDoubleComplex)*(grid->jtot * (grid->itot/2+1)));
  hipMalloc((void **)&fftj_complex_g, sizeof(hipfftDoubleComplex)*(grid->itot * (grid->jtot/2+1)));

  // Make cuFFT plan
  int rank = 1;
  int ni[] = {grid->itot};
  int nj[] = {grid->jtot};
  int istride = 1;
  int jstride = grid->itot;
  int idist = grid->itot;
  int jdist = 1;

  hipfftPlanMany(&iplanf, rank, ni, ni, istride, idist, ni, istride, idist, HIPFFT_D2Z, grid->jtot);
  hipfftPlanMany(&jplanf, rank, nj, nj, jstride, jdist, nj, jstride, jdist, HIPFFT_D2Z, grid->itot);
  hipfftPlanMany(&iplanb, rank, ni, ni, istride, idist, ni, istride, idist, HIPFFT_Z2D, grid->jtot);
  hipfftPlanMany(&jplanb, rank, nj, nj, jstride, jdist, nj, jstride, jdist, HIPFFT_Z2D, grid->itot);

  return 0;
}

__global__ void pres_2_calcdiv(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                               double * __restrict__ div, double * __restrict__ dzi, double dxi, double dyi, 
                               int jj, int kk, int istart, int jstart, int kstart,
                               int iend, int jend, int kend,
                               int icells, int jcells, int kcells)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x; 
  int j = blockIdx.y*blockDim.y + threadIdx.y; 
  int k = blockIdx.z; 
  int ii = 1;
  int ijk = i + j*jj + k*kk;

  if(i >= istart && i < iend && j >= jstart && j < jend && k >= kstart && k < kend)
    div[ijk] = (u[ijk+ii]-u[ijk])*dxi + (v[ijk+jj]-v[ijk])*dyi + (w[ijk+kk]-w[ijk])*dzi[k];
  else if(i < icells && j < jcells && k < kcells) 
    div[ijk] = 0.;
}

#ifdef USECUDA
int cpres_2::exec(double dt)
{
  //fields->forwardGPU();

  int kk;
  const int blocki  = 128;
  const int blockj  = 2;
  const int gridi   = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj   = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  dim3 grid2dGPU (gridi, gridj);
  dim3 block2dGPU(blocki, blockj);

  // calculate the cyclic BCs first
  //grid->boundary_cyclic(fields->ut->data_g);
  //grid->boundary_cyclic(fields->vt->data_g);
  //grid->boundary_cyclic(fields->wt->data_g);

  grid->boundary_cyclic_gpu(fields->ut->data_g);
  grid->boundary_cyclic_gpu(fields->vt->data_g);
  grid->boundary_cyclic_gpu(fields->wt->data_g);

  pres_2_presin<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g,
                                       fields->u->data_g, fields->v->data_g, fields->w->data_g,
                                       fields->ut->data_g, fields->vt->data_g, fields->wt->data_g,
                                       grid->dzi_g, 1./grid->dx, 1./grid->dy, 1./dt,
                                       grid->icells, grid->ijcells, grid->imax, grid->imax*grid->jmax, 
                                       grid->imax, grid->jmax, grid->kmax,
                                       grid->igc, grid->jgc, grid->kgc);

  //fields->backwardGPU();
  //grid->fftforward(fields->sd["p"]->data, fields->sd["tmp1"]->data,
  //                grid->fftini, grid->fftouti, grid->fftinj, grid->fftoutj);
  //fields->forwardGPU();

  // Forward FFT -> how to get rid of the loop at the host side....
  // A massive FFT (e.g. 3D field) would require large host fields for the FFT output
  for (int k=0; k<grid->ktot; ++k)
  {
    kk = k*grid->itot*grid->jtot;

    hipfftExecD2Z(iplanf, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk], ffti_complex_g);
    hipDeviceSynchronize();
    pres_2_complex_double_x<<<grid2dGPU,block2dGPU>>>(ffti_complex_g, &fields->sd["p"]->data_g[kk],grid->itot,grid->jtot, true); 

    hipfftExecD2Z(jplanf, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk], fftj_complex_g);
    hipDeviceSynchronize();
    pres_2_complex_double_y<<<grid2dGPU,block2dGPU>>>(fftj_complex_g, &fields->sd["p"]->data_g[kk],grid->itot,grid->jtot, true); 
  } 

  pres_2_solvein<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g,
                                        fields->sd["tmp1"]->data_g, fields->sd["tmp2"]->data_g,
                                        a_g, c_g,
                                        grid->dz_g, bmati_g, bmatj_g,
                                        grid->imax, grid->imax*grid->jmax,
                                        grid->imax, grid->jmax, grid->kmax,
                                        grid->kstart);

  pres_2_tdma<<<grid2dGPU, block2dGPU>>>(a_g, fields->sd["tmp2"]->data_g, c_g,
                                         fields->sd["p"]->data_g, fields->sd["tmp1"]->data_g,
                                         grid->imax, grid->imax*grid->jmax,
                                         grid->imax, grid->jmax, grid->kmax);

  //fields->backwardGPU();
  //grid->fftbackward(fields->sd["p"]->data, fields->sd["tmp1"]->data,
  //                  grid->fftini, grid->fftouti, grid->fftinj, grid->fftoutj);
  //fields->forwardGPU();

  // Backward FFT 
  for (int k=0; k<grid->ktot; ++k)
  {
    kk = k*grid->itot*grid->jtot;

    pres_2_complex_double_y<<<grid2dGPU,block2dGPU>>>(fftj_complex_g, &fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, false); 
    hipfftExecZ2D(jplanb, fftj_complex_g, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk]);
    hipDeviceSynchronize();
    pres_2_normalize<<<grid2dGPU,block2dGPU>>>(&fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, 1./grid->jtot);

    pres_2_complex_double_x<<<grid2dGPU,block2dGPU>>>(ffti_complex_g, &fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, false); 
    hipfftExecZ2D(iplanb, ffti_complex_g, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk]);
    hipDeviceSynchronize();
    pres_2_normalize<<<grid2dGPU,block2dGPU>>>(&fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, 1./grid->itot);
  } 

  hipMemcpy(fields->sd["tmp1"]->data_g, fields->sd["p"]->data_g, grid->ncells*sizeof(double), hipMemcpyDeviceToDevice);

  pres_2_solveout<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g, fields->sd["tmp1"]->data_g,
                                         grid->imax, grid->imax*grid->jmax,
                                         grid->icells, grid->ijcells,
                                         grid->istart, grid->jstart, grid->kstart,
                                         grid->imax, grid->jmax, grid->kmax);

  //grid->boundary_cyclic(fields->sd["p"]->data_g);
  grid->boundary_cyclic_gpu(fields->sd["p"]->data_g);

  pres_2_presout<<<gridGPU, blockGPU>>>(fields->ut->data_g, fields->vt->data_g, fields->wt->data_g,
                                        fields->sd["p"]->data_g,
                                        grid->dzhi_g, 1./grid->dx, 1./grid->dy,
                                        grid->icells, grid->ijcells,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend, grid->jend, grid->kend);
  //fields->backwardGPU();

  return 0;
}
#endif

#ifdef USECUDA
double cpres_2::calcdivergence(double * restrict u, double * restrict v, double * restrict w, double * restrict dzi)
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
  const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);
  double divmax = 0;

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const double dxi = 1./grid->dx;
  const double dyi = 1./grid->dy;

  pres_2_calcdiv<<<gridGPU, blockGPU>>>(fields->u->data_g, fields->v->data_g, fields->w->data_g, 
                                        fields->a["tmp1"]->data_g, grid->dzi_g, dxi, dyi,
                                        grid->icells, grid->ijcells,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend,   grid->jend, grid->kend,
                                        grid->icells, grid->jcells, grid->kcells);

  divmax = maximum_gpu(fields->a["tmp1"]->data_g, grid->ncells);
  grid->getmax(&divmax);

  return divmax;
}
#endif

