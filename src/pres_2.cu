#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <fftw3.h>
#include <hipfft/hipfft.h>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "pres_2.h"
#include "defines.h"
#include "model.h"
#include "tools.h"

__global__ void pres_2_presin(double * __restrict__ p,
                              double * __restrict__ u ,  double * __restrict__ v ,     double * __restrict__ w ,
                              double * __restrict__ ut,  double * __restrict__ vt,     double * __restrict__ wt,
                              double * __restrict__ dzi, double * __restrict__ rhoref, double * __restrict__ rhorefh,
                              double dxi, double dyi, double dti,
                              const int jj, const int kk,
                              const int jjp, const int kkp,
                              const int imax, const int jmax, const int kmax,
                              const int igc, const int jgc, const int kgc)
{
  const int ii = 1;
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const int j = blockIdx.y*blockDim.y + threadIdx.y;
  const int k = blockIdx.z;

  if(i < imax && j < jmax && k < kmax)
  {
    const int ijkp = i + j*jjp + k*kkp;
    const int ijk  = i+igc + (j+jgc)*jj + (k+kgc)*kk;

    p[ijkp] = rhoref [k+kgc]   * ( (ut[ijk+ii] + u[ijk+ii] * dti) - (ut[ijk] + u[ijk] * dti) ) * dxi
            + rhoref [k+kgc]   * ( (vt[ijk+jj] + v[ijk+jj] * dti) - (vt[ijk] + v[ijk] * dti) ) * dyi
          + ( rhorefh[k+kgc+1] * (  wt[ijk+kk] + w[ijk+kk] * dti) 
            - rhorefh[k+kgc  ] * (  wt[ijk   ] + w[ijk   ] * dti) ) * dzi[k+kgc];
  }
}

__global__ void pres_2_presout(double * __restrict__ ut, double * __restrict__ vt, double * __restrict__ wt,
                               double * __restrict__ p,
                               double * __restrict__ dzhi, const double dxi, const double dyi,
                               const int jj, const int kk,
                               const int istart, const int jstart, const int kstart,
                               const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  const int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] -= (p[ijk] - p[ijk-ii]) * dxi;
    vt[ijk] -= (p[ijk] - p[ijk-jj]) * dyi;
    wt[ijk] -= (p[ijk] - p[ijk-kk]) * dzhi[k];
  }
}

__global__ void pres_2_solveout(double * __restrict__ p, double * __restrict__ work3d,
                                const int jj, const int kk,
                                const int jjp, const int kkp,
                                const int istart, const int jstart, const int kstart,
                                const int imax, const int jmax, const int kmax)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const int j = blockIdx.y*blockDim.y + threadIdx.y;
  const int k = blockIdx.z;

  if(i < imax && j < jmax && k < kmax)
  {
    const int ijk  = i + j*jj + k*kk;
    const int ijkp = i+istart + (j+jstart)*jjp + (k+kstart)*kkp;

    p[ijkp] = work3d[ijk];

    if(k == 0)
      p[ijkp-kkp] = p[ijkp];
  }
}

__global__ void pres_2_solvein(double * __restrict__ p,
                               double * __restrict__ work3d, double * __restrict__ b,
                               double * __restrict__ a, double * __restrict__ c,
                               double * __restrict__ dz, double * __restrict__ rhoref, 
                               double * __restrict__ bmati, double * __restrict__ bmatj,
                               const int jj, const int kk, 
                               const int imax, const int jmax, const int kmax,
                               const int kstart)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z;

  if(i < imax && j < jmax && k < kmax)
  {
    int ijk = i + j*jj + k*kk;

    // CvH this needs to be taken into account in case of an MPI run
    // iindex = mpi->mpicoordy * iblock + i;
    // jindex = mpi->mpicoordx * jblock + j;
    // b[ijk] = dz[k+kgc]*dz[k+kgc] * (bmati[iindex]+bmatj[jindex]) - (a[k]+c[k]);
    //  if(iindex == 0 && jindex == 0)

    b[ijk] = dz[k+kstart]*dz[k+kstart] * rhoref[k+kstart]*(bmati[i]+bmatj[j]) - (a[k]+c[k]);
    p[ijk] = dz[k+kstart]*dz[k+kstart] * p[ijk];

    if(k == 0)
    {
      // substitute BC's
      // ijk = i + j*jj;
      b[ijk] += a[0];
    }
    else if(k == kmax-1)
    {
      // for wave number 0, which contains average, set pressure at top to zero
      if(i == 0 && j == 0)
        b[ijk] -= c[k];
      // set dp/dz at top to zero
      else
        b[ijk] += c[k];
    }
  }
}

__global__ void pres_2_tdma(double * __restrict__ a, double * __restrict__ b, double * __restrict__ c, 
                            double * __restrict__ p, double * __restrict__ work3d,
                            const int jj, const int kk,
                            const int imax, const int jmax, const int kmax)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  const int j = blockIdx.y*blockDim.y + threadIdx.y;

  if(i < imax && j < jmax)
  {
    const int ij = i + j*jj;
    int k,ijk;

    double work2d = b[ij];
    p[ij] /= work2d;

    for(k=1; k<kmax; k++)
    {
      ijk = ij + k*kk;
      work3d[ijk] = c[k-1] / work2d;
      work2d = b[ijk] - a[k]*work3d[ijk];
      p[ijk] -= a[k]*p[ijk-kk];
      p[ijk] /= work2d;
    }

    for(k=kmax-2; k>=0; k--)
    {
      ijk = ij + k*kk;
      p[ijk] -= work3d[ijk+kk]*p[ijk+kk];
    }
  }
}

__global__ void pres_2_complex_double_x(hipfftDoubleComplex * __restrict__ cdata, double * __restrict__ ddata, const unsigned int itot, const unsigned int jtot, bool forward)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int ij   = i + j * itot;        // index real part in ddata
  int ij2  = (itot-i) + j*itot;   // index complex part in ddata
  int imax = itot/2+1;
  int ijc  = i + j * imax;        // index in cdata

  if((j < jtot) && (i < imax))
  {
    if(forward) // complex -> double
    {
      ddata[ij]  = cdata[ijc].x;
      if(i>0 && i<imax-1) 
        ddata[ij2] = cdata[ijc].y;
    }
    else // double -> complex
    {
      cdata[ijc].x = ddata[ij];
      if(i>0 && i<imax-1) 
        cdata[ijc].y = ddata[ij2];
    }
  }
} 

__global__ void pres_2_complex_double_y(hipfftDoubleComplex * __restrict__ cdata, double * __restrict__ ddata, const unsigned int itot, const unsigned int jtot, bool forward)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int ij   = i + j * itot;        // index real part in ddata
  int ij2 = i + (jtot-j)*itot;    // index complex part in ddata
  int jmax = jtot/2+1; 
  // ijc equals ij

  if((i < itot) && (j < jmax))
  {
    if(forward) // complex -> double
    {
      ddata[ij] = cdata[ij].x;
      if(j>0 && j<jmax-1) 
        ddata[ij2] = cdata[ij].y;
    }
    else // double -> complex
    {
      cdata[ij].x = ddata[ij];
      if(j>0 && j<jmax-1) 
        cdata[ij].y = ddata[ij2];
    }
  }
}

 __global__ void pres_2_normalize(double * __restrict__ data, const unsigned int itot, const unsigned int jtot, const double in)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  int ij = i + j * itot;
  if((i < itot) && (j < jtot))
    data[ij] = data[ij] * in;
} 

__global__ void pres_2_calcdivergence(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                                      double * __restrict__ div, double * __restrict__ dzi, 
                                      double * __restrict__ rhoref, double * __restrict__ rhorefh,
                                      double dxi, double dyi, 
                                      int jj, int kk, int istart, int jstart, int kstart,
                                      int iend, int jend, int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 
  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    div[ijk] = rhoref[k]*((u[ijk+ii]-u[ijk])*dxi + (v[ijk+jj]-v[ijk])*dyi) 
            + (rhorefh[k+1]*w[ijk+kk]-rhorefh[k]*w[ijk])*dzi[k];
  }
}

int cpres_2::prepareDevice()
{
  const int kmemsize = grid->kmax*sizeof(double);
  const int imemsize = grid->itot*sizeof(double);
  const int jmemsize = grid->jtot*sizeof(double);

  const int ijmemsize = grid->imax*grid->jmax*sizeof(double);

  hipMalloc((void**)&bmati_g, imemsize);
  hipMalloc((void**)&bmatj_g, jmemsize);
  hipMalloc((void**)&a_g, kmemsize);
  hipMalloc((void**)&c_g, kmemsize);
  hipMalloc((void**)&work2d_g, ijmemsize);

  hipMemcpy(bmati_g, bmati, imemsize, hipMemcpyHostToDevice);
  hipMemcpy(bmatj_g, bmatj, jmemsize, hipMemcpyHostToDevice);
  hipMemcpy(a_g, a, kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(c_g, c, kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(work2d_g, work2d, ijmemsize, hipMemcpyHostToDevice);

  // cuFFT
  hipMalloc((void **)&ffti_complex_g, sizeof(hipfftDoubleComplex)*(grid->jtot * (grid->itot/2+1))); // sizeof(complex) = 16
  hipMalloc((void **)&fftj_complex_g, sizeof(hipfftDoubleComplex)*(grid->itot * (grid->jtot/2+1)));

  // Make cuFFT plan
  int rank      = 1;

  // Double input
  int i_ni[]    = {grid->itot}; 
  int i_nj[]    = {grid->jtot};  
  int i_istride = 1;
  int i_jstride = grid->itot;
  int i_idist   = grid->itot;
  int i_jdist   = 1;

  // Double-complex output
  int o_ni[]    = {grid->itot/2+1};
  int o_nj[]    = {grid->jtot/2+1};
  int o_istride = 1;
  int o_jstride = grid->itot;
  int o_idist   = grid->itot/2+1;
  int o_jdist   = 1;

  // Forward FFTs
  hipfftPlanMany(&iplanf, rank, i_ni, i_ni, i_istride, i_idist, o_ni, o_istride, o_idist, HIPFFT_D2Z, grid->jtot);
  hipfftPlanMany(&jplanf, rank, i_nj, i_nj, i_jstride, i_jdist, o_nj, o_jstride, o_jdist, HIPFFT_D2Z, grid->itot);

  // Backward FFTs
  // NOTE: input size is always the 'logical' size of the FFT, so itot or jtot, not itot/2+1 or jtot/2+1 
  hipfftPlanMany(&iplanb, rank, i_ni, o_ni, o_istride, o_idist, i_ni, i_istride, i_idist, HIPFFT_Z2D, grid->jtot);
  hipfftPlanMany(&jplanb, rank, i_nj, o_nj, o_jstride, o_jdist, i_nj, i_jstride, i_jdist, HIPFFT_Z2D, grid->itot);

  return 0;
}

#ifdef USECUDA
void cpres_2::exec(double dt)
{
  int kk;
  const int blocki  = 128;
  const int blockj  = 2;
  const int gridi   = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj   = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  dim3 grid2dGPU (gridi, gridj);
  dim3 block2dGPU(blocki, blockj);

  const int offs = grid->memoffset;

  // calculate the cyclic BCs first
  grid->boundary_cyclic_g(&fields->ut->data_g[offs]);
  grid->boundary_cyclic_g(&fields->vt->data_g[offs]);
  grid->boundary_cyclic_g(&fields->wt->data_g[offs]);

  pres_2_presin<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g,
                                       &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                       &fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                       grid->dzi_g, fields->rhoref_g, fields->rhorefh_g, 
                                       1./grid->dx, 1./grid->dy, 1./dt,
                                       grid->icellsp, grid->ijcellsp, grid->imax, grid->imax*grid->jmax, 
                                       grid->imax, grid->jmax, grid->kmax,
                                       grid->igc, grid->jgc, grid->kgc);

  // Forward FFT -> how to get rid of the loop at the host side....
  // A massive FFT (e.g. 3D field) would require large host fields for the FFT output
  for (int k=0; k<grid->ktot; ++k)
  {
    kk = k*grid->itot*grid->jtot;

    hipfftExecD2Z(iplanf, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk], ffti_complex_g);
    hipDeviceSynchronize();
    pres_2_complex_double_x<<<grid2dGPU,block2dGPU>>>(ffti_complex_g, &fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, true); 

    hipfftExecD2Z(jplanf, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk], fftj_complex_g);
    hipDeviceSynchronize();
    pres_2_complex_double_y<<<grid2dGPU,block2dGPU>>>(fftj_complex_g, &fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, true); 
  } 

  pres_2_solvein<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g,
                                        fields->sd["tmp1"]->data_g, fields->sd["tmp2"]->data_g,
                                        a_g, c_g,
                                        grid->dz_g, fields->rhoref_g, bmati_g, bmatj_g,
                                        grid->imax, grid->imax*grid->jmax,
                                        grid->imax, grid->jmax, grid->kmax,
                                        grid->kstart);

  pres_2_tdma<<<grid2dGPU, block2dGPU>>>(a_g, fields->sd["tmp2"]->data_g, c_g,
                                         fields->sd["p"]->data_g, fields->sd["tmp1"]->data_g,
                                         grid->imax, grid->imax*grid->jmax,
                                         grid->imax, grid->jmax, grid->kmax);

  // Backward FFT 
  for (int k=0; k<grid->ktot; ++k)
  {
    kk = k*grid->itot*grid->jtot;

    pres_2_complex_double_y<<<grid2dGPU,block2dGPU>>>(fftj_complex_g, &fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, false); 
    hipfftExecZ2D(jplanb, fftj_complex_g, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk]);
    hipDeviceSynchronize();
    pres_2_normalize<<<grid2dGPU,block2dGPU>>>(&fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, 1./grid->jtot);

    pres_2_complex_double_x<<<grid2dGPU,block2dGPU>>>(ffti_complex_g, &fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, false); 
    hipfftExecZ2D(iplanb, ffti_complex_g, (hipfftDoubleReal*)&fields->sd["p"]->data_g[kk]);
    hipDeviceSynchronize();
    pres_2_normalize<<<grid2dGPU,block2dGPU>>>(&fields->sd["p"]->data_g[kk], grid->itot, grid->jtot, 1./grid->itot);
  } 

  hipMemcpy(fields->sd["tmp1"]->data_g, fields->sd["p"]->data_g, grid->ncellsp*sizeof(double), hipMemcpyDeviceToDevice);

  pres_2_solveout<<<gridGPU, blockGPU>>>(&fields->sd["p"]->data_g[offs], fields->sd["tmp1"]->data_g,
                                         grid->imax, grid->imax*grid->jmax,
                                         grid->icellsp, grid->ijcellsp,
                                         grid->istart, grid->jstart, grid->kstart,
                                         grid->imax, grid->jmax, grid->kmax);

  grid->boundary_cyclic_g(&fields->sd["p"]->data_g[offs]);

  pres_2_presout<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                        &fields->sd["p"]->data_g[offs],
                                        grid->dzhi_g, 1./grid->dx, 1./grid->dy,
                                        grid->icellsp, grid->ijcellsp,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend, grid->jend, grid->kend);
}
#endif

#ifdef USECUDA
double cpres_2::calcdivergence(double * restrict u, double * restrict v, double * restrict w, 
                               double * restrict dzi, double * restrict rhoref, double * restrict rhorefh)
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);
  double divmax = 0;

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const double dxi = 1./grid->dx;
  const double dyi = 1./grid->dy;

  const int offs = grid->memoffset;

  pres_2_calcdivergence<<<gridGPU, blockGPU>>>(&fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs], 
                                               &fields->a["tmp1"]->data_g[offs], grid->dzi_g, 
                                               fields->rhoref_g, fields->rhorefh_g, dxi, dyi,
                                               grid->icellsp, grid->ijcellsp,
                                               grid->istart,  grid->jstart, grid->kstart,
                                               grid->iend,    grid->jend,   grid->kend);

  divmax = grid->getmax_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp2"]->data_g);
  grid->getmax(&divmax);

  return divmax;
}
#endif
