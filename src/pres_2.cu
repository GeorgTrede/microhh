#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <fftw3.h>
#include <hipfft/hipfft.h>
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "pres.h"
#include "pres_2.h"
#include "defines.h"
#include "model.h"
#include "tools.h"
#include "constants.h"

namespace Pres_2_g
{
  __global__ void presin(double * __restrict__ p,
                         double * __restrict__ u ,  double * __restrict__ v ,     double * __restrict__ w ,
                         double * __restrict__ ut,  double * __restrict__ vt,     double * __restrict__ wt,
                         double * __restrict__ dzi, double * __restrict__ rhoref, double * __restrict__ rhorefh,
                         double dxi, double dyi, double dti,
                         const int jj, const int kk,
                         const int jjp, const int kkp,
                         const int imax, const int jmax, const int kmax,
                         const int igc, const int jgc, const int kgc)
  {
    const int ii = 1;
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;
    const int k = blockIdx.z;

    if(i < imax && j < jmax && k < kmax)
    {
      const int ijkp = i + j*jjp + k*kkp;
      const int ijk  = i+igc + (j+jgc)*jj + (k+kgc)*kk;

      p[ijkp] = rhoref [k+kgc]   * ( (ut[ijk+ii] + u[ijk+ii] * dti) - (ut[ijk] + u[ijk] * dti) ) * dxi
              + rhoref [k+kgc]   * ( (vt[ijk+jj] + v[ijk+jj] * dti) - (vt[ijk] + v[ijk] * dti) ) * dyi
            + ( rhorefh[k+kgc+1] * (  wt[ijk+kk] + w[ijk+kk] * dti)
              - rhorefh[k+kgc  ] * (  wt[ijk   ] + w[ijk   ] * dti) ) * dzi[k+kgc];
    }
  }

  __global__ void presout(double * __restrict__ ut, double * __restrict__ vt, double * __restrict__ wt,
                          double * __restrict__ p,
                          double * __restrict__ dzhi, const double dxi, const double dyi,
                          const int jj, const int kk,
                          const int istart, const int jstart, const int kstart,
                          const int iend, const int jend, const int kend)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;

    const int ii = 1;

    if(i < iend && j < jend && k < kend)
    {
      int ijk = i + j*jj + k*kk;
      ut[ijk] -= (p[ijk] - p[ijk-ii]) * dxi;
      vt[ijk] -= (p[ijk] - p[ijk-jj]) * dyi;
      wt[ijk] -= (p[ijk] - p[ijk-kk]) * dzhi[k];
    }
  }

  __global__ void solveout(double * __restrict__ p, double * __restrict__ work3d,
                           const int jj, const int kk,
                           const int jjp, const int kkp,
                           const int istart, const int jstart, const int kstart,
                           const int imax, const int jmax, const int kmax)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;
    const int k = blockIdx.z;

    if(i < imax && j < jmax && k < kmax)
    {
      const int ijk  = i + j*jj + k*kk;
      const int ijkp = i+istart + (j+jstart)*jjp + (k+kstart)*kkp;

      p[ijkp] = work3d[ijk];

      if(k == 0)
        p[ijkp-kkp] = p[ijkp];
    }
  }

  __global__ void solvein(double * __restrict__ p,
                          double * __restrict__ work3d, double * __restrict__ b,
                          double * __restrict__ a, double * __restrict__ c,
                          double * __restrict__ dz, double * __restrict__ rhoref,
                          double * __restrict__ bmati, double * __restrict__ bmatj,
                          const int jj, const int kk,
                          const int imax, const int jmax, const int kmax,
                          const int kstart)
  {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z;

    if(i < imax && j < jmax && k < kmax)
    {
      int ijk = i + j*jj + k*kk;

      // CvH this needs to be taken into account in case of an MPI run
      // iindex = mpi->mpicoordy * iblock + i;
      // jindex = mpi->mpicoordx * jblock + j;
      // b[ijk] = dz[k+kgc]*dz[k+kgc] * (bmati[iindex]+bmatj[jindex]) - (a[k]+c[k]);
      //  if(iindex == 0 && jindex == 0)

      b[ijk] = dz[k+kstart]*dz[k+kstart] * rhoref[k+kstart]*(bmati[i]+bmatj[j]) - (a[k]+c[k]);
      p[ijk] = dz[k+kstart]*dz[k+kstart] * p[ijk];

      if(k == 0)
      {
        // substitute BC's
        // ijk = i + j*jj;
        b[ijk] += a[0];
      }
      else if(k == kmax-1)
      {
        // for wave number 0, which contains average, set pressure at top to zero
        if(i == 0 && j == 0)
          b[ijk] -= c[k];
        // set dp/dz at top to zero
        else
          b[ijk] += c[k];
      }
    }
  }

  __global__ void tdma(double * __restrict__ a, double * __restrict__ b, double * __restrict__ c,
                       double * __restrict__ p, double * __restrict__ work3d,
                       const int jj, const int kk,
                       const int imax, const int jmax, const int kmax)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;

    if(i < imax && j < jmax)
    {
      const int ij = i + j*jj;
      int k,ijk;

      double work2d = b[ij];
      p[ij] /= work2d;

      for(k=1; k<kmax; k++)
      {
        ijk = ij + k*kk;
        work3d[ijk] = c[k-1] / work2d;
        work2d = b[ijk] - a[k]*work3d[ijk];
        p[ijk] -= a[k]*p[ijk-kk];
        p[ijk] /= work2d;
      }

      for(k=kmax-2; k>=0; k--)
      {
        ijk = ij + k*kk;
        p[ijk] -= work3d[ijk+kk]*p[ijk+kk];
      }
    }
  }

  __global__ void calcdivergence(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ div, double * __restrict__ dzi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh,
                                 double dxi, double dyi,
                                 int jj, int kk, int istart, int jstart, int kstart,
                                 int iend, int jend, int kend)
  {
    int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    int k = blockIdx.z + kstart;
    int ii = 1;

    if(i < iend && j < jend && k < kend)
    {
      int ijk = i + j*jj + k*kk;
      div[ijk] = rhoref[k]*((u[ijk+ii]-u[ijk])*dxi + (v[ijk+jj]-v[ijk])*dyi)
               + (rhorefh[k+1]*w[ijk+kk]-rhorefh[k]*w[ijk])*dzi[k];
    }
  }
} // End namespace.

void Pres_2::prepare_device()
{
  const int kmemsize = grid->kmax*sizeof(double);
  const int imemsize = grid->itot*sizeof(double);
  const int jmemsize = grid->jtot*sizeof(double);

  const int ijmemsize = grid->imax*grid->jmax*sizeof(double);

  cudaSafeCall(hipMalloc((void**)&bmati_g, imemsize  ));
  cudaSafeCall(hipMalloc((void**)&bmatj_g, jmemsize  ));
  cudaSafeCall(hipMalloc((void**)&a_g, kmemsize      ));
  cudaSafeCall(hipMalloc((void**)&c_g, kmemsize      ));
  cudaSafeCall(hipMalloc((void**)&work2d_g, ijmemsize));

  cudaSafeCall(hipMemcpy(bmati_g, bmati, imemsize, hipMemcpyHostToDevice   ));
  cudaSafeCall(hipMemcpy(bmatj_g, bmatj, jmemsize, hipMemcpyHostToDevice   ));
  cudaSafeCall(hipMemcpy(a_g, a, kmemsize, hipMemcpyHostToDevice           ));
  cudaSafeCall(hipMemcpy(c_g, c, kmemsize, hipMemcpyHostToDevice           ));
  cudaSafeCall(hipMemcpy(work2d_g, work2d, ijmemsize, hipMemcpyHostToDevice));

  makeCufftPlan();
}

void Pres_2::clear_device()
{
  cudaSafeCall(hipFree(bmati_g ));
  cudaSafeCall(hipFree(bmatj_g ));
  cudaSafeCall(hipFree(a_g     ));
  cudaSafeCall(hipFree(c_g     ));
  cudaSafeCall(hipFree(work2d_g));
}

#ifdef USECUDA
void Pres_2::exec(double dt)
{
  const int blocki = grid->iThreadBlock;
  const int blockj = grid->jThreadBlock;
  const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

  // 3D grid
  dim3 gridGPU (gridi,  gridj,  grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  // 2D grid
  dim3 grid2dGPU (gridi,  gridj);
  dim3 block2dGPU(blocki, blockj);

  const int offs = grid->memoffset;

  // calculate the cyclic BCs first
  grid->boundary_cyclic_g(&fields->ut->data_g[offs]);
  grid->boundary_cyclic_g(&fields->vt->data_g[offs]);
  grid->boundary_cyclic_g(&fields->wt->data_g[offs]);

  Pres_2_g::presin<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g,
                                         &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                         &fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                         grid->dzi_g, fields->rhoref_g, fields->rhorefh_g,
                                         1./grid->dx, 1./grid->dy, 1./dt,
                                         grid->icellsp, grid->ijcellsp, grid->imax, grid->imax*grid->jmax,
                                         grid->imax, grid->jmax, grid->kmax,
                                         grid->igc, grid->jgc, grid->kgc);
  cudaCheckError();

  fft_forward(fields->sd["p"]->data_g, fields->atmp["tmp1"]->data_g, fields->atmp["tmp2"]->data_g);

  Pres_2_g::solvein<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g,
                                          fields->atmp["tmp1"]->data_g, fields->atmp["tmp2"]->data_g,
                                          a_g, c_g,
                                          grid->dz_g, fields->rhoref_g, bmati_g, bmatj_g,
                                          grid->imax, grid->imax*grid->jmax,
                                          grid->imax, grid->jmax, grid->kmax,
                                          grid->kstart);
  cudaCheckError();

  Pres_2_g::tdma<<<grid2dGPU, block2dGPU>>>(a_g, fields->atmp["tmp2"]->data_g, c_g,
                                           fields->sd["p"]->data_g, fields->atmp["tmp1"]->data_g,
                                           grid->imax, grid->imax*grid->jmax,
                                           grid->imax, grid->jmax, grid->kmax);
  cudaCheckError();

  fft_backward(fields->sd["p"]->data_g, fields->atmp["tmp1"]->data_g, fields->atmp["tmp2"]->data_g);

  cudaSafeCall(hipMemcpy(fields->atmp["tmp1"]->data_g, fields->sd["p"]->data_g, grid->ncellsp*sizeof(double), hipMemcpyDeviceToDevice));

  Pres_2_g::solveout<<<gridGPU, blockGPU>>>(&fields->sd["p"]->data_g[offs], fields->atmp["tmp1"]->data_g,
                                           grid->imax, grid->imax*grid->jmax,
                                           grid->icellsp, grid->ijcellsp,
                                           grid->istart, grid->jstart, grid->kstart,
                                           grid->imax, grid->jmax, grid->kmax);
  cudaCheckError();

  grid->boundary_cyclic_g(&fields->sd["p"]->data_g[offs]);

  Pres_2_g::presout<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                          &fields->sd["p"]->data_g[offs],
                                          grid->dzhi_g, 1./grid->dx, 1./grid->dy,
                                          grid->icellsp, grid->ijcellsp,
                                          grid->istart, grid->jstart, grid->kstart,
                                          grid->iend, grid->jend, grid->kend);
  cudaCheckError();
}
#endif

#ifdef USECUDA
double Pres_2::checkDivergence()
{
  const int blocki = grid->iThreadBlock;
  const int blockj = grid->jThreadBlock;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  double divmax = 0;

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const double dxi = 1./grid->dx;
  const double dyi = 1./grid->dy;

  const int offs = grid->memoffset;

  Pres_2_g::calcdivergence<<<gridGPU, blockGPU>>>(&fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs],
                                                 &fields->atmp["tmp1"]->data_g[offs], grid->dzi_g,
                                                 fields->rhoref_g, fields->rhorefh_g, dxi, dyi,
                                                 grid->icellsp, grid->ijcellsp,
                                                 grid->istart,  grid->jstart, grid->kstart,
                                                 grid->iend,    grid->jend,   grid->kend);
  cudaCheckError();

  divmax = grid->get_max_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g);
  grid->get_max(&divmax);

  return divmax;
}
#endif
