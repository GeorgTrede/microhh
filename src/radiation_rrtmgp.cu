#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <numeric>
#include <boost/algorithm/string.hpp>

#include "radiation_rrtmgp.h"
#include "grid.h"
#include "fields.h"
#include "timeloop.h"
#include "thermo.h"
#include "stats.h"
#include "netcdf_interface.h"
#include "constants.h"

#include "Array.h"
#include "Fluxes.h"
#include "subset_kernel_launcher_cuda.h"


namespace
{
    __global__
    void calc_tendency(
            Float* __restrict__ thlt_rad,  const Float* __restrict__ flux_up,
            const Float* __restrict flux_dn, const Float* __restrict__ rho,
            const Float* __restrict__ exner, const Float* __restrict__ dz,
            const int istart, const int jstart, const int kstart,
            const int iend,   const int jend,   const int kend,
            const int igc, const int jgc, const int kgc,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z*blockDim.z + threadIdx.z + kstart;

        if ( (i < iend) && (j < jend) && (k < kend) )
        {
            const Float fac = Float(1.) / (rho[k] * Constants::cp<Float> * exner[k] * dz[k]);

            const int ijk = i + j*jj + k*kk;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;

            thlt_rad[ijk] = fac * (flux_up[ijk_nogc + kk_nogc] - flux_up[ijk_nogc] -
                                   flux_dn[ijk_nogc + kk_nogc] + flux_dn[ijk_nogc] );
        }
    }


    __global__
    void add_tendency(
            Float* __restrict__ thlt,  const Float* __restrict__ thlt_rad,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z*blockDim.z + threadIdx.z + kstart;

        if ( (i < iend) && (j < jend) && (k < kend) )
        {
            const int ijk = i + j*jj + k*kk;
            thlt[ijk] = thlt_rad[ijk];
        }
    }

    __global__
    void store_surface_fluxes(
            Float* __restrict__ flux_up_sfc, Float* __restrict__ flux_dn_sfc,
            const Float* __restrict__ flux_up, const Float* __restrict__ flux_dn,
            const int istart, const int iend,
            const int jstart, const int jend,
            const int igc, const int jgc,
            const int jj, const int kk,
            const int jj_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

        if ( (i < iend) && (j < jend) )
        {
            const int ij = i + j*jj;
            const int ij_nogc = (i-igc) + (j-jgc)*jj_nogc;
            flux_up_sfc[ij] = flux_up[ij_nogc];
            flux_dn_sfc[ij] = flux_dn[ij_nogc];
        }
    }

    __global__
    void effective_radius_and_ciwp_to_gm2(
            Float* __restrict__ rel, Float* __restrict__ rei,
            Float* __restrict__ clwp, Float* __restrict__ ciwp,
            const Float* __restrict__ dz,
            const int ncol, const int nlay, const int kstart,
            const Float four_third_pi_N0_rho_w,
            const Float four_third_pi_N0_rho_i,
            const Float sig_g_fac)
    {
        const int icol = blockIdx.x*blockDim.x + threadIdx.x;
        const int ilay = blockIdx.y*blockDim.y + threadIdx.y;

        if ( (icol < ncol) && (ilay < nlay) )
        {
            const int idx = icol + ilay*ncol;
            const int idx_z = ilay + kstart; 
            const Float rel_local = clwp[idx] > Float(0.) ? Float(1.e6) * sig_g_fac * pow(clwp[idx] / dz[idx_z] / four_third_pi_N0_rho_w, Float(1.)/Float(3.)) : Float(0.);
            const Float rei_local = ciwp[idx] > Float(0.) ? Float(1.e6) * sig_g_fac * pow(ciwp[idx] / dz[idx_z] / four_third_pi_N0_rho_i, Float(1.)/Float(3.)) : Float(0.);

            rel[idx] = max(Float(2.5), min(rel_local, Float(21.5)));
            rei[idx] = max(Float(10.), min(rei_local, Float(180.)));
            
            clwp[idx] *= Float(1.e3);
            ciwp[idx] *= Float(1.e3);
        }
    }


    std::vector<std::string> get_variable_string(
            const std::string& var_name,
            std::vector<int> i_count,
            Netcdf_handle& input_nc,
            const int string_len,
            bool trim=true)
    {
        // Multiply all elements in i_count.
        int total_count = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Add the string length as the rightmost dimension.
        i_count.push_back(string_len);

        // Multiply all elements in i_count.
        // int total_count_char = std::accumulate(i_count.begin(), i_count.end(), 1, std::multiplies<>());

        // Read the entire char array;
        std::vector<char> var_char;
        var_char = input_nc.get_variable<char>(var_name, i_count);

        std::vector<std::string> var;

        for (int n=0; n<total_count; ++n)
        {
            std::string s(var_char.begin()+n*string_len, var_char.begin()+(n+1)*string_len);
            if (trim)
                boost::trim(s);
            var.push_back(s);
        }

        return var;
    }


    Gas_optics_rrtmgp_gpu load_and_init_gas_optics(
            Master& master,
            const Gas_concs_gpu& gas_concs,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read k-distribution information.
        const int n_temps = coef_nc.get_dimension_size("temperature");
        const int n_press = coef_nc.get_dimension_size("pressure");
        const int n_absorbers = coef_nc.get_dimension_size("absorber");
        const int n_char = coef_nc.get_dimension_size("string_len");
        const int n_minorabsorbers = coef_nc.get_dimension_size("minor_absorber");
        const int n_extabsorbers = coef_nc.get_dimension_size("absorber_ext");
        const int n_mixingfracs = coef_nc.get_dimension_size("mixing_fraction");
        const int n_layers = coef_nc.get_dimension_size("atmos_layer");
        const int n_bnds = coef_nc.get_dimension_size("bnd");
        const int n_gpts = coef_nc.get_dimension_size("gpt");
        const int n_pairs = coef_nc.get_dimension_size("pair");
        const int n_minor_absorber_intervals_lower = coef_nc.get_dimension_size("minor_absorber_intervals_lower");
        const int n_minor_absorber_intervals_upper = coef_nc.get_dimension_size("minor_absorber_intervals_upper");
        const int n_contributors_lower = coef_nc.get_dimension_size("contributors_lower");
        const int n_contributors_upper = coef_nc.get_dimension_size("contributors_upper");

        // Read gas names.
        Array<std::string,1> gas_names(
                get_variable_string("gas_names", {n_absorbers}, coef_nc, n_char, true), {n_absorbers});

        Array<int,3> key_species(
                coef_nc.get_variable<int>("key_species", {n_bnds, n_layers, 2}),
                {2, n_layers, n_bnds});
        Array<Float,2> band_lims(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_bnds, 2}), {2, n_bnds});
        Array<int,2> band2gpt(coef_nc.get_variable<int>("bnd_limits_gpt", {n_bnds, 2}), {2, n_bnds});
        Array<Float,1> press_ref(coef_nc.get_variable<Float>("press_ref", {n_press}), {n_press});
        Array<Float,1> temp_ref(coef_nc.get_variable<Float>("temp_ref", {n_temps}), {n_temps});

        Float temp_ref_p = coef_nc.get_variable<Float>("absorption_coefficient_ref_P");
        Float temp_ref_t = coef_nc.get_variable<Float>("absorption_coefficient_ref_T");
        Float press_ref_trop = coef_nc.get_variable<Float>("press_ref_trop");

        Array<Float,3> kminor_lower(
                coef_nc.get_variable<Float>("kminor_lower", {n_temps, n_mixingfracs, n_contributors_lower}),
                {n_contributors_lower, n_mixingfracs, n_temps});
        Array<Float,3> kminor_upper(
                coef_nc.get_variable<Float>("kminor_upper", {n_temps, n_mixingfracs, n_contributors_upper}),
                {n_contributors_upper, n_mixingfracs, n_temps});

        Array<std::string,1> gas_minor(get_variable_string("gas_minor", {n_minorabsorbers}, coef_nc, n_char),
                                       {n_minorabsorbers});

        Array<std::string,1> identifier_minor(
                get_variable_string("identifier_minor", {n_minorabsorbers}, coef_nc, n_char), {n_minorabsorbers});

        Array<std::string,1> minor_gases_lower(
                get_variable_string("minor_gases_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> minor_gases_upper(
                get_variable_string("minor_gases_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,2> minor_limits_gpt_lower(
                coef_nc.get_variable<int>("minor_limits_gpt_lower", {n_minor_absorber_intervals_lower, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_lower});
        Array<int,2> minor_limits_gpt_upper(
                coef_nc.get_variable<int>("minor_limits_gpt_upper", {n_minor_absorber_intervals_upper, n_pairs}),
                {n_pairs, n_minor_absorber_intervals_upper});

        Array<Bool,1> minor_scales_with_density_lower(
                coef_nc.get_variable<Bool>("minor_scales_with_density_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> minor_scales_with_density_upper(
                coef_nc.get_variable<Bool>("minor_scales_with_density_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Bool,1> scale_by_complement_lower(
                coef_nc.get_variable<Bool>("scale_by_complement_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<Bool,1> scale_by_complement_upper(
                coef_nc.get_variable<Bool>("scale_by_complement_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<std::string,1> scaling_gas_lower(
                get_variable_string("scaling_gas_lower", {n_minor_absorber_intervals_lower}, coef_nc, n_char),
                {n_minor_absorber_intervals_lower});
        Array<std::string,1> scaling_gas_upper(
                get_variable_string("scaling_gas_upper", {n_minor_absorber_intervals_upper}, coef_nc, n_char),
                {n_minor_absorber_intervals_upper});

        Array<int,1> kminor_start_lower(
                coef_nc.get_variable<int>("kminor_start_lower", {n_minor_absorber_intervals_lower}),
                {n_minor_absorber_intervals_lower});
        Array<int,1> kminor_start_upper(
                coef_nc.get_variable<int>("kminor_start_upper", {n_minor_absorber_intervals_upper}),
                {n_minor_absorber_intervals_upper});

        Array<Float,3> vmr_ref(
                coef_nc.get_variable<Float>("vmr_ref", {n_temps, n_extabsorbers, n_layers}),
                {n_layers, n_extabsorbers, n_temps});

        Array<Float,4> kmajor(
                coef_nc.get_variable<Float>("kmajor", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                {n_gpts, n_mixingfracs, n_press+1, n_temps});

        // Keep the size at zero, if it does not exist.
        Array<Float,3> rayl_lower;
        Array<Float,3> rayl_upper;

        if (coef_nc.variable_exists("rayl_lower"))
        {
            rayl_lower.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_upper.set_dims({n_gpts, n_mixingfracs, n_temps});
            rayl_lower = coef_nc.get_variable<Float>("rayl_lower", {n_temps, n_mixingfracs, n_gpts});
            rayl_upper = coef_nc.get_variable<Float>("rayl_upper", {n_temps, n_mixingfracs, n_gpts});
        }

        // Is it really LW if so read these variables as well.
        if (coef_nc.variable_exists("totplnk"))
        {
            int n_internal_sourcetemps = coef_nc.get_dimension_size("temperature_Planck");

            Array<Float,2> totplnk(
                    coef_nc.get_variable<Float>( "totplnk", {n_bnds, n_internal_sourcetemps}),
                    {n_internal_sourcetemps, n_bnds});
            Array<Float,4> planck_frac(
                    coef_nc.get_variable<Float>("plank_fraction", {n_temps, n_press+1, n_mixingfracs, n_gpts}),
                    {n_gpts, n_mixingfracs, n_press+1, n_temps});

            // Construct the k-distribution.
            return Gas_optics_rrtmgp_gpu(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    totplnk,
                    planck_frac,
                    rayl_lower,
                    rayl_upper);
        }
        else
        {
            Array<Float,1> solar_src_quiet(
                    coef_nc.get_variable<Float>("solar_source_quiet", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_facular(
                    coef_nc.get_variable<Float>("solar_source_facular", {n_gpts}), {n_gpts});
            Array<Float,1> solar_src_sunspot(
                    coef_nc.get_variable<Float>("solar_source_sunspot", {n_gpts}), {n_gpts});

            Float tsi = coef_nc.get_variable<Float>("tsi_default");
            Float mg_index = coef_nc.get_variable<Float>("mg_default");
            Float sb_index = coef_nc.get_variable<Float>("sb_default");

            return Gas_optics_rrtmgp_gpu(
                    gas_concs,
                    gas_names,
                    key_species,
                    band2gpt,
                    band_lims,
                    press_ref,
                    press_ref_trop,
                    temp_ref,
                    temp_ref_p,
                    temp_ref_t,
                    vmr_ref,
                    kmajor,
                    kminor_lower,
                    kminor_upper,
                    gas_minor,
                    identifier_minor,
                    minor_gases_lower,
                    minor_gases_upper,
                    minor_limits_gpt_lower,
                    minor_limits_gpt_upper,
                    minor_scales_with_density_lower,
                    minor_scales_with_density_upper,
                    scaling_gas_lower,
                    scaling_gas_upper,
                    scale_by_complement_lower,
                    scale_by_complement_upper,
                    kminor_start_lower,
                    kminor_start_upper,
                    solar_src_quiet,
                    solar_src_facular,
                    solar_src_sunspot,
                    tsi,
                    mg_index,
                    sb_index,
                    rayl_lower,
                    rayl_upper);
        }
        // End reading of k-distribution.
    }


    Cloud_optics_gpu load_and_init_cloud_optics(
            Master& master,
            const std::string& coef_file)
    {
        // READ THE COEFFICIENTS FOR THE OPTICAL SOLVER.
        Netcdf_file coef_nc(master, coef_file, Netcdf_mode::Read);

        // Read look-up table coefficient dimensions
        int n_band     = coef_nc.get_dimension_size("nband");
        int n_rghice   = coef_nc.get_dimension_size("nrghice");
        int n_size_liq = coef_nc.get_dimension_size("nsize_liq");
        int n_size_ice = coef_nc.get_dimension_size("nsize_ice");

        Array<Float,2> band_lims_wvn(coef_nc.get_variable<Float>("bnd_limits_wavenumber", {n_band, 2}), {2, n_band});

        // Read look-up table constants.
        Float radliq_lwr = coef_nc.get_variable<Float>("radliq_lwr");
        Float radliq_upr = coef_nc.get_variable<Float>("radliq_upr");
        Float radliq_fac = coef_nc.get_variable<Float>("radliq_fac");

        Float radice_lwr = coef_nc.get_variable<Float>("radice_lwr");
        Float radice_upr = coef_nc.get_variable<Float>("radice_upr");
        Float radice_fac = coef_nc.get_variable<Float>("radice_fac");

        Array<Float,2> lut_extliq(
                coef_nc.get_variable<Float>("lut_extliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_ssaliq(
                coef_nc.get_variable<Float>("lut_ssaliq", {n_band, n_size_liq}), {n_size_liq, n_band});
        Array<Float,2> lut_asyliq(
                coef_nc.get_variable<Float>("lut_asyliq", {n_band, n_size_liq}), {n_size_liq, n_band});

        Array<Float,3> lut_extice(
                coef_nc.get_variable<Float>("lut_extice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_ssaice(
                coef_nc.get_variable<Float>("lut_ssaice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});
        Array<Float,3> lut_asyice(
                coef_nc.get_variable<Float>("lut_asyice", {n_rghice, n_band, n_size_ice}), {n_size_ice, n_band, n_rghice});

        return Cloud_optics_gpu(
                band_lims_wvn,
                radliq_lwr, radliq_upr, radliq_fac,
                radice_lwr, radice_upr, radice_fac,
                lut_extliq, lut_ssaliq, lut_asyliq,
                lut_extice, lut_ssaice, lut_asyice);
    }
}


#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp<TF>::prepare_device()
{
    this->gas_concs_gpu = std::make_unique<Gas_concs_gpu>(gas_concs);

    this->kdist_lw_gpu = std::make_unique<Gas_optics_rrtmgp_gpu>(
            load_and_init_gas_optics(master, *gas_concs_gpu, "coefficients_lw.nc"));

    this->cloud_lw_gpu = std::make_unique<Cloud_optics_gpu>(
            load_and_init_cloud_optics(master, "cloud_coefficients_lw.nc"));

    auto& gd = grid.get_grid_data();
    const int nsfcsize = gd.ijcells*sizeof(Float);

    cuda_safe_call(hipMalloc(&lw_flux_dn_sfc_g, nsfcsize));
    cuda_safe_call(hipMalloc(&lw_flux_up_sfc_g, nsfcsize));
    cuda_safe_call(hipMalloc(&sw_flux_dn_sfc_g, nsfcsize));
    cuda_safe_call(hipMalloc(&sw_flux_up_sfc_g, nsfcsize));
}
#endif


#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp<TF>::exec_longwave(
        Thermo<TF>& thermo, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Array_gpu<Float,2>& flux_up, Array_gpu<Float,2>& flux_dn, Array_gpu<Float,2>& flux_net,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev, const Array_gpu<Float,1>& t_sfc,
        const Array_gpu<Float,2>& h2o, const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const bool compute_clouds)
{
    constexpr int n_col_block = 1024;

    auto& gd = grid.get_grid_data();

    const int n_col = gd.imax*gd.jmax;
    const int n_lay = gd.ktot;
    const int n_lev = gd.ktot+1;

    const int n_blocks = n_col / n_col_block;
    const int n_col_block_residual = n_col % n_col_block;

    const int n_gpt = this->kdist_lw_gpu->get_ngpt();
    const int n_bnd = this->kdist_lw_gpu->get_nband();

    const Bool top_at_1 = 0;

    // Define the pointers for the subsetting.
    std::unique_ptr<Optical_props_arry_gpu> optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_subset =
            std::make_unique<Source_func_lw_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *cloud_lw_gpu);

    std::unique_ptr<Optical_props_arry_gpu> optical_props_residual =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_residual, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_residual =
            std::make_unique<Source_func_lw_gpu>(n_col_block_residual, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_residual =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_residual, n_lay, *cloud_lw_gpu);

    // Make views to the base state pointer.
    auto p_lay = Array_gpu<Float,2>(thermo.get_basestate_fld_g("pref") + gd.kstart, {1, n_lay});
    auto p_lev = Array_gpu<Float,2>(thermo.get_basestate_fld_g("prefh") + gd.kstart, {1, n_lev});

    // CvH: this can be improved by creating a fill function for the GPU.
    Array<Float,2> emis_sfc_cpu(std::vector<Float>(n_bnd, this->emis_sfc), {n_bnd, 1});
    Array_gpu<Float,2> emis_sfc(emis_sfc_cpu);

    gas_concs_gpu->set_vmr("h2o", h2o);

    // CvH: This can be done better: we now allocate a complete array.
    Array_gpu<Float,2> col_dry({n_col, n_lay});
    Gas_optics_rrtmgp_gpu::get_col_dry(col_dry, gas_concs_gpu->get_vmr("h2o"), p_lev.subset({{ {1, n_col}, {1, n_lev} }}));

    // Constants for computation of liquid and ice droplet effective radius
    const Float sig_g = 1.34;
    const Float fac = std::exp(std::log(sig_g)*std::log(sig_g)); // no conversion to micron yet.
    
    const Float Nc0 = 100.e6; 
    const Float Ni0 = 1.e5;

    const Float four_third_pi_N0_rho_w = (4./3.)*M_PI*Nc0*Constants::rho_w<Float>;
    const Float four_third_pi_N0_rho_i = (4./3.)*M_PI*Ni0*Constants::rho_i<Float>; 
    
    const int block_col = 16;
    const int block_lay = 16;
    const int grid_col  = n_col_block/block_col + (n_col_block%block_col > 0);
    const int grid_lay  = n_lay/block_lay + (n_lay%block_lay > 0);

    dim3 gridGPU_re (grid_col, grid_lay, 1);
    dim3 blockGPU_re (block_col, block_lay, 1);

    // Lambda function for solving optical properties subset.
    auto call_kernels = [&](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu>& optical_props_subset_in,
            std::unique_ptr<Optical_props_1scl_gpu>& cloud_optical_props_subset_in,
            Source_func_lw_gpu& sources_subset_in,
            const Array_gpu<Float,2>& emis_sfc_subset_in,
            Fluxes_broadband_gpu& fluxes,
            Fluxes_broadband_gpu& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu gas_concs_subset(*gas_concs_gpu, col_s_in, n_col_in);

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});

        kdist_lw_gpu->gas_optics(
                p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                p_lev_subset,
                t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                t_sfc.subset({{ {col_s_in, col_e_in} }}),
                gas_concs_subset,
                optical_props_subset_in,
                sources_subset_in,
                col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lev} }}),
                t_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }}) );

        
        if (compute_clouds)
        {
            auto clwp_subset = clwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            auto ciwp_subset = ciwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});
            Array_gpu<Float,2> rel({n_col_in, n_lay});
            Array_gpu<Float,2> rei({n_col_in, n_lay});
    
            effective_radius_and_ciwp_to_gm2<<<gridGPU_re, blockGPU_re>>>(
                    rel.ptr(), rei.ptr(), 
                    clwp_subset.ptr(), ciwp_subset.ptr(),
                    gd.dz.data(),
                    n_col_in, n_lay, gd.kstart,
                    four_third_pi_N0_rho_w, four_third_pi_N0_rho_w, fac);
            
            cloud_lw_gpu->cloud_optics(
                    clwp_subset,
                    ciwp_subset,
                    rel,
                    rei,
                    *cloud_optical_props_subset_in);

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_1scl_gpu&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_1scl_gpu&>(*cloud_optical_props_subset_in));
        }

        Array_gpu<Float,3> gpt_flux_up({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn({n_col_in, n_lev, n_gpt});

        constexpr int n_ang = 1;

        rte_lw_gpu.rte_lw(
                optical_props_subset_in,
                top_at_1,
                sources_subset_in,
                emis_sfc_subset_in,
                Array_gpu<Float,2>(), // Add an empty array, no inc_flux.
                gpt_flux_up, gpt_flux_dn,
                n_ang);

        fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        subset_kernel_launcher_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, flux_up.ptr(), flux_dn.ptr(), flux_net.ptr(),
                fluxes.get_flux_up().ptr(), fluxes.get_flux_dn().ptr(), fluxes.get_flux_net().ptr());
    };

    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        Array_gpu<Float,2> emis_sfc_subset = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu>(n_col_block, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                *sources_subset,
                emis_sfc_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);
    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        Array_gpu<Float,2> emis_sfc_residual = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});
        std::unique_ptr<Fluxes_broadband_gpu> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                *sources_residual,
                emis_sfc_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
}
#endif


#ifdef USECUDA
template <typename TF>
void Radiation_rrtmgp<TF>::exec(Thermo<TF>& thermo, double time, Timeloop<TF>& timeloop, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU_3d (gridi, gridj, gd.kmax+1);
    dim3 blockGPU_3d(blocki, blockj, 1);
    dim3 gridGPU_2d (gridi, gridj, 1);
    dim3 blockGPU_2d(blocki, blockj, 1);

    const bool do_radiation = ((timeloop.get_itime() % idt_rad == 0) && !timeloop.in_substep()) ;

    if (do_radiation)
    {
        // Set the tendency to zero.
        // std::fill(fields.sd.at("thlt_rad")->fld.begin(), fields.sd.at("thlt_rad")->fld.end(), Float(0.));
        hipMemset(fields.sd.at("thlt_rad")->fld_g, 0, gd.ncells*sizeof(Float));

        auto t_lay = fields.get_tmp_g();
        auto t_lev = fields.get_tmp_g();
        auto h2o   = fields.get_tmp_g(); // This is the volume mixing ratio, not the specific humidity of vapor.
        auto clwp  = fields.get_tmp_g();
        auto ciwp  = fields.get_tmp_g();

        // Set the input to the radiation on a 3D grid without ghost cells.
        thermo.get_radiation_fields_g(*t_lay, *t_lev, *h2o, *clwp, *ciwp);

        const int nmaxh = gd.imax*gd.jmax*(gd.ktot+1);
        const int ijmax = gd.imax*gd.jmax;

        // Create views on existing variables.
        Array_gpu<Float,2> t_lay_a(t_lay->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> t_lev_a(t_lev->fld_g, {gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,1> t_sfc_a(t_lev->fld_bot_g, {gd.imax*gd.jmax});
        Array_gpu<Float,2> h2o_a(h2o->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> clwp_a(clwp->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> ciwp_a(ciwp->fld_g, {gd.imax*gd.jmax, gd.ktot});

        // Flux fields.
        Array_gpu<Float,2> flux_up ({gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,2> flux_dn ({gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,2> flux_net({gd.imax*gd.jmax, gd.ktot+1});

        const bool compute_clouds = true;

        try
        {
            if (sw_longwave)
            {
                exec_longwave(
                        thermo, timeloop, stats,
                        flux_up, flux_dn, flux_net,
                        t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                        compute_clouds);

                calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                        fields.sd.at("thlt_rad")->fld_g,
                        flux_up.ptr(), flux_dn.ptr(),
                        fields.rhoref_g, thermo.get_basestate_fld_g("exner"),
                        gd.dz.data(),
                        gd.istart, gd.iend, gd.jstart, gd.jend, gd.kstart, gd.kend,
                        gd.igc, gd.jgc, gd.kgc,
                        gd.icells, gd.ijcells,
                        gd.imax, gd.imax*gd.jmax);
                cuda_check_error();

                store_surface_fluxes<<<gridGPU_2d, blockGPU_2d>>>(
                        lw_flux_up_sfc_g, lw_flux_dn_sfc_g,
                        flux_up.ptr(), flux_dn.ptr(),
                        gd.istart, gd.iend,
                        gd.jstart, gd.jend,
                        gd.igc, gd.jgc,
                        gd.icells, gd.ijcells,
                        gd.imax);
                cuda_check_error();
            }

            /*
            if (sw_shortwave)
            {
                if (!sw_fixed_sza)
                {
                    // Update the solar zenith angle, and calculate new shortwave reference column
                    const int day_of_year = int(timeloop.calc_day_of_year());
                    const int year = timeloop.get_year();
                    const Float seconds_after_midnight = Float(timeloop.calc_hour_of_day()*3600);
                    this->mu0 = calc_cos_zenith_angle(lat, lon, day_of_year, seconds_after_midnight, year);

                    // Calculate correction factor for impact Sun's distance on the solar "constant"
                    const Float frac_day_of_year = Float(day_of_year) + seconds_after_midnight / Float(86400);
                    this->tsi_scaling = calc_sun_distance_factor(frac_day_of_year);

                    if (is_day(this->mu0))
                    {
                        const int n_bnd = kdist_sw->get_nband();

                        // Set the solar zenith angle and albedo.
                        Array<Float,2> sfc_alb_dir({n_bnd, n_col});
                        Array<Float,2> sfc_alb_dif({n_bnd, n_col});

                        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
                        {
                            sfc_alb_dir({ibnd, 1}) = this->sfc_alb_dir;
                            sfc_alb_dif({ibnd, 1}) = this->sfc_alb_dif;
                        }

                        Array<Float,1> mu0({n_col});
                        mu0({1}) = this->mu0;

                        solve_shortwave_column(
                                optical_props_sw,
                                sw_flux_up_col, sw_flux_dn_col, sw_flux_dn_dir_col, sw_flux_net_col,
                                sw_flux_dn_dir_inc, sw_flux_dn_dif_inc, thermo.get_basestate_vector("ph")[gd.kend],
                                gas_concs_col,
                                *kdist_sw,
                                col_dry,
                                p_lay_col, p_lev_col,
                                t_lay_col, t_lev_col,
                                mu0,
                                sfc_alb_dir, sfc_alb_dif,
                                tsi_scaling,
                                n_lay_col);
                    }
                }

                if (is_day(this->mu0))
                {
                    Array<Float,2> flux_dn_dir({gd.imax*gd.jmax, gd.ktot+1});

                    exec_shortwave(
                            thermo, timeloop, stats,
                            flux_up, flux_dn, flux_dn_dir, flux_net,
                            t_lay_a, t_lev_a, h2o_a, clwp_a, ciwp_a,
                            compute_clouds);

                    calc_tendency(
                            fields.sd.at("thlt_rad")->fld.data(),
                            flux_up.ptr(), flux_dn.ptr(),
                            fields.rhoref.data(), thermo.get_basestate_vector("exner").data(),
                            gd.dz.data(),
                            gd.istart, gd.iend, gd.jstart, gd.jend, gd.kstart, gd.kend,
                            gd.igc, gd.jgc, gd.kgc,
                            gd.icells, gd.ijcells,
                            gd.imax, gd.imax*gd.jmax);

                    store_surface_fluxes(
                            sw_flux_up_sfc.data(), sw_flux_dn_sfc.data(),
                            flux_up.ptr(), flux_dn.ptr(),
                            gd.istart, gd.iend,
                            gd.jstart, gd.jend,
                            gd.igc, gd.jgc,
                            gd.icells, gd.ijcells,
                            gd.imax);
                }
                else
                {
                    // Set the surface fluxes to zero, for (e.g.) the land-surface model.
                    std::fill(sw_flux_up_sfc.begin(), sw_flux_up_sfc.end(), Float(0));
                    std::fill(sw_flux_dn_sfc.begin(), sw_flux_dn_sfc.end(), Float(0));
                }
            }
            */
        } // End try block.
        catch (std::exception& e)
        {
            #ifdef USEMPI
            std::cout << "SINGLE PROCESS EXCEPTION: " << e.what() << std::endl;
            MPI_Abort(MPI_COMM_WORLD, 1);
            #else
            throw;
            #endif
        }

        fields.release_tmp_g(t_lay);
        fields.release_tmp_g(t_lev);
        fields.release_tmp_g(h2o);
        fields.release_tmp_g(clwp);
        fields.release_tmp_g(ciwp);
    }

    // Always add the tendency.
    add_tendency<<<gridGPU_3d, blockGPU_3d>>>(
            fields.st.at("thl")->fld_g,
            fields.sd.at("thlt_rad")->fld_g,
            gd.istart, gd.iend, gd.jstart, gd.jend, gd.kstart, gd.kend,
            gd.icells, gd.ijcells);
    cuda_check_error();

    hipDeviceSynchronize();
    stats.calc_tend(*fields.st.at("thl"), tend_name);
}


template <typename TF>
std::vector<TF>& Radiation_rrtmgp<TF>::get_surface_radiation(const std::string& name)
{
    throw std::runtime_error("Radiation_rrtmgp is not implemented yet on the GPU");
}


template <typename TF>
void Radiation_rrtmgp<TF>::clear_device()
{
    cuda_safe_call(hipFree(lw_flux_dn_sfc_g));
    cuda_safe_call(hipFree(lw_flux_up_sfc_g));
    cuda_safe_call(hipFree(sw_flux_dn_sfc_g));
    cuda_safe_call(hipFree(sw_flux_up_sfc_g));
}
#endif


#ifdef FLOAT_SINGLE
template class Radiation_rrtmgp<float>;
#else
template class Radiation_rrtmgp<double>;
#endif
