#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "radiation_rrtmgp.h"
#include "grid.h"
#include "fields.h"
#include "timeloop.h"
#include "thermo.h"
#include "stats.h"
#include "constants.h"

#include "Array.h"


namespace
{
    #ifdef USECUDA

    __global__
    void calc_tendency(
            Float* __restrict__ thlt_rad,  const Float* __restrict__ flux_up, 
            const Float* __restrict flux_dn, const Float* __restrict__ rho, 
            const Float* __restrict__ exner, const Float* __restrict__ dz,
            const int istart, const int jstart, const int kstart,
            const int iend,   const int jend,   const int kend,
            const int igc, const int jgc, const int kgc,
            const int jj, const int kk,
            const int jj_nogc, const int kk_nogc)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z*blockDim.z + threadIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const Float fac = Float(1.) / (rho[k] * Constants::cp<Float> * exner[k] * dz[k]);

            const int ijk = i + j*jj + k*kk;
            const int ijk_nogc = (i-igc) + (j-jgc)*jj_nogc + (k-kgc)*kk_nogc;

            thlt_rad[ijk] = fac * (flux_up[ijk_nogc + kk_nogc] - flux_up[ijk_nogc] -
                                   flux_dn[ijk_nogc + kk_nogc] + flux_dn[ijk_nogc] ); 
        }
    }
            
    __global__
    void add_tendency(
            Float* __restrict__ thlt,  const Float* __restrict__ thlt_rad,
            const int istart, const int jstart, const int kstart,
            const int iend, const int jend, const int kend,
            const int jj, const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z*blockDim.z + threadIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            thlt[ijk] = thlt_rad[ijk];
        }
    }
    #endif
}


#ifdef USECUDA
template<typename TF>
void Radiation_rrtmgp<TF>::exec_longwave(
        Thermo<TF>& thermo, Timeloop<TF>& timeloop, Stats<TF>& stats,
        Array_gpu<Float,2>& flux_up, Array_gpu<Float,2>& flux_dn, Array_gpu<Float,2>& flux_net,
        const Array_gpu<Float,2>& t_lay, const Array_gpu<Float,2>& t_lev, const Array_gpu<Float,1>& t_sfc,
        const Array_gpu<Float,2>& h2o, const Array_gpu<Float,2>& clwp, const Array_gpu<Float,2>& ciwp,
        const bool compute_clouds)
{
    constexpr int n_col_block = 1024;

    auto& gd = grid.get_grid_data();

    const int n_col = gd.imax*gd.jmax;
    const int n_lay = gd.ktot;
    const int n_lev = gd.ktot+1;

    const int n_blocks = n_col / n_col_block;
    const int n_col_block_left = n_col % n_col_block;

    const int n_gpt = this->kdist_lw_gpu->get_ngpt();
    const int n_bnd = this->kdist_lw_gpu->get_nband();

    const Bool top_at_1 = 0;

    // Define the pointers for the subsetting.
    std::unique_ptr<Optical_props_arry_gpu> optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_subset =
            std::make_unique<Source_func_lw_gpu>(n_col_block, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_subset =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block, n_lay, *cloud_lw_gpu);

    std::unique_ptr<Optical_props_arry_gpu> optical_props_left =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_left, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Source_func_lw_gpu> sources_left =
            std::make_unique<Source_func_lw_gpu>(n_col_block_left, n_lay, *kdist_lw_gpu);
    std::unique_ptr<Optical_props_1scl_gpu> cloud_optical_props_left =
            std::make_unique<Optical_props_1scl_gpu>(n_col_block_left, n_lay, *cloud_lw_gpu);

    /*
    // Lambda function for solving optical properties subset.
    auto call_kernels = [&](
            const int col_s_in, const int col_e_in,
            std::unique_ptr<Optical_props_arry_gpu>& optical_props_subset_in,
            std::unique_ptr<Optical_props_1scl_gpu>& cloud_optical_props_subset_in,
            Source_func_lw_gpu& sources_subset_in,
            const Array_gpu<Float,2>& emis_sfc_subset_in,
            Fluxes_broadband_gpu& fluxes,
            Fluxes_broadband_gpu& bnd_fluxes)
    {
        const int n_col_in = col_e_in - col_s_in + 1;
        Gas_concs_gpu gas_concs_subset(gas_concs, col_s_in, n_col_in);

        auto p_lev_subset = p_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }});

        Array_gpu<Float,2> col_dry_subset({n_col_in, n_lay});
        if (col_dry.size() == 0)
            Gas_optics_rrtmgp_gpu::get_col_dry(col_dry_subset, gas_concs_subset.get_vmr("h2o"), p_lev_subset);
        else
            col_dry_subset = col_dry.subset({{ {col_s_in, col_e_in}, {1, n_lay} }});

        kdist_gpu->gas_optics(
                p_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                p_lev_subset,
                t_lay.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                t_sfc.subset({{ {col_s_in, col_e_in} }}),
                gas_concs_subset,
                optical_props_subset_in,
                sources_subset_in,
                col_dry_subset,
                t_lev.subset({{ {col_s_in, col_e_in}, {1, n_lev} }}) );

        if (switch_cloud_optics)
        {
            cloud_optics_gpu->cloud_optics(
                    lwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    iwp.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    rel.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    rei.subset({{ {col_s_in, col_e_in}, {1, n_lay} }}),
                    *cloud_optical_props_subset_in);

            // cloud->delta_scale();

            // Add the cloud optical props to the gas optical properties.
            add_to(
                    dynamic_cast<Optical_props_1scl_gpu&>(*optical_props_subset_in),
                    dynamic_cast<Optical_props_1scl_gpu&>(*cloud_optical_props_subset_in));
        }

        // Store the optical properties, if desired.
        if (switch_output_optical)
        {
            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_lay, n_gpt, n_col_in, col_s_in, tau.ptr(), lay_source.ptr(), lev_source_inc.ptr(), lev_source_dec.ptr(),
                    optical_props_subset_in->get_tau().ptr(), sources_subset_in.get_lay_source().ptr(),
                    sources_subset_in.get_lev_source_inc().ptr(), sources_subset_in.get_lev_source_dec().ptr());

            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_gpt, n_col_in, col_s_in, sfc_source.ptr(), sources_subset_in.get_sfc_source().ptr());
        }

        if (!switch_fluxes)
            return;

        Array_gpu<Float,3> gpt_flux_up({n_col_in, n_lev, n_gpt});
        Array_gpu<Float,3> gpt_flux_dn({n_col_in, n_lev, n_gpt});

        constexpr int n_ang = 1;

        rte_lw.rte_lw(
                optical_props_subset_in,
                top_at_1,
                sources_subset_in,
                emis_sfc_subset_in,
                Array_gpu<Float,2>(), // Add an empty array, no inc_flux.
                gpt_flux_up, gpt_flux_dn,
                n_ang);

        fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

        // Copy the data to the output.
        subset_kernel_launcher_cuda::get_from_subset(
                n_col, n_lev, n_col_in, col_s_in, lw_flux_up.ptr(), lw_flux_dn.ptr(), lw_flux_net.ptr(),
                fluxes.get_flux_up().ptr(), fluxes.get_flux_dn().ptr(), fluxes.get_flux_net().ptr());

        if (switch_output_bnd_fluxes)
        {
            bnd_fluxes.reduce(gpt_flux_up, gpt_flux_dn, optical_props_subset_in, top_at_1);

            subset_kernel_launcher_cuda::get_from_subset(
                    n_col, n_lev, n_bnd, n_col_in, col_s_in, lw_bnd_flux_up.ptr(), lw_bnd_flux_dn.ptr(), lw_bnd_flux_net.ptr(),
                    bnd_fluxes.get_bnd_flux_up().ptr(), bnd_fluxes.get_bnd_flux_dn().ptr(), bnd_fluxes.get_bnd_flux_net().ptr());
        }
    };

    for (int b=1; b<=n_blocks; ++b)
    {
        const int col_s = (b-1) * n_col_block + 1;
        const int col_e =  b    * n_col_block;

        Array_gpu<Float,2> emis_sfc_subset = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});

        std::unique_ptr<Fluxes_broadband_gpu> fluxes_subset =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_subset =
                std::make_unique<Fluxes_byband_gpu>(n_col_block, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_subset,
                cloud_optical_props_subset,
                *sources_subset,
                emis_sfc_subset,
                *fluxes_subset,
                *bnd_fluxes_subset);
    }

    if (n_col_block_residual > 0)
    {
        const int col_s = n_col - n_col_block_residual + 1;
        const int col_e = n_col;

        Array_gpu<Float,2> emis_sfc_residual = emis_sfc.subset({{ {1, n_bnd}, {col_s, col_e} }});
        std::unique_ptr<Fluxes_broadband_gpu> fluxes_residual =
                std::make_unique<Fluxes_broadband_gpu>(n_col_block_residual, n_lev);
        std::unique_ptr<Fluxes_broadband_gpu> bnd_fluxes_residual =
                std::make_unique<Fluxes_byband_gpu>(n_col_block_residual, n_lev, n_bnd);

        call_kernels(
                col_s, col_e,
                optical_props_residual,
                cloud_optical_props_residual,
                *sources_residual,
                emis_sfc_residual,
                *fluxes_residual,
                *bnd_fluxes_residual);
    }
    */
}
#endif


#ifdef USECUDA
template <typename TF>
void Radiation_rrtmgp<TF>::exec(Thermo<TF>& thermo, double time, Timeloop<TF>& timeloop, Stats<TF>& stats)
{
    auto& gd = grid.get_grid_data();

    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU_3d (gridi, gridj, gd.kmax+1);
    dim3 blockGPU_3d(blocki, blockj, 1);
    
    const bool do_radiation = ((timeloop.get_itime() % idt_rad == 0) && !timeloop.in_substep()) ;

    if (do_radiation)
    {
        // Set the tendency to zero.
        std::fill(fields.sd.at("thlt_rad")->fld.begin(), fields.sd.at("thlt_rad")->fld.end(), Float(0.));

        auto t_lay = fields.get_tmp_g();
        auto t_lev = fields.get_tmp_g();
        auto h2o   = fields.get_tmp_g(); // This is the volume mixing ratio, not the specific humidity of vapor.
        auto clwp  = fields.get_tmp_g();
        auto ciwp  = fields.get_tmp_g();

        // Set the input to the radiation on a 3D grid without ghost cells.
        thermo.get_radiation_fields_g(*t_lay, *t_lev, *h2o, *clwp, *ciwp);

        const int nmaxh = gd.imax*gd.jmax*(gd.ktot+1);
        const int ijmax = gd.imax*gd.jmax;

        // Create views on existing variables.
        Array_gpu<Float,2> t_lay_a(t_lay->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> t_lev_a(t_lev->fld_g, {gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,1> t_sfc_a(t_lev->fld_bot_g, {gd.imax*gd.jmax});
        Array_gpu<Float,2> h2o_a(h2o->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> clwp_a(clwp->fld_g, {gd.imax*gd.jmax, gd.ktot});
        Array_gpu<Float,2> ciwp_a(ciwp->fld_g, {gd.imax*gd.jmax, gd.ktot});

        // Flux fields.
        Array_gpu<Float,2> flux_up ({gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,2> flux_dn ({gd.imax*gd.jmax, gd.ktot+1});
        Array_gpu<Float,2> flux_net({gd.imax*gd.jmax, gd.ktot+1});

        const bool compute_clouds = true;
 
        try
        {
            if (sw_longwave)
            {
                /*
                exec_longwave(
                        thermo, timeloop, stats,
                        flux_up, flux_dn, flux_net,
                        t_lay_a, t_lev_a, t_sfc_a, h2o_a, clwp_a, ciwp_a,
                        compute_clouds); */
                calc_tendency<<<gridGPU_3d, blockGPU_3d>>>(
                        fields.sd.at("thlt_rad")->fld.data(),
                        flux_up.ptr(), flux_dn.ptr(),
                        fields.rhoref.data(), thermo.get_basestate_vector("exner").data(),
                        gd.dz.data(),
                        gd.istart, gd.iend, gd.jstart, gd.jend, gd.kstart, gd.kend,
                        gd.igc, gd.jgc, gd.kgc,
                        gd.icells, gd.ijcells,
                        gd.imax, gd.imax*gd.jmax);
                /*
                store_surface_fluxes(
                        lw_flux_up_sfc.data(), lw_flux_dn_sfc.data(),
                        flux_up.ptr(), flux_dn.ptr(),
                        gd.istart, gd.iend,
                        gd.jstart, gd.jend,
                        gd.igc, gd.jgc,
                        gd.icells, gd.ijcells,
                        gd.imax);
                        */
            }

            /*
            if (sw_shortwave)
            {
                if (!sw_fixed_sza)
                {
                    // Update the solar zenith angle, and calculate new shortwave reference column
                    const int day_of_year = int(timeloop.calc_day_of_year());
                    const int year = timeloop.get_year();
                    const Float seconds_after_midnight = Float(timeloop.calc_hour_of_day()*3600);
                    this->mu0 = calc_cos_zenith_angle(lat, lon, day_of_year, seconds_after_midnight, year);

                    // Calculate correction factor for impact Sun's distance on the solar "constant"
                    const Float frac_day_of_year = Float(day_of_year) + seconds_after_midnight / Float(86400);
                    this->tsi_scaling = calc_sun_distance_factor(frac_day_of_year);

                    if (is_day(this->mu0))
                    {
                        const int n_bnd = kdist_sw->get_nband();

                        // Set the solar zenith angle and albedo.
                        Array<Float,2> sfc_alb_dir({n_bnd, n_col});
                        Array<Float,2> sfc_alb_dif({n_bnd, n_col});

                        for (int ibnd=1; ibnd<=n_bnd; ++ibnd)
                        {
                            sfc_alb_dir({ibnd, 1}) = this->sfc_alb_dir;
                            sfc_alb_dif({ibnd, 1}) = this->sfc_alb_dif;
                        }

                        Array<Float,1> mu0({n_col});
                        mu0({1}) = this->mu0;

                        solve_shortwave_column(
                                optical_props_sw,
                                sw_flux_up_col, sw_flux_dn_col, sw_flux_dn_dir_col, sw_flux_net_col,
                                sw_flux_dn_dir_inc, sw_flux_dn_dif_inc, thermo.get_basestate_vector("ph")[gd.kend],
                                gas_concs_col,
                                *kdist_sw,
                                col_dry,
                                p_lay_col, p_lev_col,
                                t_lay_col, t_lev_col,
                                mu0,
                                sfc_alb_dir, sfc_alb_dif,
                                tsi_scaling,
                                n_lay_col);
                    }
                }

                if (is_day(this->mu0))
                {
                    Array<Float,2> flux_dn_dir({gd.imax*gd.jmax, gd.ktot+1});

                    exec_shortwave(
                            thermo, timeloop, stats,
                            flux_up, flux_dn, flux_dn_dir, flux_net,
                            t_lay_a, t_lev_a, h2o_a, clwp_a, ciwp_a,
                            compute_clouds);

                    calc_tendency(
                            fields.sd.at("thlt_rad")->fld.data(),
                            flux_up.ptr(), flux_dn.ptr(),
                            fields.rhoref.data(), thermo.get_basestate_vector("exner").data(),
                            gd.dz.data(),
                            gd.istart, gd.iend, gd.jstart, gd.jend, gd.kstart, gd.kend,
                            gd.igc, gd.jgc, gd.kgc,
                            gd.icells, gd.ijcells,
                            gd.imax, gd.imax*gd.jmax);

                    store_surface_fluxes(
                            sw_flux_up_sfc.data(), sw_flux_dn_sfc.data(),
                            flux_up.ptr(), flux_dn.ptr(),
                            gd.istart, gd.iend,
                            gd.jstart, gd.jend,
                            gd.igc, gd.jgc,
                            gd.icells, gd.ijcells,
                            gd.imax);
                }
                else
                {
                    // Set the surface fluxes to zero, for (e.g.) the land-surface model.
                    std::fill(sw_flux_up_sfc.begin(), sw_flux_up_sfc.end(), Float(0));
                    std::fill(sw_flux_dn_sfc.begin(), sw_flux_dn_sfc.end(), Float(0));
                }
            }
            */
        } // End try block.
        catch (std::exception& e)
        {
            #ifdef USEMPI
            std::cout << "SINGLE PROCESS EXCEPTION: " << e.what() << std::endl;
            MPI_Abort(MPI_COMM_WORLD, 1);
            #else
            throw;
            #endif
        }

        fields.release_tmp_g(t_lay);
        fields.release_tmp_g(t_lev);
        fields.release_tmp_g(h2o);
        fields.release_tmp_g(clwp);
        fields.release_tmp_g(ciwp);
    }

    // Always add the tendency.
    add_tendency<<<gridGPU_3d, blockGPU_3d>>>(
            fields.st.at("thl")->fld_g,
            fields.sd.at("thlt_rad")->fld_g,
            gd.istart, gd.iend, gd.jstart, gd.jend, gd.kstart, gd.kend,
            gd.icells, gd.ijcells);

    stats.calc_tend(*fields.st.at("thl"), tend_name);
}


template <typename TF>
std::vector<TF>& Radiation_rrtmgp<TF>::get_surface_radiation(const std::string& name)
{
    throw std::runtime_error("Radiation_rrtmgp is not implemented yet on the GPU");
}


template <typename TF>
void Radiation_rrtmgp<TF>::prepare_device()
{
}


template <typename TF>
void Radiation_rrtmgp<TF>::clear_device()
{
}
#endif


#ifdef FLOAT_SINGLE
template class Radiation_rrtmgp<float>;
#else
template class Radiation_rrtmgp<double>;
#endif
