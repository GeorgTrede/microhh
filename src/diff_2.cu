#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "grid.h"
#include "fields.h"
#include "diff_2.h"
#include "defines.h"
#include "constants.h"
#include "tools.h"

__global__ void diff_2_diffc(double * __restrict__ const at, const double * __restrict__ const a,
                             const double * __restrict__ const dzi, const double * __restrict__ const dzhi,
                             const double dxidxi, const double dyidyi, const double visc,
                             const int jj, const int kk,
                             const int istart, const int jstart, const int kstart,
                             const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;
    const int ii = 1;

    at[ijk] += visc * (
          + (  (a[ijk+ii] - a[ijk   ]) 
             - (a[ijk   ] - a[ijk-ii]) ) * dxidxi 
          + (  (a[ijk+jj] - a[ijk   ]) 
             - (a[ijk   ] - a[ijk-jj]) ) * dyidyi
          + (  (a[ijk+kk] - a[ijk   ]) * dzhi[k+1]
             - (a[ijk   ] - a[ijk-kk]) * dzhi[k]   ) * dzi[k] );
  }
}

__global__ void diff_2_diffw(double * __restrict__ const at, const double * __restrict__ const a,
                             const double * __restrict__ const dzi, const double * __restrict__ const dzhi,
                             const double dxidxi, const double dyidyi, const double visc,
                             const int jj, const int kk,
                             const int istart, const int jstart, const int kstart,
                             const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k > kstart && k < kend)
  {
    const int ijk = i + j*jj + k*kk;
    const int ii = 1;

    at[ijk] += visc * (
          + (  (a[ijk+ii] - a[ijk   ])
             - (a[ijk   ] - a[ijk-ii]) ) * dxidxi
          + (  (a[ijk+jj] - a[ijk   ])
             - (a[ijk   ] - a[ijk-jj]) ) * dyidyi
          + (  (a[ijk+kk] - a[ijk   ]) * dzi[k]
             - (a[ijk   ] - a[ijk-kk]) * dzi[k-1] ) * dzhi[k] );
  }
}

#ifdef USECUDA
int cdiff_2::exec()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const double dxidxi = 1./(grid->dx*grid->dx);
  const double dyidyi = 1./(grid->dy*grid->dy);

  const int offs = grid->memoffset;

  diff_2_diffc<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->u->data_g[offs],
                                      grid->dzi_g, grid->dzhi_g,
                                      dxidxi, dyidyi, fields->visc,
                                      grid->icellsp, grid->ijcellsp,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend);
  cudaCheckError();

  diff_2_diffc<<<gridGPU, blockGPU>>>(&fields->vt->data_g[offs], &fields->v->data_g[offs],
                                      grid->dzi_g, grid->dzhi_g,
                                      dxidxi, dyidyi, fields->visc,
                                      grid->icellsp, grid->ijcellsp,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend);
  cudaCheckError();

  diff_2_diffw<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->w->data_g[offs],
                                      grid->dzi_g, grid->dzhi_g,
                                      dxidxi, dyidyi, fields->visc,
                                      grid->icellsp, grid->ijcellsp,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend);
  cudaCheckError();


  for(fieldmap::const_iterator it = fields->st.begin(); it!=fields->st.end(); it++)
    diff_2_diffc<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], &fields->sp[it->first]->data_g[offs],
                                        grid->dzi_g, grid->dzhi_g,
                                        dxidxi, dyidyi, fields->sp[it->first]->visc,
                                        grid->icellsp, grid->ijcellsp,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend, grid->jend, grid->kend);
  cudaCheckError();

  return 0;
}
#endif
