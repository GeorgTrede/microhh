#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2013 Chiel van Heerwaarden
 * Copyright (c) 2011-2013 Thijs Heus
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "grid.h"
#include "fields.h"
#include "diff_2.h"
#include "defines.h"

__global__ void diff_2_diffc(double * __restrict__ const at, const double * __restrict__ const a,
                             const double * __restrict__ const dzi, const double * __restrict__ const dzhi,
                             const double dx, const double dy, const double visc,
                             const int jj, const int kk,
                             const int istart, const int jstart, const int kstart,
                             const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;

    const int ii = 1;

    const double dxidxi = 1./(dx*dx);
    const double dyidyi = 1./(dy*dy);

    at[ijk] += visc * (
          + (  (a[ijk+ii] - a[ijk   ]) 
             - (a[ijk   ] - a[ijk-ii]) ) * dxidxi 
          + (  (a[ijk+jj] - a[ijk   ]) 
             - (a[ijk   ] - a[ijk-jj]) ) * dyidyi
          + (  (a[ijk+kk] - a[ijk   ]) * dzhi[k+1]
             - (a[ijk   ] - a[ijk-kk]) * dzhi[k]   ) * dzi[k] );
  }
}

__global__ void diff_2_diffw(double * __restrict__ const at, const double * __restrict__ const a,
                             const double * __restrict__ const dzi, const double * __restrict__ const dzhi,
                             const double dx, const double dy, const double visc,
                             const int jj, const int kk,
                             const int istart, const int jstart, const int kstart,
                             const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k > kstart && k < kend)
  {
    const int ijk = i + j*jj + k*kk;

    const int ii = 1;

    const double dxidxi = 1./(dx*dx);
    const double dyidyi = 1./(dy*dy);

    at[ijk] += visc * (
          + (  (a[ijk+ii] - a[ijk   ])
             - (a[ijk   ] - a[ijk-ii]) ) * dxidxi
          + (  (a[ijk+jj] - a[ijk   ])
             - (a[ijk   ] - a[ijk-jj]) ) * dyidyi
          + (  (a[ijk+kk] - a[ijk   ]) * dzi[k]
             - (a[ijk   ] - a[ijk-kk]) * dzi[k-1] ) * dzhi[k] );
  }
}

#ifdef USECUDA
int cdiff_2::exec()
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  //fields->forwardGPU();

  diff_2_diffc<<<gridGPU, blockGPU>>>(fields->ut->data_g, fields->u->data_g,
                                      grid->dzi_g, grid->dzhi_g,
                                      grid->dx, grid->dy, fields->visc,
                                      grid->icells, grid->ijcells,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend);

  diff_2_diffc<<<gridGPU, blockGPU>>>(fields->vt->data_g, fields->v->data_g,
                                      grid->dzi_g, grid->dzhi_g,
                                      grid->dx, grid->dy, fields->visc,
                                      grid->icells, grid->ijcells,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend);

  diff_2_diffw<<<gridGPU, blockGPU>>>(fields->wt->data_g, fields->w->data_g,
                                      grid->dzi_g, grid->dzhi_g,
                                      grid->dx, grid->dy, fields->visc,
                                      grid->icells, grid->ijcells,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend);


  for(fieldmap::const_iterator it = fields->st.begin(); it!=fields->st.end(); it++)
    diff_2_diffc<<<gridGPU, blockGPU>>>(it->second->data_g, fields->sp[it->first]->data_g,
                                        grid->dzi_g, grid->dzhi_g,
                                        grid->dx, grid->dy, fields->sp[it->first]->visc,
                                        grid->icells, grid->ijcells,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend, grid->jend, grid->kend);

  //fields->backwardGPU();

  return 0;
}
#endif
