#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "timeloop.h"
#include "grid.h"
#include "master.h"
#include "constants.h"

#define cA0 0.
#define cA1 -5./9.
#define cA2 -153./128.
#define cB0 1./3.
#define cB1 15./16.
#define cB2 8./15.

__global__ void rk3_kernel(double * __restrict__ a, double * __restrict__ at, double dt,
                           const int substep, const int jj, const int kk,
                           const int istart, const int jstart, const int kstart,
                           const int iend, const int jend, const int kend)
{
  const double cA[] = {0., -5./9., -153./128.};
  const double cB[] = {1./3., 15./16., 8./15.};

  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;
    a[ijk] = a[ijk] + cB[substep]*dt*at[ijk];

    const int substepn = (substep+1) % 3;
    // substep 0 resets the tendencies, because cA[0] == 0
    at[ijk] = cA[substepn]*at[ijk];
  }
}

template<int substep>
__global__ void rk3_kernel2(double * __restrict__ a, double * __restrict__ at, double dt,
                           const int jj, const int kk,
                           const int istart, const int jstart, const int kstart,
                           const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;

    switch(substep)
    {
      case 0:
        a[ijk]  = a[ijk] + cB0*dt*at[ijk];
        at[ijk] = cA1*at[ijk];
        break;
      case 1:
        a[ijk]  = a[ijk] + cB1*dt*at[ijk];
        at[ijk] = cA2*at[ijk];
        break;
      case 2:
        a[ijk]  = a[ijk] + cB2*dt*at[ijk];
        at[ijk] = 0.; 
        break;
    }
  }
}

__global__ void rk4_kernel(double * __restrict__ a, double * __restrict__ at, double dt,
                           const int substep, const int jj, const int kk,
                           const int istart, const int jstart, const int kstart,
                           const int iend, const int jend, const int kend)
{
  const double cA [] = {
      0.,
    - 567301805773./1357537059087.,
    -2404267990393./2016746695238.,
    -3550918686646./2091501179385.,
    -1275806237668./ 842570457699.};

  const double cB [] = {
    1432997174477./ 9575080441755.,
    5161836677717./13612068292357.,
    1720146321549./ 2090206949498.,
    3134564353537./ 4481467310338.,
    2277821191437./14882151754819.};
  
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;
    a[ijk] = a[ijk] + cB[substep]*dt*at[ijk];

    const int substepn = (substep+1) % 5;
    // substep 0 resets the tendencies, because cA[0] == 0
    at[ijk] = cA[substepn]*at[ijk];
  }
}

int ctimeloop::rk3_GPU(double *a, double *at, double dt)
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  if(substep==0) {
    rk3_kernel2<0><<<gridGPU, blockGPU>>>(&a[offs], &at[offs], dt,
                                      grid->icellsp, grid->ijcellsp,
                                      grid->istart,  grid->jstart, grid->kstart,
                                      grid->iend,    grid->jend,   grid->kend); }
  else if(substep==1) {
    rk3_kernel2<1><<<gridGPU, blockGPU>>>(&a[offs], &at[offs], dt,
                                      grid->icellsp, grid->ijcellsp,
                                      grid->istart,  grid->jstart, grid->kstart,
                                      grid->iend,    grid->jend,   grid->kend); }
  else if(substep==2) {
    rk3_kernel2<2><<<gridGPU, blockGPU>>>(&a[offs], &at[offs], dt,
                                      grid->icellsp, grid->ijcellsp,
                                      grid->istart,  grid->jstart, grid->kstart,
                                      grid->iend,    grid->jend,   grid->kend); }

  //rk3_kernel<<<gridGPU, blockGPU>>>(a, at, dt,
  //                                  substep, grid->icells, grid->ijcells,
  //                                  grid->istart, grid->jstart, grid->kstart,
  //                                  grid->iend, grid->jend, grid->kend);

  return 0;
}

int ctimeloop::rk4_GPU(double *a, double *at, double dt)
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  rk4_kernel<<<gridGPU, blockGPU>>>(a, at, dt,
                                    substep, grid->icells, grid->ijcells,
                                    grid->istart, grid->jstart, grid->kstart,
                                    grid->iend, grid->jend, grid->kend);

  return 0;
}
