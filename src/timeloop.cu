#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "timeloop.h"
#include "grid.h"
#include "master.h"
#include "fields.h"
#include "constants.h"
#include "tools.h"

namespace
{
    /*
       __global__ void rk3_kernel(double * __restrict__ a, double * __restrict__ at, double dt,
       const int substep, const int jj, const int kk,
       const int istart, const int jstart, const int kstart,
       const int iend, const int jend, const int kend)
       {
       const double cA[] = {0., -5./9., -153./128.};
       const double cB[] = {1./3., 15./16., 8./15.};

       const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
       const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
       const int k = blockIdx.z + kstart;

       if(i < iend && j < jend && k < kend)
       {
       const int ijk = i + j*jj + k*kk;
       a[ijk] = a[ijk] + cB[substep]*dt*at[ijk];

       const int substepn = (substep+1) % 3;
    // substep 0 resets the tendencies, because cA[0] == 0
    at[ijk] = cA[substepn]*at[ijk];
    }
    }
     */

    template<int substep> __global__ 
    void rk3_g(double* __restrict__ a, double* __restrict__ at, double dt,
               const int jj, const int kk,
               const int istart, const int jstart, const int kstart,
               const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        // const double cA0 =  0.;
        const double cA1 = -5./9.;
        const double cA2 = -153./128.;

        const double cB0 =  1./ 3.;
        const double cB1 = 15./16.;
        const double cB2 =  8./15.;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            switch (substep)
            {
                case 0:
                    a [ijk] = a[ijk] + cB0*dt*at[ijk];
                    at[ijk] = cA1*at[ijk];
                    break;
                case 1:
                    a [ijk] = a[ijk] + cB1*dt*at[ijk];
                    at[ijk] = cA2*at[ijk];
                    break;
                case 2:
                    a [ijk] = a[ijk] + cB2*dt*at[ijk];
                    at[ijk] = 0.; 
                    break;
            }
        }
    }

    template<int substep> __global__ 
    void rk4_g(double* __restrict__ a, double* __restrict__ at, double dt,
               const int jj, const int kk,
               const int istart, const int jstart, const int kstart,
               const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        // const double cA0 =   0.;
        const double cA1 = - 567301805773./1357537059087.;
        const double cA2 = -2404267990393./2016746695238.;
        const double cA3 = -3550918686646./2091501179385.;
        const double cA4 = -1275806237668./ 842570457699.;

        const double cB0 = 1432997174477./ 9575080441755.;
        const double cB1 = 5161836677717./13612068292357.;
        const double cB2 = 1720146321549./ 2090206949498.;
        const double cB3 = 3134564353537./ 4481467310338.;
        const double cB4 = 2277821191437./14882151754819.;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            switch (substep)
            {
                case 0:
                    a [ijk] = a[ijk] + cB0*dt*at[ijk];
                    at[ijk] = cA1*at[ijk];
                    break;
                case 1:
                    a [ijk] = a[ijk] + cB1*dt*at[ijk];
                    at[ijk] = cA2*at[ijk];
                    break;
                case 2:
                    a [ijk] = a[ijk] + cB2*dt*at[ijk];
                    at[ijk] = cA3*at[ijk]; 
                    break;
                case 3:
                    a [ijk] = a[ijk] + cB3*dt*at[ijk];
                    at[ijk] = cA4*at[ijk]; 
                    break;
                case 4:
                    a [ijk] = a[ijk] + cB4*dt*at[ijk];
                    at[ijk] = 0;
                    break;
            }
        }
    }
}

#ifdef USECUDA
void Timeloop::exec()
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (rkorder == 3)
    {
        for (FieldMap::const_iterator it = fields->at.begin(); it!=fields->at.end(); ++it)
        {
            if (substep == 0)
                rk3_g<0><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
            else if (substep == 1)
                rk3_g<1><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
            else if (substep == 2)
                rk3_g<2><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
        }

        substep = (substep+1) % 3;

        /*
           rk3_kernel<<<gridGPU, blockGPU>>>(a, at, dt,
           substep, grid->icells, grid->ijcells,
           grid->istart, grid->jstart, grid->kstart,
           grid->iend, grid->jend, grid->kend);
         */
    }

    else if (rkorder == 4)
    {
        for (FieldMap::const_iterator it = fields->at.begin(); it!=fields->at.end(); ++it)
        {
            if (substep==0)
                rk4_g<0><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
            else if (substep==1)
                rk4_g<1><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
            else if (substep==2)
                rk4_g<2><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
            else if (substep==3)
                rk4_g<3><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
            else if (substep==4)
                rk4_g<4><<<gridGPU, blockGPU>>>(
                    &fields->ap[it->first]->data_g[offs], &it->second->data_g[offs], dt,
                    grid->icellsp, grid->ijcellsp,
                    grid->istart,  grid->jstart, grid->kstart,
                    grid->iend,    grid->jend,   grid->kend);
        }

        substep = (substep+1) % 5;

        /*
           rk4_kernel<<<gridGPU, blockGPU>>>(a, at, dt,
           substep, grid->icells, grid->ijcells,
           grid->istart, grid->jstart, grid->kstart,
           grid->iend, grid->jend, grid->kend);
         */
    }

    cuda_check_error();
}
#endif
