#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include "master.h"
#include "input.h"
#include "grid.h"
#include "fields.h"
#include "boundary.h"
#include "defines.h"
#include "model.h"
#include "timeloop.h"
#include "fd.h"
#include "constants.h"
#include "tools.h"

using namespace fd::o4;

namespace
{
    __global__ 
    void set_bc_value_g(double* __restrict__ a, double aval, 
                  const int icells, const int icellsp, const int jcells)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*icellsp;
            a[ij] = aval;
        }
    } 

    __global__ 
    void calc_ghost_cells_bot_2nd_g(double* __restrict__ a, double* __restrict__ dzh, Boundary::Boundary_type sw, 
                                    double* __restrict__ abot, double* __restrict__ agradbot,
                                    const int icells, const int icellsp,
                                    const int jcells, const int kstart)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        const int kk  = icellsp*jcells;
        const int ij  = i + j*icellsp;
        const int ijk = i + j*icellsp + kstart*kk;

        if (i < icells && j < jcells)
        {
            if (sw == Boundary::Dirichlet_type)
                a[ijk-kk] = 2.*abot[ij] - a[ijk];

            else if (sw == Boundary::Neumann_type || sw == Boundary::Flux_type)
                a[ijk-kk] = -agradbot[ij]*dzh[kstart] + a[ijk];
        }
    } 

    __global__ 
    void calc_ghost_cells_top_2nd_g(double* __restrict__ a, double* __restrict__ dzh, const Boundary::Boundary_type sw,
                                    double* __restrict__ atop, double* __restrict__ agradtop,
                                    const int icells, const int icellsp,
                                    const int jcells, const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        const int kk  = icellsp*jcells;
        const int ij  = i + j*icellsp;
        const int ijk = i + j*icellsp + (kend-1)*kk;

        if (i < icells && j < jcells)
        {
            if (sw == Boundary::Dirichlet_type)
                a[ijk+kk] = 2.*atop[ij] - a[ijk];

            else if (sw == Boundary::Neumann_type || sw == Boundary::Flux_type)
                a[ijk+kk] = agradtop[ij]*dzh[kend] + a[ijk];
        }
    }

    __global__ 
    void calc_ghost_cells_bot_4th_g(double* __restrict__ a, const Boundary::Boundary_type sw,
                                    double* __restrict__ abot, double* __restrict__ agradbot,
                                    double* __restrict__ z,
                                    const int icells, const int icellsp,
                                    const int jcells, const int kstart)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        const int kk1 = 1*icellsp*jcells;
        const int kk2 = 2*icellsp*jcells;

        const int ij  = i + j*icellsp;
        const int ijk = i + j*icellsp + kstart*kk1;

        if (i < icells && j < jcells)
        {
            if (sw == Boundary::Dirichlet_type)
            {
                a[ijk-kk1] = (8./3.)*abot[ij] - 2.*a[ijk] + (1./3.)*a[ijk+kk1];
                a[ijk-kk2] = 8.*abot[ij] - 9.*a[ijk] + 2.*a[ijk+kk1];
            }

            else if (sw == Boundary::Neumann_type || sw == Boundary::Flux_type)
            {
                a[ijk-kk1] = -(1./24.)*grad4x(z[kstart-2], z[kstart-1], z[kstart], z[kstart+1])*agradbot[ij] + a[ijk    ];
                a[ijk-kk2] = -(1./ 8.)*grad4x(z[kstart-2], z[kstart-1], z[kstart], z[kstart+1])*agradbot[ij] + a[ijk+kk1];
            }
        }
    } 

    __global__ 
    void calc_ghost_cells_top_4th_g(double* __restrict__ a, const Boundary::Boundary_type sw,
                                    double* __restrict__ atop, double* __restrict__ agradtop,
                                    double* __restrict__ z,
                                    const int icells, const int icellsp,
                                    const int jcells, const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        const int kk1 = 1*icellsp*jcells;
        const int kk2 = 2*icellsp*jcells;

        const int ij  = i + j*icellsp;
        const int ijk = i + j*icellsp + (kend-1)*kk1;

        if( i < icells && j < jcells)
        {
            if (sw == Boundary::Dirichlet_type)
            {
                a[ijk+kk1] = (8./3.)*atop[ij] - 2.*a[ijk] + (1./3.)*a[ijk-kk1];
                a[ijk+kk2] = 8.*atop[ij] - 9.*a[ijk] + 2.*a[ijk-kk1];
            }

            else if (sw == Boundary::Neumann_type || sw == Boundary::Flux_type)
            {
                a[ijk+kk1] = (1./24.)*grad4x(z[kend-2], z[kend-1], z[kend], z[kend+1])*agradtop[ij] + a[ijk    ];
                a[ijk+kk2] = (1./ 8.)*grad4x(z[kend-2], z[kend-1], z[kend], z[kend+1])*agradtop[ij] + a[ijk-kk1];
            }
        }
    } 

    __global__ 
    void calc_ghost_cells_bot_w_4th_g(double* __restrict__ w,
                                      const int icells, const int icellsp,
                                      const int jcells, const int kstart)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        const int kk1 = 1*icellsp*jcells;
        const int kk2 = 2*icellsp*jcells;

        const int ijk = i + j*icellsp + kstart*kk1;

        if (i < icells && j < jcells)
        {
            w[ijk-kk1] = -w[ijk+kk1];
            w[ijk-kk2] = -w[ijk+kk2];
        }
    }

    __global__ 
    void calc_ghost_cells_top_w_4th_g(double* __restrict__ w,
                                      const int icells, const int icellsp,
                                      const int jcells, const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x;
        const int j = blockIdx.y*blockDim.y + threadIdx.y;

        const int kk1 = 1*icellsp*jcells;
        const int kk2 = 2*icellsp*jcells;

        const int ijk = i + j*icellsp + kend*kk1;

        if (i < icells && j < jcells)
        {
            w[ijk+kk1] = -w[ijk-kk1];
            w[ijk+kk2] = -w[ijk-kk2];
        }
    }
}

#ifdef USECUDA
void Boundary::exec()
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 grid2dGPU (gridi, gridj);
    dim3 block2dGPU(blocki, blockj);

    const int offs = grid->memoffset;

    // Cyclic boundary conditions, do this before the bottom BC's.
    grid->boundary_cyclic_g(&fields->u->data_g[offs]);
    grid->boundary_cyclic_g(&fields->v->data_g[offs]);
    grid->boundary_cyclic_g(&fields->w->data_g[offs]);

    for (FieldMap::const_iterator it = fields->sp.begin(); it!=fields->sp.end(); ++it)
        grid->boundary_cyclic_g(&it->second->data_g[offs]);

    // Calculate the boundary values.
    update_bcs();

    if(grid->swspatialorder == "2")
    {
        calc_ghost_cells_bot_2nd_g<<<grid2dGPU, block2dGPU>>>(
            &fields->u->data_g[offs], grid->dzh_g, mbcbot, 
            &fields->u->databot_g[offs], &fields->u->datagradbot_g[offs],
            grid->icells, grid->icellsp,
            grid->jcells, grid->kstart);
        cuda_check_error(); 

        calc_ghost_cells_top_2nd_g<<<grid2dGPU, block2dGPU>>>(
            &fields->u->data_g[offs], grid->dzh_g, mbctop, 
            &fields->u->datatop_g[offs], &fields->u->datagradtop_g[offs],
            grid->icells, grid->icellsp,
            grid->jcells, grid->kend);
        cuda_check_error(); 

        calc_ghost_cells_bot_2nd_g<<<grid2dGPU, block2dGPU>>>(
            &fields->v->data_g[offs], grid->dzh_g, mbcbot, 
            &fields->v->databot_g[offs], &fields->v->datagradbot_g[offs],
            grid->icells, grid->icellsp,
            grid->jcells, grid->kstart);
        cuda_check_error(); 

        calc_ghost_cells_top_2nd_g<<<grid2dGPU, block2dGPU>>>(
            &fields->v->data_g[offs], grid->dzh_g, mbctop, 
            &fields->v->datatop_g[offs], &fields->v->datagradtop_g[offs],
            grid->icells, grid->icellsp,
            grid->jcells, grid->kend);
        cuda_check_error(); 

        for (FieldMap::const_iterator it=fields->sp.begin(); it!=fields->sp.end(); ++it)
        {
            calc_ghost_cells_bot_2nd_g<<<grid2dGPU, block2dGPU>>>(
                &it->second->data_g[offs], grid->dzh_g, sbc[it->first]->bcbot, 
                &it->second->databot_g[offs], &it->second->datagradbot_g[offs],
                grid->icells, grid->icellsp,
                grid->jcells, grid->kstart);
            cuda_check_error(); 

            calc_ghost_cells_top_2nd_g<<<grid2dGPU, block2dGPU>>>(
                &it->second->data_g[offs], grid->dzh_g, sbc[it->first]->bctop, 
                &it->second->datatop_g[offs], &it->second->datagradtop_g[offs],
                grid->icells, grid->icellsp,
                grid->jcells, grid->kend);
            cuda_check_error(); 
        }
    }
    else if(grid->swspatialorder == "4")
    {
        calc_ghost_cells_bot_4th_g<<<grid2dGPU, block2dGPU>>>(
            &fields->u->data_g[offs], mbcbot, 
            &fields->u->databot_g[offs], &fields->u->datagradbot_g[offs],
            grid->z_g,
            grid->icells, grid->icellsp,
            grid->jcells, grid->kstart);
        cuda_check_error(); 

        calc_ghost_cells_top_4th_g<<<grid2dGPU, block2dGPU>>>(
            &fields->u->data_g[offs], mbctop, 
            &fields->u->datatop_g[offs], &fields->u->datagradtop_g[offs],
            grid->z_g,
            grid->icells, grid->icellsp,
            grid->jcells, grid->kend);
        cuda_check_error(); 

        calc_ghost_cells_bot_4th_g<<<grid2dGPU, block2dGPU>>>(
            &fields->v->data_g[offs], mbcbot, 
            &fields->v->databot_g[offs], &fields->v->datagradbot_g[offs],
            grid->z_g,
            grid->icells, grid->icellsp,
            grid->jcells, grid->kstart);
        cuda_check_error(); 

        calc_ghost_cells_top_4th_g<<<grid2dGPU, block2dGPU>>>(
            &fields->v->data_g[offs], mbctop, 
            &fields->v->datatop_g[offs], &fields->v->datagradtop_g[offs],
            grid->z_g,
            grid->icells, grid->icellsp,
            grid->jcells, grid->kend);
        cuda_check_error(); 

        calc_ghost_cells_bot_w_4th_g<<<grid2dGPU, block2dGPU>>>(
            &fields->w->data_g[offs],
            grid->icells, grid->icellsp,
            grid->jcells, grid->kstart);
        cuda_check_error(); 

        calc_ghost_cells_top_w_4th_g<<<grid2dGPU, block2dGPU>>>(
            &fields->w->data_g[offs],
            grid->icells, grid->icellsp,
            grid->jcells, grid->kend);
        cuda_check_error(); 

        for (FieldMap::const_iterator it=fields->sp.begin(); it!=fields->sp.end(); ++it)
        {
            calc_ghost_cells_bot_4th_g<<<grid2dGPU, block2dGPU>>>(
                &it->second->data_g[offs], sbc[it->first]->bcbot,
                &it->second->databot_g[offs], &it->second->datagradbot_g[offs],
                grid->z_g,
                grid->icells, grid->icellsp,
                grid->jcells, grid->kstart);
            cuda_check_error(); 

            calc_ghost_cells_top_4th_g<<<grid2dGPU, block2dGPU>>>(
                &it->second->data_g[offs], sbc[it->first]->bctop, 
                &it->second->datatop_g[offs], &it->second->datagradtop_g[offs],
                grid->z_g,
                grid->icells, grid->icellsp,
                grid->jcells, grid->kend);
            cuda_check_error(); 
        }
    }
}
#endif

void Boundary::set_bc_g(double* restrict a, double* restrict agrad, double* restrict aflux, 
                        Boundary_type sw, double aval, double visc, double offset)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 grid2dGPU (gridi, gridj);
    dim3 block2dGPU(blocki, blockj);

    const int offs = grid->memoffset;

    if (sw == Dirichlet_type)
    {
        set_bc_value_g<<<grid2dGPU, block2dGPU>>>(&a[offs], aval-offset,    grid->icells, grid->icellsp, grid->jcells);
        cuda_check_error(); 
    }
    else if (sw == Neumann_type)
    {
        set_bc_value_g<<<grid2dGPU, block2dGPU>>>(&agrad[offs], aval,       grid->icells, grid->icellsp, grid->jcells);
        set_bc_value_g<<<grid2dGPU, block2dGPU>>>(&aflux[offs], -aval*visc, grid->icells, grid->icellsp, grid->jcells);
        cuda_check_error(); 
    }
    else if (sw == Flux_type)
    {
        set_bc_value_g<<<grid2dGPU, block2dGPU>>>(&aflux[offs], aval,       grid->icells, grid->icellsp, grid->jcells);
        set_bc_value_g<<<grid2dGPU, block2dGPU>>>(&agrad[offs], -aval*visc, grid->icells, grid->icellsp, grid->jcells);
        cuda_check_error(); 
    }
}
