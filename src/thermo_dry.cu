#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_dry.h"
#include "defines.h"
#include "constants.h"
#include "master.h"
#include "tools.h"

namespace Thermo_dry_g
{
    __global__ 
    void calc_buoyancy_tend_2nd(double* __restrict__ wt, 
                                double* __restrict__ th, double* __restrict__ threfh, 
                                int istart, int jstart, int kstart,
                                int iend,   int jend,   int kend,
                                int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart; 

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += constants::grav/threfh[k] * (0.5*(th[ijk-kk]+th[ijk]) - threfh[k]);
        }
    }


    __global__ 
    void calc_buoyancy(double* __restrict__ b,
                       double* __restrict__ th, double* __restrict__ thref, 
                       int istart, int jstart,
                       int iend,   int jend,   int kcells,
                       int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z; 

        if (i < iend && j < jend && k < kcells)
        {
            const int ijk = i + j*jj + k*kk;
            b[ijk] = constants::grav/thref[k] * (th[ijk] - thref[k]);
        }
    }

    __global__ 
    void calc_buoyancy_bot(double* __restrict__ b,     double* __restrict__ bbot,
                           double* __restrict__ th,    double* __restrict__ thbot, 
                           double* __restrict__ thref, double* __restrict__ threfh,
                           double grav, int kstart, int icells, int jcells,  
                           int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y; 

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            bbot[ij] = grav/threfh[kstart] * (thbot[ij] - threfh[kstart]);
            b[ijk]   = grav/thref [kstart] * (th[ijk]   - thref [kstart]);
        }
    }

    __global__ 
    void calc_buoyancy_flux_bot(double* __restrict__ bfluxbot, double* __restrict__ thfluxbot,
                                double* __restrict__ threfh, 
                                double grav, int kstart, int icells, int jcells,  
                                int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y; 

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            bfluxbot[ij] = grav/threfh[kstart]*thfluxbot[ij];
        }
    }

    __global__ 
    void calc_N2(double* __restrict__ N2,    double* __restrict__ th,
                 double* __restrict__ thref, double* __restrict__ dzi, 
                 int istart, int jstart, int kstart,
                 int iend,   int jend,   int kend,
                 int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart; 

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            N2[ijk] = constants::grav/thref[k]*0.5*(th[ijk+kk] - th[ijk-kk])*dzi[k];
        }
    }
} // end namespace

void Thermo_dry::prepare_device()
{
    const int nmemsize = grid->kcells*sizeof(double);

    // Allocate fields for Boussinesq and anelastic solver
    cudaSafeCall(hipMalloc(&thref_g,  nmemsize));
    cudaSafeCall(hipMalloc(&threfh_g, nmemsize));
    cudaSafeCall(hipMalloc(&pref_g,   nmemsize));
    cudaSafeCall(hipMalloc(&prefh_g,  nmemsize));
    cudaSafeCall(hipMalloc(&exner_g,  nmemsize));
    cudaSafeCall(hipMalloc(&exnerh_g, nmemsize));

    // Copy fields to device
    cudaSafeCall(hipMemcpy(thref_g,  thref,  nmemsize, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(threfh_g, threfh, nmemsize, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(pref_g,   pref,   nmemsize, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(prefh_g,  prefh,  nmemsize, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(exner_g,  exner,  nmemsize, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(exnerh_g, exnerh, nmemsize, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(thref_g,  thref,  nmemsize, hipMemcpyHostToDevice));
}

void Thermo_dry::clear_device()
{
    cudaSafeCall(hipFree(thref_g ));
    cudaSafeCall(hipFree(threfh_g));
    cudaSafeCall(hipFree(pref_g  ));
    cudaSafeCall(hipFree(prefh_g ));
    cudaSafeCall(hipFree(exner_g ));
    cudaSafeCall(hipFree(exnerh_g));
}

#ifdef USECUDA
void Thermo_dry::exec()
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (grid->swspatialorder== "2")
    {
        Thermo_dry_g::calc_buoyancy_tend_2nd<<<gridGPU, blockGPU>>>(
            &fields->wt->data_g[offs], &fields->sp["th"]->data_g[offs], threfh_g, 
            grid->istart,  grid->jstart, grid->kstart+1,
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);

        cudaCheckError();
    }
    else if (grid->swspatialorder == "4")
    {
        master->print_message("4th order thermo_dry not (yet) implemented\n");  
        throw 1;
    }
}
#endif

#ifdef USECUDA
void Thermo_dry::get_thermo_field(Field3d *fld, Field3d *tmp, std::string name)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kcells);
    dim3 blockGPU(blocki, blockj, 1);

    dim3 gridGPU2 (gridi, gridj, grid->kmax);
    dim3 blockGPU2(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (name == "b")
    {
        Thermo_dry_g::calc_buoyancy<<<gridGPU, blockGPU>>>(
            &fld->data_g[offs], &fields->sp["th"]->data_g[offs], thref_g, 
            grid->istart, grid->jstart, 
            grid->iend, grid->jend, grid->kcells,
            grid->icellsp, grid->ijcellsp);
        cudaCheckError();
    }
    else if (name == "N2")
    {
        Thermo_dry_g::calc_N2<<<gridGPU2, blockGPU2>>>(
            &fld->data_g[offs], &fields->sp["th"]->data_g[offs], thref_g, grid->dzi_g, 
            grid->istart,  grid->jstart, grid->kstart, 
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);
        cudaCheckError();
    }
    else
        throw 1;
}
#endif

#ifdef USECUDA
void Thermo_dry::get_buoyancy_fluxbot(Field3d *bfield)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    Thermo_dry_g::calc_buoyancy_flux_bot<<<gridGPU, blockGPU>>>(
        &bfield->datafluxbot_g[offs], &fields->sp["th"]->datafluxbot_g[offs], 
        threfh_g, constants::grav, grid->kstart, grid->icells, grid->jcells, 
        grid->icellsp, grid->ijcellsp);
    cudaCheckError();
}
#endif

#ifdef USECUDA
void Thermo_dry::get_buoyancy_surf(Field3d *bfield)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    Thermo_dry_g::calc_buoyancy_bot<<<gridGPU, blockGPU>>>(
        &bfield->data_g[offs], &bfield->databot_g[offs], 
        &fields->sp["th"]->data_g[offs], &fields->sp["th"]->databot_g[offs],
        thref_g, threfh_g, constants::grav, grid->kstart, grid->icells, grid->jcells, 
        grid->icellsp, grid->ijcellsp);
    cudaCheckError();

    Thermo_dry_g::calc_buoyancy_flux_bot<<<gridGPU, blockGPU>>>(
        &bfield->datafluxbot_g[offs], &fields->sp["th"]->datafluxbot_g[offs], 
        threfh_g, constants::grav, grid->kstart, grid->icells, grid->jcells, 
        grid->icellsp, grid->ijcellsp);
    cudaCheckError();
}
#endif
