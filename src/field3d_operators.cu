/*
 * MicroHH
 * Copyright (c) 2011-2017 Chiel van Heerwaarden
 * Copyright (c) 2011-2017 Thijs Heus
 * Copyright (c) 2014-2017 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <iostream>
#include <cmath>
#include "master.h"
#include "grid.h"
#include "field3d.h"
#include "fields.h"
#include "defines.h"
#include "tools.h"
#include "field3d_operators.h"

#ifdef USECUDA
template<typename TF>
void Field3d_operators<TF>::calc_mean_profile(TF* const restrict prof, const TF* const restrict fld)
{
    using namespace Tools_g;

    const Grid_data<TF>& gd = grid.get_grid_data();
    const TF scalefac = 1./(gd.itot*gd.jtot);
    auto tmp = fields.get_tmp_g();
    // Reduce 3D field excluding ghost cells and padding to jtot*kcells values
    reduce_interior<TF>(&fld[gd.memoffset], &tmp->fld_g[gd.memoffset], gd.itot, gd.istart, gd.iend, gd.jtot, gd.jstart, gd.jend, gd.kcells, 0, gd.icellsp, gd.ijcellsp, Sum_type);
    // Reduce jtot*kcells to kcells values
    reduce_all<TF>     (&tmp->fld_g[gd.memoffset], prof, gd.jtot*gd.kcells, gd.kcells, gd.jtot, Sum_type, scalefac);
    fields.release_tmp_g(tmp);
}

template<typename TF>
TF Field3d_operators<TF>::calc_mean(const TF* const restrict fld)
{
    using namespace Tools_g;

    const Grid_data<TF>& gd = grid.get_grid_data();
    const TF scalefac = 1./(gd.itot*gd.jtot*gd.ktot);
    TF mean_value;

    auto tmp = fields.get_tmp_g();
    // Reduce 3D field excluding ghost cells and padding to jtot*ktot values
    reduce_interior<TF>(&fld[gd.memoffset], &tmp->fld_g[gd.memoffset], gd.itot, gd.istart, gd.iend, gd.jtot, gd.jstart, gd.jend, gd.ktot, gd.kstart, gd.icellsp, gd.ijcellsp, Sum_type);
    // Reduce jtot*ktot to ktot values
    reduce_all<TF>     (&tmp->fld_g[gd.memoffset], &tmp->fld_g[gd.memoffset+gd.jtot*gd.ktot], gd.jtot*gd.ktot, gd.ktot, gd.jtot, Sum_type, 1.);
    // Reduce ktot values to a single value
    reduce_all<TF>     (&tmp->fld_g[gd.memoffset+gd.jtot*gd.ktot], &tmp->fld_g[gd.memoffset], gd.ktot, 1, gd.ktot, Sum_type, scalefac);
    // Copy back result from GPU
    cuda_safe_call(hipMemcpy(&mean_value, &tmp->fld_g[gd.memoffset], sizeof(TF), hipMemcpyDeviceToHost));
    fields.release_tmp_g(tmp);
    return mean_value;
}

template<typename TF>
TF Field3d_operators<TF>::calc_max(const TF* const restrict fld)
{
    using namespace Tools_g;

    const Grid_data<TF>& gd = grid.get_grid_data();
    const TF scalefac = 1.;
    TF max_value;

    auto tmp = fields.get_tmp_g();
    // Reduce 3D field excluding ghost cells and padding to jtot*ktot values
    reduce_interior<TF>(&fld[gd.memoffset], &tmp->fld_g[gd.memoffset], gd.itot, gd.istart, gd.iend, gd.jtot, gd.jstart, gd.jend, gd.ktot, gd.kstart, gd.icellsp, gd.ijcellsp, Max_type);
    // Reduce jtot*ktot to ktot values
    reduce_all<TF>     (&tmp->fld_g[gd.memoffset], &tmp->fld_g[gd.memoffset+gd.jtot*gd.ktot], gd.jtot*gd.ktot, gd.ktot, gd.jtot, Max_type, scalefac);
    // Reduce ktot values to a single value
    reduce_all<TF>     (&tmp->fld_g[gd.memoffset+gd.jtot*gd.ktot], &tmp->fld_g[gd.memoffset], gd.ktot, 1, gd.ktot, Max_type, scalefac);
    // Copy back result from GPU
    cuda_safe_call(hipMemcpy(&max_value, &tmp->fld_g[gd.memoffset], sizeof(TF), hipMemcpyDeviceToHost));
    fields.release_tmp_g(tmp);

    return max_value;
}
#endif

template class Field3d_operators<double>;
template class Field3d_operators<float>;
