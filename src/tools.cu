#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include "float.h"
#include "tools.h"

#define MAXTHREADS 512 // Maximum number of threads used in reduce algoritms
#define SUM 0
#define MAX 1
#define MIN 2

int nextpow2(unsigned int x)
{
  return (int)pow(2,ceil(log(x)/log(2)));
}

template <int func>
__device__ double reduction(double v1, double v2)
{
  double rval;
  if (func == SUM)
    rval = v1+v2;
  else if (func == MAX)
    rval = fmax(v1,v2);
  else if (func == MIN)
    rval = fmin(v1,v2);
  return rval;
} 

// Reduce one block of data
template <int func, int blockSize> 
__device__ void reduceBlock(volatile double *as, const unsigned int tid)
{
  /* Loop is completely unrolled for performance */
  if (blockSize >= 512) { if (tid < 256) { as[tid] = reduction<func>(as[tid],as[tid + 256]); } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { as[tid] = reduction<func>(as[tid],as[tid + 128]); } __syncthreads(); }
  if (blockSize >= 128) { if (tid <  64) { as[tid] = reduction<func>(as[tid],as[tid +  64]); } __syncthreads(); }

  /* Once we get to the last 32 values (1 thread warp), the __syncthreads() is no longer necessary */
  if (tid < 32)
  {
    if (blockSize >=  64) { if (tid < 32) { as[tid] = reduction<func>(as[tid],as[tid + 32]); }}
    if (blockSize >=  32) { if (tid < 16) { as[tid] = reduction<func>(as[tid],as[tid + 16]); }}
    if (blockSize >=  16) { if (tid <  8) { as[tid] = reduction<func>(as[tid],as[tid +  8]); }}
    if (blockSize >=   8) { if (tid <  4) { as[tid] = reduction<func>(as[tid],as[tid +  4]); }}
    if (blockSize >=   4) { if (tid <  2) { as[tid] = reduction<func>(as[tid],as[tid +  2]); }}
    if (blockSize >=   2) { if (tid <  1) { as[tid] = reduction<func>(as[tid],as[tid +  1]); }}
  }
}

// Reduce field from 3D to 2D, excluding ghost cells and padding
template <int func, int blockSize> 
__global__ void deviceReduceInterior(const double *a, double *a2d, 
                                     unsigned int istart, unsigned int jstart, unsigned int kstart, 
                                     unsigned int iend,   unsigned int jend,   
                                     unsigned int icells, unsigned int ijcells)
{
  extern __shared__ double as[];

  unsigned int tid  = threadIdx.x;
  unsigned int i    = istart + threadIdx.x;
  unsigned int j    = jstart + blockIdx.y;
  unsigned int k    = kstart + blockIdx.z; 
  unsigned int jk   = blockIdx.y+blockIdx.z*(jend-jstart);   // Index in 2D "a2d"
  unsigned int ijk  = i + j*icells + k*ijcells;              // Index in 3D "a"
  unsigned int ijkm = ijkm = iend + j*icells + k*ijcells;    // Max index in X-direction

  double tmpval;
  if (func == MAX)
    tmpval = -DBL_MAX;
  else if (func == MIN)
    tmpval = DBL_MAX;
  else if (func == SUM)
    tmpval = 0;
  
  int ii = ijk;
  while (ii < ijkm)
  {
    tmpval = reduction<func>(tmpval,a[ii]);
    if(ii + blockDim.x < ijkm)
      tmpval = reduction<func>(tmpval,a[ii+blockDim.x]);
    ii += 2*blockDim.x;
  }
  as[tid] = tmpval;

  __syncthreads();

  reduceBlock<func, blockSize>(as, tid);

  if (tid == 0)
    a2d[jk] = as[0];
}

// Reduce array, not accounting from ghost cells or padding 
template <int func, int blockSize> 
__global__ void deviceReduceAll(const double *a, double *aout, unsigned int ncells, unsigned int nvaluesperblock)  
{
  extern __shared__ double as[];

  unsigned int tid  = threadIdx.x;
  unsigned int ii   = nvaluesperblock *  blockIdx.x + threadIdx.x;
  unsigned int iim  = nvaluesperblock * (blockIdx.x+1);

  double tmpval;
  if (func == MAX)
    tmpval = -DBL_MAX;
  else if (func == MIN)
    tmpval = DBL_MAX;
  else if (func == SUM)
    tmpval = 0;
  
  while (ii < iim)
  {
    tmpval = reduction<func>(tmpval,a[ii]);
    if(ii + blockDim.x < iim && ii + blockDim.x < ncells)
      tmpval = reduction<func>(tmpval,a[ii+blockDim.x]);
    ii += 2*blockDim.x;
  }
  as[tid] = tmpval;

  /* Make sure all threads are synchronised before reducing the shared array */
  __syncthreads();

  /* Reduce block in shared memory */
  reduceBlock<func, blockSize>(as, tid);

  /* First value in shared array now holds the reduced value. Write back to global memory */
  if (tid == 0)
    aout[blockIdx.x] = as[0];
}

void reduceInterior(double *a, double *a2d, 
                    int itot, int istart, int iend,
                    int jtot, int jstart, int jend,
                    int ktot, int kstart, int kend,
                    int icells, int ijcells)
{
  int nthreads = min(MAXTHREADS, nextpow2(itot/2));
  dim3 gridGPU (1, jtot, ktot);
  dim3 blockGPU(nthreads, 1, 1);

  // HACK BVS: for now mode hardcoded at MAX for testing......
  switch (nthreads)
  {
    case 512:
      deviceReduceInterior<MAX, 512><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 256:
      deviceReduceInterior<MAX, 256><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 128:
      deviceReduceInterior<MAX, 128><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 64:
      deviceReduceInterior<MAX,  64><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 32:
      deviceReduceInterior<MAX,  32><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 16:
      deviceReduceInterior<MAX,  16><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 8:
      deviceReduceInterior<MAX,   8><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 4:
      deviceReduceInterior<MAX,   4><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 2:
      deviceReduceInterior<MAX,   2><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
    case 1:
      deviceReduceInterior<MAX,   1><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, a2d, istart, jstart, kstart, iend, jend, icells, ijcells); break;
  }
}

void reduceAll(double *a, double *aout, int ncells, int nblocks, int nvaluesperblock)
{
  int nthreads = min(MAXTHREADS, nextpow2(nvaluesperblock/2));
  dim3 gridGPU (nblocks,  1, 1);
  dim3 blockGPU(nthreads, 1, 1);

  // HACK BVS: for now mode hardcoded at MAX for testing......
  switch (nthreads)
  {
    case 512:
      deviceReduceAll<MAX, 512><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 256:
      deviceReduceAll<MAX, 256><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 128:
      deviceReduceAll<MAX, 128><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 64:
      deviceReduceAll<MAX,  64><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 32:
      deviceReduceAll<MAX,  32><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 16:
      deviceReduceAll<MAX,  16><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 8:
      deviceReduceAll<MAX,   8><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 4:
      deviceReduceAll<MAX,   4><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 2:
      deviceReduceAll<MAX,   2><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
    case 1:
      deviceReduceAll<MAX,   1><<<gridGPU, blockGPU, nthreads*sizeof(double)>>>(a, aout, ncells, nvaluesperblock); break;
  }
}

// CUDA error checking. 
void CudaCheckError()
{
  hipError_t err = hipGetLastError();
  if(hipSuccess != err)
    printf("CUDA error : %s\n",hipGetErrorString(err));

  err = hipDeviceSynchronize();
  if(hipSuccess != err)
    printf("CUDA error with sync : %s\n",hipGetErrorString(err));
 
  return;
}
