#include "grid.h"

int cgrid::prepareGPU()
{
  const int kmemsize = kcells*sizeof(double);

  hipMalloc((void**)&dz_g   , kmemsize);
  hipMalloc((void**)&dzh_g  , kmemsize);
  hipMalloc((void**)&dzi_g  , kmemsize);
  hipMalloc((void**)&dzhi_g , kmemsize);
  hipMalloc((void**)&dzi4_g , kmemsize);
  hipMalloc((void**)&dzhi4_g, kmemsize);

  hipMemcpy(dz_g   , dz   , kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(dzh_g  , dzh  , kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(dzi_g  , dzi  , kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(dzhi_g , dzhi , kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(dzi4_g , dzi4 , kmemsize, hipMemcpyHostToDevice);
  hipMemcpy(dzhi4_g, dzhi4, kmemsize, hipMemcpyHostToDevice);

  return 0;
}
