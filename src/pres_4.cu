#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "master.h"
#include "grid.h"
#include "fields.h"
#include "pres.h"
#include "pres_4.h"
#include "fd.h"
#include "tools.h"

using namespace fd::o4;

namespace Pres_4_g
{
  __global__ void gcwt(double * const __restrict__ wt,
                       const int jj, const int kk,
                       const int istart, const int jstart, const int kstart,
                       const int iend, const int jend, const int kend)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;

    if(i < iend && j < jend)
    {
      int ijk = i + j*jj + kstart*kk;
      wt[ijk-kk] = -wt[ijk+kk];

      ijk = i + j*jj + kend*kk;
      wt[ijk+kk] = -wt[ijk-kk];
    }
  }

  __global__ void presin(double * const __restrict__ p,
                         const double * const __restrict__ u , const double * const __restrict__ v , const double * const __restrict__ w ,
                         const double * const __restrict__ ut, const double * const __restrict__ vt, const double * const __restrict__ wt,
                         const double * const __restrict__ dzi4,
                         const double dxi, const double dyi, const double dti,
                         const int jj, const int kk,
                         const int jjp, const int kkp,
                         const int imax, const int jmax, const int kmax,
                         const int igc, const int jgc, const int kgc)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;
    const int k = blockIdx.z;

    const int ii1 = 1;
    const int ii2 = 2;
    const int jj1 = 1*jj;
    const int jj2 = 2*jj;
    const int kk1 = 1*kk;
    const int kk2 = 2*kk;

    if(i < imax && j < jmax && k < kmax)
    {
      const int ijkp = i + j*jjp + k*kkp;
      const int ijk  = i+igc + (j+jgc)*jj + (k+kgc)*kk;

      p[ijkp] = (cg0*(ut[ijk-ii1] + u[ijk-ii1]*dti) + cg1*(ut[ijk] + u[ijk]*dti) + cg2*(ut[ijk+ii1] + u[ijk+ii1]*dti) + cg3*(ut[ijk+ii2] + u[ijk+ii2]*dti)) * cgi*dxi
              + (cg0*(vt[ijk-jj1] + v[ijk-jj1]*dti) + cg1*(vt[ijk] + v[ijk]*dti) + cg2*(vt[ijk+jj1] + v[ijk+jj1]*dti) + cg3*(vt[ijk+jj2] + v[ijk+jj2]*dti)) * cgi*dyi
              + (cg0*(wt[ijk-kk1] + w[ijk-kk1]*dti) + cg1*(wt[ijk] + w[ijk]*dti) + cg2*(wt[ijk+kk1] + w[ijk+kk1]*dti) + cg3*(wt[ijk+kk2] + w[ijk+kk2]*dti)) * dzi4[k+kgc];
    }
  }

  __global__ void solvein(const double * const __restrict__ p,
                          const double * const __restrict__ m1, const double * const __restrict__ m2, const double * const __restrict__ m3, const double * const __restrict__ m4,
                          const double * const __restrict__ m5, const double * const __restrict__ m6, const double * const __restrict__ m7,
                          double * const __restrict__ m1temp, double * const __restrict__ m2temp, double * __restrict__ const m3temp, double * const __restrict__ m4temp,
                          double * const __restrict__ m5temp, double * const __restrict__ m6temp, double * __restrict__ const m7temp, double * const __restrict__ ptemp,
                          const double * const __restrict__ bmati, const double * const __restrict__ bmatj,
                          const int mpicoordx, const int mpicoordy,
                          const int iblock, const int jblock,
                          const int kmax,
                          const int n, const int jslice)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;

    const int jj = iblock;
    const int kk = iblock*jblock;

    const int kki1 = 1*iblock*jslice;
    const int kki2 = 2*iblock*jslice;
    const int kki3 = 3*iblock*jslice;

    int ik,ijk,iindex,jindex;

    if(i < iblock && j < jslice)
    {
      // Swap the mpicoords, because domain is turned 90 degrees to avoid two mpi transposes.
      iindex = mpicoordy*iblock + i;
      jindex = mpicoordx*jblock + n*jslice + j;

      // Set a zero gradient bc at the bottom.
      ik = i + j*jj;
      m1temp[ik] =  0.;
      m2temp[ik] =  0.;
      m3temp[ik] =  0.;
      m4temp[ik] =  1.;
      m5temp[ik] =  0.;
      m6temp[ik] =  0.;
      m7temp[ik] = -1.;
      ptemp [ik] =  0.;

      m1temp[ik+kki1] =  0.;
      m2temp[ik+kki1] =  0.;
      m3temp[ik+kki1] =  0.;
      m4temp[ik+kki1] =  1.;
      m5temp[ik+kki1] = -1.;
      m6temp[ik+kki1] =  0.;
      m7temp[ik+kki1] =  0.;
      ptemp [ik+kki1] =  0.;

      for(int k=0; k<kmax; ++k)
      {
        // Swap the mpicoords, because domain is turned 90 degrees to avoid two mpi transposes.
        ijk = i + (j + n*jslice)*jj + k*kk;
        ik  = i + j*jj + k*kki1;
        m1temp[ik+kki2] = m1[k];
        m2temp[ik+kki2] = m2[k];
        m3temp[ik+kki2] = m3[k];
        m4temp[ik+kki2] = m4[k] + bmati[iindex] + bmatj[jindex];
        m5temp[ik+kki2] = m5[k];
        m6temp[ik+kki2] = m6[k];
        m7temp[ik+kki2] = m7[k];
        ptemp [ik+kki2] = p[ijk];
      }

      // Set the top boundary.
      ik = i + j*jj + kmax*kki1;
      if(iindex == 0 && jindex == 0)
      {
        m1temp[ik+kki2] =    0.;
        m2temp[ik+kki2] = -1/3.;
        m3temp[ik+kki2] =    2.;
        m4temp[ik+kki2] =    1.;

        m1temp[ik+kki3] =   -2.;
        m2temp[ik+kki3] =    9.;
        m3temp[ik+kki3] =    0.;
        m4temp[ik+kki3] =    1.;
      }

      // Set dp/dz at top to zero.
      else
      {
        m1temp[ik+kki2] =  0.;
        m2temp[ik+kki2] =  0.;
        m3temp[ik+kki2] = -1.;
        m4temp[ik+kki2] =  1.;

        m1temp[ik+kki3] = -1.;
        m2temp[ik+kki3] =  0.;
        m3temp[ik+kki3] =  0.;
        m4temp[ik+kki3] =  1.;
      }

      // Set the top boundary.
      m5temp[ik+kki2] = 0.;
      m6temp[ik+kki2] = 0.;
      m7temp[ik+kki2] = 0.;
      ptemp [ik+kki2] = 0.;

      m5temp[ik+kki3] = 0.;
      m6temp[ik+kki3] = 0.;
      m7temp[ik+kki3] = 0.;
      ptemp [ik+kki3] = 0.;
    }
  }

  __global__ void solveputback(double * const __restrict__ p,
                               const double * const __restrict__ ptemp,
                               const int iblock, const int jblock,
                               const int kmax,
                               const int n, const int jslice)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;

    const int jj = iblock;
    const int kk = iblock*jblock;

    const int kki1 = 1*iblock*jslice;
    const int kki2 = 2*iblock*jslice;

    if(i < iblock && j < jslice)
    {
      // Put back the solution.
      for(int k=0; k<kmax; ++k)
      {
        const int ik  = i + j*jj + k*kki1;
        const int ijk = i + (j + n*jslice)*jj + k*kk;
        p[ijk] = ptemp[ik+kki2];
      }
    }
  }

  __global__ void hdma(double * const __restrict__ m1, double * const __restrict__ m2, double * const __restrict__ m3, double * const __restrict__ m4,
                       double * const __restrict__ m5, double * const __restrict__ m6, double * const __restrict__ m7, double * const __restrict__ p,
                       const int iblock, const int kmax, const int jslice)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;

    const int jj = iblock;

    const int kk1 = 1*iblock*jslice;
    const int kk2 = 2*iblock*jslice;
    const int kk3 = 3*iblock*jslice;

    int k,ik;

    if(i < iblock && j < jslice)
    {
      // Use LU factorization.
      k = 0;
      ik = i + j*jj;
      m1[ik] = 1.;
      m2[ik] = 1.;
      m3[ik] = 1.            / m4[ik];
      m4[ik] = 1.;
      m5[ik] = m5[ik]*m3[ik];
      m6[ik] = m6[ik]*m3[ik];
      m7[ik] = m7[ik]*m3[ik];

      k = 1;
      ik = i + j*jj + k*kk1;
      m1[ik] = 1.;
      m2[ik] = 1.;
      m3[ik] = m3[ik]                     / m4[ik-kk1];
      m4[ik] = m4[ik] - m3[ik]*m5[ik-kk1];
      m5[ik] = m5[ik] - m3[ik]*m6[ik-kk1];
      m6[ik] = m6[ik] - m3[ik]*m7[ik-kk1];

      k = 2;
      ik = i + j*jj + k*kk1;
      m1[ik] = 1.;
      m2[ik] =   m2[ik]                                           / m4[ik-kk2];
      m3[ik] = ( m3[ik]                     - m2[ik]*m5[ik-kk2] ) / m4[ik-kk1];
      m4[ik] =   m4[ik] - m3[ik]*m5[ik-kk1] - m2[ik]*m6[ik-kk2];
      m5[ik] =   m5[ik] - m3[ik]*m6[ik-kk1] - m2[ik]*m7[ik-kk2];
      m6[ik] =   m6[ik] - m3[ik]*m7[ik-kk1];

      for(k=3; k<kmax+2; ++k)
      {
        ik = i + j*jj + k*kk1;
        m1[ik] = ( m1[ik]                                                            ) / m4[ik-kk3];
        m2[ik] = ( m2[ik]                                         - m1[ik]*m5[ik-kk3]) / m4[ik-kk2];
        m3[ik] = ( m3[ik]                     - m2[ik]*m5[ik-kk2] - m1[ik]*m6[ik-kk3]) / m4[ik-kk1];
        m4[ik] =   m4[ik] - m3[ik]*m5[ik-kk1] - m2[ik]*m6[ik-kk2] - m1[ik]*m7[ik-kk3];
        m5[ik] =   m5[ik] - m3[ik]*m6[ik-kk1] - m2[ik]*m7[ik-kk2];
        m6[ik] =   m6[ik] - m3[ik]*m7[ik-kk1];
      }

      k = kmax+1;
      ik = i + j*jj + k*kk1;
      m7[ik] = 1.;

      k = kmax+2;
      ik = i + j*jj + k*kk1;
      m1[ik] = ( m1[ik]                                                            ) / m4[ik-kk3];
      m2[ik] = ( m2[ik]                                         - m1[ik]*m5[ik-kk3]) / m4[ik-kk2];
      m3[ik] = ( m3[ik]                     - m2[ik]*m5[ik-kk2] - m1[ik]*m6[ik-kk3]) / m4[ik-kk1];
      m4[ik] =   m4[ik] - m3[ik]*m5[ik-kk1] - m2[ik]*m6[ik-kk2] - m1[ik]*m7[ik-kk3];
      m5[ik] =   m5[ik] - m3[ik]*m6[ik-kk1] - m2[ik]*m7[ik-kk2];
      m6[ik] = 1.;
      m7[ik] = 1.;

      k = kmax+3;
      ik = i + j*jj + k*kk1;
      m1[ik] = ( m1[ik]                                                            ) / m4[ik-kk3];
      m2[ik] = ( m2[ik]                                         - m1[ik]*m5[ik-kk3]) / m4[ik-kk2];
      m3[ik] = ( m3[ik]                     - m2[ik]*m5[ik-kk2] - m1[ik]*m6[ik-kk3]) / m4[ik-kk1];
      m4[ik] =   m4[ik] - m3[ik]*m5[ik-kk1] - m2[ik]*m6[ik-kk2] - m1[ik]*m7[ik-kk3];
      m5[ik] = 1.;
      m6[ik] = 1.;
      m7[ik] = 1.;

      // Do the backward substitution.
      // First, solve Ly = p, forward.
      ik = i + j*jj;
      p[ik    ] =             p[ik    ]*m3[ik    ];
      p[ik+kk1] = p[ik+kk1] - p[ik    ]*m3[ik+kk1];
      p[ik+kk2] = p[ik+kk2] - p[ik+kk1]*m3[ik+kk2] - p[ik]*m2[ik+kk2];

      for(k=3; k<kmax+4; ++k)
      {
        ik = i + j*jj + k*kk1;
        p[ik] = p[ik] - p[ik-kk1]*m3[ik] - p[ik-kk2]*m2[ik] - p[ik-kk3]*m1[ik];
      }

      // Second, solve Ux=y, backward.
      k = kmax+3;
      ik = i + j*jj + k*kk1;
      p[ik    ] =   p[ik    ]                                             / m4[ik    ];
      p[ik-kk1] = ( p[ik-kk1] - p[ik    ]*m5[ik-kk1] )                    / m4[ik-kk1];
      p[ik-kk2] = ( p[ik-kk2] - p[ik-kk1]*m5[ik-kk2] - p[ik]*m6[ik-kk2] ) / m4[ik-kk2];

      for(k=kmax; k>=0; --k)
      {
        ik = i + j*jj + k*kk1;
        p[ik] = ( p[ik] - p[ik+kk1]*m5[ik] - p[ik+kk2]*m6[ik] - p[ik+kk3]*m7[ik] ) / m4[ik];
      }
    }
  }

  __global__ void solveout(double * __restrict__ p, double * __restrict__ work3d,
                           const int jj, const int kk,
                           const int jjp, const int kkp,
                           const int istart, const int jstart, const int kstart,
                           const int imax, const int jmax, const int kmax)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const int j = blockIdx.y*blockDim.y + threadIdx.y;
    const int k = blockIdx.z;

    const int kkp1 = 1*kkp;
    const int kkp2 = 2*kkp;

    if(i < imax && j < jmax && k < kmax)
    {
      const int ijk  = i + j*jj + k*kk;
      const int ijkp = i+istart + (j+jstart)*jjp + (k+kstart)*kkp;

      p[ijkp] = work3d[ijk];

      // set the BC
      if(k == 0)
      {
        p[ijkp-kkp1] = p[ijkp     ];
        p[ijkp-kkp2] = p[ijkp+kkp1];
      }
      else if(k == kmax-1)
      {
        p[ijkp+kkp1] = p[ijkp     ];
        p[ijkp+kkp2] = p[ijkp-kkp1];
      }
    }
  }

  __global__ void presout(double * const __restrict__ ut, double * const __restrict__ vt, double * const __restrict__ wt,
                          const double * const __restrict__ p,
                          const double * const __restrict__ dzhi4,
                          const double dxi, const double dyi,
                          const int jj, const int kk,
                          const int istart, const int jstart, const int kstart,
                          const int iend, const int jend, const int kend)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;

    const int ii1 = 1;
    const int ii2 = 2;
    const int jj1 = 1*jj;
    const int jj2 = 2*jj;
    const int kk1 = 1*kk;
    const int kk2 = 2*kk;

    if(i < iend && j < jend && k == kstart)
    {
      const int ijk = i + j*jj + k*kk;
      ut[ijk] -= (cg0*p[ijk-ii2] + cg1*p[ijk-ii1] + cg2*p[ijk] + cg3*p[ijk+ii1]) * cgi*dxi;
      vt[ijk] -= (cg0*p[ijk-jj2] + cg1*p[ijk-jj1] + cg2*p[ijk] + cg3*p[ijk+jj1]) * cgi*dyi;
    }
    else if(i < iend && j < jend && k < kend)
    {
      const int ijk = i + j*jj1 + k*kk1;
      ut[ijk] -= (cg0*p[ijk-ii2] + cg1*p[ijk-ii1] + cg2*p[ijk] + cg3*p[ijk+ii1]) * cgi*dxi;
      vt[ijk] -= (cg0*p[ijk-jj2] + cg1*p[ijk-jj1] + cg2*p[ijk] + cg3*p[ijk+jj1]) * cgi*dyi;
      wt[ijk] -= (cg0*p[ijk-kk2] + cg1*p[ijk-kk1] + cg2*p[ijk] + cg3*p[ijk+kk1]) * dzhi4[k];
    }
  }

  __global__ void calcdivergence(double * __restrict__ div,
                                 double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ dzi4,
                                 double dxi, double dyi,
                                 int jj, int kk,
                                 int istart, int jstart, int kstart,
                                 int iend, int jend, int kend)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;

    const int ii1 = 1;
    const int ii2 = 2;
    const int jj1 = 1*jj;
    const int jj2 = 2*jj;
    const int kk1 = 1*kk;
    const int kk2 = 2*kk;

    if(i < iend && j < jend && k < kend)
    {
      const int ijk = i + j*jj + k*kk;
      div[ijk] = (cg0*u[ijk-ii1] + cg1*u[ijk] + cg2*u[ijk+ii1] + cg3*u[ijk+ii2]) * cgi*dxi
               + (cg0*v[ijk-jj1] + cg1*v[ijk] + cg2*v[ijk+jj1] + cg3*v[ijk+jj2]) * cgi*dyi
               + (cg0*w[ijk-kk1] + cg1*w[ijk] + cg2*w[ijk+kk1] + cg3*w[ijk+kk2]) * dzi4[k];
    }
  }
} // End namespace.

#ifdef USECUDA
void Pres_4::exec(double dt)
{
  // 1. Create the input for the pressure solver.
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  dim3 grid2dGPU (gridi, gridj);
  dim3 block2dGPU(blocki, blockj);

  dim3 grid1dGPU (gridi);
  dim3 block1dGPU(blocki);

  const int offs = grid->memoffset;

  // calculate the cyclic BCs first
  grid->boundaryCyclic_g(&fields->ut->data_g[offs]);
  grid->boundaryCyclic_g(&fields->vt->data_g[offs]);
  grid->boundaryCyclic_g(&fields->wt->data_g[offs]);

  Pres_4_g::gcwt<<<grid2dGPU, block2dGPU>>>(&fields->wt->data_g[offs],
                                           grid->icellsp, grid->ijcellsp,
                                           grid->istart, grid->jstart, grid->kstart,
                                           grid->iend, grid->jend, grid->kend);
  cudaCheckError();

  Pres_4_g::presin<<<gridGPU, blockGPU>>>(fields->sd["p"]->data_g,
                                         &fields->u ->data_g[offs], &fields->v ->data_g[offs], &fields->w ->data_g[offs],
                                         &fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                         grid->dzi4_g,
                                         1./grid->dx, 1./grid->dy, 1./dt,
                                         grid->icellsp, grid->ijcellsp,
                                         grid->imax, grid->imax*grid->jmax,
                                         grid->imax, grid->jmax, grid->kmax,
                                         grid->igc, grid->jgc, grid->kgc);
  cudaCheckError();

  fftForward(fields->sd["p"]->data_g, fields->atmp["tmp1"]->data_g, fields->atmp["tmp2"]->data_g);

  double *tmp1_g = fields->atmp["tmp1"]->data_g;
  double *tmp2_g = fields->atmp["tmp2"]->data_g;

  // Set jslice to a higher value
  const int jslice = std::max(grid->jblock/4, 1);

  const int blockis = 128;
  const int blockjs = 1;
  const int gridis  = grid->iblock/blockis + (grid->iblock%blockis > 0);
  const int gridjs  =       jslice/blockjs + (      jslice%blockjs > 0);

  dim3 grid2dsGPU (gridis , gridjs );
  dim3 block2dsGPU(blockis, blockjs);

  const int ns = grid->iblock*jslice*(grid->kmax+4);
  const int nj = grid->jblock/jslice;

  for(int n=0; n<nj; ++n)
  {
    // Prepare the fields that go into the matrix solver
    Pres_4_g::solvein<<<grid2dsGPU,block2dsGPU>>>(fields->sd["p"]->data_g,
                                                 m1_g, m2_g, m3_g, m4_g,
                                                 m5_g, m6_g, m7_g,
                                                 &tmp1_g[0*ns], &tmp1_g[1*ns], &tmp1_g[2*ns], &tmp1_g[3*ns],
                                                 &tmp2_g[0*ns], &tmp2_g[1*ns], &tmp2_g[2*ns], &tmp2_g[3*ns],
                                                 bmati_g, bmatj_g,
                                                 master->mpicoordx, master->mpicoordy,
                                                 grid->iblock, grid->jblock,
                                                 grid->kmax,
                                                 n, jslice);
    cudaCheckError();

    // Solve the sevenbanded matrix
    Pres_4_g::hdma<<<grid2dsGPU,block2dsGPU>>>(&tmp1_g[0*ns], &tmp1_g[1*ns], &tmp1_g[2*ns], &tmp1_g[3*ns],
                                              &tmp2_g[0*ns], &tmp2_g[1*ns], &tmp2_g[2*ns], &tmp2_g[3*ns],
                                              grid->iblock, grid->kmax, jslice);
    cudaCheckError();

    // Put the solution back into the pressure field
    Pres_4_g::solveputback<<<grid2dsGPU,block2dsGPU>>>(fields->sd["p"]->data_g,
                                                      &tmp2_g[3*ns],
                                                      grid->iblock, grid->jblock,
                                                      grid->kmax,
                                                      n, jslice);
    cudaCheckError();
  }

  fftBackward(fields->sd["p"]->data_g, fields->atmp["tmp1"]->data_g, fields->atmp["tmp2"]->data_g);

  cudaSafeCall(hipMemcpy(fields->atmp["tmp1"]->data_g, fields->sd["p"]->data_g, grid->ncellsp*sizeof(double), hipMemcpyDeviceToDevice));

  Pres_4_g::solveout<<<gridGPU, blockGPU>>>(&fields->sd["p"]->data_g[offs], fields->atmp["tmp1"]->data_g,
                                           grid->imax, grid->imax*grid->jmax,
                                           grid->icellsp, grid->ijcellsp,
                                           grid->istart, grid->jstart, grid->kstart,
                                           grid->imax, grid->jmax, grid->kmax);
  cudaCheckError();

  grid->boundaryCyclic_g(&fields->sd["p"]->data_g[offs]);

  // 3. Get the pressure tendencies from the pressure field.
  Pres_4_g::presout<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                          &fields->sd["p"]->data_g[offs],
                                          grid->dzhi4_g,
                                          1./grid->dx, 1./grid->dy,
                                          grid->icellsp, grid->ijcellsp,
                                          grid->istart, grid->jstart, grid->kstart,
                                          grid->iend, grid->jend, grid->kend);
  cudaCheckError();
}

double Pres_4::checkDivergence()
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  Pres_4_g::calcdivergence<<<gridGPU, blockGPU>>>(&fields->atmp["tmp1"]->data_g[offs],
                                                 &fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs],
                                                 grid->dzi4_g,
                                                 grid->dxi, grid->dyi,
                                                 grid->icellsp, grid->ijcellsp,
                                                 grid->istart, grid->jstart, grid->kstart,
                                                 grid->iend, grid->jend, grid->kend);
  cudaCheckError();

  double divmax = grid->getMax_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g);
  grid->getMax(&divmax);

  return divmax;
}

void Pres_4::prepare_device()
{
  const int kmemsize = grid->kmax*sizeof(double);
  const int imemsize = grid->itot*sizeof(double);
  const int jmemsize = grid->jtot*sizeof(double);

  cudaSafeCall(hipMalloc((void**)&bmati_g, imemsize));
  cudaSafeCall(hipMalloc((void**)&bmatj_g, jmemsize));

  cudaSafeCall(hipMalloc((void**)&m1_g, kmemsize));
  cudaSafeCall(hipMalloc((void**)&m2_g, kmemsize));
  cudaSafeCall(hipMalloc((void**)&m3_g, kmemsize));
  cudaSafeCall(hipMalloc((void**)&m4_g, kmemsize));
  cudaSafeCall(hipMalloc((void**)&m5_g, kmemsize));
  cudaSafeCall(hipMalloc((void**)&m6_g, kmemsize));
  cudaSafeCall(hipMalloc((void**)&m7_g, kmemsize));

  cudaSafeCall(hipMemcpy(bmati_g, bmati, imemsize, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(bmatj_g, bmatj, jmemsize, hipMemcpyHostToDevice));

  cudaSafeCall(hipMemcpy(m1_g, m1, kmemsize, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(m2_g, m2, kmemsize, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(m3_g, m3, kmemsize, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(m4_g, m4, kmemsize, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(m5_g, m5, kmemsize, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(m6_g, m6, kmemsize, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(m7_g, m7, kmemsize, hipMemcpyHostToDevice));

  makeCufftPlan();
}

void Pres_4::clear_device()
{
  cudaSafeCall(hipFree(bmati_g));
  cudaSafeCall(hipFree(bmatj_g));

  cudaSafeCall(hipFree(m1_g));
  cudaSafeCall(hipFree(m2_g));
  cudaSafeCall(hipFree(m3_g));
  cudaSafeCall(hipFree(m4_g));
  cudaSafeCall(hipFree(m5_g));
  cudaSafeCall(hipFree(m6_g));
  cudaSafeCall(hipFree(m7_g));
}
#endif
