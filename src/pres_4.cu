#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "grid.h"
#include "fields.h"
#include "pres_4.h"
#include "fd.h"

using namespace fd::o4;

__global__ void pres_4_calcdivergence(double * __restrict__ div,
                                      double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                      double * __restrict__ dzi4,
                                      double dxi, double dyi,
                                      int jj, int kk,
                                      int istart, int jstart, int kstart,
                                      int iend, int jend, int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  const int ii1 = 1;
  const int ii2 = 2;
  const int jj1 = 1*jj;
  const int jj2 = 2*jj;
  const int kk1 = 1*kk;
  const int kk2 = 2*kk;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;
    div[ijk] = (cg0*u[ijk-ii1] + cg1*u[ijk] + cg2*u[ijk+ii1] + cg3*u[ijk+ii2]) * cgi*dxi
             + (cg0*v[ijk-jj1] + cg1*v[ijk] + cg2*v[ijk+jj1] + cg3*v[ijk+jj2]) * cgi*dyi
             + (cg0*w[ijk-kk1] + cg1*w[ijk] + cg2*w[ijk+kk1] + cg3*w[ijk+kk2]) * dzi4[k];
  }
}

#ifdef USECUDA
void cpres_4::exec(double dt)
{
  // 1. Create the input for the pressure solver.
  pres_in(fields->sd["p"]->data,
          fields->u ->data, fields->v ->data, fields->w ->data,
          fields->ut->data, fields->vt->data, fields->wt->data, 
          grid->dzi4, dt);

  // 2. Solve the Poisson equation using FFTs and a heptadiagonal solver
  // Take slices out of a temporary field to save memory. The temp arrays
  // are always big enough, this cannot fail.
  double *tmp2 = fields->sd["tmp2"]->data;
  double *tmp3 = fields->sd["tmp3"]->data;
  const int ns = grid->iblock*(grid->kmax+4);
  pres_solve(fields->sd["p"]->data, fields->sd["tmp1"]->data, grid->dz,
             m1, m2, m3, m4,
             m5, m6, m7,
             &tmp2[0*ns], &tmp2[1*ns], &tmp2[2*ns], &tmp2[3*ns], 
             &tmp3[0*ns], &tmp3[1*ns], &tmp3[2*ns], &tmp3[3*ns], 
             bmati, bmatj);

  // 3. Get the pressure tendencies from the pressure field.
  pres_out(fields->ut->data, fields->vt->data, fields->wt->data, 
           fields->sd["p"]->data, grid->dzhi4);
}

double cpres_4::check()
{
  fields->forwardGPU();

  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  pres_4_calcdivergence<<<gridGPU, blockGPU>>>(&fields->a["tmp1"]->data_g[offs],
                                               &fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs],
                                               grid->dzi4_g,
                                               grid->dxi, grid->dyi,
                                               grid->icellsp, grid->ijcellsp,
                                               grid->istart,  grid->jstart, grid->kstart,
                                               grid->iend,    grid->jend,   grid->kend);

  double divmax = grid->getmax_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp2"]->data_g);
  grid->getmax(&divmax);

  fields->backwardGPU();

  return divmax;
}
#endif
