#include <algorithm>
#include "grid.h"
#include "tools.h"
#include "timedep.h"
#include "aerosol.h"

template<typename TF>
void Aerosol<TF>::prepare_device()
{
    if (sw_aerosol && sw_timedep)
    {
        auto &gd = grid.get_grid_data();
        const int nmemsize = gd.kcells * sizeof(TF);

        cuda_safe_call(hipMalloc(&aermr01_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr02_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr03_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr04_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr05_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr06_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr07_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr08_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr09_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr10_g, nmemsize));
        cuda_safe_call(hipMalloc(&aermr11_g, nmemsize));

        cuda_safe_call(hipMemcpy(aermr01_g, aermr01.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr02_g, aermr02.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr03_g, aermr03.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr04_g, aermr04.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr05_g, aermr05.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr06_g, aermr06.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr07_g, aermr07.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr08_g, aermr08.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr09_g, aermr09.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr10_g, aermr10.data(), nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(aermr11_g, aermr11.data(), nmemsize, hipMemcpyHostToDevice));
    }
}

template<typename TF>
void Aerosol<TF>::clear_device()
{
    if (sw_aerosol && sw_timedep)
    {
        cuda_safe_call(hipFree(aermr01_g));
        cuda_safe_call(hipFree(aermr02_g));
        cuda_safe_call(hipFree(aermr03_g));
        cuda_safe_call(hipFree(aermr04_g));
        cuda_safe_call(hipFree(aermr05_g));
        cuda_safe_call(hipFree(aermr06_g));
        cuda_safe_call(hipFree(aermr07_g));
        cuda_safe_call(hipFree(aermr08_g));
        cuda_safe_call(hipFree(aermr09_g));
        cuda_safe_call(hipFree(aermr10_g));
        cuda_safe_call(hipFree(aermr11_g));

        tdep_aermr01->clear_device();
        tdep_aermr02->clear_device();
        tdep_aermr03->clear_device();
        tdep_aermr04->clear_device();
        tdep_aermr05->clear_device();
        tdep_aermr06->clear_device();
        tdep_aermr07->clear_device();
        tdep_aermr08->clear_device();
        tdep_aermr09->clear_device();
        tdep_aermr10->clear_device();
        tdep_aermr11->clear_device();
    }
}

#ifdef USECUDA
template <typename TF>
void Aerosol<TF>::update_time_dependent(Timeloop<TF>& timeloop)
{
    if (!sw_aerosol)
        return;

    if (sw_timedep)
    {
        auto& gd = grid.get_grid_data();
        const int nmemsize  = gd.kcells*sizeof(TF);

        tdep_aermr01 ->update_time_dependent_prof_g(aermr01_g, timeloop);
        tdep_aermr02 ->update_time_dependent_prof_g(aermr02_g, timeloop);
        tdep_aermr03 ->update_time_dependent_prof_g(aermr03_g, timeloop);
        tdep_aermr04 ->update_time_dependent_prof_g(aermr04_g, timeloop);
        tdep_aermr05 ->update_time_dependent_prof_g(aermr05_g, timeloop);
        tdep_aermr06 ->update_time_dependent_prof_g(aermr06_g, timeloop);
        tdep_aermr07 ->update_time_dependent_prof_g(aermr07_g, timeloop);
        tdep_aermr08 ->update_time_dependent_prof_g(aermr08_g, timeloop);
        tdep_aermr09 ->update_time_dependent_prof_g(aermr09_g, timeloop);
        tdep_aermr10 ->update_time_dependent_prof_g(aermr10_g, timeloop);
        tdep_aermr11 ->update_time_dependent_prof_g(aermr11_g, timeloop);

        cuda_safe_call(hipMemcpy(aermr01.data(), aermr01_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr02.data(), aermr02_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr03.data(), aermr03_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr04.data(), aermr04_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr05.data(), aermr05_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr06.data(), aermr06_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr07.data(), aermr07_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr08.data(), aermr08_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr09.data(), aermr09_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr10.data(), aermr10_g, nmemsize, hipMemcpyDeviceToHost));
        cuda_safe_call(hipMemcpy(aermr11.data(), aermr11_g, nmemsize, hipMemcpyDeviceToHost));
    }
}
#endif

template class Aerosol<double>;
template class Aerosol<float>;