#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2017 Chiel van Heerwaarden
 * Copyright (c) 2011-2017 Thijs Heus
 * Copyright (c) 2014-2017 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>
#include "master.h"
#include "force.h"
#include "grid.h"
#include "fields.h"
#include "field3d_operators.h"
#include "finite_difference.h"
#include "constants.h"
#include "tools.h"
#include "boundary.h"
#include "timeloop.h"

using namespace Finite_difference::O2;

namespace
{
    template<typename TF> __global__
    void flux_step_1_g(TF* const __restrict__ aSum, const TF* const __restrict__ a,
                       const TF* const __restrict__ dz,
                       const int jj, const int kk,
                       const int istart, const int jstart, const int kstart,
                       const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            aSum [ijk] = a[ijk]*dz[k];
        }
    }

    template<typename TF> __global__
    void flux_step_2_g(TF* const __restrict__ ut,
                       const TF fbody,
                       const int jj, const int kk,
                       const int istart, const int jstart, const int kstart,
                       const int iend,   const int jend,   const int kend)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fbody;
        }
    }
/*
    __global__
    void coriolis_2nd_g(double* const __restrict__ ut, double* const __restrict__ vt,
                        double* const __restrict__ u,  double* const __restrict__ v,
                        double* const __restrict__ ug, double* const __restrict__ vg,
                        const double fc, const double ugrid, const double vgrid,
                        const int jj, const int kk,
                        const int istart, const int jstart, const int kstart,
                        const int iend,   const int jend,   const int kend)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k  = blockIdx.z + kstart;
        const int ii = 1;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fc * (0.25*(v[ijk-ii] + v[ijk] + v[ijk-ii+jj] + v[ijk+jj]) + vgrid - vg[k]);
            vt[ijk] -= fc * (0.25*(u[ijk-jj] + u[ijk] + u[ijk+ii-jj] + u[ijk+ii]) + ugrid - ug[k]);
        }
    }

    __global__
    void coriolis_4th_g(double* const __restrict__ ut, double* const __restrict__ vt,
                        double* const __restrict__ u,  double* const __restrict__ v,
                        double* const __restrict__ ug, double* const __restrict__ vg,
                        const double fc, const double ugrid, const double vgrid,
                        const int jj, const int kk,
                        const int istart, const int jstart, const int kstart,
                        const int iend,   const int jend,   const int kend)
    {
        using namespace Finite_difference::O4;

        const int i   = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j   = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k   = blockIdx.z + kstart;
        const int ii  = 1;
        const int ii2 = 2;
        const int jj2 = 2*jj;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            ut[ijk] += fc * ( ( ci0*(ci0*v[ijk-ii2-jj ] + ci1*v[ijk-ii-jj ] + ci2*v[ijk-jj    ] + ci3*v[ijk+ii-jj  ])
                              + ci1*(ci0*v[ijk-ii2    ] + ci1*v[ijk-ii    ] + ci2*v[ijk       ] + ci3*v[ijk+ii     ])
                              + ci2*(ci0*v[ijk-ii2+jj ] + ci1*v[ijk-ii+jj ] + ci2*v[ijk+jj    ] + ci3*v[ijk+ii+jj  ])
                              + ci3*(ci0*v[ijk-ii2+jj2] + ci1*v[ijk-ii+jj2] + ci2*v[ijk+jj2   ] + ci3*v[ijk+ii+jj2 ]) )
                       + vgrid - vg[k] );

            vt[ijk] -= fc * ( ( ci0*(ci0*u[ijk-ii-jj2 ] + ci1*u[ijk-jj2   ] + ci2*u[ijk+ii-jj2] + ci3*u[ijk+ii2-jj2])
                              + ci1*(ci0*u[ijk-ii-jj  ] + ci1*u[ijk-jj    ] + ci2*u[ijk+ii-jj ] + ci3*u[ijk+ii2-jj ])
                              + ci2*(ci0*u[ijk-ii     ] + ci1*u[ijk       ] + ci2*u[ijk+ii    ] + ci3*u[ijk+ii2    ])
                              + ci3*(ci0*u[ijk-ii+jj  ] + ci1*u[ijk+jj    ] + ci2*u[ijk+ii+jj ] + ci3*u[ijk+ii2+jj ]) )
                       + ugrid - ug[k]);
        }
    }

    __global__
    void advec_wls_2nd_g(double* const __restrict__ st, double* const __restrict__ s,
                         const double* const __restrict__ wls, const double* const __restrict__ dzhi,
                         const int istart, const int jstart, const int kstart,
                         const int iend,   const int jend,   const int kend,
                         const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            if (wls[k] > 0.)
                st[ijk] -=  wls[k] * (s[k]-s[k-1])*dzhi[k];
            else
                st[ijk] -=  wls[k] * (s[k+1]-s[k])*dzhi[k+1];
        }
    }

    __global__
    void large_scale_source_g(double* const __restrict__ st, double* const __restrict__ sls,
                              const int istart, const int jstart, const int kstart,
                              const int iend,   const int jend,   const int kend,
                              const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            st[ijk] += sls[k];
        }
    }

    __global__
    void nudging_tendency_g(double* const __restrict__ st, double* const __restrict__ smn,
			    double* const __restrict__ snudge, double* const __restrict__ nudge_fac,
                            const int istart, const int jstart, const int kstart,
                            const int iend,   const int jend,   const int kend,
                            const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;

            st[ijk] += - nudge_fac[k] * (smn[k]-snudge[k]);

        }
    }

    __global__
    void update_time_dependent_prof_g(double* const __restrict__ prof, const double* const __restrict__ data,
                                      const double fac0, const double fac1,
                                      const int index0,  const int index1,
                                      const int kmax,    const int kgc)
    {
        const int k = blockIdx.x*blockDim.x + threadIdx.x;
        const int kk = kmax;

        if (k < kmax)
            prof[k+kgc] = fac0*data[index0*kk+k] + fac1*data[index1*kk+k];
    }
*/
} // end namespace

template<typename TF>
void Force<TF>::prepare_device()
{
/*
    const int nmemsize  = gd.kcells*sizeof(TF);

    if (swlspres == "geo")
    {
        cuda_safe_call(hipMalloc(&ug_g, nmemsize));
        cuda_safe_call(hipMalloc(&vg_g, nmemsize));

        cuda_safe_call(hipMemcpy(ug_g, ug, nmemsize, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(vg_g, vg, nmemsize, hipMemcpyHostToDevice));
        if (swtimedep_geo == "1")
        {

            for (std::map<std::string, double *>::const_iterator it=timedepdata_geo.begin(); it!=timedepdata_geo.end(); ++it)
            {
                int nmemsize2 = gd.kmax*timedeptime_geo[it->first].size()*sizeof(double);
                cuda_safe_call(hipMalloc(&timedepdata_geo_g[it->first], nmemsize2));
                cuda_safe_call(hipMemcpy(timedepdata_geo_g[it->first], timedepdata_geo[it->first], nmemsize2, hipMemcpyHostToDevice));
            }
        }
    }

    if (swls == "1")
    {
        for (std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
        {
            cuda_safe_call(hipMalloc(&lsprofs_g[*it], nmemsize));
            cuda_safe_call(hipMemcpy(lsprofs_g[*it], lsprofs[*it], nmemsize, hipMemcpyHostToDevice));
        }
        if (swtimedep_ls == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_ls.begin(); it!=timedepdata_ls.end(); ++it)
            {
                int nmemsize2 = gd.kmax*timedeptime_ls[it->first].size()*sizeof(double);
                cuda_safe_call(hipMalloc(&timedepdata_ls_g[it->first], nmemsize2));
                cuda_safe_call(hipMemcpy(timedepdata_ls_g[it->first], timedepdata_ls[it->first], nmemsize2, hipMemcpyHostToDevice));
            }
        }
    }

    if (swnudge == "1")
    {
        for (std::vector<std::string>::const_iterator it=nudgelist.begin(); it!=nudgelist.end(); ++it)
        {
            cuda_safe_call(hipMalloc(&nudgeprofs_g[*it], nmemsize));
            cuda_safe_call(hipMemcpy(nudgeprofs_g[*it], nudgeprofs[*it], nmemsize, hipMemcpyHostToDevice));
        }
        cuda_safe_call(hipMalloc(&nudge_factor_g, nmemsize));
        cuda_safe_call(hipMemcpy(nudge_factor_g, nudge_factor, nmemsize, hipMemcpyHostToDevice));
        if (swtimedep_nudge == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_nudge.begin(); it!=timedepdata_nudge.end(); ++it)
            {
                int nmemsize2 = gd.kmax*timedeptime_nudge[it->first].size()*sizeof(double);
                cuda_safe_call(hipMalloc(&timedepdata_nudge_g[it->first], nmemsize2));
                cuda_safe_call(hipMemcpy(timedepdata_nudge_g[it->first], timedepdata_nudge[it->first], nmemsize2, hipMemcpyHostToDevice));
            }
        }
    }

    if (swwls == "1")
    {
        cuda_safe_call(hipMalloc(&wls_g, nmemsize));
        cuda_safe_call(hipMemcpy(wls_g, wls, nmemsize, hipMemcpyHostToDevice));
        if (swtimedep_wls == "1")
        {
            int nmemsize2 = gd.kmax*timedeptime_wls.size()*sizeof(double);
            cuda_safe_call(hipMalloc(&timedepdata_wls_g, nmemsize2));
            cuda_safe_call(hipMemcpy(timedepdata_wls_g, timedepdata_wls, nmemsize2, hipMemcpyHostToDevice));
        }
    }
*/
}

template<typename TF>
void Force<TF>::clear_device()
{
/*    if (swlspres == "geo")
    {
        cuda_safe_call(hipFree(ug_g));
        cuda_safe_call(hipFree(vg_g));
        if (swtimedep_geo == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_geo.begin(); it!=timedepdata_geo.end(); ++it)
                cuda_safe_call(hipFree(timedepdata_geo_g[it->first]));
        }
    }

    if (swls == "1")
    {
        for(std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
            cuda_safe_call(hipFree(lsprofs_g[*it]));
        if (swtimedep_ls == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_ls.begin(); it!=timedepdata_ls.end(); ++it)
                cuda_safe_call(hipFree(timedepdata_ls_g[it->first]));
        }
    }

    if (swnudge == "1")
    {
        for(std::vector<std::string>::const_iterator it=nudgelist.begin(); it!=nudgelist.end(); ++it)
            cuda_safe_call(hipFree(nudgeprofs_g[*it]));
        cuda_safe_call(hipFree(nudge_factor_g));
        if (swtimedep_nudge == "1")
        {
            for (std::map<std::string, double *>::const_iterator it=timedepdata_nudge.begin(); it!=timedepdata_nudge.end(); ++it)
                cuda_safe_call(hipFree(timedepdata_nudge_g[it->first]));
        }
    }

    if (swwls == "1")
    {
        cuda_safe_call(hipFree(wls_g));
        if (swtimedep_wls == "1")
        {
            cuda_safe_call(hipFree(timedepdata_wls_g));
        }
   }
*/
}

#ifdef USECUDA
template<typename TF>
void Force<TF>::exec(double dt)
{
    auto& gd = grid.get_grid_data();
    const int blocki = gd.ithread_block;
    const int blockj = gd.jthread_block;
    const int gridi  = gd.imax/blocki + (gd.imax%blocki > 0);
    const int gridj  = gd.jmax/blockj + (gd.jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, gd.kcells);
    dim3 blockGPU(blocki, blockj, 1);

    if (swlspres == Large_scale_pressure_type::fixed_flux)
    {
        auto tmp = fields.get_tmp_g();
/*
        flux_step_1_g<TF><<<gridGPU, blockGPU>>>(
            &tmp->fld_g[offs], &fields.mp.at("u")->fld_g[offs],
            gd.dz_g,
            gd.icells, gd.ijcells,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend,    gd.jend,   gd.kend);
        cuda_check_error();
*/
        TF uavg  = field3d_operators.calc_mean(fields.mp.at("u")->fld_g);
        TF utavg = field3d_operators.calc_mean(fields.mt.at("u")->fld_g);
/*        flux_step_1_g<TF><<<gridGPU, blockGPU>>>(
            &tmp->fld_g[offs], &fields.mt.at("u")->fld_g[offs],
            gd.dz_g,
            gd.icells, gd.ijcells,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend,    gd.jend,   gd.kend);
        cuda_check_error();
*/
//        TF utavg  = field3d_operators.calc_mean(&tmp->fld_g[offs]);
        fields.release_tmp_g(tmp);


        const TF fbody = (uflux - uavg - grid.utrans) / dt - utavg;

        flux_step_2_g<TF><<<gridGPU, blockGPU>>>(
            fields.mt.at("u")->fld_g,
            fbody,
            gd.icells, gd.ijcells,
            gd.istart,  gd.jstart, gd.kstart,
            gd.iend,    gd.jend,   gd.kend);
        cuda_check_error();
    }
/*    else if (swlspres == "geo")
    {
        if (gd.swspatialorder == "2")
        {
            coriolis_2nd_g<<<gridGPU, blockGPU>>>(
                &fields->ut->data_g[offs], &fields->vt->data_g[offs],
                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                ug_g, vg_g, fc, grid.utrans, grid.vtrans,
                gd.icells, gd.ijcells,
                gd.istart,  gd.jstart, gd.kstart,
                gd.iend,    gd.jend,   gd.kend);
            cuda_check_error();
        }
        else if (gd.swspatialorder == "4")
        {
            coriolis_4th_g<<<gridGPU, blockGPU>>>(
                &fields->ut->data_g[offs], &fields->vt->data_g[offs],
                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                ug_g, vg_g, fc, grid.utrans, grid.vtrans,
                gd.icells, gd.ijcells,
                gd.istart,  gd.jstart, gd.kstart,
                gd.iend,    gd.jend,   gd.kend);
            cuda_check_error();
        }
    }
*/
/*
    if (swls == "1")
    {
        for (std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
        {
            large_scale_source_g<<<gridGPU, blockGPU>>>(
                &fields->st[*it]->data_g[offs], lsprofs_g[*it],
                gd.istart,  gd.jstart, gd.kstart,
                gd.iend,    gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
        }
    }

    if (swnudge == "1")
    {
        for(std::vector<std::string>::const_iterator it=nudgelist.begin(); it!=nudgelist.end(); ++it)
        {
            nudging_tendency_g<<<gridGPU, blockGPU>>>(
                &fields->at[*it]->data_g[offs],  fields->ap[*it]->datamean_g,
                nudgeprofs_g[*it], nudge_factor_g,
                gd.istart,  gd.jstart, gd.kstart,
                gd.iend,    gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
        }
    }

    if (swwls == "1")
    {
        for (FieldMap::iterator it = fields->st.begin(); it!=fields->st.end(); it++)
        {
            advec_wls_2nd_g<<<gridGPU, blockGPU>>>(
                &it->second->data_g[offs], fields->sp[it->first]->datamean_g, wls_g, gd.dzhi_g,
                gd.istart,  gd.jstart, gd.kstart,
                gd.iend,    gd.jend,   gd.kend,
                gd.icells, gd.ijcells);
            cuda_check_error();
        }
    }
*/
}
#endif
/*
#ifdef USECUDA
void Force::update_time_dependent_profs(std::map<std::string, double*>& profiles, std::map<std::string, double*> time_profiles,
                                        std::map<std::string, std::vector<double>> times, std::string suffix)
{
    const int blockk = 128;
    const int gridk  = gd.kmax/blockk + (gd.kmax%blockk > 0);

    // Loop over all profiles which might be time dependent
    for (auto& it : profiles)
    {
        std::string name = it.first + suffix;

        // Check if they have time dependent data
        if (time_profiles.find(name) != time_profiles.end())
        {
            // Get/calculate the interpolation indexes/factors
            int index0, index1;
            double fac0, fac1;

            model->timeloop->get_interpolation_factors(index0, index1, fac0, fac1, times[name]);

            // Calculate the new vertical profile
            update_time_dependent_prof_g<<<gridk, blockk>>>(
                it.second, time_profiles[name], fac0, fac1, index0, index1, gd.kmax, gd.kgc);
            cuda_check_error();
        }
    }

}
#endif
*/
/*
#ifdef USECUDA
void Force::update_time_dependent_prof(double* const prof, const double* const data, std::vector<double> times)
{
    const int blockk = 128;
    const int gridk  = gd.kmax/blockk + (gd.kmax%blockk > 0);

    int index0, index1;
    double fac0, fac1;

    model->timeloop->get_interpolation_factors(index0, index1, fac0, fac1, times);

    // Calculate the new vertical profile
    update_time_dependent_prof_g<<<gridk, blockk>>>(
        prof, data, fac0, fac1, index0, index1, gd.kmax, gd.kgc);
    cuda_check_error();

}
#endif
*/

template class Force<double>;
template class Force<float>;
