#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "master.h"
#include "force.h"
#include "grid.h"
#include "fields.h"
#include "fd.h"
#include "constants.h"
#include "tools.h"

__global__ void force_flux_step1(double * const __restrict__ aSum, const double * const __restrict__ a,
                                 const double * const __restrict__ dz,
                                 const int jj, const int kk, 
                                 const int istart, const int jstart, const int kstart,
                                 const int iend,   const int jend,   const int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    aSum [ijk] = a[ijk]*dz[k];
  }
}

__global__ void force_flux_step2(double * const __restrict__ ut,
                                 const double fbody,
                                 const int jj, const int kk, 
                                 const int istart, const int jstart, const int kstart,
                                 const int iend,   const int jend,   const int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] += fbody;
  }
}

__global__ void force_coriolis_2nd(double * const __restrict__ ut, double * const __restrict__ vt,
                                   double * const __restrict__ u,  double * const __restrict__ v, 
                                   double * const __restrict__ ug, double * const __restrict__ vg, 
                                   const double fc, const double ugrid, const double vgrid,
                                   const int jj, const int kk, 
                                   const int istart, const int jstart, const int kstart,
                                   const int iend,   const int jend,   const int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] += fc * (0.25*(v[ijk-ii] + v[ijk] + v[ijk-ii+jj] + v[ijk+jj]) + vgrid - vg[k]);
    vt[ijk] -= fc * (0.25*(u[ijk-jj] + u[ijk] + u[ijk+ii-jj] + u[ijk+ii]) + ugrid - ug[k]);
  }
}

__global__ void force_coriolis_4th(double * const __restrict__ ut, double * const __restrict__ vt,
                                   double * const __restrict__ u,  double * const __restrict__ v, 
                                   double * const __restrict__ ug, double * const __restrict__ vg, 
                                   const double fc, const double ugrid, const double vgrid,
                                   const int jj, const int kk, 
                                   const int istart, const int jstart, const int kstart,
                                   const int iend,   const int jend,   const int kend)
{
  using namespace fd::o4;

  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  int ii  = 1;
  int ii2 = 2;
  int jj2 = 2*jj;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] += fc * ( ( ci0*(ci0*v[ijk-ii2-jj ] + ci1*v[ijk-ii-jj ] + ci2*v[ijk-jj    ] + ci3*v[ijk+ii-jj  ])
                      + ci1*(ci0*v[ijk-ii2    ] + ci1*v[ijk-ii    ] + ci2*v[ijk       ] + ci3*v[ijk+ii     ])
                      + ci2*(ci0*v[ijk-ii2+jj ] + ci1*v[ijk-ii+jj ] + ci2*v[ijk+jj    ] + ci3*v[ijk+ii+jj  ])
                      + ci3*(ci0*v[ijk-ii2+jj2] + ci1*v[ijk-ii+jj2] + ci2*v[ijk+jj2   ] + ci3*v[ijk+ii+jj2 ]) )
                      + vgrid - vg[k] );

    vt[ijk] -= fc * ( ( ci0*(ci0*u[ijk-ii-jj2 ] + ci1*u[ijk-jj2   ] + ci2*u[ijk+ii-jj2] + ci3*u[ijk+ii2-jj2])
                      + ci1*(ci0*u[ijk-ii-jj  ] + ci1*u[ijk-jj    ] + ci2*u[ijk+ii-jj ] + ci3*u[ijk+ii2-jj ])
                      + ci2*(ci0*u[ijk-ii     ] + ci1*u[ijk       ] + ci2*u[ijk+ii    ] + ci3*u[ijk+ii2    ])
                      + ci3*(ci0*u[ijk-ii+jj  ] + ci1*u[ijk+jj    ] + ci2*u[ijk+ii+jj ] + ci3*u[ijk+ii2+jj ]) )
                      + ugrid - ug[k]);
  }
}

__global__ void force_advecwls_2nd(double * const __restrict__ st, double * const __restrict__ s,
                                   const double * const __restrict__ wls, const double * const __restrict__ dzhi,
                                   const int istart, const int jstart, const int kstart,
                                   const int iend,   const int jend,   const int kend,
                                   const int jj, const int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;

    if(wls[k] > 0.)
      st[ijk] -=  wls[k] * (s[k]-s[k-1])*dzhi[k];
    else
      st[ijk] -=  wls[k] * (s[k+1]-s[k])*dzhi[k+1];
  }
}

__global__ void force_lssource(double * const __restrict__ st, double * const __restrict__ sls,
                               const int istart, const int jstart, const int kstart,
                               const int iend,   const int jend,   const int kend,
                               const int jj, const int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    st[ijk] += sls[k];
  }
}

__global__ void force_updatetimedepprof(double * const __restrict__ sls, double * const __restrict__ slstd,
                                        const double fac0, const double fac1, const int index0, const int index1, const int kmax, const int kgc)
{
  int k = blockIdx.x*blockDim.x + threadIdx.x;
  int kk = kmax;

  if(k < kmax)
  {
    sls[k+kgc] = fac0*slstd[index0*kk+k] + fac1*slstd[index1*kk+k];
  }
}

int Force::prepareDevice()
{
  const int nmemsize  = grid->kcells*sizeof(double);

  if(swlspres == "geo")
  {
    cudaSafeCall(hipMalloc(&ug_g, nmemsize));
    cudaSafeCall(hipMalloc(&vg_g, nmemsize));

    cudaSafeCall(hipMemcpy(ug_g, ug, nmemsize, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(vg_g, vg, nmemsize, hipMemcpyHostToDevice));
  }

  if(swls == "1")
  {
    for(std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
    {
      cudaSafeCall(hipMalloc(&lsprofs_g[*it], nmemsize));
      cudaSafeCall(hipMemcpy(lsprofs_g[*it], lsprofs[*it], nmemsize, hipMemcpyHostToDevice));
    }
  }

  if(swwls == "1")
  {
    cudaSafeCall(hipMalloc(&wls_g, nmemsize));
    cudaSafeCall(hipMemcpy(wls_g, wls, nmemsize, hipMemcpyHostToDevice));
  }

  if(swtimedep == "1")
  {
    int nmemsize2 = grid->kmax*timedeptime.size()*sizeof(double);
    for(std::map<std::string, double *>::const_iterator it=timedepdata.begin(); it!=timedepdata.end(); ++it)
    {
      cudaSafeCall(hipMalloc(&timedepdata_g[it->first], nmemsize2));
      cudaSafeCall(hipMemcpy(timedepdata_g[it->first], timedepdata[it->first], nmemsize2, hipMemcpyHostToDevice));
    }
  }

  return 0;
}

int Force::clearDevice()
{
  if(swlspres == "geo")
  {
    cudaSafeCall(hipFree(ug_g));
    cudaSafeCall(hipFree(vg_g));
  }

  if(swls == "1")
  {
    for(std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
      cudaSafeCall(hipFree(lsprofs_g[*it]));
  }

  if(swwls == "1")
    cudaSafeCall(hipFree(wls_g));

  if(swtimedep == "1")
  {
    for(std::map<std::string, double *>::const_iterator it=timedepdata.begin(); it!=timedepdata.end(); ++it)
      cudaSafeCall(hipFree(timedepdata_g[it->first]));
  }

  return 0; 
}

#ifdef USECUDA
int Force::exec(double dt)
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  if(swlspres == "uflux")
  {
    force_flux_step1<<<gridGPU, blockGPU>>>(&fields->a["tmp1"]->data_g[offs], &fields->u->data_g[offs],
                                            grid->dz_g,
                                            grid->icellsp, grid->ijcellsp,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend);
    cudaCheckError();

    double uavg  = grid->getsum_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp2"]->data_g); 

    force_flux_step1<<<gridGPU, blockGPU>>>(&fields->a["tmp1"]->data_g[offs], &fields->ut->data_g[offs],
                                            grid->dz_g,
                                            grid->icellsp, grid->ijcellsp,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend);
    cudaCheckError();

    double utavg = grid->getsum_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp2"]->data_g); 

    uavg  = uavg  / (grid->itot*grid->jtot*grid->zsize);
    utavg = utavg / (grid->itot*grid->jtot*grid->zsize);

    double fbody = (uflux - uavg - grid->utrans) / dt - utavg;

    force_flux_step2<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs],
                                            fbody,
                                            grid->icellsp, grid->ijcellsp,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend);
    cudaCheckError();
  }
  else if(swlspres == "geo")
  {
    if(grid->swspatialorder == "2")
    {
      force_coriolis_2nd<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs],
                                                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                                                ug_g, vg_g, fc, grid->utrans, grid->vtrans, 
                                                grid->icellsp, grid->ijcellsp,
                                                grid->istart,  grid->jstart, grid->kstart,
                                                grid->iend,    grid->jend,   grid->kend);
      cudaCheckError();
    }
    else if(grid->swspatialorder == "4")
    {
      force_coriolis_4th<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs],
                                                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                                                ug_g, vg_g, fc, grid->utrans, grid->vtrans, 
                                                grid->icellsp, grid->ijcellsp,
                                                grid->istart,  grid->jstart, grid->kstart,
                                                grid->iend,    grid->jend,   grid->kend);
      cudaCheckError();
    }
  }

  if(swls == "1")
  {
    for(std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
    {
      force_lssource<<<gridGPU, blockGPU>>>(&fields->st[*it]->data_g[offs], lsprofs_g[*it],
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend,
                                            grid->icellsp, grid->ijcellsp);
      cudaCheckError();
    }
  }

  if(swwls == "1")
  {
    for(fieldmap::iterator it = fields->st.begin(); it!=fields->st.end(); it++)
    {
      force_advecwls_2nd<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], fields->s[it->first]->datamean_g, wls_g, grid->dzhi_g,
                                                grid->istart,  grid->jstart, grid->kstart,
                                                grid->iend,    grid->jend,   grid->kend,
                                                grid->icellsp, grid->ijcellsp);
      cudaCheckError();
    }
  }

  return 0;
}
#endif

#ifdef USECUDA
int Force::settimedepprofiles(double fac0, double fac1, int index0, int index1)
{
  const int blockk = 128;
  const int gridk  = grid->kmax/blockk + (grid->kmax%blockk > 0);

  for(std::vector<std::string>::const_iterator it1=lslist.begin(); it1!=lslist.end(); ++it1)
  {
    std::string name = *it1 + "ls";
    std::map<std::string, double *>::const_iterator it2 = timedepdata_g.find(name);

    // update the profile
    if(it2 != timedepdata.end())
    {
      force_updatetimedepprof<<<gridk, blockk>>>(lsprofs_g[*it1], it2->second, fac0, fac1, index0, index1, grid->kmax, grid->kgc);
      cudaCheckError();
    }
  }

  return 0;
}
#endif
