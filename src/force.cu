#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "master.h"
#include "force.h"
#include "grid.h"
#include "fields.h"
#include "fd.h"

__global__ void force_flux_step1(double * const __restrict__ usum, double * const __restrict__ utsum,
                                 const double * const __restrict__ u, const double * const __restrict__ ut,
                                 const double * const __restrict__ dz,
                                 const int jj, const int kk, 
                                 const int istart, const int jstart, const int kstart,
                                 const int iend,   const int jend,   const int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    usum [ijk] = u [ijk]*dz[k];
    utsum[ijk] = ut[ijk]*dz[k];
  }
}

__global__ void force_flux_step2(double * const __restrict__ ut,
                                 const double fbody,
                                 const int jj, const int kk, 
                                 const int istart, const int jstart, const int kstart,
                                 const int iend,   const int jend,   const int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] += fbody;
  }
}

__global__ void force_coriolis_2nd(double * const __restrict__ ut, double * const __restrict__ vt,
                                   double * const __restrict__ u,  double * const __restrict__ v, 
                                   double * const __restrict__ ug, double * const __restrict__ vg, 
                                   const double fc, const double ugrid, const double vgrid,
                                   const int jj, const int kk, 
                                   const int istart, const int jstart, const int kstart,
                                   const int iend,   const int jend,   const int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] += fc * (0.25*(v[ijk-ii] + v[ijk] + v[ijk-ii+jj] + v[ijk+jj]) + vgrid - vg[k]);
    vt[ijk] -= fc * (0.25*(u[ijk-jj] + u[ijk] + u[ijk+ii-jj] + u[ijk+ii]) + ugrid - ug[k]);
  }
}

__global__ void force_coriolis_4th(double * const __restrict__ ut, double * const __restrict__ vt,
                                   double * const __restrict__ u,  double * const __restrict__ v, 
                                   double * const __restrict__ ug, double * const __restrict__ vg, 
                                   const double fc, const double ugrid, const double vgrid,
                                   const int jj, const int kk, 
                                   const int istart, const int jstart, const int kstart,
                                   const int iend,   const int jend,   const int kend)
{
  using namespace fd::o4;

  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  int ii  = 1;
  int ii2 = 2;
  int jj2 = 2*jj;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    ut[ijk] += fc * ( ( ci0*(ci0*v[ijk-ii2-jj ] + ci1*v[ijk-ii-jj ] + ci2*v[ijk-jj    ] + ci3*v[ijk+ii-jj  ])
                      + ci1*(ci0*v[ijk-ii2    ] + ci1*v[ijk-ii    ] + ci2*v[ijk       ] + ci3*v[ijk+ii     ])
                      + ci2*(ci0*v[ijk-ii2+jj ] + ci1*v[ijk-ii+jj ] + ci2*v[ijk+jj    ] + ci3*v[ijk+ii+jj  ])
                      + ci3*(ci0*v[ijk-ii2+jj2] + ci1*v[ijk-ii+jj2] + ci2*v[ijk+jj2   ] + ci3*v[ijk+ii+jj2 ]) )
                      + vgrid - vg[k] );

    vt[ijk] -= fc * ( ( ci0*(ci0*u[ijk-ii-jj2 ] + ci1*u[ijk-jj2   ] + ci2*u[ijk+ii-jj2] + ci3*u[ijk+ii2-jj2])
                      + ci1*(ci0*u[ijk-ii-jj  ] + ci1*u[ijk-jj    ] + ci2*u[ijk+ii-jj ] + ci3*u[ijk+ii2-jj ])
                      + ci2*(ci0*u[ijk-ii     ] + ci1*u[ijk       ] + ci2*u[ijk+ii    ] + ci3*u[ijk+ii2    ])
                      + ci3*(ci0*u[ijk-ii+jj  ] + ci1*u[ijk+jj    ] + ci2*u[ijk+ii+jj ] + ci3*u[ijk+ii2+jj ]) )
                      + ugrid - ug[k]);
  }
}

__global__ void force_advecwls_2nd(double * const __restrict__ st, double * const __restrict__ s,
                                   const double * const __restrict__ wls, const double * const __restrict__ dzhi,
                                   const int istart, const int jstart, const int kstart,
                                   const int iend,   const int jend,   const int kend,
                                   const int jj, const int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;

    if(wls[k] > 0.)
      st[ijk] -=  wls[k] * (s[k]-s[k-1])*dzhi[k];
    else
      st[ijk] -=  wls[k] * (s[k+1]-s[k])*dzhi[k+1];
  }
}



//int cforce::advecwls_2nd(double * const restrict st, const double * const restrict s,
//                         const double * const restrict wls, const double * const dzhi)
//{
//  int ijk,jj,kk;
//
//  jj = grid->icells;
//  kk = grid->ijcells;
//
//  // use an upwind differentiation
//  for(int k=grid->kstart; k<grid->kend; ++k)
//  {
//    if(wls[k] > 0.)
//    {
//      for(int j=grid->jstart; j<grid->jend; ++j)
//        for(int i=grid->istart; i<grid->iend; ++i)
//        {
//          ijk = i + j*jj + k*kk;
//          st[ijk] -=  wls[k] * (s[k]-s[k-1])*dzhi[k];
//        }
//    }
//    else
//    {
//      for(int j=grid->jstart; j<grid->jend; ++j)
//        for(int i=grid->istart; i<grid->iend; ++i)
//        {
//          ijk = i + j*jj + k*kk;
//          st[ijk] -=  wls[k] * (s[k+1]-s[k])*dzhi[k+1];
//        }
//    }
//  }


int cforce::prepareDevice()
{
  const int nmemsize = grid->kcells*sizeof(double);

  if(swlspres == "geo")
  {
    hipMalloc(&ug_g, nmemsize);
    hipMalloc(&vg_g, nmemsize);

    hipMemcpy(ug_g, ug, nmemsize, hipMemcpyHostToDevice);
    hipMemcpy(vg_g, vg, nmemsize, hipMemcpyHostToDevice);
  }

  //if(swls == "1")
  //{
  //  for(std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
  //    lsprofs[*it] = new double[grid->kcells];
  //}

  if(swwls == "1")
  {
    hipMalloc(&wls_g, nmemsize);
    hipMemcpy(wls_g, wls, nmemsize, hipMemcpyHostToDevice);
  }

  return 0;
}

int cforce::clearDevice()
{
  if(swlspres == "geo")
  {
    hipFree(ug_g);
    hipFree(vg_g);
  }

  //if(swls == "1")
  //{
  //}

  if(swwls == "1")
    hipFree(wls_g);

  return 0; 
}

#ifdef USECUDA
int cforce::exec(double dt)
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  if(swlspres == "uflux")
  {
    force_flux_step1<<<gridGPU, blockGPU>>>(&fields->a["tmp1"]->data_g[offs], &fields->a["tmp2"]->data_g[offs],
                                            &fields->u->data_g[offs], &fields->ut->data_g[offs],
                                            grid->dz_g,
                                            grid->icellsp, grid->ijcellsp,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend);

    double uavg  = grid->getsum_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp3"]->data_g); 
    double utavg = grid->getsum_g(&fields->a["tmp2"]->data_g[offs], fields->a["tmp3"]->data_g); 

    uavg  = uavg  / (grid->itot*grid->jtot*grid->zsize);
    utavg = utavg / (grid->itot*grid->jtot*grid->zsize);

    double fbody = (uflux - uavg - grid->utrans) / dt - utavg;

    force_flux_step2<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs],
                                            fbody,
                                            grid->icellsp, grid->ijcellsp,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend);
  }
  else if(swlspres == "geo")
  {
    if(grid->swspatialorder == "2")
      force_coriolis_2nd<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs],
                                                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                                                ug_g, vg_g, fc, grid->utrans, grid->vtrans, 
                                                grid->icellsp, grid->ijcellsp,
                                                grid->istart,  grid->jstart, grid->kstart,
                                                grid->iend,    grid->jend,   grid->kend);
    else if(grid->swspatialorder == "4")
      force_coriolis_4th<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs],
                                                &fields->u->data_g[offs],  &fields->v->data_g[offs],
                                                ug_g, vg_g, fc, grid->utrans, grid->vtrans, 
                                                grid->icellsp, grid->ijcellsp,
                                                grid->istart,  grid->jstart, grid->kstart,
                                                grid->iend,    grid->jend,   grid->kend);
  }

  //if(swls == "1")
  //{
  //  for(std::vector<std::string>::const_iterator it=lslist.begin(); it!=lslist.end(); ++it)
  //    lssource(fields->st[*it]->data, lsprofs[*it]);
  //}

  if(swwls == "1")
  {
    for(fieldmap::iterator it = fields->st.begin(); it!=fields->st.end(); it++)
      force_advecwls_2nd<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], fields->s[it->first]->datamean_g, wls_g, grid->dzhi_g,
                                                grid->istart,  grid->jstart, grid->kstart,
                                                grid->iend,    grid->jend,   grid->kend,
                                                grid->icellsp, grid->ijcellsp);
  }

  return 0;
}
#endif

