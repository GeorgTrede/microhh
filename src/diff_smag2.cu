#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include "grid.h"
#include "fields.h"
#include "master.h"
#include "diff_smag2.h"
#include "boundary_surface.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "model.h"
#include "tools.h"
#include "most.h"

namespace DiffSmag2_g
{
  __global__ void strain2(double * __restrict__ strain2,
                          double * __restrict__ u,  double * __restrict__ v,  double * __restrict__ w,
                          double * __restrict__ ufluxbot, double * __restrict__ vfluxbot,
                          double * __restrict__ ustar, double * __restrict__ obuk, 
                          double * __restrict__ z, double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi, 
                          int istart, int jstart, int kstart, int iend, int jend, int kend, 
                          int jj, int kk)
  
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;
    const int ii = 1;
  
    if(i < iend && j < jend && k < kend)
    {
      const int ij  = i + j*jj;
      const int ijk = i + j*jj + k*kk;
  
      if(k == kstart)
      {
        strain2[ijk] = 2.*(
          // du/dz
          + 0.5*pow(-0.5*(ufluxbot[ij]+ufluxbot[ij+ii])/(constants::kappa*z[k]*ustar[ij])*most::phim(z[k]/obuk[ij]), 2)
          // dv/dz
          + 0.5*pow(-0.5*(vfluxbot[ij]+vfluxbot[ij+jj])/(constants::kappa*z[k]*ustar[ij])*most::phim(z[k]/obuk[ij]), 2) );
         // add a small number to avoid zero divisions
         strain2[ijk] += constants::dsmall;  
      }
      else
      {
        strain2[ijk] = 2.*(
          // du/dx + du/dx
          + pow((u[ijk+ii]-u[ijk])*dxi, 2)
          // dv/dy + dv/dy
          + pow((v[ijk+jj]-v[ijk])*dyi, 2)
          // dw/dz + dw/dz
          + pow((w[ijk+kk]-w[ijk])*dzi[k], 2)
          // du/dy + dv/dx
          + 0.125*pow((u[ijk      ]-u[ijk   -jj])*dyi  + (v[ijk      ]-v[ijk-ii   ])*dxi, 2)
          + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-jj])*dyi  + (v[ijk+ii   ]-v[ijk      ])*dxi, 2)
          + 0.125*pow((u[ijk   +jj]-u[ijk      ])*dyi  + (v[ijk   +jj]-v[ijk-ii+jj])*dxi, 2)
          + 0.125*pow((u[ijk+ii+jj]-u[ijk+ii   ])*dyi  + (v[ijk+ii+jj]-v[ijk   +jj])*dxi, 2)
          // du/dz + dw/dx
          + 0.125*pow((u[ijk      ]-u[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-ii   ])*dxi, 2)
          + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-kk])*dzhi[k  ] + (w[ijk+ii   ]-w[ijk      ])*dxi, 2)
          + 0.125*pow((u[ijk   +kk]-u[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-ii+kk])*dxi, 2)
          + 0.125*pow((u[ijk+ii+kk]-u[ijk+ii   ])*dzhi[k+1] + (w[ijk+ii+kk]-w[ijk   +kk])*dxi, 2)
          // dv/dz + dw/dy
          + 0.125*pow((v[ijk      ]-v[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-jj   ])*dyi, 2)
          + 0.125*pow((v[ijk+jj   ]-v[ijk+jj-kk])*dzhi[k  ] + (w[ijk+jj   ]-w[ijk      ])*dyi, 2)
          + 0.125*pow((v[ijk   +kk]-v[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-jj+kk])*dyi, 2)
          + 0.125*pow((v[ijk+jj+kk]-v[ijk+jj   ])*dzhi[k+1] + (w[ijk+jj+kk]-w[ijk   +kk])*dyi, 2) );
        // add a small number to avoid zero divisions
        strain2[ijk] += constants::dsmall;
      }
    }
  }
  
  __global__ void evisc(double * __restrict__ evisc, double * __restrict__ N2,
                        double * __restrict__ bfluxbot, double * __restrict__ ustar, double * __restrict__ obuk,
                        double * __restrict__ mlen,
                        double tPri, double z0m, double zsl,
                        int istart, int jstart, int kstart, int iend, int jend, int kend, 
                        int jj, int kk)
  
  {
    //__shared__ double fac;
    
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;
  
    if(i < iend && j < jend && k < kend)
    {
      const int ij  = i + j*jj;
      const int ijk = i + j*jj + k*kk;
  
      if(k == kstart)
      {
        // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
        double RitPrratio = -bfluxbot[ij]/(constants::kappa*zsl*ustar[ij])*most::phih(zsl/obuk[ij]) / evisc[ijk] * tPri;
        RitPrratio        = fmin(RitPrratio, 1.-constants::dsmall);
        evisc[ijk]        = mlen[k] * sqrt(evisc[ijk] * (1.-RitPrratio));
      }
      else
      {
        // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
        double RitPrratio = N2[ijk] / evisc[ijk] * tPri;
        RitPrratio        = fmin(RitPrratio, 1.-constants::dsmall);
        evisc[ijk]        = mlen[k] * sqrt(evisc[ijk] * (1.-RitPrratio));
      }
    }
  }
  
  __global__ void evisc_neutral(double * __restrict__ evisc, double * __restrict__ mlen,
                                int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                int jj, int kk)
  
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;
  
    if(i < iend && j < jend && k < kend)
    {
      const int ijk = i + j*jj + k*kk;
      evisc[ijk]    = mlen[k] * sqrt(evisc[ijk]);
    }
  }
  
  __global__ void diffuvw(double * __restrict__ ut, double * __restrict__ vt, double * __restrict__ wt, 
                          double * __restrict__ evisc,
                          double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                          double * __restrict__ fluxbotu, double * __restrict__ fluxtopu, 
                          double * __restrict__ fluxbotv, double * __restrict__ fluxtopv, 
                          double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                          double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                          int istart, int jstart, int kstart, int iend, int jend, int kend, 
                          int jj, int kk)
  
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;
    double eviscnu, eviscsu, eviscbu, evisctu;
    double eviscev, eviscwv, eviscbv, evisctv;
    double eviscew, eviscww, eviscnw, eviscsw;
  
    if(i < iend && j < jend && k < kend)
    {
      const int ii  = 1;
      const int ij  = i + j*jj;
      const int ijk = i + j*jj + k*kk;
  
      // U
      eviscnu = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
      eviscsu = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
      evisctu = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
      eviscbu = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);
  
      // V
      eviscev = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
      eviscwv = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
      evisctv = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
      eviscbv = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);
  
      // W
      eviscew = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+ii-kk] + evisc[ijk+ii]);
      eviscww = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-ii] + evisc[ijk   -kk] + evisc[ijk   ]);
      eviscnw = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+jj-kk] + evisc[ijk+jj]);
      eviscsw = 0.25*(evisc[ijk-jj-kk] + evisc[ijk-jj] + evisc[ijk   -kk] + evisc[ijk   ]);
  
      if(k == kstart)
      {
        ut[ijk] +=
              // du/dx + du/dx
              + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
                 - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
              // du/dy + dv/dx
              + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
                 - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
              // du/dz + dw/dx
              + (  rhorefh[kstart+1] * evisctu*((u[ijk+kk]-u[ijk   ])* dzhi[kstart+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
                 + rhorefh[kstart  ] * fluxbotu[ij] ) / rhoref[kstart] * dzi[kstart];
  
        vt[ijk] +=
              // dv/dx + du/dy
              + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
                 - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
              // dv/dy + dv/dy
              + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
                 - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
              // dv/dz + dw/dy
              + (  rhorefh[k+1] * evisctv*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
                 + rhorefh[k  ] * fluxbotv[ij] ) / rhoref[k] * dzi[k];
      }
      else if(k == kend-1)
      {
        ut[ijk] +=
              // du/dx + du/dx
              + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
                 - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
              // du/dy + dv/dx
              + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
                 - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
              // du/dz + dw/dx
              + (- rhorefh[kend  ] * fluxtopu[ij]
                 - rhorefh[kend-1] * eviscbu*((u[ijk   ]-u[ijk-kk])* dzhi[kend-1] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[kend-1] * dzi[kend-1];
  
        vt[ijk] +=
              // dv/dx + du/dy
              + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
                 - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
              // dv/dy + dv/dy
              + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
                 - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
              // dv/dz + dw/dy
              + (- rhorefh[k  ] * fluxtopv[ij]
                 - rhorefh[k-1] * eviscbv*((v[ijk   ]-v[ijk-kk])*dzhi[k-1] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k-1] * dzi[k-1];
  
        wt[ijk] +=
              // dw/dx + du/dz
              + (  eviscew*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
                 - eviscww*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
              // dw/dy + dv/dz
              + (  eviscnw*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
                 - eviscsw*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
              // dw/dz + dw/dz
              + (  rhoref[k  ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzi[k  ]
                 - rhoref[k-1] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzi[k-1] ) / rhorefh[k] * 2.* dzhi[k];
      }
      else
      {
        ut[ijk] +=
              // du/dx + du/dx
              + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
                 - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
              // du/dy + dv/dx
              + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
                 - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
              // du/dz + dw/dx
              + (  rhorefh[k+1] * evisctu*((u[ijk+kk]-u[ijk   ])* dzhi[k+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
                 - rhorefh[k  ] * eviscbu*((u[ijk   ]-u[ijk-kk])* dzhi[k  ] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[k] * dzi[k];
  
        vt[ijk] +=
              // dv/dx + du/dy
              + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
                 - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
              // dv/dy + dv/dy
              + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
                 - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
              // dv/dz + dw/dy
              + (  rhorefh[k+1] * evisctv*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
                 - rhorefh[k  ] * eviscbv*((v[ijk   ]-v[ijk-kk])*dzhi[k  ] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k] * dzi[k];
  
        wt[ijk] +=
              // dw/dx + du/dz
              + (  eviscew*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
                 - eviscww*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
              // dw/dy + dv/dz
              + (  eviscnw*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
                 - eviscsw*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
              // dw/dz + dw/dz
              + (  rhoref[k  ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzi[k  ]
                 - rhoref[k-1] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzi[k-1] ) / rhorefh[k] * 2.* dzhi[k];
      }
    }
  }
 
  __global__ void diffc(double * __restrict__ at, double * __restrict__ a, double * __restrict__ evisc,
                        double * __restrict__ fluxbot, double * __restrict__ fluxtop, 
                        double * __restrict__ dzi, double * __restrict__ dzhi, double dxidxi, double dyidyi,
                        double * __restrict__ rhoref, double * __restrict__ rhorefh, double tPri, 
                        int istart, int jstart, int kstart, int iend, int jend, int kend, 
                        int jj, int kk)
  
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;
    double evisce,eviscw,eviscn,eviscs,evisct,eviscb;
  
    if(i < iend && j < jend && k < kend)
    {
      const int ii  = 1;
      const int ij  = i + j*jj;
      const int ijk = i + j*jj + k*kk;
  
      if(k == kstart)
      {
        evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
        eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
        eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
        eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
        evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;
        eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;
  
        at[ijk] +=
              + (  evisce*(a[ijk+ii]-a[ijk   ]) 
                 - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
              + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
                 - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
              + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
                 + rhorefh[k  ] * fluxbot[ij] ) / rhoref[k] * dzi[k];
      }
      else if(k == kend-1)
      {
        evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
        eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
        eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
        eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
        evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;
        eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;
  
        at[ijk] +=
              + (  evisce*(a[ijk+ii]-a[ijk   ]) 
                 - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
              + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
                 - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
              + (- rhorefh[k  ] * fluxtop[ij]
                 - rhorefh[k-1] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k-1] ) / rhoref[k-1] * dzi[k-1];
      }
      else
      {
        evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
        eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
        eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
        eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
        evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;
        eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;
  
        at[ijk] +=
              + (  evisce*(a[ijk+ii]-a[ijk   ]) 
                 - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
              + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
                 - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
              + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
                 - rhorefh[k  ] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k]  ) / rhoref[k] * dzi[k];
      }
    }
  }
  
  __global__ void calcdnmul(double * __restrict__ dnmul, double * __restrict__ evisc, 
                            double * __restrict__ dzi, double tPrfac, double dxidxi, double dyidyi,
                            int istart, int jstart, int kstart, int iend, int jend, int kend, int jj, int kk)
  
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
    const int k = blockIdx.z + kstart;
  
    if(i < iend && j < jend && k < kend)
    {
      const int ijk = i + j*jj + k*kk;
      dnmul[ijk] = fabs(tPrfac*evisc[ijk]*(dxidxi + dyidyi + dzi[k]*dzi[k]));
    }
  }
}

/* Calculate the mixing length (mlen) offline, and put on GPU */
#ifdef USECUDA
void DiffSmag2::prepareDevice()
{
  BoundarySurface *boundaryptr = static_cast<BoundarySurface *>(model->boundary);

  const double n=2.;
  double mlen0;
  double *mlen = new double[grid->kcells];
  for(int k=0; k<grid->kcells; ++k) 
  {
    mlen0   = cs * pow(grid->dx*grid->dy*grid->dz[k], 1./3.);
    mlen[k] = pow(pow(1./(1./pow(mlen0, n) + 1./(pow(constants::kappa*(grid->z[k]+boundaryptr->z0m), n))), 1./n), 2);
  }

  const int nmemsize = grid->kcells*sizeof(double);
  cudaSafeCall(hipMalloc(&mlen_g, nmemsize));
  cudaSafeCall(hipMemcpy(mlen_g, mlen, nmemsize, hipMemcpyHostToDevice));

  delete[] mlen;
}
#endif

void DiffSmag2::clearDevice()
{
  cudaSafeCall(hipFree(mlen_g));
}

#ifdef USECUDA
void DiffSmag2::execViscosity()
{
  // do a cast because the base boundary class does not have the MOST related variables
  BoundarySurface *boundaryptr = static_cast<BoundarySurface *>(model->boundary);

  const int blocki = grid->iThreadBlock;
  const int blockj = grid->jThreadBlock;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  // Calculate total strain rate
  DiffSmag2_g::strain2<<<gridGPU, blockGPU>>>(&fields->sd["evisc"]->data_g[offs], 
                                              &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                              &fields->u->datafluxbot_g[offs],  &fields->v->datafluxbot_g[offs],
                                              &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
                                              grid->z_g, grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                              grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                              grid->icellsp, grid->ijcellsp);  
  cudaCheckError();

  // start with retrieving the stability information
  if(model->thermo->getSwitch() == "0")
  {
    DiffSmag2_g::evisc_neutral<<<gridGPU, blockGPU>>>(&fields->sd["evisc"]->data_g[offs], mlen_g,
                                                      grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                                      grid->icellsp, grid->ijcellsp);  
    cudaCheckError();

    grid->boundaryCyclic_g(&fields->sd["evisc"]->data_g[offs]);
  }
  // assume buoyancy calculation is needed
  else
  {
    // store the buoyancyflux in datafluxbot of tmp1
    model->thermo->getBuoyancyFluxbot(fields->atmp["tmp1"]);
    // store the Brunt-vaisala frequency in data of tmp1 
    model->thermo->getThermoField(fields->atmp["tmp1"], fields->atmp["tmp2"], "N2");

    // Calculate eddy viscosity
    double tPri = 1./tPr;
    DiffSmag2_g::evisc<<<gridGPU, blockGPU>>>(&fields->sd["evisc"]->data_g[offs], &fields->atmp["tmp1"]->data_g[offs], 
                                              &fields->atmp["tmp1"]->datafluxbot_g[offs], &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
                                              mlen_g, tPri, boundaryptr->z0m, grid->z[grid->kstart],
                                              grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                              grid->icellsp, grid->ijcellsp);  
    cudaCheckError();

    grid->boundaryCyclic_g(&fields->sd["evisc"]->data_g[offs]);
  }
}
#endif

#ifdef USECUDA
void DiffSmag2::exec()
{
  const int blocki = grid->iThreadBlock;
  const int blockj = grid->jThreadBlock;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;
  const double dxidxi = 1./(grid->dx * grid->dx);
  const double dyidyi = 1./(grid->dy * grid->dy);
  const double tPri = 1./tPr;

  DiffSmag2_g::diffuvw<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                              &fields->sd["evisc"]->data_g[offs], 
                                              &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                              &fields->u->datafluxbot_g[offs], &fields->u->datafluxtop_g[offs],
                                              &fields->v->datafluxbot_g[offs], &fields->v->datafluxtop_g[offs],
                                              grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                              fields->rhoref_g, fields->rhorefh_g,
                                              grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                              grid->icellsp, grid->ijcellsp);  
  cudaCheckError();

  for(FieldMap::const_iterator it = fields->st.begin(); it!=fields->st.end(); ++it)
    DiffSmag2_g::diffc<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], &fields->sp[it->first]->data_g[offs], &fields->sd["evisc"]->data_g[offs], 
                                              &fields->sp[it->first]->datafluxbot_g[offs], &fields->sp[it->first]->datafluxtop_g[offs],
                                              grid->dzi_g, grid->dzhi_g, dxidxi, dyidyi,
                                              fields->rhoref_g, fields->rhorefh_g, tPri,
                                              grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                              grid->icellsp, grid->ijcellsp);  
  cudaCheckError();
}
#endif

#ifdef USECUDA
unsigned long DiffSmag2::getTimeLimit(unsigned long idt, double dt)
{
  const int blocki = grid->iThreadBlock;
  const int blockj = grid->jThreadBlock;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  double dnmul;
  unsigned long idtlim;
  const int offs = grid->memoffset;
  const double dxidxi = 1./(grid->dx * grid->dx);
  const double dyidyi = 1./(grid->dy * grid->dy);
  const double tPrfac = std::min(1., tPr);

  // Calculate dnmul in tmp1 field
  DiffSmag2_g::calcdnmul<<<gridGPU, blockGPU>>>(&fields->atmp["tmp1"]->data_g[offs], &fields->sd["evisc"]->data_g[offs],
                                                grid->dzi_g, tPrfac, dxidxi, dyidyi,  
                                                grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                                grid->icellsp, grid->ijcellsp);  
  cudaCheckError();

  // Get maximum from tmp1 field
  dnmul = grid->getMax_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g); 
  dnmul = std::max(constants::dsmall, dnmul);
  idtlim = idt * dnmax/(dnmul*dt);

  return idtlim;
}
#endif

#ifdef USECUDA
double DiffSmag2::get_dn(double dt)
{
  const int blocki = grid->iThreadBlock;
  const int blockj = grid->jThreadBlock;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  double dnmul;
  const int offs = grid->memoffset;
  const double dxidxi = 1./(grid->dx * grid->dx);
  const double dyidyi = 1./(grid->dy * grid->dy);
  const double tPrfac = std::min(1., tPr);

  // Calculate dnmul in tmp1 field
  DiffSmag2_g::calcdnmul<<<gridGPU, blockGPU>>>(&fields->atmp["tmp1"]->data_g[offs], &fields->sd["evisc"]->data_g[offs],
                                                grid->dzi_g, tPrfac, dxidxi, dyidyi,  
                                                grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                                grid->icellsp, grid->ijcellsp);  
  cudaCheckError();

  // Get maximum from tmp1 field
  dnmul = grid->getMax_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g); 

  return dnmul*dt;
}
#endif
