#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2018 Chiel van Heerwaarden
 * Copyright (c) 2011-2018 Thijs Heus
 * Copyright (c) 2014-2018 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include "grid.h"
#include "fields.h"
#include "master.h"
#include "diff_smag2.h"
#include "boundary_surface.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "model.h"
#include "tools.h"
#include "monin_obukhov.h"

namespace
{
    namespace most = Monin_obukhov;

    template<TF> __global__ 
    void strain2_g(double* __restrict__ strain2,
                   double* __restrict__ u,  double* __restrict__ v,  double* __restrict__ w,
                   double* __restrict__ ufluxbot, double* __restrict__ vfluxbot,
                   double* __restrict__ ustar, double* __restrict__ obuk, 
                   double* __restrict__ z, double* __restrict__ dzi, double* __restrict__ dzhi, const double dxi, const double dyi, 
                   const int istart, const int jstart, const int kstart, 
                   const int iend,   const int jend,   const int kend, 
                   const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;
        const int ii = 1;

        if (i < iend && j < jend && k < kend)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + k*kk;

            if (k == kstart)
            {
                strain2[ijk] = 2.*(
                   // du/dz
                   + 0.5*pow(-0.5*(ufluxbot[ij]+ufluxbot[ij+ii])/(Constants::kappa*z[k]*ustar[ij])*most::phim(z[k]/obuk[ij]), 2)
                   // dv/dz
                   + 0.5*pow(-0.5*(vfluxbot[ij]+vfluxbot[ij+jj])/(Constants::kappa*z[k]*ustar[ij])*most::phim(z[k]/obuk[ij]), 2) );
                // add a small number to avoid zero divisions
                strain2[ijk] += Constants::dsmall;  
            }
            else
            {
                strain2[ijk] = 2.*(
                    // du/dx + du/dx
                    + pow((u[ijk+ii]-u[ijk])*dxi, 2)
                    // dv/dy + dv/dy
                    + pow((v[ijk+jj]-v[ijk])*dyi, 2)
                    // dw/dz + dw/dz
                    + pow((w[ijk+kk]-w[ijk])*dzi[k], 2)
                    // du/dy + dv/dx
                    + 0.125*pow((u[ijk      ]-u[ijk   -jj])*dyi  + (v[ijk      ]-v[ijk-ii   ])*dxi, 2)
                    + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-jj])*dyi  + (v[ijk+ii   ]-v[ijk      ])*dxi, 2)
                    + 0.125*pow((u[ijk   +jj]-u[ijk      ])*dyi  + (v[ijk   +jj]-v[ijk-ii+jj])*dxi, 2)
                    + 0.125*pow((u[ijk+ii+jj]-u[ijk+ii   ])*dyi  + (v[ijk+ii+jj]-v[ijk   +jj])*dxi, 2)
                    // du/dz + dw/dx
                    + 0.125*pow((u[ijk      ]-u[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-ii   ])*dxi, 2)
                    + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-kk])*dzhi[k  ] + (w[ijk+ii   ]-w[ijk      ])*dxi, 2)
                    + 0.125*pow((u[ijk   +kk]-u[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-ii+kk])*dxi, 2)
                    + 0.125*pow((u[ijk+ii+kk]-u[ijk+ii   ])*dzhi[k+1] + (w[ijk+ii+kk]-w[ijk   +kk])*dxi, 2)
                    // dv/dz + dw/dy
                    + 0.125*pow((v[ijk      ]-v[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-jj   ])*dyi, 2)
                    + 0.125*pow((v[ijk+jj   ]-v[ijk+jj-kk])*dzhi[k  ] + (w[ijk+jj   ]-w[ijk      ])*dyi, 2)
                    + 0.125*pow((v[ijk   +kk]-v[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-jj+kk])*dyi, 2)
                    + 0.125*pow((v[ijk+jj+kk]-v[ijk+jj   ])*dzhi[k+1] + (w[ijk+jj+kk]-w[ijk   +kk])*dyi, 2) );
                // add a small number to avoid zero divisions
                strain2[ijk] += Constants::dsmall;
            }
        }
    }

    template<typename TF> __global__ 
    void evisc_g(double* __restrict__ evisc, double* __restrict__ N2,
                 double* __restrict__ bfluxbot, double* __restrict__ ustar, double* __restrict__ obuk,
                 double* __restrict__ mlen,
                 const double tPri, const double z0m, const double zsl,
                 const int istart,  const int jstart, const int kstart,
                 const int iend,    const int jend,   const int kend, 
                 const int jj,      const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + k*kk;

            if (k == kstart)
            {
                // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
                double RitPrratio = -bfluxbot[ij]/(Constants::kappa*zsl*ustar[ij])*most::phih(zsl/obuk[ij]) / evisc[ijk] * tPri;
                RitPrratio        = fmin(RitPrratio, 1.-Constants::dsmall);
                evisc[ijk]        = mlen[k] * sqrt(evisc[ijk] * (1.-RitPrratio));
            }
            else
            {
                // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
                double RitPrratio = N2[ijk] / evisc[ijk] * tPri;
                RitPrratio        = fmin(RitPrratio, 1.-Constants::dsmall);
                evisc[ijk]        = mlen[k] * sqrt(evisc[ijk] * (1.-RitPrratio));
            }
        }
    }

    template<typename TF> __global__ 
    void evisc_neutral_g(double* __restrict__ evisc, double* __restrict__ mlen,
                         const int istart, const int jstart, const int kstart, 
                         const int iend,   const int jend,   const int kend, 
                         const int jj,     const int kk)

    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            evisc[ijk]    = mlen[k] * sqrt(evisc[ijk]);
        }
    }

    template<typename TF> __global__ 
    void diff_uvw_g(double* __restrict__ ut, double* __restrict__ vt, double* __restrict__ wt, 
                    double* __restrict__ evisc,
                    double* __restrict__ u, double* __restrict__ v, double* __restrict__ w,
                    double* __restrict__ fluxbotu, double* __restrict__ fluxtopu, 
                    double* __restrict__ fluxbotv, double* __restrict__ fluxtopv, 
                    double* __restrict__ dzi, double* __restrict__ dzhi, const double dxi, const double dyi,
                    double* __restrict__ rhoref, double* __restrict__ rhorefh, 
                    const int istart, const int jstart, const int kstart, 
                    const int iend,   const int jend,   const int kend, 
                    const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ii  = 1;
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + k*kk;

            // U
            const double eviscnu = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
            const double eviscsu = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
            const double evisctu = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
            const double eviscbu = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);

            // V
            const double eviscev = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
            const double eviscwv = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
            const double evisctv = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
            const double eviscbv = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);

            // W
            const double eviscew = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+ii-kk] + evisc[ijk+ii]);
            const double eviscww = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-ii] + evisc[ijk   -kk] + evisc[ijk   ]);
            const double eviscnw = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+jj-kk] + evisc[ijk+jj]);
            const double eviscsw = 0.25*(evisc[ijk-jj-kk] + evisc[ijk-jj] + evisc[ijk   -kk] + evisc[ijk   ]);

            if (k == kstart)
            {
                ut[ijk] +=
                    // du/dx + du/dx
                    + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
                       - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
                    // du/dy + dv/dx
                    + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
                       - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
                    // du/dz + dw/dx
                    + (  rhorefh[kstart+1] * evisctu*((u[ijk+kk]-u[ijk   ])* dzhi[kstart+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
                       + rhorefh[kstart  ] * fluxbotu[ij] ) / rhoref[kstart] * dzi[kstart];

                vt[ijk] +=
                    // dv/dx + du/dy
                    + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
                       - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
                    // dv/dy + dv/dy
                    + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
                       - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
                    // dv/dz + dw/dy
                    + (  rhorefh[k+1] * evisctv*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
                       + rhorefh[k  ] * fluxbotv[ij] ) / rhoref[k] * dzi[k];
            }
            else if (k == kend-1)
            {
                ut[ijk] +=
                    // du/dx + du/dx
                    + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
                       - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
                    // du/dy + dv/dx
                    + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
                       - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
                    // du/dz + dw/dx
                    + (- rhorefh[kend  ] * fluxtopu[ij]
                       - rhorefh[kend-1] * eviscbu*((u[ijk   ]-u[ijk-kk])* dzhi[kend-1] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[kend-1] * dzi[kend-1];

                vt[ijk] +=
                    // dv/dx + du/dy
                    + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
                       - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
                    // dv/dy + dv/dy
                    + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
                       - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
                    // dv/dz + dw/dy
                    + (- rhorefh[k  ] * fluxtopv[ij]
                       - rhorefh[k-1] * eviscbv*((v[ijk   ]-v[ijk-kk])*dzhi[k-1] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k-1] * dzi[k-1];

                wt[ijk] +=
                    // dw/dx + du/dz
                    + (  eviscew*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
                       - eviscww*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
                    // dw/dy + dv/dz
                    + (  eviscnw*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
                       - eviscsw*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
                    // dw/dz + dw/dz
                    + (  rhoref[k  ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzi[k  ]
                       - rhoref[k-1] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzi[k-1] ) / rhorefh[k] * 2.* dzhi[k];
            }
            else
            {
                ut[ijk] +=
                    // du/dx + du/dx
                    + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
                       - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
                    // du/dy + dv/dx
                    + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
                       - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
                    // du/dz + dw/dx
                    + (  rhorefh[k+1] * evisctu*((u[ijk+kk]-u[ijk   ])* dzhi[k+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
                       - rhorefh[k  ] * eviscbu*((u[ijk   ]-u[ijk-kk])* dzhi[k  ] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[k] * dzi[k];

                vt[ijk] +=
                    // dv/dx + du/dy
                    + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
                       - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
                    // dv/dy + dv/dy
                    + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
                       - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
                    // dv/dz + dw/dy
                    + (  rhorefh[k+1] * evisctv*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
                       - rhorefh[k  ] * eviscbv*((v[ijk   ]-v[ijk-kk])*dzhi[k  ] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k] * dzi[k];

                wt[ijk] +=
                    // dw/dx + du/dz
                    + (  eviscew*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
                       - eviscww*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
                    // dw/dy + dv/dz
                    + (  eviscnw*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
                       - eviscsw*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
                    // dw/dz + dw/dz
                    + (  rhoref[k  ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzi[k  ]
                       - rhoref[k-1] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzi[k-1] ) / rhorefh[k] * 2.* dzhi[k];
            }
        }
    }

    template<typename TF> __global__ 
    void diff_c_g(double* __restrict__ at, double* __restrict__ a, double* __restrict__ evisc,
                  double* __restrict__ fluxbot, double* __restrict__ fluxtop, 
                  double* __restrict__ dzi, double* __restrict__ dzhi, const double dxidxi, const double dyidyi,
                  double* __restrict__ rhoref, double* __restrict__ rhorefh, const double tPri, 
                  const int istart, const int jstart, const int kstart, 
                  const int iend,   const int jend,   const int kend, 
                  const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ii  = 1;
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + k*kk;

            if (k == kstart)
            {
                const double evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
                const double eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
                const double eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
                const double eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
                const double evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;

                at[ijk] +=
                    + (  evisce*(a[ijk+ii]-a[ijk   ]) 
                       - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
                    + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
                       - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
                    + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
                       + rhorefh[k  ] * fluxbot[ij] ) / rhoref[k] * dzi[k];
            }
            else if (k == kend-1)
            {
                const double evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
                const double eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
                const double eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
                const double eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
                const double eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;

                at[ijk] +=
                    + (  evisce*(a[ijk+ii]-a[ijk   ]) 
                       - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
                    + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
                       - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
                    + (- rhorefh[k  ] * fluxtop[ij]
                       - rhorefh[k-1] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k-1] ) / rhoref[k-1] * dzi[k-1];
            }
            else
            {
                const double evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
                const double eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
                const double eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
                const double eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
                const double evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;
                const double eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;

                at[ijk] +=
                    + (  evisce*(a[ijk+ii]-a[ijk   ]) 
                       - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
                    + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
                       - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
                    + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
                       - rhorefh[k  ] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k]  ) / rhoref[k] * dzi[k];
            }
        }
    }

    template<typename TF> __global__ 
    void calc_dnmul_g(double* __restrict__ dnmul, double* __restrict__ evisc, 
                      double* __restrict__ dzi, double tPrfac, const double dxidxi, const double dyidyi,
                      const int istart, const int jstart, const int kstart, 
                      const int iend,   const int jend,   const int kend, 
                      const int jj,     const int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
        const int k = blockIdx.z + kstart;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            dnmul[ijk] = fabs(tPrfac*evisc[ijk]*(dxidxi + dyidyi + dzi[k]*dzi[k]));
        }
    }
}

/* Calculate the mixing length (mlen) offline, and put on GPU */
#ifdef USECUDA
template<typename TF>
void Diff_smag_2<TF>::prepare_device()
{
    Boundary_surface *boundaryptr = static_cast<Boundary_surface *>(model->boundary);

    const double n=2.;
    double mlen0;
    double *mlen = new double[grid->kcells];
    for (int k=0; k<grid->kcells; ++k) 
    {
        mlen0   = cs * pow(grid->dx*grid->dy*grid->dz[k], 1./3.);
        mlen[k] = pow(pow(1./(1./pow(mlen0, n) + 1./(pow(Constants::kappa*(grid->z[k]+boundaryptr->z0m), n))), 1./n), 2);
    }

    const int nmemsize = grid->kcells*sizeof(double);
    cuda_safe_call(hipMalloc(&mlen_g, nmemsize));
    cuda_safe_call(hipMemcpy(mlen_g, mlen, nmemsize, hipMemcpyHostToDevice));

    delete[] mlen;
}
#endif

template<typename TF>
void Diff_smag_2<TF>::clear_device()
{
    cuda_safe_call(hipFree(mlen_g));
}

#ifdef USECUDA
template<typename TF>
void Diff_smag_2<TF>::exec_viscosity()
{
    // do a cast because the base boundary class does not have the MOST related variables
    Boundary_surface *boundaryptr = static_cast<Boundary_surface *>(model->boundary);

    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kcells);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    // Calculate total strain rate
    strain2_g<<<gridGPU, blockGPU>>>(
        &fields->sd["evisc"]->data_g[offs], 
        &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
        &fields->u->datafluxbot_g[offs],  &fields->v->datafluxbot_g[offs],
        &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
        grid->z_g, grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
        grid->istart,  grid->jstart, grid->kstart, 
        grid->iend,    grid->jend,   grid->kend,
        grid->icellsp, grid->ijcellsp);  
    cuda_check_error();

    // start with retrieving the stability information
    if (model->thermo->get_switch() == "0")
    {
        evisc_neutral_g<<<gridGPU, blockGPU>>>(
            &fields->sd["evisc"]->data_g[offs], mlen_g,
            grid->istart,  grid->jstart, grid->kstart, 
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);  
        cuda_check_error();

        grid->boundary_cyclic_g(&fields->sd["evisc"]->data_g[offs]);
    }
    // assume buoyancy calculation is needed
    else
    {
        // store the buoyancyflux in datafluxbot of tmp1
        model->thermo->get_buoyancy_fluxbot(fields->atmp["tmp1"]);
        // store the Brunt-vaisala frequency in data of tmp1 
        model->thermo->get_thermo_field(fields->atmp["tmp1"], fields->atmp["tmp2"], "N2", false);

        // Calculate eddy viscosity
        double tPri = 1./tPr;
        evisc_g<<<gridGPU, blockGPU>>>(
            &fields->sd["evisc"]->data_g[offs], &fields->atmp["tmp1"]->data_g[offs], 
            &fields->atmp["tmp1"]->datafluxbot_g[offs], &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
            mlen_g, tPri, boundaryptr->z0m, grid->z[grid->kstart],
            grid->istart,  grid->jstart, grid->kstart, 
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);  
        cuda_check_error();

        grid->boundary_cyclic_g(&fields->sd["evisc"]->data_g[offs]);
    }
}
#endif

#ifdef USECUDA
template<typename TF>
void Diff_smag_2<TF>::exec()
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;
    const double dxidxi = 1./(grid->dx * grid->dx);
    const double dyidyi = 1./(grid->dy * grid->dy);
    const double tPri = 1./tPr;

    diff_uvw_g<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
            &fields->sd["evisc"]->data_g[offs], 
            &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
            &fields->u->datafluxbot_g[offs], &fields->u->datafluxtop_g[offs],
            &fields->v->datafluxbot_g[offs], &fields->v->datafluxtop_g[offs],
            grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
            fields->rhoref_g, fields->rhorefh_g,
            grid->istart,  grid->jstart, grid->kstart, 
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);  
    cuda_check_error();

    for (FieldMap::const_iterator it = fields->st.begin(); it!=fields->st.end(); ++it)
        diff_c_g<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], &fields->sp[it->first]->data_g[offs], &fields->sd["evisc"]->data_g[offs], 
                &fields->sp[it->first]->datafluxbot_g[offs], &fields->sp[it->first]->datafluxtop_g[offs],
                grid->dzi_g, grid->dzhi_g, dxidxi, dyidyi,
                fields->rhoref_g, fields->rhorefh_g, tPri,
                grid->istart,  grid->jstart, grid->kstart, 
                grid->iend,    grid->jend,   grid->kend,
                grid->icellsp, grid->ijcellsp);  
    cuda_check_error();
}
#endif

#ifdef USECUDA
template<typename TF>
unsigned long Diff_smag_2<TF>::get_time_limit(unsigned long idt, double dt)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    const double dxidxi = 1./(grid->dx * grid->dx);
    const double dyidyi = 1./(grid->dy * grid->dy);
    const double tPrfac = std::min(1., tPr);

    // Calculate dnmul in tmp1 field
    calc_dnmul_g<<<gridGPU, blockGPU>>>(
        &fields->atmp["tmp1"]->data_g[offs], &fields->sd["evisc"]->data_g[offs],
        grid->dzi_g, tPrfac, dxidxi, dyidyi,  
        grid->istart,  grid->jstart, grid->kstart, 
        grid->iend,    grid->jend,   grid->kend,
        grid->icellsp, grid->ijcellsp);  
    cuda_check_error();

    // Get maximum from tmp1 field
    double dnmul = grid->get_max_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g); 
    dnmul = std::max(Constants::dsmall, dnmul);
    const unsigned long idtlim = idt * dnmax/(dnmul*dt);

    return idtlim;
}
#endif

#ifdef USECUDA
template<typename TF>
double Diff_smag_2<TF>::get_dn(double dt)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    const double dxidxi = 1./(grid->dx * grid->dx);
    const double dyidyi = 1./(grid->dy * grid->dy);
    const double tPrfac = std::min(1., tPr);

    // Calculate dnmul in tmp1 field
    calc_dnmul_g<<<gridGPU, blockGPU>>>(
        &fields->atmp["tmp1"]->data_g[offs], &fields->sd["evisc"]->data_g[offs],
        grid->dzi_g, tPrfac, dxidxi, dyidyi,  
        grid->istart,  grid->jstart, grid->kstart, 
        grid->iend,    grid->jend,   grid->kend,
        grid->icellsp, grid->ijcellsp);  
    cuda_check_error();

    // Get maximum from tmp1 field
    double dnmul = grid->get_max_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g); 

    return dnmul*dt;
}
#endif
