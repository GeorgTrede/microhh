#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "advec_4.h"
#include "grid.h"
#include "fields.h"
#include "defines.h"
#include "fd.h"

using namespace fd::o4;

__global__ void advec_4_advecu(double * __restrict__ ut, double * __restrict__ u, 
                               double * __restrict__ v, double * __restrict__ w,
                               double * __restrict__ dzi4, double dxi, double dyi, 
                               int jj, int kk,
                               int istart, int jstart, int kstart,
                               int iend,   int jend,   int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  int ii1 = 1;
  int ii2 = 2;
  int ii3 = 3;
  int jj1 = 1*jj;
  int jj2 = 2*jj;
  int jj3 = 3*jj;
  int kk1 = 1*kk;
  int kk2 = 2*kk;
  int kk3 = 3*kk;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      ut[ijk] -= ( cg0*((ci0*u[ijk-ii3] + ci1*u[ijk-ii2] + ci2*u[ijk-ii1] + ci3*u[ijk    ]) * (ci0*u[ijk-ii3] + ci1*u[ijk-ii2] + ci2*u[ijk-ii1] + ci3*u[ijk    ]))
                 + cg1*((ci0*u[ijk-ii2] + ci1*u[ijk-ii1] + ci2*u[ijk    ] + ci3*u[ijk+ii1]) * (ci0*u[ijk-ii2] + ci1*u[ijk-ii1] + ci2*u[ijk    ] + ci3*u[ijk+ii1]))
                 + cg2*((ci0*u[ijk-ii1] + ci1*u[ijk    ] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2]) * (ci0*u[ijk-ii1] + ci1*u[ijk    ] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2]))
                 + cg3*((ci0*u[ijk    ] + ci1*u[ijk+ii1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii3]) * (ci0*u[ijk    ] + ci1*u[ijk+ii1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii3])) ) * cgi*dxi;

      ut[ijk] -= ( cg0*((ci0*v[ijk-ii2-jj1] + ci1*v[ijk-ii1-jj1] + ci2*v[ijk-jj1] + ci3*v[ijk+ii1-jj1]) * (ci0*u[ijk-jj3] + ci1*u[ijk-jj2] + ci2*u[ijk-jj1] + ci3*u[ijk    ]))
                 + cg1*((ci0*v[ijk-ii2    ] + ci1*v[ijk-ii1    ] + ci2*v[ijk    ] + ci3*v[ijk+ii1    ]) * (ci0*u[ijk-jj2] + ci1*u[ijk-jj1] + ci2*u[ijk    ] + ci3*u[ijk+jj1]))
                 + cg2*((ci0*v[ijk-ii2+jj1] + ci1*v[ijk-ii1+jj1] + ci2*v[ijk+jj1] + ci3*v[ijk+ii1+jj1]) * (ci0*u[ijk-jj1] + ci1*u[ijk    ] + ci2*u[ijk+jj1] + ci3*u[ijk+jj2]))
                 + cg3*((ci0*v[ijk-ii2+jj2] + ci1*v[ijk-ii1+jj2] + ci2*v[ijk+jj2] + ci3*v[ijk+ii1+jj2]) * (ci0*u[ijk    ] + ci1*u[ijk+jj1] + ci2*u[ijk+jj2] + ci3*u[ijk+jj3])) ) * cgi*dyi;

      ut[ijk] -= ( cg0*((ci0*w[ijk-ii2-kk1] + ci1*w[ijk-ii1-kk1] + ci2*w[ijk-kk1] + ci3*w[ijk+ii1-kk1]) * (bi0*u[ijk-kk2] + bi1*u[ijk-kk1] + bi2*u[ijk    ] + bi3*u[ijk+kk1]))
                 + cg1*((ci0*w[ijk-ii2    ] + ci1*w[ijk-ii1    ] + ci2*w[ijk    ] + ci3*w[ijk+ii1    ]) * (ci0*u[ijk-kk2] + ci1*u[ijk-kk1] + ci2*u[ijk    ] + ci3*u[ijk+kk1]))
                 + cg2*((ci0*w[ijk-ii2+kk1] + ci1*w[ijk-ii1+kk1] + ci2*w[ijk+kk1] + ci3*w[ijk+ii1+kk1]) * (ci0*u[ijk-kk1] + ci1*u[ijk    ] + ci2*u[ijk+kk1] + ci3*u[ijk+kk2]))
                 + cg3*((ci0*w[ijk-ii2+kk2] + ci1*w[ijk-ii1+kk2] + ci2*w[ijk+kk2] + ci3*w[ijk+ii1+kk2]) * (ci0*u[ijk    ] + ci1*u[ijk+kk1] + ci2*u[ijk+kk2] + ci3*u[ijk+kk3])) )
                 * dzi4[k];
    }
    else if(k == kend-1)
    {
      ut[ijk] -= ( cg0*((ci0*u[ijk-ii3] + ci1*u[ijk-ii2] + ci2*u[ijk-ii1] + ci3*u[ijk    ]) * (ci0*u[ijk-ii3] + ci1*u[ijk-ii2] + ci2*u[ijk-ii1] + ci3*u[ijk    ]))
                 + cg1*((ci0*u[ijk-ii2] + ci1*u[ijk-ii1] + ci2*u[ijk    ] + ci3*u[ijk+ii1]) * (ci0*u[ijk-ii2] + ci1*u[ijk-ii1] + ci2*u[ijk    ] + ci3*u[ijk+ii1]))
                 + cg2*((ci0*u[ijk-ii1] + ci1*u[ijk    ] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2]) * (ci0*u[ijk-ii1] + ci1*u[ijk    ] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2]))
                 + cg3*((ci0*u[ijk    ] + ci1*u[ijk+ii1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii3]) * (ci0*u[ijk    ] + ci1*u[ijk+ii1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii3])) ) * cgi*dxi;

      ut[ijk] -= ( cg0*((ci0*v[ijk-ii2-jj1] + ci1*v[ijk-ii1-jj1] + ci2*v[ijk-jj1] + ci3*v[ijk+ii1-jj1]) * (ci0*u[ijk-jj3] + ci1*u[ijk-jj2] + ci2*u[ijk-jj1] + ci3*u[ijk    ]))
                 + cg1*((ci0*v[ijk-ii2    ] + ci1*v[ijk-ii1    ] + ci2*v[ijk    ] + ci3*v[ijk+ii1    ]) * (ci0*u[ijk-jj2] + ci1*u[ijk-jj1] + ci2*u[ijk    ] + ci3*u[ijk+jj1]))
                 + cg2*((ci0*v[ijk-ii2+jj1] + ci1*v[ijk-ii1+jj1] + ci2*v[ijk+jj1] + ci3*v[ijk+ii1+jj1]) * (ci0*u[ijk-jj1] + ci1*u[ijk    ] + ci2*u[ijk+jj1] + ci3*u[ijk+jj2]))
                 + cg3*((ci0*v[ijk-ii2+jj2] + ci1*v[ijk-ii1+jj2] + ci2*v[ijk+jj2] + ci3*v[ijk+ii1+jj2]) * (ci0*u[ijk    ] + ci1*u[ijk+jj1] + ci2*u[ijk+jj2] + ci3*u[ijk+jj3])) ) * cgi*dyi;

      ut[ijk] -= ( cg0*((ci0*w[ijk-ii2-kk1] + ci1*w[ijk-ii1-kk1] + ci2*w[ijk-kk1] + ci3*w[ijk+ii1-kk1]) * (ci0*u[ijk-kk3] + ci1*u[ijk-kk2] + ci2*u[ijk-kk1] + ci3*u[ijk    ]))
                 + cg1*((ci0*w[ijk-ii2    ] + ci1*w[ijk-ii1    ] + ci2*w[ijk    ] + ci3*w[ijk+ii1    ]) * (ci0*u[ijk-kk2] + ci1*u[ijk-kk1] + ci2*u[ijk    ] + ci3*u[ijk+kk1]))
                 + cg2*((ci0*w[ijk-ii2+kk1] + ci1*w[ijk-ii1+kk1] + ci2*w[ijk+kk1] + ci3*w[ijk+ii1+kk1]) * (ci0*u[ijk-kk1] + ci1*u[ijk    ] + ci2*u[ijk+kk1] + ci3*u[ijk+kk2]))
                 + cg3*((ci0*w[ijk-ii2+kk2] + ci1*w[ijk-ii1+kk2] + ci2*w[ijk+kk2] + ci3*w[ijk+ii1+kk2]) * (ti0*u[ijk-kk1] + ti1*u[ijk    ] + ti2*u[ijk+kk1] + ti3*u[ijk+kk2])) )
                 * dzi4[k];
    }
    else
    {
      ut[ijk] -= ( cg0*((ci0*u[ijk-ii3] + ci1*u[ijk-ii2] + ci2*u[ijk-ii1] + ci3*u[ijk    ]) * (ci0*u[ijk-ii3] + ci1*u[ijk-ii2] + ci2*u[ijk-ii1] + ci3*u[ijk    ]))
                 + cg1*((ci0*u[ijk-ii2] + ci1*u[ijk-ii1] + ci2*u[ijk    ] + ci3*u[ijk+ii1]) * (ci0*u[ijk-ii2] + ci1*u[ijk-ii1] + ci2*u[ijk    ] + ci3*u[ijk+ii1]))
                 + cg2*((ci0*u[ijk-ii1] + ci1*u[ijk    ] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2]) * (ci0*u[ijk-ii1] + ci1*u[ijk    ] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2]))
                 + cg3*((ci0*u[ijk    ] + ci1*u[ijk+ii1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii3]) * (ci0*u[ijk    ] + ci1*u[ijk+ii1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii3])) ) * cgi*dxi;

      ut[ijk] -= ( cg0*((ci0*v[ijk-ii2-jj1] + ci1*v[ijk-ii1-jj1] + ci2*v[ijk-jj1] + ci3*v[ijk+ii1-jj1]) * (ci0*u[ijk-jj3] + ci1*u[ijk-jj2] + ci2*u[ijk-jj1] + ci3*u[ijk    ]))
                 + cg1*((ci0*v[ijk-ii2    ] + ci1*v[ijk-ii1    ] + ci2*v[ijk    ] + ci3*v[ijk+ii1    ]) * (ci0*u[ijk-jj2] + ci1*u[ijk-jj1] + ci2*u[ijk    ] + ci3*u[ijk+jj1]))
                 + cg2*((ci0*v[ijk-ii2+jj1] + ci1*v[ijk-ii1+jj1] + ci2*v[ijk+jj1] + ci3*v[ijk+ii1+jj1]) * (ci0*u[ijk-jj1] + ci1*u[ijk    ] + ci2*u[ijk+jj1] + ci3*u[ijk+jj2]))
                 + cg3*((ci0*v[ijk-ii2+jj2] + ci1*v[ijk-ii1+jj2] + ci2*v[ijk+jj2] + ci3*v[ijk+ii1+jj2]) * (ci0*u[ijk    ] + ci1*u[ijk+jj1] + ci2*u[ijk+jj2] + ci3*u[ijk+jj3])) ) * cgi*dyi;

      ut[ijk] -= ( cg0*((ci0*w[ijk-ii2-kk1] + ci1*w[ijk-ii1-kk1] + ci2*w[ijk-kk1] + ci3*w[ijk+ii1-kk1]) * (ci0*u[ijk-kk3] + ci1*u[ijk-kk2] + ci2*u[ijk-kk1] + ci3*u[ijk    ]))
                 + cg1*((ci0*w[ijk-ii2    ] + ci1*w[ijk-ii1    ] + ci2*w[ijk    ] + ci3*w[ijk+ii1    ]) * (ci0*u[ijk-kk2] + ci1*u[ijk-kk1] + ci2*u[ijk    ] + ci3*u[ijk+kk1]))
                 + cg2*((ci0*w[ijk-ii2+kk1] + ci1*w[ijk-ii1+kk1] + ci2*w[ijk+kk1] + ci3*w[ijk+ii1+kk1]) * (ci0*u[ijk-kk1] + ci1*u[ijk    ] + ci2*u[ijk+kk1] + ci3*u[ijk+kk2]))
                 + cg3*((ci0*w[ijk-ii2+kk2] + ci1*w[ijk-ii1+kk2] + ci2*w[ijk+kk2] + ci3*w[ijk+ii1+kk2]) * (ci0*u[ijk    ] + ci1*u[ijk+kk1] + ci2*u[ijk+kk2] + ci3*u[ijk+kk3])) )
                 * dzi4[k];
    }
  }
}

__global__ void advec_4_advecv(double * __restrict__ vt, double * __restrict__ u, 
                               double * __restrict__ v, double * __restrict__ w,
                               double * __restrict__ dzi4, double dxi, double dyi, 
                               int jj, int kk,
                               int istart, int jstart, int kstart,
                               int iend,   int jend,   int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  int ii1 = 1;
  int ii2 = 2;
  int ii3 = 3;
  int jj1 = 1*jj;
  int jj2 = 2*jj;
  int jj3 = 3*jj;
  int kk1 = 1*kk;
  int kk2 = 2*kk;
  int kk3 = 3*kk;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    if(k == kstart)
    {
      vt[ijk] -= ( cg0*((ci0*u[ijk-ii1-jj2] + ci1*u[ijk-ii1-jj1] + ci2*u[ijk-ii1] + ci3*u[ijk-ii1+jj1]) * (ci0*v[ijk-ii3] + ci1*v[ijk-ii2] + ci2*v[ijk-ii1] + ci3*v[ijk    ]))
                 + cg1*((ci0*u[ijk    -jj2] + ci1*u[ijk    -jj1] + ci2*u[ijk    ] + ci3*u[ijk    +jj1]) * (ci0*v[ijk-ii2] + ci1*v[ijk-ii1] + ci2*v[ijk    ] + ci3*v[ijk+ii1]))
                 + cg2*((ci0*u[ijk+ii1-jj2] + ci1*u[ijk+ii1-jj1] + ci2*u[ijk+ii1] + ci3*u[ijk+ii1+jj1]) * (ci0*v[ijk-ii1] + ci1*v[ijk    ] + ci2*v[ijk+ii1] + ci3*v[ijk+ii2]))
                 + cg3*((ci0*u[ijk+ii2-jj2] + ci1*u[ijk+ii2-jj1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii2+jj1]) * (ci0*v[ijk    ] + ci1*v[ijk+ii1] + ci2*v[ijk+ii2] + ci3*v[ijk+ii3])) ) * cgi*dxi;

      vt[ijk] -= ( cg0*((ci0*v[ijk-jj3] + ci1*v[ijk-jj2] + ci2*v[ijk-jj1] + ci3*v[ijk    ]) * (ci0*v[ijk-jj3] + ci1*v[ijk-jj2] + ci2*v[ijk-jj1] + ci3*v[ijk    ]))
                 + cg1*((ci0*v[ijk-jj2] + ci1*v[ijk-jj1] + ci2*v[ijk    ] + ci3*v[ijk+jj1]) * (ci0*v[ijk-jj2] + ci1*v[ijk-jj1] + ci2*v[ijk    ] + ci3*v[ijk+jj1]))
                 + cg2*((ci0*v[ijk-jj1] + ci1*v[ijk    ] + ci2*v[ijk+jj1] + ci3*v[ijk+jj2]) * (ci0*v[ijk-jj1] + ci1*v[ijk    ] + ci2*v[ijk+jj1] + ci3*v[ijk+jj2]))
                 + cg3*((ci0*v[ijk    ] + ci1*v[ijk+jj1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj3]) * (ci0*v[ijk    ] + ci1*v[ijk+jj1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj3])) ) * cgi*dyi;

      vt[ijk] -= ( cg0*((ci0*w[ijk-jj2-kk1] + ci1*w[ijk-jj1-kk1] + ci2*w[ijk-kk1] + ci3*w[ijk+jj1-kk1]) * (bi0*v[ijk-kk2] + bi1*v[ijk-kk1] + bi2*v[ijk    ] + bi3*v[ijk+kk1]))
                 + cg1*((ci0*w[ijk-jj2    ] + ci1*w[ijk-jj1    ] + ci2*w[ijk    ] + ci3*w[ijk+jj1    ]) * (ci0*v[ijk-kk2] + ci1*v[ijk-kk1] + ci2*v[ijk    ] + ci3*v[ijk+kk1]))
                 + cg2*((ci0*w[ijk-jj2+kk1] + ci1*w[ijk-jj1+kk1] + ci2*w[ijk+kk1] + ci3*w[ijk+jj1+kk1]) * (ci0*v[ijk-kk1] + ci1*v[ijk    ] + ci2*v[ijk+kk1] + ci3*v[ijk+kk2]))
                 + cg3*((ci0*w[ijk-jj2+kk2] + ci1*w[ijk-jj1+kk2] + ci2*w[ijk+kk2] + ci3*w[ijk+jj1+kk2]) * (ci0*v[ijk    ] + ci1*v[ijk+kk1] + ci2*v[ijk+kk2] + ci3*v[ijk+kk3])) )
                 * dzi4[k];
    }
    else if(k == kend-1)
    {
      vt[ijk] -= ( cg0*((ci0*u[ijk-ii1-jj2] + ci1*u[ijk-ii1-jj1] + ci2*u[ijk-ii1] + ci3*u[ijk-ii1+jj1]) * (ci0*v[ijk-ii3] + ci1*v[ijk-ii2] + ci2*v[ijk-ii1] + ci3*v[ijk    ]))
                 + cg1*((ci0*u[ijk    -jj2] + ci1*u[ijk    -jj1] + ci2*u[ijk    ] + ci3*u[ijk    +jj1]) * (ci0*v[ijk-ii2] + ci1*v[ijk-ii1] + ci2*v[ijk    ] + ci3*v[ijk+ii1]))
                 + cg2*((ci0*u[ijk+ii1-jj2] + ci1*u[ijk+ii1-jj1] + ci2*u[ijk+ii1] + ci3*u[ijk+ii1+jj1]) * (ci0*v[ijk-ii1] + ci1*v[ijk    ] + ci2*v[ijk+ii1] + ci3*v[ijk+ii2]))
                 + cg3*((ci0*u[ijk+ii2-jj2] + ci1*u[ijk+ii2-jj1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii2+jj1]) * (ci0*v[ijk    ] + ci1*v[ijk+ii1] + ci2*v[ijk+ii2] + ci3*v[ijk+ii3])) ) * cgi*dxi;

      vt[ijk] -= ( cg0*((ci0*v[ijk-jj3] + ci1*v[ijk-jj2] + ci2*v[ijk-jj1] + ci3*v[ijk    ]) * (ci0*v[ijk-jj3] + ci1*v[ijk-jj2] + ci2*v[ijk-jj1] + ci3*v[ijk    ]))
                 + cg1*((ci0*v[ijk-jj2] + ci1*v[ijk-jj1] + ci2*v[ijk    ] + ci3*v[ijk+jj1]) * (ci0*v[ijk-jj2] + ci1*v[ijk-jj1] + ci2*v[ijk    ] + ci3*v[ijk+jj1]))
                 + cg2*((ci0*v[ijk-jj1] + ci1*v[ijk    ] + ci2*v[ijk+jj1] + ci3*v[ijk+jj2]) * (ci0*v[ijk-jj1] + ci1*v[ijk    ] + ci2*v[ijk+jj1] + ci3*v[ijk+jj2]))
                 + cg3*((ci0*v[ijk    ] + ci1*v[ijk+jj1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj3]) * (ci0*v[ijk    ] + ci1*v[ijk+jj1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj3])) ) * cgi*dyi;

      vt[ijk] -= ( cg0*((ci0*w[ijk-jj2-kk1] + ci1*w[ijk-jj1-kk1] + ci2*w[ijk-kk1] + ci3*w[ijk+jj1-kk1]) * (ci0*v[ijk-kk3] + ci1*v[ijk-kk2] + ci2*v[ijk-kk1] + ci3*v[ijk    ]))
                 + cg1*((ci0*w[ijk-jj2    ] + ci1*w[ijk-jj1    ] + ci2*w[ijk    ] + ci3*w[ijk+jj1    ]) * (ci0*v[ijk-kk2] + ci1*v[ijk-kk1] + ci2*v[ijk    ] + ci3*v[ijk+kk1]))
                 + cg2*((ci0*w[ijk-jj2+kk1] + ci1*w[ijk-jj1+kk1] + ci2*w[ijk+kk1] + ci3*w[ijk+jj1+kk1]) * (ci0*v[ijk-kk1] + ci1*v[ijk    ] + ci2*v[ijk+kk1] + ci3*v[ijk+kk2]))
                 + cg3*((ci0*w[ijk-jj2+kk2] + ci1*w[ijk-jj1+kk2] + ci2*w[ijk+kk2] + ci3*w[ijk+jj1+kk2]) * (ti0*v[ijk-kk1] + ti1*v[ijk    ] + ti2*v[ijk+kk1] + ti3*v[ijk+kk2])) )
                 * dzi4[k];
    }
    else
    {
      vt[ijk] -= ( cg0*((ci0*u[ijk-ii1-jj2] + ci1*u[ijk-ii1-jj1] + ci2*u[ijk-ii1] + ci3*u[ijk-ii1+jj1]) * (ci0*v[ijk-ii3] + ci1*v[ijk-ii2] + ci2*v[ijk-ii1] + ci3*v[ijk    ]))
                 + cg1*((ci0*u[ijk    -jj2] + ci1*u[ijk    -jj1] + ci2*u[ijk    ] + ci3*u[ijk    +jj1]) * (ci0*v[ijk-ii2] + ci1*v[ijk-ii1] + ci2*v[ijk    ] + ci3*v[ijk+ii1]))
                 + cg2*((ci0*u[ijk+ii1-jj2] + ci1*u[ijk+ii1-jj1] + ci2*u[ijk+ii1] + ci3*u[ijk+ii1+jj1]) * (ci0*v[ijk-ii1] + ci1*v[ijk    ] + ci2*v[ijk+ii1] + ci3*v[ijk+ii2]))
                 + cg3*((ci0*u[ijk+ii2-jj2] + ci1*u[ijk+ii2-jj1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii2+jj1]) * (ci0*v[ijk    ] + ci1*v[ijk+ii1] + ci2*v[ijk+ii2] + ci3*v[ijk+ii3])) ) * cgi*dxi;

      vt[ijk] -= ( cg0*((ci0*v[ijk-jj3] + ci1*v[ijk-jj2] + ci2*v[ijk-jj1] + ci3*v[ijk    ]) * (ci0*v[ijk-jj3] + ci1*v[ijk-jj2] + ci2*v[ijk-jj1] + ci3*v[ijk    ]))
                 + cg1*((ci0*v[ijk-jj2] + ci1*v[ijk-jj1] + ci2*v[ijk    ] + ci3*v[ijk+jj1]) * (ci0*v[ijk-jj2] + ci1*v[ijk-jj1] + ci2*v[ijk    ] + ci3*v[ijk+jj1]))
                 + cg2*((ci0*v[ijk-jj1] + ci1*v[ijk    ] + ci2*v[ijk+jj1] + ci3*v[ijk+jj2]) * (ci0*v[ijk-jj1] + ci1*v[ijk    ] + ci2*v[ijk+jj1] + ci3*v[ijk+jj2]))
                 + cg3*((ci0*v[ijk    ] + ci1*v[ijk+jj1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj3]) * (ci0*v[ijk    ] + ci1*v[ijk+jj1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj3])) ) * cgi*dyi;

      vt[ijk] -= ( cg0*((ci0*w[ijk-jj2-kk1] + ci1*w[ijk-jj1-kk1] + ci2*w[ijk-kk1] + ci3*w[ijk+jj1-kk1]) * (ci0*v[ijk-kk3] + ci1*v[ijk-kk2] + ci2*v[ijk-kk1] + ci3*v[ijk    ]))
                 + cg1*((ci0*w[ijk-jj2    ] + ci1*w[ijk-jj1    ] + ci2*w[ijk    ] + ci3*w[ijk+jj1    ]) * (ci0*v[ijk-kk2] + ci1*v[ijk-kk1] + ci2*v[ijk    ] + ci3*v[ijk+kk1]))
                 + cg2*((ci0*w[ijk-jj2+kk1] + ci1*w[ijk-jj1+kk1] + ci2*w[ijk+kk1] + ci3*w[ijk+jj1+kk1]) * (ci0*v[ijk-kk1] + ci1*v[ijk    ] + ci2*v[ijk+kk1] + ci3*v[ijk+kk2]))
                 + cg3*((ci0*w[ijk-jj2+kk2] + ci1*w[ijk-jj1+kk2] + ci2*w[ijk+kk2] + ci3*w[ijk+jj1+kk2]) * (ci0*v[ijk    ] + ci1*v[ijk+kk1] + ci2*v[ijk+kk2] + ci3*v[ijk+kk3])) )
                 * dzi4[k];
    }
  }
}

__global__ void advec_4_advecw(double * __restrict__ wt, double * __restrict__ u, 
                               double * __restrict__ v, double * __restrict__ w,
                               double * __restrict__ dzhi4, double dxi, double dyi, 
                               int jj, int kk, int istart,
                               int jstart, int kstart,
                               int iend,   int jend,   int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart + 1;

  int ii1 = 1;
  int ii2 = 2;
  int ii3 = 3;
  int jj1 = 1*jj;
  int jj2 = 2*jj;
  int jj3 = 3*jj;
  int kk1 = 1*kk;
  int kk2 = 2*kk;
  int kk3 = 3*kk;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    if(k == kstart+1)
    {
      wt[ijk] -= ( cg0*((ci0*u[ijk-ii1-kk2] + ci1*u[ijk-ii1-kk1] + ci2*u[ijk-ii1] + ci3*u[ijk-ii1+kk1]) * (ci0*w[ijk-ii3] + ci1*w[ijk-ii2] + ci2*w[ijk-ii1] + ci3*w[ijk    ]))
                 + cg1*((ci0*u[ijk    -kk2] + ci1*u[ijk    -kk1] + ci2*u[ijk    ] + ci3*u[ijk    +kk1]) * (ci0*w[ijk-ii2] + ci1*w[ijk-ii1] + ci2*w[ijk    ] + ci3*w[ijk+ii1]))
                 + cg2*((ci0*u[ijk+ii1-kk2] + ci1*u[ijk+ii1-kk1] + ci2*u[ijk+ii1] + ci3*u[ijk+ii1+kk1]) * (ci0*w[ijk-ii1] + ci1*w[ijk    ] + ci2*w[ijk+ii1] + ci3*w[ijk+ii2]))
                 + cg3*((ci0*u[ijk+ii2-kk2] + ci1*u[ijk+ii2-kk1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii2+kk1]) * (ci0*w[ijk    ] + ci1*w[ijk+ii1] + ci2*w[ijk+ii2] + ci3*w[ijk+ii3])) ) * cgi*dxi;

      wt[ijk] -= ( cg0*((ci0*v[ijk-jj1-kk2] + ci1*v[ijk-jj1-kk1] + ci2*v[ijk-jj1] + ci3*v[ijk-jj1+kk1]) * (ci0*w[ijk-jj3] + ci1*w[ijk-jj2] + ci2*w[ijk-jj1] + ci3*w[ijk    ]))
                 + cg1*((ci0*v[ijk    -kk2] + ci1*v[ijk    -kk1] + ci2*v[ijk    ] + ci3*v[ijk    +kk1]) * (ci0*w[ijk-jj2] + ci1*w[ijk-jj1] + ci2*w[ijk    ] + ci3*w[ijk+jj1]))
                 + cg2*((ci0*v[ijk+jj1-kk2] + ci1*v[ijk+jj1-kk1] + ci2*v[ijk+jj1] + ci3*v[ijk+jj1+kk1]) * (ci0*w[ijk-jj1] + ci1*w[ijk    ] + ci2*w[ijk+jj1] + ci3*w[ijk+jj2]))
                 + cg3*((ci0*v[ijk+jj2-kk2] + ci1*v[ijk+jj2-kk1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj2+kk1]) * (ci0*w[ijk    ] + ci1*w[ijk+jj1] + ci2*w[ijk+jj2] + ci3*w[ijk+jj3])) ) * cgi*dyi;

      wt[ijk] -= ( cg0*((bi0*w[ijk-kk2] + bi1*w[ijk-kk1] + bi2*w[ijk    ] + bi3*w[ijk+kk1]) * (bi0*w[ijk-kk2] + bi1*w[ijk-kk1] + bi2*w[ijk    ] + bi3*w[ijk+kk1]))
                 + cg1*((ci0*w[ijk-kk2] + ci1*w[ijk-kk1] + ci2*w[ijk    ] + ci3*w[ijk+kk1]) * (ci0*w[ijk-kk2] + ci1*w[ijk-kk1] + ci2*w[ijk    ] + ci3*w[ijk+kk1]))
                 + cg2*((ci0*w[ijk-kk1] + ci1*w[ijk    ] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2]) * (ci0*w[ijk-kk1] + ci1*w[ijk    ] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2]))
                 + cg3*((ci0*w[ijk    ] + ci1*w[ijk+kk1] + ci2*w[ijk+kk2] + ci3*w[ijk+kk3]) * (ci0*w[ijk    ] + ci1*w[ijk+kk1] + ci2*w[ijk+kk2] + ci3*w[ijk+kk3])) )
                 * dzhi4[k];
    }
    else if(k == kend-1)
    {
      wt[ijk] -= ( cg0*((ci0*u[ijk-ii1-kk2] + ci1*u[ijk-ii1-kk1] + ci2*u[ijk-ii1] + ci3*u[ijk-ii1+kk1]) * (ci0*w[ijk-ii3] + ci1*w[ijk-ii2] + ci2*w[ijk-ii1] + ci3*w[ijk    ]))
                 + cg1*((ci0*u[ijk    -kk2] + ci1*u[ijk    -kk1] + ci2*u[ijk    ] + ci3*u[ijk    +kk1]) * (ci0*w[ijk-ii2] + ci1*w[ijk-ii1] + ci2*w[ijk    ] + ci3*w[ijk+ii1]))
                 + cg2*((ci0*u[ijk+ii1-kk2] + ci1*u[ijk+ii1-kk1] + ci2*u[ijk+ii1] + ci3*u[ijk+ii1+kk1]) * (ci0*w[ijk-ii1] + ci1*w[ijk    ] + ci2*w[ijk+ii1] + ci3*w[ijk+ii2]))
                 + cg3*((ci0*u[ijk+ii2-kk2] + ci1*u[ijk+ii2-kk1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii2+kk1]) * (ci0*w[ijk    ] + ci1*w[ijk+ii1] + ci2*w[ijk+ii2] + ci3*w[ijk+ii3])) ) * cgi*dxi;

      wt[ijk] -= ( cg0*((ci0*v[ijk-jj1-kk2] + ci1*v[ijk-jj1-kk1] + ci2*v[ijk-jj1] + ci3*v[ijk-jj1+kk1]) * (ci0*w[ijk-jj3] + ci1*w[ijk-jj2] + ci2*w[ijk-jj1] + ci3*w[ijk    ]))
                 + cg1*((ci0*v[ijk    -kk2] + ci1*v[ijk    -kk1] + ci2*v[ijk    ] + ci3*v[ijk    +kk1]) * (ci0*w[ijk-jj2] + ci1*w[ijk-jj1] + ci2*w[ijk    ] + ci3*w[ijk+jj1]))
                 + cg2*((ci0*v[ijk+jj1-kk2] + ci1*v[ijk+jj1-kk1] + ci2*v[ijk+jj1] + ci3*v[ijk+jj1+kk1]) * (ci0*w[ijk-jj1] + ci1*w[ijk    ] + ci2*w[ijk+jj1] + ci3*w[ijk+jj2]))
                 + cg3*((ci0*v[ijk+jj2-kk2] + ci1*v[ijk+jj2-kk1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj2+kk1]) * (ci0*w[ijk    ] + ci1*w[ijk+jj1] + ci2*w[ijk+jj2] + ci3*w[ijk+jj3])) ) * cgi*dyi;

      wt[ijk] -= ( cg0*((ci0*w[ijk-kk3] + ci1*w[ijk-kk2] + ci2*w[ijk-kk1] + ci3*w[ijk    ]) * (ci0*w[ijk-kk3] + ci1*w[ijk-kk2] + ci2*w[ijk-kk1] + ci3*w[ijk    ]))
                 + cg1*((ci0*w[ijk-kk2] + ci1*w[ijk-kk1] + ci2*w[ijk    ] + ci3*w[ijk+kk1]) * (ci0*w[ijk-kk2] + ci1*w[ijk-kk1] + ci2*w[ijk    ] + ci3*w[ijk+kk1]))
                 + cg2*((ci0*w[ijk-kk1] + ci1*w[ijk    ] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2]) * (ci0*w[ijk-kk1] + ci1*w[ijk    ] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2]))
                 + cg3*((ti0*w[ijk-kk1] + ti1*w[ijk    ] + ti2*w[ijk+kk1] + ti3*w[ijk+kk2]) * (ti0*w[ijk-kk1] + ti1*w[ijk    ] + ti2*w[ijk+kk1] + ti3*w[ijk+kk2])) )
                 * dzhi4[k];
    }
    else
    {
      wt[ijk] -= ( cg0*((ci0*u[ijk-ii1-kk2] + ci1*u[ijk-ii1-kk1] + ci2*u[ijk-ii1] + ci3*u[ijk-ii1+kk1]) * (ci0*w[ijk-ii3] + ci1*w[ijk-ii2] + ci2*w[ijk-ii1] + ci3*w[ijk    ]))
                 + cg1*((ci0*u[ijk    -kk2] + ci1*u[ijk    -kk1] + ci2*u[ijk    ] + ci3*u[ijk    +kk1]) * (ci0*w[ijk-ii2] + ci1*w[ijk-ii1] + ci2*w[ijk    ] + ci3*w[ijk+ii1]))
                 + cg2*((ci0*u[ijk+ii1-kk2] + ci1*u[ijk+ii1-kk1] + ci2*u[ijk+ii1] + ci3*u[ijk+ii1+kk1]) * (ci0*w[ijk-ii1] + ci1*w[ijk    ] + ci2*w[ijk+ii1] + ci3*w[ijk+ii2]))
                 + cg3*((ci0*u[ijk+ii2-kk2] + ci1*u[ijk+ii2-kk1] + ci2*u[ijk+ii2] + ci3*u[ijk+ii2+kk1]) * (ci0*w[ijk    ] + ci1*w[ijk+ii1] + ci2*w[ijk+ii2] + ci3*w[ijk+ii3])) ) * cgi*dxi;

      wt[ijk] -= ( cg0*((ci0*v[ijk-jj1-kk2] + ci1*v[ijk-jj1-kk1] + ci2*v[ijk-jj1] + ci3*v[ijk-jj1+kk1]) * (ci0*w[ijk-jj3] + ci1*w[ijk-jj2] + ci2*w[ijk-jj1] + ci3*w[ijk    ]))
                 + cg1*((ci0*v[ijk    -kk2] + ci1*v[ijk    -kk1] + ci2*v[ijk    ] + ci3*v[ijk    +kk1]) * (ci0*w[ijk-jj2] + ci1*w[ijk-jj1] + ci2*w[ijk    ] + ci3*w[ijk+jj1]))
                 + cg2*((ci0*v[ijk+jj1-kk2] + ci1*v[ijk+jj1-kk1] + ci2*v[ijk+jj1] + ci3*v[ijk+jj1+kk1]) * (ci0*w[ijk-jj1] + ci1*w[ijk    ] + ci2*w[ijk+jj1] + ci3*w[ijk+jj2]))
                 + cg3*((ci0*v[ijk+jj2-kk2] + ci1*v[ijk+jj2-kk1] + ci2*v[ijk+jj2] + ci3*v[ijk+jj2+kk1]) * (ci0*w[ijk    ] + ci1*w[ijk+jj1] + ci2*w[ijk+jj2] + ci3*w[ijk+jj3])) ) * cgi*dyi;

      wt[ijk] -= ( cg0*((ci0*w[ijk-kk3] + ci1*w[ijk-kk2] + ci2*w[ijk-kk1] + ci3*w[ijk    ]) * (ci0*w[ijk-kk3] + ci1*w[ijk-kk2] + ci2*w[ijk-kk1] + ci3*w[ijk    ]))
                 + cg1*((ci0*w[ijk-kk2] + ci1*w[ijk-kk1] + ci2*w[ijk    ] + ci3*w[ijk+kk1]) * (ci0*w[ijk-kk2] + ci1*w[ijk-kk1] + ci2*w[ijk    ] + ci3*w[ijk+kk1]))
                 + cg2*((ci0*w[ijk-kk1] + ci1*w[ijk    ] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2]) * (ci0*w[ijk-kk1] + ci1*w[ijk    ] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2]))
                 + cg3*((ci0*w[ijk    ] + ci1*w[ijk+kk1] + ci2*w[ijk+kk2] + ci3*w[ijk+kk3]) * (ci0*w[ijk    ] + ci1*w[ijk+kk1] + ci2*w[ijk+kk2] + ci3*w[ijk+kk3])) )
                 * dzhi4[k];
    }
  }
}

__global__ void advec_4_advecs(double * __restrict__ st, double * __restrict__ s, 
                               double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                               double * __restrict__ dzi4, double dxi, double dyi, 
                               int jj, int kk,
                               int istart, int jstart, int kstart,
                               int iend,   int jend,   int kend)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  int ii1 = 1;
  int ii2 = 2;
  int ii3 = 3;
  int jj1 = 1*jj;
  int jj2 = 2*jj;
  int jj3 = 3*jj;
  int kk1 = 1*kk;
  int kk2 = 2*kk;
  int kk3 = 3*kk;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    if(k == kstart)
    {
      st[ijk] -= ( cg0*(u[ijk-ii1] * (ci0*s[ijk-ii3] + ci1*s[ijk-ii2] + ci2*s[ijk-ii1] + ci3*s[ijk    ]))
                 + cg1*(u[ijk    ] * (ci0*s[ijk-ii2] + ci1*s[ijk-ii1] + ci2*s[ijk    ] + ci3*s[ijk+ii1]))
                 + cg2*(u[ijk+ii1] * (ci0*s[ijk-ii1] + ci1*s[ijk    ] + ci2*s[ijk+ii1] + ci3*s[ijk+ii2]))
                 + cg3*(u[ijk+ii2] * (ci0*s[ijk    ] + ci1*s[ijk+ii1] + ci2*s[ijk+ii2] + ci3*s[ijk+ii3])) ) * cgi*dxi;

      st[ijk] -= ( cg0*(v[ijk-jj1] * (ci0*s[ijk-jj3] + ci1*s[ijk-jj2] + ci2*s[ijk-jj1] + ci3*s[ijk    ]))
                 + cg1*(v[ijk    ] * (ci0*s[ijk-jj2] + ci1*s[ijk-jj1] + ci2*s[ijk    ] + ci3*s[ijk+jj1]))
                 + cg2*(v[ijk+jj1] * (ci0*s[ijk-jj1] + ci1*s[ijk    ] + ci2*s[ijk+jj1] + ci3*s[ijk+jj2]))
                 + cg3*(v[ijk+jj2] * (ci0*s[ijk    ] + ci1*s[ijk+jj1] + ci2*s[ijk+jj2] + ci3*s[ijk+jj3])) ) * cgi*dyi;

      st[ijk] -= ( cg0*(w[ijk-kk1] * (bi0*s[ijk-kk2] + bi1*s[ijk-kk1] + bi2*s[ijk    ] + bi3*s[ijk+kk1]))
                 + cg1*(w[ijk    ] * (ci0*s[ijk-kk2] + ci1*s[ijk-kk1] + ci2*s[ijk    ] + ci3*s[ijk+kk1]))
                 + cg2*(w[ijk+kk1] * (ci0*s[ijk-kk1] + ci1*s[ijk    ] + ci2*s[ijk+kk1] + ci3*s[ijk+kk2]))
                 + cg3*(w[ijk+kk2] * (ci0*s[ijk    ] + ci1*s[ijk+kk1] + ci2*s[ijk+kk2] + ci3*s[ijk+kk3])) )
                 * dzi4[k];
    }
    else if(k == kend-1)
    {
      st[ijk] -= ( cg0*(u[ijk-ii1] * (ci0*s[ijk-ii3] + ci1*s[ijk-ii2] + ci2*s[ijk-ii1] + ci3*s[ijk    ]))
                 + cg1*(u[ijk    ] * (ci0*s[ijk-ii2] + ci1*s[ijk-ii1] + ci2*s[ijk    ] + ci3*s[ijk+ii1]))
                 + cg2*(u[ijk+ii1] * (ci0*s[ijk-ii1] + ci1*s[ijk    ] + ci2*s[ijk+ii1] + ci3*s[ijk+ii2]))
                 + cg3*(u[ijk+ii2] * (ci0*s[ijk    ] + ci1*s[ijk+ii1] + ci2*s[ijk+ii2] + ci3*s[ijk+ii3])) ) * cgi*dxi;

      st[ijk] -= ( cg0*(v[ijk-jj1] * (ci0*s[ijk-jj3] + ci1*s[ijk-jj2] + ci2*s[ijk-jj1] + ci3*s[ijk    ]))
                 + cg1*(v[ijk    ] * (ci0*s[ijk-jj2] + ci1*s[ijk-jj1] + ci2*s[ijk    ] + ci3*s[ijk+jj1]))
                 + cg2*(v[ijk+jj1] * (ci0*s[ijk-jj1] + ci1*s[ijk    ] + ci2*s[ijk+jj1] + ci3*s[ijk+jj2]))
                 + cg3*(v[ijk+jj2] * (ci0*s[ijk    ] + ci1*s[ijk+jj1] + ci2*s[ijk+jj2] + ci3*s[ijk+jj3])) ) * cgi*dyi;

      st[ijk] -= ( cg0*(w[ijk-kk1] * (ci0*s[ijk-kk3] + ci1*s[ijk-kk2] + ci2*s[ijk-kk1] + ci3*s[ijk    ]))
                 + cg1*(w[ijk    ] * (ci0*s[ijk-kk2] + ci1*s[ijk-kk1] + ci2*s[ijk    ] + ci3*s[ijk+kk1]))
                 + cg2*(w[ijk+kk1] * (ci0*s[ijk-kk1] + ci1*s[ijk    ] + ci2*s[ijk+kk1] + ci3*s[ijk+kk2]))
                 + cg3*(w[ijk+kk2] * (ti0*s[ijk-kk1] + ti1*s[ijk    ] + ti2*s[ijk+kk1] + ti3*s[ijk+kk2])) )
                 * dzi4[k];
    }
    else
    {
      st[ijk] -= ( cg0*(u[ijk-ii1] * (ci0*s[ijk-ii3] + ci1*s[ijk-ii2] + ci2*s[ijk-ii1] + ci3*s[ijk    ]))
                 + cg1*(u[ijk    ] * (ci0*s[ijk-ii2] + ci1*s[ijk-ii1] + ci2*s[ijk    ] + ci3*s[ijk+ii1]))
                 + cg2*(u[ijk+ii1] * (ci0*s[ijk-ii1] + ci1*s[ijk    ] + ci2*s[ijk+ii1] + ci3*s[ijk+ii2]))
                 + cg3*(u[ijk+ii2] * (ci0*s[ijk    ] + ci1*s[ijk+ii1] + ci2*s[ijk+ii2] + ci3*s[ijk+ii3])) ) * cgi*dxi;

      st[ijk] -= ( cg0*(v[ijk-jj1] * (ci0*s[ijk-jj3] + ci1*s[ijk-jj2] + ci2*s[ijk-jj1] + ci3*s[ijk    ]))
                 + cg1*(v[ijk    ] * (ci0*s[ijk-jj2] + ci1*s[ijk-jj1] + ci2*s[ijk    ] + ci3*s[ijk+jj1]))
                 + cg2*(v[ijk+jj1] * (ci0*s[ijk-jj1] + ci1*s[ijk    ] + ci2*s[ijk+jj1] + ci3*s[ijk+jj2]))
                 + cg3*(v[ijk+jj2] * (ci0*s[ijk    ] + ci1*s[ijk+jj1] + ci2*s[ijk+jj2] + ci3*s[ijk+jj3])) ) * cgi*dyi;

      st[ijk] -= ( cg0*(w[ijk-kk1] * (ci0*s[ijk-kk3] + ci1*s[ijk-kk2] + ci2*s[ijk-kk1] + ci3*s[ijk    ]))
                 + cg1*(w[ijk    ] * (ci0*s[ijk-kk2] + ci1*s[ijk-kk1] + ci2*s[ijk    ] + ci3*s[ijk+kk1]))
                 + cg2*(w[ijk+kk1] * (ci0*s[ijk-kk1] + ci1*s[ijk    ] + ci2*s[ijk+kk1] + ci3*s[ijk+kk2]))
                 + cg3*(w[ijk+kk2] * (ci0*s[ijk    ] + ci1*s[ijk+kk1] + ci2*s[ijk+kk2] + ci3*s[ijk+kk3])) )
                 * dzi4[k];
    }
  }
}

__global__ void advec_4_calccfl(double * const __restrict__ tmp1,
                                const double * const __restrict__ u, const double * const __restrict__ v, const double * const __restrict__ w, 
                                const double * const __restrict__ dzi, const double dxi, const double dyi,
                                const int jj, const int kk,
                                const int istart, const int jstart, const int kstart,
                                const int iend, const int jend, const int kend)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x; 
  const int j = blockIdx.y*blockDim.y + threadIdx.y; 
  const int k = blockIdx.z; 

  const int ii1 = 1;
  const int ii2 = 2;
  const int jj1 = 1*jj;
  const int jj2 = 2*jj;
  const int kk1 = 1*kk;
  const int kk2 = 2*kk;

  const int ijk = i + j*jj + k*kk;

  if(i < iend && j < jend && k < kend)
    tmp1[ijk] = std::abs(ci0*u[ijk-ii1] + ci1*u[ijk] + ci2*u[ijk+ii1] + ci3*u[ijk+ii2])*dxi + 
                std::abs(ci0*v[ijk-jj1] + ci1*v[ijk] + ci2*v[ijk+jj1] + ci3*v[ijk+jj2])*dyi + 
                std::abs(ci0*w[ijk-kk1] + ci1*w[ijk] + ci2*w[ijk+kk1] + ci3*w[ijk+kk2])*dzi[k];
}

#ifdef USECUDA
void cadvec_4::exec()
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const double dxi = 1./grid->dx;
  const double dyi = 1./grid->dy;

  const int offs = grid->memoffset;

  advec_4_advecu<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->u->data_g[offs], &fields->v->data_g[offs], 
                                        &fields->w->data_g[offs], grid->dzi4_g, dxi, dyi,
                                        grid->icellsp, grid->ijcellsp,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend,   grid->jend, grid->kend);

  advec_4_advecv<<<gridGPU, blockGPU>>>(&fields->vt->data_g[offs], &fields->u->data_g[offs], &fields->v->data_g[offs], 
                                        &fields->w->data_g[offs], grid->dzi4_g, dxi, dyi,
                                        grid->icellsp, grid->ijcellsp,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend,   grid->jend, grid->kend);

  advec_4_advecw<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->u->data_g[offs], &fields->v->data_g[offs], 
                                        &fields->w->data_g[offs], grid->dzhi4_g, dxi, dyi,
                                        grid->icellsp, grid->ijcellsp,
                                        grid->istart, grid->jstart, grid->kstart,
                                        grid->iend,   grid->jend, grid->kend);

  for(fieldmap::const_iterator it = fields->st.begin(); it!=fields->st.end(); it++)
    advec_4_advecs<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], &fields->s[it->first]->data_g[offs], 
                                          &fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs], 
                                          grid->dzi4_g, dxi, dyi,
                                          grid->icellsp, grid->ijcellsp,
                                          grid->istart, grid->jstart, grid->kstart,
                                          grid->iend,   grid->jend, grid->kend);

  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
    printf("CUDA ERROR ADV: %s\n", hipGetErrorString(error));
}
#endif

#ifdef USECUDA
double cadvec_4::calccfl(double * u, double * v, double * w, double * dzi, double dt)
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
  const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);
  double cfl = 0;

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const double dxi = 1./grid->dx;
  const double dyi = 1./grid->dy;

  const int offs = grid->memoffset;

  advec_4_calccfl<<<gridGPU, blockGPU>>>(&fields->a["tmp1"]->data_g[offs],
                                         &fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs],
                                         grid->dzi_g, dxi, dyi,
                                         grid->icellsp, grid->ijcellsp,
                                         grid->istart, grid->jstart, grid->kstart,
                                         grid->iend,   grid->jend,   grid->kend);

  cfl = grid->getmax_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp2"]->data_g); 
  grid->getmax(&cfl); 
  cfl = cfl*dt;

  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
    printf("CUDA ERROR CFL: %s\n", hipGetErrorString(error));

  return cfl;
}
#endif
