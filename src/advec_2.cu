#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2017 Chiel van Heerwaarden
 * Copyright (c) 2011-2017 Thijs Heus
 * Copyright (c) 2014-2017 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "advec_2.h"
#include "master.h"
#include "grid.h"
#include "fields.h"
#include "tools.h"
#include "constants.h"
#include "tools.h"
#include "finite_difference.h"

using namespace Finite_difference::O2;

//namespace
//{
//    __global__ 
//    void advec_uvw_g(double* __restrict__ ut, double* __restrict__ vt, double * __restrict__ wt, 
//                     double* __restrict__ u,  double* __restrict__ v,  double * __restrict__ w,
//                     double* __restrict__ rhoref, double* __restrict__ rhorefh,
//                     double* __restrict__ dzi,    double* __restrict__ dzhi, double dxi, double dyi, 
//                     int jj, int kk, 
//                     int istart, int jstart, int kstart,
//                     int iend,   int jend,   int kend)
//    {
//        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
//        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
//        const int k  = blockIdx.z + kstart;
//        const int ii = 1;
//
//        if (i < iend && j < jend && k < kend)
//        {
//            const int ijk = i + j*jj + k*kk;
//            ut[ijk] += 
//                - (  interp2(u[ijk   ], u[ijk+ii]) * interp2(u[ijk   ], u[ijk+ii])
//                   - interp2(u[ijk-ii], u[ijk   ]) * interp2(u[ijk-ii], u[ijk   ]) ) * dxi
//
//                - (  interp2(v[ijk-ii+jj], v[ijk+jj]) * interp2(u[ijk   ], u[ijk+jj])
//                   - interp2(v[ijk-ii   ], v[ijk   ]) * interp2(u[ijk-jj], u[ijk   ]) ) * dyi 
//
//                - (  rhorefh[k+1] * interp2(w[ijk-ii+kk], w[ijk+kk]) * interp2(u[ijk   ], u[ijk+kk])
//                   - rhorefh[k  ] * interp2(w[ijk-ii   ], w[ijk   ]) * interp2(u[ijk-kk], u[ijk   ]) ) / rhoref[k] * dzi[k];
//
//            vt[ijk] += 
//                - (  interp2(u[ijk+ii-jj], u[ijk+ii]) * interp2(v[ijk   ], v[ijk+ii])
//                   - interp2(u[ijk   -jj], u[ijk   ]) * interp2(v[ijk-ii], v[ijk   ]) ) * dxi
//
//                - (  interp2(v[ijk   ], v[ijk+jj]) * interp2(v[ijk   ], v[ijk+jj])
//                   - interp2(v[ijk-jj], v[ijk   ]) * interp2(v[ijk-jj], v[ijk   ]) ) * dyi
//
//                - (  rhorefh[k+1] * interp2(w[ijk-jj+kk], w[ijk+kk]) * interp2(v[ijk   ], v[ijk+kk])
//                   - rhorefh[k  ] * interp2(w[ijk-jj   ], w[ijk   ]) * interp2(v[ijk-kk], v[ijk   ]) ) / rhoref[k] * dzi[k];
//
//            if (k > kstart)
//            {
//                wt[ijk] += 
//                    - (  interp2(u[ijk+ii-kk], u[ijk+ii]) * interp2(w[ijk   ], w[ijk+ii])
//                       - interp2(u[ijk   -kk], u[ijk   ]) * interp2(w[ijk-ii], w[ijk   ]) ) * dxi
//
//                    - (  interp2(v[ijk+jj-kk], v[ijk+jj]) * interp2(w[ijk   ], w[ijk+jj])
//                       - interp2(v[ijk   -kk], v[ijk   ]) * interp2(w[ijk-jj], w[ijk   ]) ) * dyi
//
//                    - (  rhoref[k  ] * interp2(w[ijk   ], w[ijk+kk]) * interp2(w[ijk   ], w[ijk+kk])
//                       - rhoref[k-1] * interp2(w[ijk-kk], w[ijk   ]) * interp2(w[ijk-kk], w[ijk   ]) ) / rhorefh[k] * dzhi[k];
//            }
//        }
//    }
//
//    __global__ 
//    void advec_s_g(double* __restrict__ st, double* __restrict__ s, 
//                   double* __restrict__ u,  double* __restrict__ v, double* __restrict__ w,
//                   double* __restrict__ rhoref, double* __restrict__ rhorefh,
//                   double* __restrict__ dzi, double dxi, double dyi, 
//                   int jj, int kk, 
//                   int istart, int jstart, int kstart,
//                   int iend,   int jend,   int kend)
//    {
//        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart;
//        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart;
//        const int k  = blockIdx.z + kstart;
//        const int ii = 1;
//
//        if (i < iend && j < jend && k < kend)
//        {
//            const int ijk = i + j*jj + k*kk;
//            st[ijk] += 
//                - (  u[ijk+ii] * interp2(s[ijk   ], s[ijk+ii])
//                   - u[ijk   ] * interp2(s[ijk-ii], s[ijk   ]) ) * dxi
//
//                - (  v[ijk+jj] * interp2(s[ijk   ], s[ijk+jj])
//                   - v[ijk   ] * interp2(s[ijk-jj], s[ijk   ]) ) * dyi 
//
//                - (  rhorefh[k+1] * w[ijk+kk] * interp2(s[ijk   ], s[ijk+kk])
//                   - rhorefh[k  ] * w[ijk   ] * interp2(s[ijk-kk], s[ijk   ]) ) / rhoref[k] * dzi[k];
//        }
//    }
//
//    __global__ 
//    void calc_cfl_g(double* __restrict__ u, double* __restrict__ v, double* __restrict__ w, 
//                    double* __restrict__ cfl, double* __restrict__ dzi, double dxi, double dyi,
//                    int jj, int kk,
//                    int istart, int jstart, int kstart,
//                    int iend, int jend, int kend)
//    {
//        const int i  = blockIdx.x*blockDim.x + threadIdx.x + istart; 
//        const int j  = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
//        const int k  = blockIdx.z + kstart; 
//        const int ii = 1;
//
//        if (i < iend && j < jend && k < kend)
//        {
//            const int ijk = i + j*jj + k*kk;
//            cfl[ijk] = std::abs(interp2(u[ijk], u[ijk+ii]))*dxi + 
//                       std::abs(interp2(v[ijk], v[ijk+jj]))*dyi + 
//                       std::abs(interp2(w[ijk], w[ijk+kk]))*dzi[k];
//        }
//    }
//}

#ifdef USECUDA
template<typename TF>
unsigned long Advec_2<TF>::get_time_limit(unsigned long idt, double dt)
{
    // Calculate cfl and prevent zero divisons.
    //double cfl = get_cfl(dt);
    //cfl = std::max(cflmin, cfl);
    //const unsigned long idtlim = idt * cflmax / cfl;

    //return idtlim;

    return 1;
}

template<typename TF>
double Advec_2<TF>::get_cfl(double dt)
{
    //const int blocki = grid->ithread_block;
    //const int blockj = grid->jthread_block;
    //const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    //const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    //dim3 gridGPU (gridi, gridj, grid->kcells);
    //dim3 blockGPU(blocki, blockj, 1);

    //const double dxi = 1./grid->dx;
    //const double dyi = 1./grid->dy;

    //const int offs = grid->memoffset;

    //calc_cfl_g<<<gridGPU, blockGPU>>>(
    //    &fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs], 
    //    &fields->atmp["tmp1"]->data_g[offs], grid->dzi_g, dxi, dyi,
    //    grid->icellsp, grid->ijcellsp,
    //    grid->istart,  grid->jstart, grid->kstart,
    //    grid->iend,    grid->jend,   grid->kend);
    //cuda_check_error(); 

    //double cfl = grid->get_max_g(&fields->atmp["tmp1"]->data_g[offs], fields->atmp["tmp2"]->data_g); 
    //grid->get_max(&cfl); 
    //cfl = cfl*dt;

    //return cfl;

    return 1;
}

template<typename TF>
void Advec_2<TF>::exec()
{
    //const int blocki = grid->ithread_block;
    //const int blockj = grid->jthread_block;
    //const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    //const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    //dim3 gridGPU (gridi, gridj, grid->kmax);
    //dim3 blockGPU(blocki, blockj, 1);

    //const double dxi = 1./grid->dx;
    //const double dyi = 1./grid->dy;

    //const int offs = grid->memoffset;

    //advec_uvw_g<<<gridGPU, blockGPU>>>(
    //    &fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs], 
    //    &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs], 
    //    fields->rhoref_g, fields->rhorefh_g, grid->dzi_g, grid->dzhi_g, dxi, dyi, 
    //    grid->icellsp, grid->ijcellsp,
    //    grid->istart,  grid->jstart, grid->kstart,
    //    grid->iend,    grid->jend,   grid->kend);
    //cuda_check_error(); 

    //for (FieldMap::iterator it = fields->st.begin(); it!=fields->st.end(); it++)
    //    advec_s_g<<<gridGPU, blockGPU>>>(
    //        &it->second->data_g[offs], &fields->sp[it->first]->data_g[offs], 
    //        &fields->u->data_g[offs], &fields->v->data_g[offs], &fields->w->data_g[offs],
    //        fields->rhoref_g, fields->rhorefh_g, grid->dzi_g, dxi, dyi,
    //        grid->icellsp, grid->ijcellsp,
    //        grid->istart,  grid->jstart, grid->kstart,
    //        grid->iend,    grid->jend,   grid->kend);
    //cuda_check_error(); 
}
#endif
