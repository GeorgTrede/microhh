#include "fields.h"
#include "grid.h"

int cfields::prepareGPU()
{
  double *a;

  const int nmemsize = grid->ncells*sizeof(double);

  hipMalloc((void**)&a, nmemsize);
  hipMemcpy(a, sp["th"]->data, nmemsize, hipMemcpyHostToDevice);

  // test CUDA-aware MPI
  grid->boundary_cyclic(a);

  return 0;
}
