#include "fields.h"
#include "grid.h"
#include "master.h"

int cfields::prepareGPU()
{
  const int nmemsize = grid->ncells*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
    hipMalloc(&it->second->data_g, nmemsize);

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMalloc(&it->second->data_g, nmemsize);

  // copy all the data to the GPU
  forwardGPU();

  return 0;
}

int cfields::forwardGPU()
{
  const int nmemsize = grid->ncells*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
    hipMemcpy(it->second->data_g, it->second->data, nmemsize, hipMemcpyHostToDevice);

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy(it->second->data_g, it->second->data, nmemsize, hipMemcpyHostToDevice);

  return 0;
}

int cfields::backwardGPU()
{
  const int nmemsize = grid->ncells*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
    hipMemcpy(it->second->data, it->second->data_g, nmemsize, hipMemcpyDeviceToHost);

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy(it->second->data, it->second->data_g, nmemsize, hipMemcpyDeviceToHost);

  return 0;
}
