#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_buoy.h"
#include "fd.h"
#include "tools.h"

namespace ThermoBuoy_g
{
  __global__ void calcBuoyancyTend_2nd(double * __restrict__ wt, double * __restrict__ b, 
                                       int istart, int jstart, int kstart,
                                       int iend,   int jend,   int kend,
                                       int jj, int kk)
  {
    int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
    int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
    int k = blockIdx.z + kstart; 
  
    if(i < iend && j < jend && k < kend)
    {
      int ijk = i + j*jj + k*kk;
      wt[ijk] += fd::o2::interp2(b[ijk-kk], b[ijk]);
    }
  }
  
  __global__ void calcBuoyancyTend_4th(double * __restrict__ wt, double * __restrict__ b, 
                                       int istart, int jstart, int kstart,
                                       int iend,   int jend,   int kend,
                                       int jj, int kk)
  {
    const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
    const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
    const int k = blockIdx.z + kstart;
  
    const int kk1 = 1*kk;
    const int kk2 = 2*kk;
  
    if(i < iend && j < jend && k < kend)
    {
      const int ijk = i + j*jj + k*kk;
      wt[ijk] += fd::o4::ci0*b[ijk-kk2] + fd::o4::ci1*b[ijk-kk1] + fd::o4::ci2*b[ijk] + fd::o4::ci3*b[ijk+kk1];
    }
  }
} // end namespace

#ifdef USECUDA
void ThermoBuoy::exec()
{
  const int blocki = grid->iThreadBlock;
  const int blockj = grid->jThreadBlock;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax-1);
  dim3 blockGPU(blocki, blockj, 1);
  
  const int offs = grid->memoffset;

  if(grid->swspatialorder== "2")
  {
    ThermoBuoy_g::calcBuoyancyTend_2nd<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
                                                              grid->istart, grid->jstart, grid->kstart+1,
                                                              grid->iend,   grid->jend, grid->kend,
                                                              grid->icellsp, grid->ijcellsp);
    cudaCheckError();
  }
  else if(grid->swspatialorder== "4")
  {
    ThermoBuoy_g::calcBuoyancyTend_4th<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
                                                              grid->istart, grid->jstart, grid->kstart+1,
                                                              grid->iend,   grid->jend, grid->kend,
                                                              grid->icellsp, grid->ijcellsp);
    cudaCheckError();
  }
}
#endif
