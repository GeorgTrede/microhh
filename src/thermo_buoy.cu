#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_buoy.h"
#include "fd.h"
#include "tools.h"

namespace
{
    __global__ 
    void calc_buoyancy_tend_2nd_g(double* __restrict__ wt, double* __restrict__ b, 
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart; 

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += fd::o2::interp2(b[ijk-kk], b[ijk]);
        }
    }

    __global__ 
    void calc_buoyancy_tend_4th_g(double* __restrict__ wt, double* __restrict__ b, 
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart;

        const int kk1 = 1*kk;
        const int kk2 = 2*kk;

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += fd::o4::ci0*b[ijk-kk2] + fd::o4::ci1*b[ijk-kk1] + fd::o4::ci2*b[ijk] + fd::o4::ci3*b[ijk+kk1];
        }
    }
} // end namespace

#ifdef USECUDA
void Thermo_buoy::exec()
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax-1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (grid->swspatialorder== "2")
    {
        calc_buoyancy_tend_2nd_g<<<gridGPU, blockGPU>>>(
            &fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
            grid->istart,  grid->jstart, grid->kstart+1,
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);
        cuda_check_error();
    }
    else if (grid->swspatialorder== "4")
    {
        calc_buoyancy_tend_4th_g<<<gridGPU, blockGPU>>>(
            &fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
            grid->istart,  grid->jstart, grid->kstart+1,
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);
        cuda_check_error();
    }
}
#endif
