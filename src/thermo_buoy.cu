#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_buoy.h"
#include "fd.h"
#include "tools.h"

// BvS: what to do with these double functions? Make generic .cu file with interp2, interp4, etc.?
__device__ double interp22(double a, double b)
{
  return 0.5*(a + b);
}

__global__ void ThermoBuoy_calcBuoyancyTend_2nd(double * __restrict__ wt, double * __restrict__ b, 
                                                int istart, int jstart, int kstart,
                                                int iend,   int jend,   int kend,
                                                int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    wt[ijk] += interp22(b[ijk-kk], b[ijk]);
  }
}

__global__ void ThermoBuoy_calcBuoyancyTend_4th(double * __restrict__ wt, double * __restrict__ b, 
                                                int istart, int jstart, int kstart,
                                                int iend,   int jend,   int kend,
                                                int jj, int kk)
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  const int k = blockIdx.z + kstart;

  const int kk1 = 1*kk;
  const int kk2 = 2*kk;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;
    wt[ijk] += fd::o4::ci0*b[ijk-kk2] + fd::o4::ci1*b[ijk-kk1] + fd::o4::ci2*b[ijk] + fd::o4::ci3*b[ijk+kk1];
  }
}

#ifdef USECUDA
void ThermoBuoy::exec()
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax-1);
  dim3 blockGPU(blocki, blockj, 1);
  
  const int offs = grid->memoffset;

  if(grid->swspatialorder== "2")
  {
    ThermoBuoy_calcBuoyancyTend_2nd<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
                                                           grid->istart, grid->jstart, grid->kstart+1,
                                                           grid->iend,   grid->jend, grid->kend,
                                                           grid->icellsp, grid->ijcellsp);
    cudaCheckError();
  }
  else if(grid->swspatialorder== "4")
  {
    ThermoBuoy_calcBuoyancyTend_4th<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->sp["b"]->data_g[offs], 
                                                           grid->istart, grid->jstart, grid->kstart+1,
                                                           grid->iend,   grid->jend, grid->kend,
                                                           grid->icellsp, grid->ijcellsp);
    cudaCheckError();
  }
}
#endif
