#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2013 Chiel van Heerwaarden
 * Copyright (c) 2011-2013 Thijs Heus
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "grid.h"
#include "fields.h"
#include "master.h"
#include "diff_4.h"
#include "defines.h"
#include "model.h"

__global__ void diffc_kernel(double * __restrict__ at, double * __restrict__ a, double visc,
                             int jj, int kk,
                             int istart, int jstart, int kstart,
                             int iend, int jend, int kend,
                             double dx, double dy,
                             double * __restrict__ dzi4, double * __restrict__ dzhi4)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    int ii1,ii2,ii3,jj1,jj2,jj3,kk1,kk2,kk3;
    double dxidxi,dyidyi;

    ii1 = 1;
    ii2 = 2;
    ii3 = 3;
    jj1 = 1*jj;
    jj2 = 2*jj;
    jj3 = 3*jj;
    kk1 = 1*kk;
    kk2 = 2*kk;
    kk3 = 3*kk;

    dxidxi = 1./(dx*dx);
    dyidyi = 1./(dy*dy);

    // bottom boundary
    if(k == kstart)
    {
      at[ijk] += visc * (cdg3*a[ijk-ii3] + cdg2*a[ijk-ii2] + cdg1*a[ijk-ii1] + cdg0*a[ijk] + cdg1*a[ijk+ii1] + cdg2*a[ijk+ii2] + cdg3*a[ijk+ii3])*dxidxi;
      at[ijk] += visc * (cdg3*a[ijk-jj3] + cdg2*a[ijk-jj2] + cdg1*a[ijk-jj1] + cdg0*a[ijk] + cdg1*a[ijk+jj1] + cdg2*a[ijk+jj2] + cdg3*a[ijk+jj3])*dyidyi;
      at[ijk] += visc * ( cg0*(bg0*a[ijk-kk2] + bg1*a[ijk-kk1] + bg2*a[ijk    ] + bg3*a[ijk+kk1]) * dzhi4[k-1]
                        + cg1*(cg0*a[ijk-kk2] + cg1*a[ijk-kk1] + cg2*a[ijk    ] + cg3*a[ijk+kk1]) * dzhi4[k  ]
                        + cg2*(cg0*a[ijk-kk1] + cg1*a[ijk    ] + cg2*a[ijk+kk1] + cg3*a[ijk+kk2]) * dzhi4[k+1]
                        + cg3*(cg0*a[ijk    ] + cg1*a[ijk+kk1] + cg2*a[ijk+kk2] + cg3*a[ijk+kk3]) * dzhi4[k+2] )
                        * dzi4[k];
    }
    // top boundary
    else if(k == kend-1)
    {
      at[ijk] += visc * (cdg3*a[ijk-ii3] + cdg2*a[ijk-ii2] + cdg1*a[ijk-ii1] + cdg0*a[ijk] + cdg1*a[ijk+ii1] + cdg2*a[ijk+ii2] + cdg3*a[ijk+ii3])*dxidxi;
      at[ijk] += visc * (cdg3*a[ijk-jj3] + cdg2*a[ijk-jj2] + cdg1*a[ijk-jj1] + cdg0*a[ijk] + cdg1*a[ijk+jj1] + cdg2*a[ijk+jj2] + cdg3*a[ijk+jj3])*dyidyi;
      at[ijk] += visc * ( cg0*(cg0*a[ijk-kk3] + cg1*a[ijk-kk2] + cg2*a[ijk-kk1] + cg3*a[ijk    ]) * dzhi4[k-1]
                        + cg1*(cg0*a[ijk-kk2] + cg1*a[ijk-kk1] + cg2*a[ijk    ] + cg3*a[ijk+kk1]) * dzhi4[k  ]
                        + cg2*(cg0*a[ijk-kk1] + cg1*a[ijk    ] + cg2*a[ijk+kk1] + cg3*a[ijk+kk2]) * dzhi4[k+1]
                        + cg3*(tg0*a[ijk-kk1] + tg1*a[ijk    ] + tg2*a[ijk+kk1] + tg3*a[ijk+kk2]) * dzhi4[k+2] )
                        * dzi4[k];
    }
    // interior
    else
    {
      at[ijk] += visc * (cdg3*a[ijk-ii3] + cdg2*a[ijk-ii2] + cdg1*a[ijk-ii1] + cdg0*a[ijk] + cdg1*a[ijk+ii1] + cdg2*a[ijk+ii2] + cdg3*a[ijk+ii3])*dxidxi;
      at[ijk] += visc * (cdg3*a[ijk-jj3] + cdg2*a[ijk-jj2] + cdg1*a[ijk-jj1] + cdg0*a[ijk] + cdg1*a[ijk+jj1] + cdg2*a[ijk+jj2] + cdg3*a[ijk+jj3])*dyidyi;
      at[ijk] += visc * ( cg0*(cg0*a[ijk-kk3] + cg1*a[ijk-kk2] + cg2*a[ijk-kk1] + cg3*a[ijk    ]) * dzhi4[k-1]
                        + cg1*(cg0*a[ijk-kk2] + cg1*a[ijk-kk1] + cg2*a[ijk    ] + cg3*a[ijk+kk1]) * dzhi4[k  ]
                        + cg2*(cg0*a[ijk-kk1] + cg1*a[ijk    ] + cg2*a[ijk+kk1] + cg3*a[ijk+kk2]) * dzhi4[k+1]
                        + cg3*(cg0*a[ijk    ] + cg1*a[ijk+kk1] + cg2*a[ijk+kk2] + cg3*a[ijk+kk3]) * dzhi4[k+2] )
                        * dzi4[k];
    }
  }
}

__global__ void diffw_kernel(double * __restrict__ at, double * __restrict__ a, double visc,
                             int jj, int kk,
                             int istart, int jstart, int kstart,
                             int iend, int jend, int kend,
                             double dx, double dy,
                             double * __restrict__ dzi4, double * __restrict__ dzhi4)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    int ii1,ii2,ii3,jj1,jj2,jj3,kk1,kk2,kk3;
    double dxidxi,dyidyi;

    ii1 = 1;
    ii2 = 2;
    ii3 = 3;
    jj1 = 1*jj;
    jj2 = 2*jj;
    jj3 = 3*jj;
    kk1 = 1*kk;
    kk2 = 2*kk;
    kk3 = 3*kk;

    dxidxi = 1./(dx*dx);
    dyidyi = 1./(dy*dy);

    // bottom boundary
    if(k == kstart+1)
    {
      at[ijk] += visc * (cdg3*a[ijk-ii3] + cdg2*a[ijk-ii2] + cdg1*a[ijk-ii1] + cdg0*a[ijk] + cdg1*a[ijk+ii1] + cdg2*a[ijk+ii2] + cdg3*a[ijk+ii3])*dxidxi;
      at[ijk] += visc * (cdg3*a[ijk-jj3] + cdg2*a[ijk-jj2] + cdg1*a[ijk-jj1] + cdg0*a[ijk] + cdg1*a[ijk+jj1] + cdg2*a[ijk+jj2] + cdg3*a[ijk+jj3])*dyidyi;
      at[ijk] += visc * ( cg0*(bg0*a[ijk-kk2] + bg1*a[ijk-kk1] + bg2*a[ijk    ] + bg3*a[ijk+kk1]) * dzi4[k-1]
                        + cg1*(cg0*a[ijk-kk2] + cg1*a[ijk-kk1] + cg2*a[ijk    ] + cg3*a[ijk+kk1]) * dzi4[k  ]
                        + cg2*(cg0*a[ijk-kk1] + cg1*a[ijk    ] + cg2*a[ijk+kk1] + cg3*a[ijk+kk2]) * dzi4[k+1]
                        + cg3*(cg0*a[ijk    ] + cg1*a[ijk+kk1] + cg2*a[ijk+kk2] + cg3*a[ijk+kk3]) * dzi4[k+2] )
                        * dzhi4[k];
    }
    else if(k == kend-1)
    {
      // top boundary
      at[ijk] += visc * (cdg3*a[ijk-ii3] + cdg2*a[ijk-ii2] + cdg1*a[ijk-ii1] + cdg0*a[ijk] + cdg1*a[ijk+ii1] + cdg2*a[ijk+ii2] + cdg3*a[ijk+ii3])*dxidxi;
      at[ijk] += visc * (cdg3*a[ijk-jj3] + cdg2*a[ijk-jj2] + cdg1*a[ijk-jj1] + cdg0*a[ijk] + cdg1*a[ijk+jj1] + cdg2*a[ijk+jj2] + cdg3*a[ijk+jj3])*dyidyi;
      at[ijk] += visc * ( cg0*(cg0*a[ijk-kk3] + cg1*a[ijk-kk2] + cg2*a[ijk-kk1] + cg3*a[ijk    ]) * dzi4[k-2]
                        + cg1*(cg0*a[ijk-kk2] + cg1*a[ijk-kk1] + cg2*a[ijk    ] + cg3*a[ijk+kk1]) * dzi4[k-1]
                        + cg2*(cg0*a[ijk-kk1] + cg1*a[ijk    ] + cg2*a[ijk+kk1] + cg3*a[ijk+kk2]) * dzi4[k  ]
                        + cg3*(tg0*a[ijk-kk1] + tg1*a[ijk    ] + tg2*a[ijk+kk1] + tg3*a[ijk+kk2]) * dzi4[k+1] )
                        * dzhi4[k];
    }
    else
    {
      // interior
      at[ijk] += visc * (cdg3*a[ijk-ii3] + cdg2*a[ijk-ii2] + cdg1*a[ijk-ii1] + cdg0*a[ijk] + cdg1*a[ijk+ii1] + cdg2*a[ijk+ii2] + cdg3*a[ijk+ii3])*dxidxi;
      at[ijk] += visc * (cdg3*a[ijk-jj3] + cdg2*a[ijk-jj2] + cdg1*a[ijk-jj1] + cdg0*a[ijk] + cdg1*a[ijk+jj1] + cdg2*a[ijk+jj2] + cdg3*a[ijk+jj3])*dyidyi;
      at[ijk] += visc * ( cg0*(cg0*a[ijk-kk3] + cg1*a[ijk-kk2] + cg2*a[ijk-kk1] + cg3*a[ijk    ]) * dzi4[k-2]
                        + cg1*(cg0*a[ijk-kk2] + cg1*a[ijk-kk1] + cg2*a[ijk    ] + cg3*a[ijk+kk1]) * dzi4[k-1]
                        + cg2*(cg0*a[ijk-kk1] + cg1*a[ijk    ] + cg2*a[ijk+kk1] + cg3*a[ijk+kk2]) * dzi4[k  ]
                        + cg3*(cg0*a[ijk    ] + cg1*a[ijk+kk1] + cg2*a[ijk+kk2] + cg3*a[ijk+kk3]) * dzi4[k+1] )
                        * dzhi4[k];
    }
  }
}

int cdiff_4::diffc_GPU(double *at, double *a, double *dzi4, double *dzhi4, double visc)
{
  const int blocki = 1;//128;
  const int blockj = 1;//2;
  const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  hipError_t err = hipGetLastError();
  master->printMessage("CvH diffc before: %s\n", hipGetErrorString(err));
  diffc_kernel<<<gridGPU, blockGPU>>>(at, a, visc,
                                      grid->icells, grid->ijcells,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend,
                                      grid->dx, grid->dy,
                                      grid->dzi4, grid->dzhi4);

  master->printMessage("CvH diffc after : %s\n", hipGetErrorString(err));

  return 0;
}

int cdiff_4::diffw_GPU(double *at, double *a, double *dzi4, double *dzhi4, double visc)
{
  const int blocki = 1;//128;
  const int blockj = 1;//2;
  const int gridi = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  hipError_t err = hipGetLastError();
  master->printMessage("CvH diffw before: %s\n", hipGetErrorString(err));
  diffw_kernel<<<gridGPU, blockGPU>>>(at, a, visc,
                                      grid->icells, grid->ijcells,
                                      grid->istart, grid->jstart, grid->kstart,
                                      grid->iend, grid->jend, grid->kend,
                                      grid->dx, grid->dy,
                                      grid->dzi4, grid->dzhi4);

  master->printMessage("CvH diffw after : %s\n", hipGetErrorString(err));

  return 0;
}
