/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "boundary_surface_lsm.h"
#include "boundary.h"
#include "tools.h"


#ifdef USECUDA
template<typename TF>
void Boundary_surface_lsm<TF>::exec(
        Thermo<TF>& thermo, Radiation<TF>& radiation,
        Microphys<TF>& microphys, Timeloop<TF>& timeloop)
{
}

template<typename TF>
void Boundary_surface_lsm<TF>::exec_column(Column<TF>& column)
{
}

template<typename TF>
void Boundary_surface_lsm<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);
    const int float_memsize_lut = nzL_lut*sizeof(float);

    // Monin-Obukhov stuff:
    cuda_safe_call(hipMalloc(&obuk_g,  tf_memsize_ij));
    cuda_safe_call(hipMalloc(&ustar_g, tf_memsize_ij));

    cuda_safe_call(hipMalloc(&z0m_g,   tf_memsize_ij));
    cuda_safe_call(hipMalloc(&z0h_g,   tf_memsize_ij));

    cuda_safe_call(hipMalloc(&dudz_mo_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&dvdz_mo_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&dbdz_mo_g, tf_memsize_ij));

    cuda_safe_call(hipMalloc(&nobuk_g, int_memsize_ij));
    cuda_safe_call(hipMalloc(&zL_sl_g, float_memsize_lut));
    cuda_safe_call(hipMalloc(&f_sl_g,  float_memsize_lut));

    // Land-surface stuff:

    // Copy data from host to device
    forward_device();
}

template<typename TF>
void Boundary_surface_lsm<TF>::forward_device()
{
    auto& gd = grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);
    const int float_memsize_lut = nzL_lut*sizeof(float);

    cuda_safe_call(hipMemcpy(obuk_g,  obuk.data(),  tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(ustar_g, ustar.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(z0m_g, z0m.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(z0h_g, z0h.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(dudz_mo_g, dudz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(dvdz_mo_g, dvdz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(dbdz_mo_g, dbdz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(nobuk_g, nobuk.data(), int_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(zL_sl_g, zL_sl.data(), float_memsize_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(f_sl_g,  f_sl.data(),  float_memsize_lut, hipMemcpyHostToDevice));
}

template<typename TF>
void Boundary_surface_lsm<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);

    // NOTE: only copy back the required/useful data...

    cuda_safe_call(hipMemcpy(obuk.data(),  obuk_g,  tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(ustar.data(), ustar_g, tf_memsize_ij, hipMemcpyDeviceToHost));

    cuda_safe_call(hipMemcpy(dudz_mo.data(), dudz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(dvdz_mo.data(), dvdz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(dbdz_mo.data(), dbdz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));

    cuda_safe_call(hipMemcpy(nobuk.data(), nobuk_g, int_memsize_ij, hipMemcpyDeviceToHost));
}

template<typename TF>
void Boundary_surface_lsm<TF>::clear_device()
{
    //
    // De-llocate fields on GPU
    //
    // Monin-Obukhov stuff:
    cuda_safe_call(hipFree(obuk_g));
    cuda_safe_call(hipFree(ustar_g));

    cuda_safe_call(hipFree(z0m_g));
    cuda_safe_call(hipFree(z0h_g));

    cuda_safe_call(hipFree(dudz_mo_g));
    cuda_safe_call(hipFree(dvdz_mo_g));
    cuda_safe_call(hipFree(dbdz_mo_g));

    cuda_safe_call(hipFree(nobuk_g));
    cuda_safe_call(hipFree(zL_sl_g));
    cuda_safe_call(hipFree(f_sl_g));

    // Land-surface stuff:
}
#endif

template class Boundary_surface_lsm<double>;
template class Boundary_surface_lsm<float>;
