/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "boundary_surface_lsm.h"
#include "boundary.h"
#include "land_surface_kernels_gpu.h"
#include "tools.h"
#include "grid.h"
#include "soil_grid.h"

namespace
{
    namespace lsmk = Land_surface_kernels_g;
}


#ifdef USECUDA
template<typename TF>
void Boundary_surface_lsm<TF>::exec(
        Thermo<TF>& thermo, Radiation<TF>& radiation,
        Microphys<TF>& microphys, Timeloop<TF>& timeloop)
{
}

template<typename TF>
void Boundary_surface_lsm<TF>::exec_column(Column<TF>& column)
{
}

template<typename TF>
void Boundary_surface_lsm<TF>::prepare_device()
{
    auto& gd = grid.get_grid_data();
    auto& sgd = soil_grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);
    const int float_memsize_mo_lut = nzL_lut*sizeof(float);

    // Surface layer / Monin-Obukhov:
    cuda_safe_call(hipMalloc(&obuk_g,  tf_memsize_ij));
    cuda_safe_call(hipMalloc(&ustar_g, tf_memsize_ij));

    cuda_safe_call(hipMalloc(&z0m_g,   tf_memsize_ij));
    cuda_safe_call(hipMalloc(&z0h_g,   tf_memsize_ij));

    cuda_safe_call(hipMalloc(&dudz_mo_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&dvdz_mo_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&dbdz_mo_g, tf_memsize_ij));

    cuda_safe_call(hipMalloc(&nobuk_g, int_memsize_ij));
    cuda_safe_call(hipMalloc(&zL_sl_g, float_memsize_mo_lut));
    cuda_safe_call(hipMalloc(&f_sl_g,  float_memsize_mo_lut));

    // Land-surface:
    // 1. Init tiles:
    for (auto& tile : tiles)
        lsmk::init_tile(tile.second, gd.ijcells);

    // 2. Init 2D surface properties:
    cuda_safe_call(hipMalloc(&gD_coeff_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&c_veg_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&lai_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&rs_veg_min_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&rs_soil_min_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&lambda_stable_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&lambda_unstable_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&cs_veg_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&water_mask_g, int_memsize_ij));
    cuda_safe_call(hipMalloc(&t_bot_water_g, tf_memsize_ij));

    cuda_safe_call(hipMalloc(&interception_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&throughfall_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&infiltration_g, tf_memsize_ij));
    cuda_safe_call(hipMalloc(&runoff_g, tf_memsize_ij));

    // 3. Init 3D soil properties:
    const int tf_memsize_ijk  = sgd.ncells*sizeof(TF);
    const int int_memsize_ijk = sgd.ncells*sizeof(int);

    cuda_safe_call(hipMalloc(&soil_index_g, int_memsize_ijk));
    cuda_safe_call(hipMalloc(&diffusivity_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&diffusivity_h_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&conductivity_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&conductivity_h_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&source_g, tf_memsize_ijk));
    cuda_safe_call(hipMalloc(&root_fraction_g, tf_memsize_ijk));

    // 4. Init lookup table with van Genuchten parameters:
    const int memsize_vg_lut = theta_res.size() * sizeof(TF);

    cuda_safe_call(hipMalloc(&theta_res_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&theta_wp_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&theta_fc_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&theta_sat_g, memsize_vg_lut));

    cuda_safe_call(hipMalloc(&gamma_theta_sat_g, memsize_vg_lut));

    cuda_safe_call(hipMalloc(&vg_a_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&vg_l_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&vg_n_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&vg_m_g, memsize_vg_lut));

    cuda_safe_call(hipMalloc(&kappa_theta_max_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&kappa_theta_min_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&gamma_theta_max_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&gamma_theta_min_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&gamma_T_dry_g, memsize_vg_lut));
    cuda_safe_call(hipMalloc(&rho_C_g, memsize_vg_lut));

    // Copy data from host to device
    forward_device();
}

template<typename TF>
void Boundary_surface_lsm<TF>::forward_device()
{
    auto& gd = grid.get_grid_data();
    auto& sgd = soil_grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);
    const int float_memsize_lut = nzL_lut*sizeof(float);

    // Surface layer / Monin-Obukhov:
    cuda_safe_call(hipMemcpy(obuk_g,  obuk.data(),  tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(ustar_g, ustar.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(z0m_g, z0m.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(z0h_g, z0h.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(dudz_mo_g, dudz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(dvdz_mo_g, dvdz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(dbdz_mo_g, dbdz_mo.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(nobuk_g, nobuk.data(), int_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(zL_sl_g, zL_sl.data(), float_memsize_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(f_sl_g,  f_sl.data(),  float_memsize_lut, hipMemcpyHostToDevice));

    // Land-surface:
    // 1. Copy tiles:
    for (auto& tile : tiles)
        lsmk::forward_device_tile(tile.second, gd.ijcells);

    // 2. Copy 2D surface properties:
    cuda_safe_call(hipMemcpy(gD_coeff_g, gD_coeff.data(), int_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(c_veg_g, c_veg.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(lai_g, lai.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(rs_veg_min_g, rs_veg_min.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(rs_soil_min_g, rs_soil_min.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(lambda_stable_g, lambda_stable.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(lambda_unstable_g, lambda_unstable.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(cs_veg_g, cs_veg.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(water_mask_g, water_mask.data(), int_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(t_bot_water_g, t_bot_water.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(interception_g, interception.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(throughfall_g, throughfall.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(infiltration_g, infiltration.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(runoff_g, runoff.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    // 3. Copy 3D soil properties:
    const int tf_memsize_ijk  = sgd.ncells*sizeof(TF);
    const int int_memsize_ijk = sgd.ncells*sizeof(int);

    cuda_safe_call(hipMemcpy(soil_index_g, soil_index.data(), int_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(diffusivity_g, diffusivity.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(diffusivity_h_g, diffusivity_h.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(conductivity_g, conductivity.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(conductivity_h_g, conductivity_h.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(source_g, source.data(), tf_memsize_ij, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(root_fraction_g, root_fraction.data(), tf_memsize_ij, hipMemcpyHostToDevice));

    // 4. Copy lookup table with van Genuchten parameters:
    const int memsize_vg_lut = theta_res.size() * sizeof(TF);

    cuda_safe_call(hipMemcpy(theta_res_g, theta_res.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(theta_wp_g, theta_wp.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(theta_fc_g, theta_fc.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(theta_sat_g, theta_sat.data(), memsize_vg_lut, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(gamma_theta_sat_g, gamma_theta_sat.data(), memsize_vg_lut, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(vg_a_g, vg_a.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(vg_l_g, vg_l.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(vg_n_g, vg_n.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(vg_m_g, vg_m.data(), memsize_vg_lut, hipMemcpyHostToDevice));

    cuda_safe_call(hipMemcpy(kappa_theta_max_g, kappa_theta_max.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(kappa_theta_min_g, kappa_theta_min.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gamma_theta_max_g, gamma_theta_max.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gamma_theta_min_g, gamma_theta_min.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(gamma_T_dry_g, gamma_T_dry.data(), memsize_vg_lut, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(rho_C_g, rho_C.data(), memsize_vg_lut, hipMemcpyHostToDevice));
}

template<typename TF>
void Boundary_surface_lsm<TF>::backward_device()
{
    auto& gd = grid.get_grid_data();

    const int tf_memsize_ij  = gd.ijcells*sizeof(TF);
    const int int_memsize_ij = gd.ijcells*sizeof(int);

    // NOTE: only copy back the required/useful data...

    cuda_safe_call(hipMemcpy(obuk.data(),  obuk_g,  tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(ustar.data(), ustar_g, tf_memsize_ij, hipMemcpyDeviceToHost));

    cuda_safe_call(hipMemcpy(dudz_mo.data(), dudz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(dvdz_mo.data(), dvdz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));
    cuda_safe_call(hipMemcpy(dbdz_mo.data(), dbdz_mo_g, tf_memsize_ij, hipMemcpyDeviceToHost));

    cuda_safe_call(hipMemcpy(nobuk.data(), nobuk_g, int_memsize_ij, hipMemcpyDeviceToHost));

    // TODO: which fields are needed from the land-surface?
}

template<typename TF>
void Boundary_surface_lsm<TF>::clear_device()
{
    //
    // De-llocate fields on GPU
    //
    // Monin-Obukhov stuff:
    cuda_safe_call(hipFree(obuk_g));
    cuda_safe_call(hipFree(ustar_g));

    cuda_safe_call(hipFree(z0m_g));
    cuda_safe_call(hipFree(z0h_g));

    cuda_safe_call(hipFree(dudz_mo_g));
    cuda_safe_call(hipFree(dvdz_mo_g));
    cuda_safe_call(hipFree(dbdz_mo_g));

    cuda_safe_call(hipFree(nobuk_g));
    cuda_safe_call(hipFree(zL_sl_g));
    cuda_safe_call(hipFree(f_sl_g));

    // Land-surface stuff:
}
#endif

template class Boundary_surface_lsm<double>;
template class Boundary_surface_lsm<float>;
