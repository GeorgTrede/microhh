#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include "grid.h"
#include "fields.h"
#include "master.h"
#include "diff_les2s.h"
#include "boundary_surface.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "model.h"

__device__ double diff_les2s_phim(double zeta)
{
  double phim;
  if(zeta <= 0.)
    phim = pow(1. + 3.6*pow(fabs(zeta), 2./3.), -1./2.);
  else
    phim = 1. + 5.*zeta;
  return phim;
}

__device__ double diff_les2s_phih(double zeta)
{
  double phih;
  if(zeta <= 0.)
    phih = pow(1. + 7.9*pow(fabs(zeta), 2./3.), -1./2.);
  else
    phih = 1. + 5.*zeta;
  return phih;
}

__global__ void diff_les2s_strain2(double * __restrict__ strain2,
                                   double * __restrict__ u,  double * __restrict__ v,  double * __restrict__ w,
                                   double * __restrict__ ufluxbot, double * __restrict__ vfluxbot,
                                   double * __restrict__ ustar, double * __restrict__ obuk, 
                                   double * __restrict__ z, double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi, 
                                   int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                   int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  const int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      strain2[ijk] = 2.*(
        // du/dz
        + 0.5*pow(-0.5*(ufluxbot[ij]+ufluxbot[ij+ii])/(constants::kappa*z[k]*ustar[ij])*diff_les2s_phim(z[k]/obuk[ij]), 2)
        // dv/dz
        + 0.5*pow(-0.5*(vfluxbot[ij]+vfluxbot[ij+jj])/(constants::kappa*z[k]*ustar[ij])*diff_les2s_phim(z[k]/obuk[ij]), 2) );
       // add a small number to avoid zero divisions
       strain2[ijk] += constants::dsmall;  
    }
    else
    {
      strain2[ijk] = 2.*(
        // du/dx + du/dx
        + pow((u[ijk+ii]-u[ijk])*dxi, 2.)
        // dv/dy + dv/dy
        + pow((v[ijk+jj]-v[ijk])*dyi, 2.)
        // dw/dz + dw/dz
        + pow((w[ijk+kk]-w[ijk])*dzi[k], 2.)
        // du/dy + dv/dx
        + 0.125*pow((u[ijk      ]-u[ijk   -jj])*dyi  + (v[ijk      ]-v[ijk-ii   ])*dxi, 2)
        + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-jj])*dyi  + (v[ijk+ii   ]-v[ijk      ])*dxi, 2)
        + 0.125*pow((u[ijk   +jj]-u[ijk      ])*dyi  + (v[ijk   +jj]-v[ijk-ii+jj])*dxi, 2)
        + 0.125*pow((u[ijk+ii+jj]-u[ijk+ii   ])*dyi  + (v[ijk+ii+jj]-v[ijk   +jj])*dxi, 2)
        // du/dz + dw/dx
        + 0.125*pow((u[ijk      ]-u[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-ii   ])*dxi, 2)
        + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-kk])*dzhi[k  ] + (w[ijk+ii   ]-w[ijk      ])*dxi, 2)
        + 0.125*pow((u[ijk   +kk]-u[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-ii+kk])*dxi, 2)
        + 0.125*pow((u[ijk+ii+kk]-u[ijk+ii   ])*dzhi[k+1] + (w[ijk+ii+kk]-w[ijk   +kk])*dxi, 2)
        // dv/dz + dw/dy
        + 0.125*pow((v[ijk      ]-v[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-jj   ])*dyi, 2)
        + 0.125*pow((v[ijk+jj   ]-v[ijk+jj-kk])*dzhi[k  ] + (w[ijk+jj   ]-w[ijk      ])*dyi, 2)
        + 0.125*pow((v[ijk   +kk]-v[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-jj+kk])*dyi, 2)
        + 0.125*pow((v[ijk+jj+kk]-v[ijk+jj   ])*dzhi[k+1] + (w[ijk+jj+kk]-w[ijk   +kk])*dyi, 2) );
      // add a small number to avoid zero divisions
      strain2[ijk] += constants::dsmall;
    }
  }
}

__global__ void diff_les2s_evisc(double * __restrict__ evisc, double * __restrict__ N2,
                                 double * __restrict__ bfluxbot, double * __restrict__ ustar, double * __restrict__ obuk,
                                 double * __restrict__ z, double * __restrict__ dz, double dx, double dy,
                                 double cs, double tPr, double z0m,
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;
    double n = 2;

    /* BvS: TODO: pre-calculate fac offline. Now every thread within one vertical slice 
       has to calculate it.... */
    if(k == kstart)
    {
      // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
      double mlen0      = cs * pow(dx*dy*dz[k], 1./3.);
      double mlen       = pow(1./(1./pow(mlen0, n) + 1./(pow(constants::kappa*(z[k]+z0m), n))), 1./n);
      double fac        = pow(mlen, 2);
      double RitPrratio = -bfluxbot[ij]/(constants::kappa*z[k]*ustar[ij])*diff_les2s_phih(z[k]/obuk[ij]) / evisc[ijk] / tPr;
      RitPrratio        = fmin(RitPrratio, 1.-constants::dsmall);
      evisc[ijk]        = fac * sqrt(evisc[ijk]) * sqrt(1.-RitPrratio);
    }
    else
    {
      // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
      double mlen0      = cs * pow(dx*dy*dz[k], 1./3.);
      double mlen       = pow(1./(1./pow(mlen0, n) + 1./(pow(constants::kappa*(z[k]+z0m), n))), 1./n);
      double fac        = std::pow(mlen, 2.);
      double RitPrratio = N2[ijk] / evisc[ijk] / tPr;
      RitPrratio        = fmin(RitPrratio, 1.-constants::dsmall);
      evisc[ijk]        = fac * sqrt(evisc[ijk]) * sqrt(1.-RitPrratio);
    }
  }
}

__global__ void diff_les2s_diffu(double * __restrict__ ut, double * __restrict__ evisc,
                                 double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ fluxbot, double * __restrict__ fluxtop, 
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  double eviscn, eviscs, eviscb, evisct;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      eviscn = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
      eviscs = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscn*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscs*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (  rhorefh[kstart+1] * evisct*((u[ijk+kk]-u[ijk   ])* dzhi[kstart+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
               + rhorefh[kstart  ] * fluxbot[ij] ) / rhoref[kstart] * dzi[kstart];
    }
    else if(k == kend-1)
    {
      eviscn = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
      eviscs = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscn*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscs*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (- rhorefh[kend  ] * fluxtop[ij]
               - rhorefh[kend-1] * eviscb*((u[ijk   ]-u[ijk-kk])* dzhi[kend-1] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[kend-1] * dzi[kend-1];
    }
    else
    {
      eviscn = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
      eviscs = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscn*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscs*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (  rhorefh[k+1] * evisct*((u[ijk+kk]-u[ijk   ])* dzhi[k+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
               - rhorefh[k  ] * eviscb*((u[ijk   ]-u[ijk-kk])* dzhi[k  ] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[k] * dzi[k];
    }
  }
}

__global__ void diff_les2s_diffv(double * __restrict__ vt, double * __restrict__ evisc,
                                 double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ fluxbot, double * __restrict__ fluxtop, 
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  double evisce,eviscw,eviscb,evisct;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      evisce = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
      eviscw = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);
      vt[ijk] +=
            // dv/dx + du/dy
            + (  evisce*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscw*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (  rhorefh[k+1] * evisct*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
               + rhorefh[k  ] * fluxbot[ij] ) / rhoref[k] * dzi[k];
    }
    else if(k == kend-1)
    {
      evisce = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
      eviscw = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);
      vt[ijk] +=
            // dv/dx + du/dy
            + (  evisce*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscw*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (- rhorefh[k  ] * fluxtop[ij]
               - rhorefh[k-1] * eviscb*((v[ijk   ]-v[ijk-kk])*dzhi[k-1] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k-1] * dzi[k-1];
    }
    else
    {
      evisce = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
      eviscw = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);
      vt[ijk] +=
            // dv/dx + du/dy
            + (  evisce*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscw*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (  rhorefh[k+1] * evisct*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
               - rhorefh[k  ] * eviscb*((v[ijk   ]-v[ijk-kk])*dzhi[k  ] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k] * dzi[k];
    }
  }
}

__global__ void diff_les2s_diffw(double * __restrict__ wt, double * __restrict__ evisc,
                                 double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart+1;
  double evisce, eviscw, eviscn, eviscs;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ijk = i + j*jj + k*kk;

    evisce = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+ii-kk] + evisc[ijk+ii]);
    eviscw = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-ii] + evisc[ijk   -kk] + evisc[ijk   ]);
    eviscn = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+jj-kk] + evisc[ijk+jj]);
    eviscs = 0.25*(evisc[ijk-jj-kk] + evisc[ijk-jj] + evisc[ijk   -kk] + evisc[ijk   ]);
    wt[ijk] +=
          // dw/dx + du/dz
          + (  evisce*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
             - eviscw*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
          // dw/dy + dv/dz
          + (  eviscn*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
             - eviscs*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
          // dw/dz + dw/dz
          + (  rhoref[k  ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzi[k  ]
             - rhoref[k-1] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzi[k-1] ) / rhorefh[k] * 2.* dzhi[k];
  }
}

__global__ void diff_les2s_diffc(double * __restrict__ at, double * __restrict__ a, double * __restrict__ evisc,
                                 double * __restrict__ fluxbot, double * __restrict__ fluxtop, 
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxidxi, double dyidyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, double tPr, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  double evisce,eviscw,eviscn,eviscs,evisct,eviscb;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])/tPr;
      eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])/tPr;
      eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])/tPr;
      eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])/tPr;
      evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])/tPr;
      eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])/tPr;

      at[ijk] +=
            + (  evisce*(a[ijk+ii]-a[ijk   ]) 
               - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
            + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
               - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
            + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
               + rhorefh[k  ] * fluxbot[ij] ) / rhoref[k] * dzi[k];
    }
    else if(k == kend-1)
    {
      evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])/tPr;
      eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])/tPr;
      eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])/tPr;
      eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])/tPr;
      evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])/tPr;
      eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])/tPr;

      at[ijk] +=
            + (  evisce*(a[ijk+ii]-a[ijk   ]) 
               - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
            + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
               - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
            + (- rhorefh[k  ] * fluxtop[ij]
               - rhorefh[k-1] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k-1] ) / rhoref[k-1] * dzi[k-1];
    }
    else
    {
      evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])/tPr;
      eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])/tPr;
      eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])/tPr;
      eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])/tPr;
      evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])/tPr;
      eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])/tPr;

      at[ijk] +=
            + (  evisce*(a[ijk+ii]-a[ijk   ]) 
               - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
            + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
               - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
            + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
               - rhorefh[k  ] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k]  ) / rhoref[k] * dzi[k];
    }
  }
}



/*
#ifdef USECUDA
int cdiff_les2s::execvisc()
{
  // do a cast because the base boundary class does not have the MOST related variables
  cboundary_surface *boundaryptr = static_cast<cboundary_surface *>(model->boundary);

  fields->forwardDevice();
  boundaryptr->forwardDevice();

  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  // Calculate total strain rate
  diff_les2s_strain2<<<gridGPU, blockGPU>>>(&fields->s["evisc"]->data_g[offs], 
                                            &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                            &fields->u->datafluxbot_g[offs],  &fields->v->datafluxbot_g[offs],
                                            &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
                                            grid->z_g, grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                            grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                            grid->icellsp, grid->ijcellsp);  

  fields->backwardDevice();

  // start with retrieving the stability information
  if(model->thermo->getsw() == "0")
  {
    master->printMessage("diff_les2s without thermo not yet supported on GPU\n");
    //evisc_neutral(fields->s["evisc"]->data,
    //              fields->u->data, fields->v->data, fields->w->data,
    //              fields->u->datafluxbot, fields->v->datafluxbot,
    //              grid->z, grid->dz, boundaryptr->z0m);
  }
  // assume buoyancy calculation is needed
  else
  {
    // store the buoyancyflux in datafluxbot of tmp1
    model->thermo->getbuoyancyfluxbot(fields->sd["tmp1"]);
    // store the Brunt-vaisala frequency in data of tmp1 
    model->thermo->getthermofield(fields->sd["tmp1"], fields->sd["tmp2"], "N2");

    fields->forwardDevice();

    // Calculate eddy viscosity
    diff_les2s_evisc<<<gridGPU, blockGPU>>>(&fields->s["evisc"]->data_g[offs], &fields->s["tmp1"]->data_g[offs], 
                                            &fields->sd["tmp1"]->datafluxbot_g[offs], &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
                                            grid->z_g, grid->dz_g, grid->dx, grid->dy, cs, tPr, boundaryptr->z0m,
                                            grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                            grid->icellsp, grid->ijcellsp);  
    grid->boundary_cyclic_g(&fields->sd["evisc"]->data_g[offs]);

  }

  fields->backwardDevice();

  return 0;
}
#endif
*/

#ifdef USECUDA
int cdiff_les2s::exec()
{
  fields->forwardDevice();

  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;
  const double dxidxi = 1./(grid->dx * grid->dx);
  const double dyidyi = 1./(grid->dy * grid->dy);

  diff_les2s_diffu<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->s["evisc"]->data_g[offs], 
                                          &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                          &fields->u->datafluxbot_g[offs], &fields->u->datafluxtop_g[offs],
                                          grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                          fields->rhoref_g, fields->rhorefh_g,
                                          grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                          grid->icellsp, grid->ijcellsp);  

  diff_les2s_diffv<<<gridGPU, blockGPU>>>(&fields->vt->data_g[offs], &fields->s["evisc"]->data_g[offs], 
                                          &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                          &fields->v->datafluxbot_g[offs], &fields->v->datafluxtop_g[offs],
                                          grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                          fields->rhoref_g, fields->rhorefh_g,
                                          grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                          grid->icellsp, grid->ijcellsp);  

  diff_les2s_diffw<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->s["evisc"]->data_g[offs], 
                                          &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                          grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                          fields->rhoref_g, fields->rhorefh_g,
                                          grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                          grid->icellsp, grid->ijcellsp);  

  for(fieldmap::const_iterator it = fields->st.begin(); it!=fields->st.end(); ++it)
    diff_les2s_diffc<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], &fields->s[it->first]->data_g[offs], &fields->s["evisc"]->data_g[offs], 
                                            &fields->s[it->first]->datafluxbot_g[offs], &fields->s[it->first]->datafluxtop_g[offs],
                                            grid->dzi_g, grid->dzhi_g, dxidxi, dyidyi,
                                            fields->rhoref_g, fields->rhorefh_g, tPr,
                                            grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                            grid->icellsp, grid->ijcellsp);  

  fields->backwardDevice();

  return 0;
}
#endif

