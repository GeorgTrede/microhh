#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include <cmath>
#include <algorithm>
#include "grid.h"
#include "fields.h"
#include "master.h"
#include "diff_les2s.h"
#include "boundary_surface.h"
#include "defines.h"
#include "constants.h"
#include "thermo.h"
#include "model.h"

__device__ double diff_les2s_phim(double zeta)
{
  double phim;
  if(zeta <= 0.)
    phim = pow(1. + 3.6*pow(fabs(zeta), 2./3.), -1./2.);
  else
    phim = 1. + 5.*zeta;
  return phim;
}

__device__ double diff_les2s_phih(double zeta)
{
  double phih;
  if(zeta <= 0.)
    phih = pow(1. + 7.9*pow(fabs(zeta), 2./3.), -1./2.);
  else
    phih = 1. + 5.*zeta;
  return phih;
}

__global__ void diff_les2s_strain2(double * __restrict__ strain2,
                                   double * __restrict__ u,  double * __restrict__ v,  double * __restrict__ w,
                                   double * __restrict__ ufluxbot, double * __restrict__ vfluxbot,
                                   double * __restrict__ ustar, double * __restrict__ obuk, 
                                   double * __restrict__ z, double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi, 
                                   int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                   int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  const int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      strain2[ijk] = 2.*(
        // du/dz
        + 0.5*pow(-0.5*(ufluxbot[ij]+ufluxbot[ij+ii])/(constants::kappa*z[k]*ustar[ij])*diff_les2s_phim(z[k]/obuk[ij]), 2)
        // dv/dz
        + 0.5*pow(-0.5*(vfluxbot[ij]+vfluxbot[ij+jj])/(constants::kappa*z[k]*ustar[ij])*diff_les2s_phim(z[k]/obuk[ij]), 2) );
       // add a small number to avoid zero divisions
       strain2[ijk] += constants::dsmall;  
    }
    else
    {
      strain2[ijk] = 2.*(
        // du/dx + du/dx
        + pow((u[ijk+ii]-u[ijk])*dxi, 2)
        // dv/dy + dv/dy
        + pow((v[ijk+jj]-v[ijk])*dyi, 2)
        // dw/dz + dw/dz
        + pow((w[ijk+kk]-w[ijk])*dzi[k], 2)
        // du/dy + dv/dx
        + 0.125*pow((u[ijk      ]-u[ijk   -jj])*dyi  + (v[ijk      ]-v[ijk-ii   ])*dxi, 2)
        + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-jj])*dyi  + (v[ijk+ii   ]-v[ijk      ])*dxi, 2)
        + 0.125*pow((u[ijk   +jj]-u[ijk      ])*dyi  + (v[ijk   +jj]-v[ijk-ii+jj])*dxi, 2)
        + 0.125*pow((u[ijk+ii+jj]-u[ijk+ii   ])*dyi  + (v[ijk+ii+jj]-v[ijk   +jj])*dxi, 2)
        // du/dz + dw/dx
        + 0.125*pow((u[ijk      ]-u[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-ii   ])*dxi, 2)
        + 0.125*pow((u[ijk+ii   ]-u[ijk+ii-kk])*dzhi[k  ] + (w[ijk+ii   ]-w[ijk      ])*dxi, 2)
        + 0.125*pow((u[ijk   +kk]-u[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-ii+kk])*dxi, 2)
        + 0.125*pow((u[ijk+ii+kk]-u[ijk+ii   ])*dzhi[k+1] + (w[ijk+ii+kk]-w[ijk   +kk])*dxi, 2)
        // dv/dz + dw/dy
        + 0.125*pow((v[ijk      ]-v[ijk   -kk])*dzhi[k  ] + (w[ijk      ]-w[ijk-jj   ])*dyi, 2)
        + 0.125*pow((v[ijk+jj   ]-v[ijk+jj-kk])*dzhi[k  ] + (w[ijk+jj   ]-w[ijk      ])*dyi, 2)
        + 0.125*pow((v[ijk   +kk]-v[ijk      ])*dzhi[k+1] + (w[ijk   +kk]-w[ijk-jj+kk])*dyi, 2)
        + 0.125*pow((v[ijk+jj+kk]-v[ijk+jj   ])*dzhi[k+1] + (w[ijk+jj+kk]-w[ijk   +kk])*dyi, 2) );
      // add a small number to avoid zero divisions
      strain2[ijk] += constants::dsmall;
    }
  }
}

__global__ void diff_les2s_evisc(double * __restrict__ evisc, double * __restrict__ N2,
                                 double * __restrict__ bfluxbot, double * __restrict__ ustar, double * __restrict__ obuk,
                                 double * __restrict__ mlen,
                                 double tPri, double z0m, double zsl,
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  //__shared__ double fac;
  
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
      double RitPrratio = -bfluxbot[ij]/(constants::kappa*zsl*ustar[ij])*diff_les2s_phih(zsl/obuk[ij]) / evisc[ijk] * tPri;
      RitPrratio        = fmin(RitPrratio, 1.-constants::dsmall);
      evisc[ijk]        = mlen[k] * sqrt(evisc[ijk] * (1.-RitPrratio));
    }
    else
    {
      // calculate smagorinsky constant times filter width squared, use wall damping according to Mason
      double RitPrratio = N2[ijk] / evisc[ijk] * tPri;
      RitPrratio        = fmin(RitPrratio, 1.-constants::dsmall);
      evisc[ijk]        = mlen[k] * sqrt(evisc[ijk] * (1.-RitPrratio));
    }
  }
}

__global__ void diff_les2s_diffu(double * __restrict__ ut, double * __restrict__ evisc,
                                 double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ fluxbot, double * __restrict__ fluxtop, 
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  double eviscn, eviscs, eviscb, evisct;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      eviscn = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
      eviscs = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscn*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscs*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (  rhorefh[kstart+1] * evisct*((u[ijk+kk]-u[ijk   ])* dzhi[kstart+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
               + rhorefh[kstart  ] * fluxbot[ij] ) / rhoref[kstart] * dzi[kstart];
    }
    else if(k == kend-1)
    {
      eviscn = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
      eviscs = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscn*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscs*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (- rhorefh[kend  ] * fluxtop[ij]
               - rhorefh[kend-1] * eviscb*((u[ijk   ]-u[ijk-kk])* dzhi[kend-1] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[kend-1] * dzi[kend-1];
    }
    else
    {
      eviscn = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
      eviscs = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscn*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscs*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (  rhorefh[k+1] * evisct*((u[ijk+kk]-u[ijk   ])* dzhi[k+1] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
               - rhorefh[k  ] * eviscb*((u[ijk   ]-u[ijk-kk])* dzhi[k  ] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhoref[k] * dzi[k];
    }
  }
}

__global__ void diff_les2s_diffv(double * __restrict__ vt, double * __restrict__ evisc,
                                 double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ fluxbot, double * __restrict__ fluxtop, 
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  double evisce,eviscw,eviscb,evisct;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      evisce = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
      eviscw = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);
      vt[ijk] +=
            // dv/dx + du/dy
            + (  evisce*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscw*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (  rhorefh[k+1] * evisct*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
               + rhorefh[k  ] * fluxbot[ij] ) / rhoref[k] * dzi[k];
    }
    else if(k == kend-1)
    {
      evisce = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
      eviscw = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);
      vt[ijk] +=
            // dv/dx + du/dy
            + (  evisce*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscw*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (- rhorefh[k  ] * fluxtop[ij]
               - rhorefh[k-1] * eviscb*((v[ijk   ]-v[ijk-kk])*dzhi[k-1] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k-1] * dzi[k-1];
    }
    else
    {
      evisce = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
      eviscw = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
      evisct = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
      eviscb = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);
      vt[ijk] +=
            // dv/dx + du/dy
            + (  evisce*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscw*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (  rhorefh[k+1] * evisct*((v[ijk+kk]-v[ijk   ])*dzhi[k+1] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
               - rhorefh[k  ] * eviscb*((v[ijk   ]-v[ijk-kk])*dzhi[k  ] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhoref[k] * dzi[k];
    }
  }
}

__global__ void diff_les2s_diffw(double * __restrict__ wt, double * __restrict__ evisc,
                                 double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart+1;
  double evisce, eviscw, eviscn, eviscs;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ijk = i + j*jj + k*kk;

    evisce = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+ii-kk] + evisc[ijk+ii]);
    eviscw = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-ii] + evisc[ijk   -kk] + evisc[ijk   ]);
    eviscn = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+jj-kk] + evisc[ijk+jj]);
    eviscs = 0.25*(evisc[ijk-jj-kk] + evisc[ijk-jj] + evisc[ijk   -kk] + evisc[ijk   ]);
    wt[ijk] +=
          // dw/dx + du/dz
          + (  evisce*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
             - eviscw*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
          // dw/dy + dv/dz
          + (  eviscn*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
             - eviscs*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
          // dw/dz + dw/dz
          + (  rhoref[k  ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzi[k  ]
             - rhoref[k-1] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzi[k-1] ) / rhorefh[k] * 2.* dzhi[k];
  }
}

__global__ void diff_les2s_diffuvw(double * __restrict__ ut, double * __restrict__ vt, double * __restrict__ wt, 
                                   double * __restrict__ evisc,
                                   double * __restrict__ u, double * __restrict__ v, double * __restrict__ w,
                                   double * __restrict__ fluxbotu, double * __restrict__ fluxtopu, 
                                   double * __restrict__ fluxbotv, double * __restrict__ fluxtopv, 
                                   double * __restrict__ dzi, double * __restrict__ dzhi, double dxi, double dyi,
                                   double * __restrict__ rhoref, double * __restrict__ rhorefh, 
                                   int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                   int jj, int kk)

{
  __shared__ double s[12]; // Contains rhoref, rhorefh, dzi, dzhi at k-1, k, k+1
  double * rhorefs  = &s[0];
  double * rhorefhs = &s[3];
  double * dzis     = &s[6];
  double * dzhis    = &s[9];

  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  double eviscnu, eviscsu, eviscbu, evisctu;
  double eviscev, eviscwv, eviscbv, evisctv;
  double eviscew, eviscww, eviscnw, eviscsw;

  const int kms = 0;
  const int ks  = 1;
  const int kps = 2;  

  if(threadIdx.x == 0 and threadIdx.y == 0)
  {
    rhorefs[kms]  = rhoref[k-1];
    rhorefs[ks]   = rhoref[k];
    rhorefs[kps]  = rhoref[k+1];
    rhorefhs[kms] = rhorefh[k-1];
    rhorefhs[ks]  = rhorefh[k];
    rhorefhs[kps] = rhorefh[k+1];
    dzis[kms]     = dzi[k-1];
    dzis[ks]      = dzi[k];
    dzis[kps]     = dzi[k+1];
    dzhis[kms]    = dzhi[k-1];
    dzhis[ks]     = dzhi[k];
    dzhis[kps]    = dzhi[k+1];
  }
  __syncthreads();

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    // U
    eviscnu = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+jj] + evisc[ijk+jj]);
    eviscsu = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-jj] + evisc[ijk-ii   ] + evisc[ijk   ]);
    evisctu = 0.25*(evisc[ijk-ii   ] + evisc[ijk   ] + evisc[ijk-ii+kk] + evisc[ijk+kk]);
    eviscbu = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-kk] + evisc[ijk-ii   ] + evisc[ijk   ]);

    // V
    eviscev = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+ii-jj] + evisc[ijk+ii]);
    eviscwv = 0.25*(evisc[ijk-ii-jj] + evisc[ijk-ii] + evisc[ijk   -jj] + evisc[ijk   ]);
    evisctv = 0.25*(evisc[ijk   -jj] + evisc[ijk   ] + evisc[ijk+kk-jj] + evisc[ijk+kk]);
    eviscbv = 0.25*(evisc[ijk-kk-jj] + evisc[ijk-kk] + evisc[ijk   -jj] + evisc[ijk   ]);

    // W
    eviscew = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+ii-kk] + evisc[ijk+ii]);
    eviscww = 0.25*(evisc[ijk-ii-kk] + evisc[ijk-ii] + evisc[ijk   -kk] + evisc[ijk   ]);
    eviscnw = 0.25*(evisc[ijk   -kk] + evisc[ijk   ] + evisc[ijk+jj-kk] + evisc[ijk+jj]);
    eviscsw = 0.25*(evisc[ijk-jj-kk] + evisc[ijk-jj] + evisc[ijk   -kk] + evisc[ijk   ]);


    if(k == kstart)
    {
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (  rhorefhs[kps] * evisctu*((u[ijk+kk]-u[ijk   ])* dzhis[kps] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
               + rhorefhs[ks ] * fluxbotu[ij] ) / rhorefs[ks] * dzis[ks];

      vt[ijk] +=
            // dv/dx + du/dy
            + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (  rhorefhs[kps] * evisctv*((v[ijk+kk]-v[ijk   ])*dzhis[kps] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
               + rhorefhs[ks ] * fluxbotv[ij] ) / rhorefs[ks] * dzis[ks];
    }
    else if(k == kend-1)
    {
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (- rhorefhs[kps] * fluxtopu[ij]
               - rhorefhs[ks ] * eviscbu*((u[ijk   ]-u[ijk-kk])* dzhis[ks] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhorefs[ks] * dzis[ks];

      vt[ijk] +=
            // dv/dx + du/dy
            + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (- rhorefhs[kps] * fluxtopv[ij]
               - rhorefhs[ks] * eviscbv*((v[ijk   ]-v[ijk-kk])* dzhis[ks] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhorefs[ks] * dzis[ks];

      wt[ijk] +=
            // dw/dx + du/dz
            + (  eviscew*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
               - eviscww*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
            // dw/dy + dv/dz
            + (  eviscnw*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
               - eviscsw*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
            // dw/dz + dw/dz
            + (  rhorefs[ks ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzis[ks ]
               - rhorefs[kms] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzis[kms] ) / rhorefhs[ks] * 2.* dzhis[ks];
    }
    else
    {
      ut[ijk] +=
            // du/dx + du/dx
            + (  evisc[ijk   ]*(u[ijk+ii]-u[ijk   ])*dxi
               - evisc[ijk-ii]*(u[ijk   ]-u[ijk-ii])*dxi ) * 2.* dxi
            // du/dy + dv/dx
            + (  eviscnu*((u[ijk+jj]-u[ijk   ])*dyi  + (v[ijk+jj]-v[ijk-ii+jj])*dxi)
               - eviscsu*((u[ijk   ]-u[ijk-jj])*dyi  + (v[ijk   ]-v[ijk-ii   ])*dxi) ) * dyi
            // du/dz + dw/dx
            + (  rhorefhs[kps] * evisctu*((u[ijk+kk]-u[ijk   ])* dzhis[kps] + (w[ijk+kk]-w[ijk-ii+kk])*dxi)
               - rhorefhs[ks ] * eviscbu*((u[ijk   ]-u[ijk-kk])* dzhis[ks ] + (w[ijk   ]-w[ijk-ii   ])*dxi) ) / rhorefs[ks] * dzis[ks];

      vt[ijk] +=
            // dv/dx + du/dy
            + (  eviscev*((v[ijk+ii]-v[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-jj])*dyi)
               - eviscwv*((v[ijk   ]-v[ijk-ii])*dxi + (u[ijk   ]-u[ijk   -jj])*dyi) ) * dxi
            // dv/dy + dv/dy
            + (  evisc[ijk   ]*(v[ijk+jj]-v[ijk   ])*dyi
               - evisc[ijk-jj]*(v[ijk   ]-v[ijk-jj])*dyi ) * 2.* dyi
            // dv/dz + dw/dy
            + (  rhorefhs[kps] * evisctv*((v[ijk+kk]-v[ijk   ])*dzhis[kps] + (w[ijk+kk]-w[ijk-jj+kk])*dyi)
               - rhorefhs[ks ] * eviscbv*((v[ijk   ]-v[ijk-kk])*dzhis[ks ] + (w[ijk   ]-w[ijk-jj   ])*dyi) ) / rhorefs[ks] * dzis[ks];

      wt[ijk] +=
            // dw/dx + du/dz
            + (  eviscew*((w[ijk+ii]-w[ijk   ])*dxi + (u[ijk+ii]-u[ijk+ii-kk])*dzhi[k])
               - eviscww*((w[ijk   ]-w[ijk-ii])*dxi + (u[ijk   ]-u[ijk+  -kk])*dzhi[k]) ) * dxi
            // dw/dy + dv/dz
            + (  eviscnw*((w[ijk+jj]-w[ijk   ])*dyi + (v[ijk+jj]-v[ijk+jj-kk])*dzhi[k])
               - eviscsw*((w[ijk   ]-w[ijk-jj])*dyi + (v[ijk   ]-v[ijk+  -kk])*dzhi[k]) ) * dyi
            // dw/dz + dw/dz
            + (  rhorefs[ks ] * evisc[ijk   ]*(w[ijk+kk]-w[ijk   ])*dzis[ks ]
               - rhorefs[kms] * evisc[ijk-kk]*(w[ijk   ]-w[ijk-kk])*dzis[kms] ) / rhorefhs[ks] * 2.* dzhis[ks];
    }
  }
}


__global__ void diff_les2s_diffc(double * __restrict__ at, double * __restrict__ a, double * __restrict__ evisc,
                                 double * __restrict__ fluxbot, double * __restrict__ fluxtop, 
                                 double * __restrict__ dzi, double * __restrict__ dzhi, double dxidxi, double dyidyi,
                                 double * __restrict__ rhoref, double * __restrict__ rhorefh, double tPri, 
                                 int istart, int jstart, int kstart, int iend, int jend, int kend, 
                                 int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;
  double evisce,eviscw,eviscn,eviscs,evisct,eviscb;

  if(i < iend && j < jend && k < kend)
  {
    const int ii  = 1;
    const int ij  = i + j*jj;
    const int ijk = i + j*jj + k*kk;

    if(k == kstart)
    {
      evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
      eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
      eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
      eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
      evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;
      eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;

      at[ijk] +=
            + (  evisce*(a[ijk+ii]-a[ijk   ]) 
               - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
            + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
               - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
            + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
               + rhorefh[k  ] * fluxbot[ij] ) / rhoref[k] * dzi[k];
    }
    else if(k == kend-1)
    {
      evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
      eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
      eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
      eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
      evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;
      eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;

      at[ijk] +=
            + (  evisce*(a[ijk+ii]-a[ijk   ]) 
               - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
            + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
               - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
            + (- rhorefh[k  ] * fluxtop[ij]
               - rhorefh[k-1] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k-1] ) / rhoref[k-1] * dzi[k-1];
    }
    else
    {
      evisce = 0.5*(evisc[ijk   ]+evisc[ijk+ii])*tPri;
      eviscw = 0.5*(evisc[ijk-ii]+evisc[ijk   ])*tPri;
      eviscn = 0.5*(evisc[ijk   ]+evisc[ijk+jj])*tPri;
      eviscs = 0.5*(evisc[ijk-jj]+evisc[ijk   ])*tPri;
      evisct = 0.5*(evisc[ijk   ]+evisc[ijk+kk])*tPri;
      eviscb = 0.5*(evisc[ijk-kk]+evisc[ijk   ])*tPri;

      at[ijk] +=
            + (  evisce*(a[ijk+ii]-a[ijk   ]) 
               - eviscw*(a[ijk   ]-a[ijk-ii]) ) * dxidxi 
            + (  eviscn*(a[ijk+jj]-a[ijk   ]) 
               - eviscs*(a[ijk   ]-a[ijk-jj]) ) * dyidyi
            + (  rhorefh[k+1] * evisct*(a[ijk+kk]-a[ijk   ])*dzhi[k+1]
               - rhorefh[k  ] * eviscb*(a[ijk   ]-a[ijk-kk])*dzhi[k]  ) / rhoref[k] * dzi[k];
    }
  }
}

__global__ void diff_les2s_calcdnmul(double * __restrict__ dnmul, double * __restrict__ evisc, 
                                     double * __restrict__ dzi, double tPrfac, double dxidxi, double dyidyi,
                                     int istart, int jstart, int kstart, int iend, int jend, int kend, int jj, int kk)

{
  const int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
  const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart;
  const int k = blockIdx.z + kstart;

  if(i < iend && j < jend && k < kend)
  {
    const int ijk = i + j*jj + k*kk;
    dnmul[ijk] = fabs(tPrfac*evisc[ijk]*(dxidxi + dyidyi + dzi[k]*dzi[k]));
  }
}

/* Calculate the mixing length (mlen) offline, and put on GPU */
#ifdef USECUDA
int cdiff_les2s::prepareDevice()
{
  cboundary_surface *boundaryptr = static_cast<cboundary_surface *>(model->boundary);

  const double n=2.;
  double mlen0;
  double *mlen = new double[grid->kcells];
  for(int k=0; k<grid->kcells; ++k) 
  {
    mlen0   = cs * pow(grid->dx*grid->dy*grid->dz[k], 1./3.);
    mlen[k] = pow(pow(1./(1./pow(mlen0, n) + 1./(pow(constants::kappa*(grid->z[k]+boundaryptr->z0m), n))), 1./n), 2);
  }

  const int nmemsize = grid->kcells*sizeof(double);
  hipMalloc(&mlen_g, nmemsize);
  hipMemcpy(mlen_g, mlen, nmemsize, hipMemcpyHostToDevice);

  delete[] mlen;

  return 0;
}
#endif

#ifdef USECUDA
int cdiff_les2s::execvisc()
{
  // do a cast because the base boundary class does not have the MOST related variables
  cboundary_surface *boundaryptr = static_cast<cboundary_surface *>(model->boundary);

  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  // Calculate total strain rate
  diff_les2s_strain2<<<gridGPU, blockGPU>>>(&fields->s["evisc"]->data_g[offs], 
                                            &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                            &fields->u->datafluxbot_g[offs],  &fields->v->datafluxbot_g[offs],
                                            &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
                                            grid->z_g, grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                            grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                            grid->icellsp, grid->ijcellsp);  

  // start with retrieving the stability information
  if(model->thermo->getsw() == "0")
  {
    master->printMessage("diff_les2s without thermo not yet supported on GPU\n");
    //evisc_neutral(fields->s["evisc"]->data,
    //              fields->u->data, fields->v->data, fields->w->data,
    //              fields->u->datafluxbot, fields->v->datafluxbot,
    //              grid->z, grid->dz, boundaryptr->z0m);
  }
  // assume buoyancy calculation is needed
  else
  {
    // store the buoyancyflux in datafluxbot of tmp1
    model->thermo->getbuoyancyfluxbot(fields->sd["tmp1"]);
    // store the Brunt-vaisala frequency in data of tmp1 
    model->thermo->getthermofield(fields->sd["tmp1"], fields->sd["tmp2"], "N2");

    // Calculate eddy viscosity
    double tPri = 1./tPr;
    diff_les2s_evisc<<<gridGPU, blockGPU>>>(&fields->s["evisc"]->data_g[offs], &fields->s["tmp1"]->data_g[offs], 
                                            &fields->sd["tmp1"]->datafluxbot_g[offs], &boundaryptr->ustar_g[offs], &boundaryptr->obuk_g[offs],
                                            mlen_g, tPri, boundaryptr->z0m, grid->z[grid->kstart],
                                            grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                            grid->icellsp, grid->ijcellsp);  
    grid->boundary_cyclic_g(&fields->sd["evisc"]->data_g[offs]);

  }

  return 0;
}
#endif

#ifdef USECUDA
int cdiff_les2s::exec()
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;
  const double dxidxi = 1./(grid->dx * grid->dx);
  const double dyidyi = 1./(grid->dy * grid->dy);
  const double tPri = 1./tPr;

  //diff_les2s_diffu<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->s["evisc"]->data_g[offs], 
  //                                        &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
  //                                        &fields->u->datafluxbot_g[offs], &fields->u->datafluxtop_g[offs],
  //                                        grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
  //                                        fields->rhoref_g, fields->rhorefh_g,
  //                                        grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
  //                                        grid->icellsp, grid->ijcellsp);  

  //diff_les2s_diffv<<<gridGPU, blockGPU>>>(&fields->vt->data_g[offs], &fields->s["evisc"]->data_g[offs], 
  //                                        &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
  //                                        &fields->v->datafluxbot_g[offs], &fields->v->datafluxtop_g[offs],
  //                                        grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
  //                                        fields->rhoref_g, fields->rhorefh_g,
  //                                        grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
  //                                        grid->icellsp, grid->ijcellsp);  

  //diff_les2s_diffw<<<gridGPU, blockGPU>>>(&fields->wt->data_g[offs], &fields->s["evisc"]->data_g[offs], 
  //                                        &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
  //                                        grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
  //                                        fields->rhoref_g, fields->rhorefh_g,
  //                                        grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
  //                                        grid->icellsp, grid->ijcellsp);  

  diff_les2s_diffuvw<<<gridGPU, blockGPU>>>(&fields->ut->data_g[offs], &fields->vt->data_g[offs], &fields->wt->data_g[offs],
                                            &fields->s["evisc"]->data_g[offs], 
                                            &fields->u->data_g[offs],  &fields->v->data_g[offs],  &fields->w->data_g[offs],
                                            &fields->u->datafluxbot_g[offs], &fields->u->datafluxtop_g[offs],
                                            &fields->v->datafluxbot_g[offs], &fields->v->datafluxtop_g[offs],
                                            grid->dzi_g, grid->dzhi_g, grid->dxi, grid->dyi,
                                            fields->rhoref_g, fields->rhorefh_g,
                                            grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                            grid->icellsp, grid->ijcellsp);  

  for(fieldmap::const_iterator it = fields->st.begin(); it!=fields->st.end(); ++it)
    diff_les2s_diffc<<<gridGPU, blockGPU>>>(&it->second->data_g[offs], &fields->s[it->first]->data_g[offs], &fields->s["evisc"]->data_g[offs], 
                                            &fields->s[it->first]->datafluxbot_g[offs], &fields->s[it->first]->datafluxtop_g[offs],
                                            grid->dzi_g, grid->dzhi_g, dxidxi, dyidyi,
                                            fields->rhoref_g, fields->rhorefh_g, tPri,
                                            grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                            grid->icellsp, grid->ijcellsp);  

  return 0;
}
#endif

#ifdef USECUDA
unsigned long cdiff_les2s::gettimelim(unsigned long idt, double dt)
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  double dnmul;
  unsigned long idtlim;
  const int offs = grid->memoffset;
  const double dxidxi = 1./(grid->dx * grid->dx);
  const double dyidyi = 1./(grid->dy * grid->dy);
  const double tPrfac = std::min(1., tPr);

  // Calculate dnmul in tmp1 field
  diff_les2s_calcdnmul<<<gridGPU, blockGPU>>>(&fields->s["tmp1"]->data_g[offs], &fields->s["evisc"]->data_g[offs],
                                              grid->dzi_g, tPrfac, dxidxi, dyidyi,  
                                              grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                              grid->icellsp, grid->ijcellsp);  

  // Get maximum from tmp1 field
  dnmul = grid->getmax_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp2"]->data_g); 
  dnmul = std::max(constants::dsmall, dnmul);
  idtlim = idt * dnmax/(dnmul*dt);

  return idtlim;
}
#endif

#ifdef USECUDA
double cdiff_les2s::getdn(double dt)
{
  const int blocki = 128;
  const int blockj = 2;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kmax);
  dim3 blockGPU(blocki, blockj, 1);

  double dnmul;
  const int offs = grid->memoffset;
  const double dxidxi = 1./(grid->dx * grid->dx);
  const double dyidyi = 1./(grid->dy * grid->dy);
  const double tPrfac = std::min(1., tPr);

  // Calculate dnmul in tmp1 field
  diff_les2s_calcdnmul<<<gridGPU, blockGPU>>>(&fields->s["tmp1"]->data_g[offs], &fields->s["evisc"]->data_g[offs],
                                              grid->dzi_g, tPrfac, dxidxi, dyidyi,  
                                              grid->istart, grid->jstart, grid->kstart, grid->iend, grid->jend, grid->kend,
                                              grid->icellsp, grid->ijcellsp);  

  // Get maximum from tmp1 field
  dnmul = grid->getmax_g(&fields->a["tmp1"]->data_g[offs], fields->a["tmp2"]->data_g); 

  return dnmul*dt;
}
#endif

