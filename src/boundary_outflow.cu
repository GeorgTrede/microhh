#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2018 Chiel van Heerwaarden
 * Copyright (c) 2011-2018 Thijs Heus
 * Copyright (c) 2014-2018 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "master.h"
#include "grid.h"
#include "boundary_outflow.h"
#include "tools.h"

namespace
{
    template<typename TF, Edge_location location> __global__
    void set_neumann_g(
            TF* const __restrict__ fld,
            const int istart, const int iend, const int igc,
            const int jstart, const int jend, const int jgc,
            const int icells, const int jcells, const int kcells,
            const int ijcells)
    {
        const int i  = blockIdx.x*blockDim.x + threadIdx.x;
        const int j  = blockIdx.y*blockDim.y + threadIdx.y;
        const int k  = blockIdx.z*blockDim.z + threadIdx.z;

        if (location == Edge_location::West || location == Edge_location::East)
        {
            if (i<igc && j<jcells && k<kcells)
            {
                if (location == Edge_location::West)
                {
                    const int ijk    = (istart+i  ) + j*icells + k*ijcells;
                    const int ijk_gc = (istart-1-i) + j*icells + k*ijcells;
                    fld[ijk_gc] = fld[ijk];
                }
                else if (location == Edge_location::East)
                {
                    const int ijk    = (iend-1-i) + j*icells + k*ijcells;
                    const int ijk_gc = (iend+i  ) + j*icells + k*ijcells;
                    fld[ijk_gc] = fld[ijk];
                }
            }
        }
        else if (location == Edge_location::North || location == Edge_location::South)
        {
            if (i<icells && j<jgc && k<kcells)
            {
                if (location == Edge_location::South)
                {
                    const int ijk    = i + (jstart+j  )*icells + k*ijcells;
                    const int ijk_gc = i + (jstart-1-j)*icells + k*ijcells;
                    fld[ijk_gc] = fld[ijk];
                }
                else if (location == Edge_location::North)
                {
                    const int ijk    = i + (jend-1-j)*icells + k*ijcells;
                    const int ijk_gc = i + (jend+j  )*icells + k*ijcells;
                    fld[ijk_gc] = fld[ijk];
                }
            }
        }
    }

    template<typename TF> __global__
    void compute_outflow_4th(
            TF* const restrict a,
            const int iend, const int icells,
            const int jcells, const int kcells,
            const int ijcells)
    {
        const int j  = blockIdx.x*blockDim.x + threadIdx.x;
        const int k  = blockIdx.y*blockDim.y + threadIdx.y;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;

        if (j < jcells && k < kcells)
        {
            const int ijk = (iend-1) + j*icells + k*ijcells;
            a[ijk+ii1] = TF(2.)*a[ijk] - TF( 3./2.)*a[ijk-ii1] + TF(1./2.)*a[ijk-ii2];
            a[ijk+ii2] = TF(3.)*a[ijk] - TF( 7./2.)*a[ijk-ii1] + TF(3./2.)*a[ijk-ii2];
            a[ijk+ii3] = TF(5.)*a[ijk] - TF(15./2.)*a[ijk-ii1] + TF(7./2.)*a[ijk-ii2];
        }
    }

    template<typename TF> __global__
    void compute_inflow_4th(
            TF* const restrict a,
            const TF value,
            const int istart, const int icells,
            const int jcells, const int kcells,
            const int ijcells)
    {
        const int j  = blockIdx.x*blockDim.x + threadIdx.x;
        const int k  = blockIdx.y*blockDim.y + threadIdx.y;

        const int ii1 = 1;
        const int ii2 = 2;
        const int ii3 = 3;

        if (j < jcells && k < kcells)
        {
            const int ijk = istart + j*icells + k*ijcells;
            a[ijk-ii1] = value + TF( 9./8.)*a[ijk] - TF( 14./8.)*a[ijk+ii1] + TF( 5./8.)*a[ijk+ii2];
            a[ijk-ii2] = value + TF(33./8.)*a[ijk] - TF( 54./8.)*a[ijk+ii1] + TF(21./8.)*a[ijk+ii2];
            a[ijk-ii3] = value + TF(65./8.)*a[ijk] - TF(110./8.)*a[ijk+ii1] + TF(45./8.)*a[ijk+ii2];
        }
    }
}

#ifdef USECUDA
template<typename TF>
void Boundary_outflow<TF>::exec(TF* const restrict data)
{
    auto& gd = grid.get_grid_data();
    auto& md = master.get_MPI_data();

    if (grid.get_spatial_order() == Grid_order::Fourth)
    {
        const int blocki = gd.jthread_block;
        const int blockj = 64;

        const int gridi  = gd.jcells/blocki + (gd.jcells%blocki > 0);
        const int gridj  = gd.kcells/blockj + (gd.kcells%blockj > 0);

        dim3 grid2dGPU (gridi, gridj);
        dim3 block2dGPU(blocki, blockj);

        // Dirichlet BCs on west boundary, Neumann on east boundary,
        // cyclic BCs on north-south boundaries
        if (md.mpicoordx == 0)
            compute_inflow_4th<<<grid2dGPU, block2dGPU>>>(
                    data, TF(0),
                    gd.istart,
                    gd.icells, gd.jcells, gd.kcells,
                    gd.ijcells);

        if (md.mpicoordx == md.npx-1)
            compute_outflow_4th<<<grid2dGPU, block2dGPU>>>(
                    data,
                    gd.iend,
                    gd.icells, gd.jcells, gd.kcells,
                    gd.ijcells);

        cuda_check_error();
    }
    else if (grid.get_spatial_order() == Grid_order::Second)
    {
        const int blockk = 4;
        const int gridk  = gd.kcells/blockk + (gd.kcells%blockk > 0);

        // Grid x-direction
        const int blocki_x = gd.igc;
        const int blockj_x = 64;

        const int gridi_x  = 1;
        const int gridj_x  = gd.jcells/blockj_x + (gd.jcells%blockj_x > 0);

        dim3 gridGPU_x (gridi_x, gridj_x, gridk);
        dim3 blockGPU_x(blocki_x, blockj_x, blockk);

        // Grid y-direction
        const int blocki_y = 64;
        const int blockj_y = gd.jgc;

        const int gridi_y  = gd.icells/blocki_y + (gd.icells%blocki_y > 0);
        const int gridj_y  = 1;

        dim3 gridGPU_y (gridi_y, gridj_y, gridk);
        dim3 blockGPU_y(blocki_y, blockj_y, blockk);

        // Neumann BCs on all boundaries
        if (md.mpicoordx == 0)
            set_neumann_g<TF, Edge_location::West><<<gridGPU_x, blockGPU_x>>>(
                    data,
                    gd.istart, gd.iend, gd.igc,
                    gd.jstart, gd.jend, gd.kgc,
                    gd.icells, gd.jcells, gd.kcells,
                    gd.ijcells);

        if (md.mpicoordx == md.npx-1)
            set_neumann_g<TF, Edge_location::East><<<gridGPU_x, blockGPU_x>>>(
                    data,
                    gd.istart, gd.iend, gd.igc,
                    gd.jstart, gd.jend, gd.kgc,
                    gd.icells, gd.jcells, gd.kcells,
                    gd.ijcells);

        if (md.mpicoordy == 0)
            set_neumann_g<TF, Edge_location::South><<<gridGPU_y, blockGPU_y>>>(
                    data,
                    gd.istart, gd.iend, gd.igc,
                    gd.jstart, gd.jend, gd.kgc,
                    gd.icells, gd.jcells, gd.kcells,
                    gd.ijcells);

        if (md.mpicoordy == md.npy-1)
            set_neumann_g<TF, Edge_location::North><<<gridGPU_y, blockGPU_y>>>(
                    data,
                    gd.istart, gd.iend, gd.igc,
                    gd.jstart, gd.jend, gd.kgc,
                    gd.icells, gd.jcells, gd.kcells,
                    gd.ijcells);

        cuda_check_error();
    }
}
#endif

template class Boundary_outflow<double>;
template class Boundary_outflow<float>;
