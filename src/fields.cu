#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "fields.h"
#include "grid.h"
#include "master.h"
#include "boundary.h" // TMP BVS
#include "constants.h"

// TODO use interp2 functions instead of manual interpolation
__global__ void fields_calcmom_2nd(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                                   double * __restrict__ mom, double * __restrict__ dz,
                                   int istart, int jstart, int kstart,
                                   int iend,   int jend,   int kend,
                                   int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 
  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    mom[ijk] = (0.5*(u[ijk]+u[ijk+ii]) + 0.5*(v[ijk]+v[ijk+jj]) + 0.5*(w[ijk]+w[ijk+kk]))*dz[k];
  }
}

__global__ void fields_calctke_2nd(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                                   double * __restrict__ tke, double * __restrict__ dz,
                                   int istart, int jstart, int kstart,
                                   int iend,   int jend,   int kend,
                                   int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 
  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    tke[ijk] = ( 0.5*(pow(u[ijk],2)+pow(u[ijk+ii],2)) 
               + 0.5*(pow(v[ijk],2)+pow(v[ijk+jj],2)) 
               + 0.5*(pow(w[ijk],2)+pow(w[ijk+kk],2)))*dz[k];
  }
}

__global__ void fields_calcmass_2nd(double * __restrict__ s, double * __restrict__ mass, double * __restrict__ dz,
                                    int istart, int jstart, int kstart,
                                    int iend,   int jend,   int kend,
                                    int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    mass[ijk] = s[ijk]*dz[k];
  }
}

#ifdef USECUDA
int cfields::exec()
{
  // calculate the means for the prognostic scalars
  if(calcprofs)
  {
    for(fieldmap::iterator it=sp.begin(); it!=sp.end(); ++it)
      grid->calcmean_g(it->second->datamean_g, &it->second->data_g[grid->memoffset], s["tmp1"]->data_g);
  }

  return 0;
}
#endif

#ifdef USECUDA
double cfields::checkmom()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  fields_calcmom_2nd<<<gridGPU, blockGPU>>>(&u->data_g[offs], &v->data_g[offs], &w->data_g[offs], 
                                            &a["tmp1"]->data_g[offs], grid->dz_g,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend,
                                            grid->icellsp, grid->ijcellsp);

  double mom = grid->getsum_g(&a["tmp1"]->data_g[offs], a["tmp2"]->data_g); 
  grid->getsum(&mom);
  mom /= (grid->itot*grid->jtot*grid->zsize);

  return mom;
}
#endif

#ifdef USECUDA
double cfields::checktke()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  fields_calctke_2nd<<<gridGPU, blockGPU>>>(&u->data_g[offs], &v->data_g[offs], &w->data_g[offs], 
                                            &a["tmp1"]->data_g[offs], grid->dz_g,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend,
                                            grid->icellsp, grid->ijcellsp);

  double tke = grid->getsum_g(&a["tmp1"]->data_g[offs], a["tmp2"]->data_g); 

  grid->getsum(&tke);
  tke /= (grid->itot*grid->jtot*grid->zsize);
  tke *= 0.5;

  return tke;
}
#endif

#ifdef USECUDA
double cfields::checkmass()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;
  double mass;

  // CvH for now, do the mass check on the first scalar... Do we want to change this?
  fieldmap::iterator itProg=sp.begin();
  if(sp.begin() != sp.end())
  {
    fields_calcmass_2nd<<<gridGPU, blockGPU>>>(&itProg->second->data_g[offs], &a["tmp1"]->data_g[offs], grid->dz_g,
                                               grid->istart,  grid->jstart, grid->kstart,
                                               grid->iend,    grid->jend,   grid->kend,
                                               grid->icellsp, grid->ijcellsp);

    mass = grid->getsum_g(&a["tmp1"]->data_g[offs], a["tmp2"]->data_g); 
    grid->getsum(&mass);
    mass /= (grid->itot*grid->jtot*grid->zsize);
  }
  else
    mass = 0; 

  return mass;
}
#endif

int cfields::prepareDevice()
{
  const int nmemsize   = grid->ncellsp*sizeof(double);
  const int nmemsize1d = grid->kcells*sizeof(double);
  const int nmemsize2d = (grid->ijcellsp+grid->memoffset)*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMalloc(&it->second->data_g,        nmemsize  );
    hipMalloc(&it->second->databot_g,     nmemsize2d);
    hipMalloc(&it->second->datatop_g,     nmemsize2d);
    hipMalloc(&it->second->datagradbot_g, nmemsize2d);
    hipMalloc(&it->second->datagradtop_g, nmemsize2d);
    hipMalloc(&it->second->datafluxbot_g, nmemsize2d);
    hipMalloc(&it->second->datafluxtop_g, nmemsize2d);
    hipMalloc(&it->second->datamean_g,    nmemsize1d);
  }

  // BvS: slightly wasteful, but make sure we have all fields...
  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipMalloc(&it->second->data_g,        nmemsize  );
    hipMalloc(&it->second->databot_g,     nmemsize2d);
    hipMalloc(&it->second->datatop_g,     nmemsize2d);
    hipMalloc(&it->second->datagradbot_g, nmemsize2d);
    hipMalloc(&it->second->datagradtop_g, nmemsize2d);
    hipMalloc(&it->second->datafluxbot_g, nmemsize2d);
    hipMalloc(&it->second->datafluxtop_g, nmemsize2d);
    hipMalloc(&it->second->datamean_g,    nmemsize1d);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipMalloc(&it->second->data_g, nmemsize);
  }

  hipMalloc(&rhoref_g,  nmemsize1d);
  hipMalloc(&rhorefh_g, nmemsize1d);

  // copy all the data to the GPU
  forwardDevice();

  return 0;
}

int cfields::forwardDevice()
{
  const int jcells     = grid->jcells;
  const int jkcells    = grid->jcells * grid->kcells;
  const int nmemsize1d = grid->kcells*sizeof(double);
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep,  it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->databot_g[grid->memoffset],     imemsizep,  it->second->databot,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datatop_g[grid->memoffset],     imemsizep,  it->second->datatop,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradbot_g[grid->memoffset], imemsizep,  it->second->datagradbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradtop_g[grid->memoffset], imemsizep,  it->second->datagradtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxbot_g[grid->memoffset], imemsizep,  it->second->datafluxbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxtop_g[grid->memoffset], imemsizep,  it->second->datafluxtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy(it->second->datamean_g, it->second->datamean, nmemsize1d, hipMemcpyHostToDevice);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep,  it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->databot_g[grid->memoffset],     imemsizep,  it->second->databot,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datatop_g[grid->memoffset],     imemsizep,  it->second->datatop,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradbot_g[grid->memoffset], imemsizep,  it->second->datagradbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradtop_g[grid->memoffset], imemsizep,  it->second->datagradtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxbot_g[grid->memoffset], imemsizep,  it->second->datafluxbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxtop_g[grid->memoffset], imemsizep,  it->second->datafluxtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy(it->second->datamean_g, it->second->datamean, nmemsize1d, hipMemcpyHostToDevice);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep, it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);

  //hipMemcpy2D(&a["p"]->data_g[grid->memoffset],              imemsizep, a["p"]->data,            imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  //hipMemcpy2D(&a["tmp1"]->data_g[grid->memoffset],           imemsizep, a["tmp1"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  //hipMemcpy2D(&a["tmp2"]->data_g[grid->memoffset],           imemsizep, a["tmp2"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  //hipMemcpy2D(&a["tmp3"]->data_g[grid->memoffset],           imemsizep, a["tmp3"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  

  hipMemcpy(rhoref_g,  rhoref,  nmemsize1d, hipMemcpyHostToDevice);
  hipMemcpy(rhorefh_g, rhorefh, nmemsize1d, hipMemcpyHostToDevice);

  //master->printMessage("Synchronized GPU with CPU (forward)\n");

  return 0;
}

int cfields::backwardDevice()
{
  const int jcells     = grid->jcells;
  const int jkcells    = grid->jcells * grid->kcells;
  const int nmemsize1d = grid->kcells*sizeof(double);
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->databot,     imemsize, &it->second->databot_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datatop,     imemsize, &it->second->datatop_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradbot, imemsize, &it->second->datagradbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradtop, imemsize, &it->second->datagradtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxbot, imemsize, &it->second->datafluxbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxtop, imemsize, &it->second->datafluxtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datamean, it->second->datamean_g, nmemsize1d, hipMemcpyDeviceToHost);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->databot,     imemsize, &it->second->databot_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datatop,     imemsize, &it->second->datatop_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradbot, imemsize, &it->second->datagradbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradtop, imemsize, &it->second->datagradtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxbot, imemsize, &it->second->datafluxbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxtop, imemsize, &it->second->datafluxtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datamean, it->second->datamean_g, nmemsize1d, hipMemcpyDeviceToHost);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);

  //hipMemcpy2D(a["p"]->data,              imemsize, &a["p"]->data_g[grid->memoffset],            imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  //hipMemcpy2D(a["tmp1"]->data,           imemsize, &a["tmp1"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  //hipMemcpy2D(a["tmp2"]->data,           imemsize, &a["tmp2"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  //hipMemcpy2D(a["tmp3"]->data,           imemsize, &a["tmp3"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  

  hipMemcpy(rhoref,  rhoref_g,  nmemsize1d, hipMemcpyDeviceToHost);
  hipMemcpy(rhorefh, rhorefh_g, nmemsize1d, hipMemcpyDeviceToHost);

  //master->printMessage("Synchronized CPU with GPU (backward)\n");

  return 0;
}

int cfields::clearDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipFree(it->second->data_g);
    hipFree(it->second->databot_g);
    hipFree(it->second->datatop_g);
    hipFree(it->second->datagradbot_g);
    hipFree(it->second->datagradtop_g);
    hipFree(it->second->datafluxbot_g);
    hipFree(it->second->datafluxtop_g);
    hipFree(it->second->datamean_g);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipFree(it->second->data_g);
    hipFree(it->second->databot_g);
    hipFree(it->second->datatop_g);
    hipFree(it->second->datagradbot_g);
    hipFree(it->second->datagradtop_g);
    hipFree(it->second->datafluxbot_g);
    hipFree(it->second->datafluxtop_g);
    hipFree(it->second->datamean_g);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipFree(it->second->data_g);
  }

  //hipFree(&a["p"]->data_g);
  //hipFree(&a["tmp1"]->data_g);
  //hipFree(&a["tmp2"]->data_g);
  //hipFree(&a["tmp3"]->data_g);

  hipFree(rhoref_g);
  hipFree(rhorefh_g);

  return 0;
}

