#include "fields.h"
#include "grid.h"
#include "master.h"

int cfields::prepareGPU()
{
  const int nmemsize = grid->ncells*sizeof(double);
  const int nmemsize2d = grid->ijcells*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMalloc(&it->second->data_g,        nmemsize);
    hipMalloc(&it->second->databot_g,     nmemsize2d);
    hipMalloc(&it->second->datatop_g,     nmemsize2d);
    hipMalloc(&it->second->datagradbot_g, nmemsize2d);
    hipMalloc(&it->second->datagradtop_g, nmemsize2d);
    hipMalloc(&it->second->datafluxbot_g, nmemsize2d);
    hipMalloc(&it->second->datafluxtop_g, nmemsize2d);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipMalloc(&it->second->data_g, nmemsize);
  }

  hipMalloc(&a["p"]->data_g, nmemsize);
  hipMalloc(&a["tmp1"]->data_g, nmemsize);
  hipMalloc(&a["tmp2"]->data_g, nmemsize);

  // copy all the data to the GPU
  forwardGPU();

  return 0;
}

int cfields::forwardGPU()
{
  const int nmemsize = grid->ncells*sizeof(double);
  const int nmemsize2d = grid->ijcells*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy(it->second->data_g,        it->second->data,        nmemsize,   hipMemcpyHostToDevice);
    hipMemcpy(it->second->databot_g,     it->second->databot,     nmemsize2d, hipMemcpyHostToDevice);
    hipMemcpy(it->second->datatop_g,     it->second->datatop,     nmemsize2d, hipMemcpyHostToDevice);
    hipMemcpy(it->second->datagradbot_g, it->second->datagradbot, nmemsize2d, hipMemcpyHostToDevice);
    hipMemcpy(it->second->datagradtop_g, it->second->datagradtop, nmemsize2d, hipMemcpyHostToDevice);
    hipMemcpy(it->second->datafluxbot_g, it->second->datafluxbot, nmemsize2d, hipMemcpyHostToDevice);
    hipMemcpy(it->second->datafluxtop_g, it->second->datafluxtop, nmemsize2d, hipMemcpyHostToDevice);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy(it->second->data_g, it->second->data, nmemsize, hipMemcpyHostToDevice);

  hipMemcpy(a["p"]->data_g, a["p"]->data, nmemsize, hipMemcpyHostToDevice);  
  hipMemcpy(a["tmp1"]->data_g, a["tmp1"]->data, nmemsize, hipMemcpyHostToDevice);  
  hipMemcpy(a["tmp2"]->data_g, a["tmp2"]->data, nmemsize, hipMemcpyHostToDevice);  

  printf("--> forwardGPU\n");

  return 0;
}

int cfields::backwardGPU()
{
  const int nmemsize = grid->ncells*sizeof(double);
  const int nmemsize2d = grid->ijcells*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy(it->second->data,        it->second->data_g,        nmemsize,   hipMemcpyDeviceToHost);
    hipMemcpy(it->second->databot,     it->second->databot_g,     nmemsize2d, hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datatop,     it->second->datatop_g,     nmemsize2d, hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datagradbot, it->second->datagradbot_g, nmemsize2d, hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datagradtop, it->second->datagradtop_g, nmemsize2d, hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datafluxbot, it->second->datafluxbot_g, nmemsize2d, hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datafluxtop, it->second->datafluxtop_g, nmemsize2d, hipMemcpyDeviceToHost);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy(it->second->data, it->second->data_g, nmemsize, hipMemcpyDeviceToHost);

  hipMemcpy(a["p"]->data, a["p"]->data_g, nmemsize, hipMemcpyDeviceToHost);  
  hipMemcpy(a["tmp1"]->data, a["tmp1"]->data_g, nmemsize, hipMemcpyDeviceToHost);  
  hipMemcpy(a["tmp2"]->data, a["tmp2"]->data_g, nmemsize, hipMemcpyDeviceToHost);  

  printf("--> backwardGPU\n");

  return 0;
}

int cfields::clearGPU()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipFree(&it->second->data_g);
    hipFree(&it->second->databot_g);
    hipFree(&it->second->datatop_g);
    hipFree(&it->second->datagradbot_g);
    hipFree(&it->second->datagradtop_g);
    hipFree(&it->second->datafluxbot_g);
    hipFree(&it->second->datafluxtop_g);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipFree(&it->second->data_g);
  }

  hipFree(&a["p"]->data_g);
  hipFree(&a["tmp1"]->data_g);
  hipFree(&a["tmp2"]->data_g);

  return 0;
}

