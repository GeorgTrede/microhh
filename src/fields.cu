#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "fields.h"
#include "grid.h"
#include "master.h"
#include "boundary.h" // TMP BVS
#include "constants.h"
#include "tools.h"

// TODO use interp2 functions instead of manual interpolation
__global__ void fields_calcmom_2nd(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                                   double * __restrict__ mom, double * __restrict__ dz,
                                   int istart, int jstart, int kstart,
                                   int iend,   int jend,   int kend,
                                   int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 
  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    mom[ijk] = (0.5*(u[ijk]+u[ijk+ii]) + 0.5*(v[ijk]+v[ijk+jj]) + 0.5*(w[ijk]+w[ijk+kk]))*dz[k];
  }
}

__global__ void fields_calctke_2nd(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                                   double * __restrict__ tke, double * __restrict__ dz,
                                   int istart, int jstart, int kstart,
                                   int iend,   int jend,   int kend,
                                   int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 
  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    tke[ijk] = ( 0.5*(pow(u[ijk],2)+pow(u[ijk+ii],2)) 
               + 0.5*(pow(v[ijk],2)+pow(v[ijk+jj],2)) 
               + 0.5*(pow(w[ijk],2)+pow(w[ijk+kk],2)))*dz[k];
  }
}

__global__ void fields_calcmass_2nd(double * __restrict__ s, double * __restrict__ mass, double * __restrict__ dz,
                                    int istart, int jstart, int kstart,
                                    int iend,   int jend,   int kend,
                                    int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    mass[ijk] = s[ijk]*dz[k];
  }
}

#ifdef USECUDA
int Fields::exec()
{
  // calculate the means for the prognostic scalars
  if(calcprofs)
  {
    for(fieldmap::iterator it=sp.begin(); it!=sp.end(); ++it)
      grid->calcMean_g(it->second->datamean_g, &it->second->data_g[grid->memoffset], s["tmp1"]->data_g);
  }

  return 0;
}
#endif

#ifdef USECUDA
double Fields::checkmom()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  fields_calcmom_2nd<<<gridGPU, blockGPU>>>(&u->data_g[offs], &v->data_g[offs], &w->data_g[offs], 
                                            &a["tmp1"]->data_g[offs], grid->dz_g,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend,
                                            grid->icellsp, grid->ijcellsp);
  cudaCheckError();

  double mom = grid->getSum_g(&a["tmp1"]->data_g[offs], a["tmp2"]->data_g); 
  grid->getSum(&mom);
  mom /= (grid->itot*grid->jtot*grid->zsize);

  return mom;
}
#endif

#ifdef USECUDA
double Fields::checktke()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  fields_calctke_2nd<<<gridGPU, blockGPU>>>(&u->data_g[offs], &v->data_g[offs], &w->data_g[offs], 
                                            &a["tmp1"]->data_g[offs], grid->dz_g,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend,
                                            grid->icellsp, grid->ijcellsp);
  cudaCheckError();

  double tke = grid->getSum_g(&a["tmp1"]->data_g[offs], a["tmp2"]->data_g); 

  grid->getSum(&tke);
  tke /= (grid->itot*grid->jtot*grid->zsize);
  tke *= 0.5;

  return tke;
}
#endif

#ifdef USECUDA
double Fields::checkmass()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;
  double mass;

  // CvH for now, do the mass check on the first scalar... Do we want to change this?
  fieldmap::iterator itProg=sp.begin();
  if(sp.begin() != sp.end())
  {
    fields_calcmass_2nd<<<gridGPU, blockGPU>>>(&itProg->second->data_g[offs], &a["tmp1"]->data_g[offs], grid->dz_g,
                                               grid->istart,  grid->jstart, grid->kstart,
                                               grid->iend,    grid->jend,   grid->kend,
                                               grid->icellsp, grid->ijcellsp);
    cudaCheckError();

    mass = grid->getSum_g(&a["tmp1"]->data_g[offs], a["tmp2"]->data_g); 
    grid->getSum(&mass);
    mass /= (grid->itot*grid->jtot*grid->zsize);
  }
  else
    mass = 0; 

  return mass;
}
#endif

int Fields::prepareDevice()
{
  const int nmemsize   = grid->ncellsp*sizeof(double);
  const int nmemsize1d = grid->kcells*sizeof(double);
  const int nmemsize2d = (grid->ijcellsp+grid->memoffset)*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    cudaSafeCall(hipMalloc(&it->second->data_g,        nmemsize  ));
    cudaSafeCall(hipMalloc(&it->second->databot_g,     nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datatop_g,     nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datagradbot_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datagradtop_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datafluxbot_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datafluxtop_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datamean_g,    nmemsize1d));
  }

  // BvS: slightly wasteful, but make sure we have all fields...
  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    cudaSafeCall(hipMalloc(&it->second->data_g,        nmemsize  ));
    cudaSafeCall(hipMalloc(&it->second->databot_g,     nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datatop_g,     nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datagradbot_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datagradtop_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datafluxbot_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datafluxtop_g, nmemsize2d));
    cudaSafeCall(hipMalloc(&it->second->datamean_g,    nmemsize1d));
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    cudaSafeCall(hipMalloc(&it->second->data_g, nmemsize));
  }

  cudaSafeCall(hipMalloc(&rhoref_g,  nmemsize1d));
  cudaSafeCall(hipMalloc(&rhorefh_g, nmemsize1d));

  // copy all the data to the GPU
  forwardDevice();

  return 0;
}

int Fields::forwardDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    forward3DFieldDevice(it->second->data_g,        it->second->data,        Offset);
    forward2DFieldDevice(it->second->databot_g,     it->second->databot,     Offset);
    forward2DFieldDevice(it->second->datatop_g,     it->second->datatop,     Offset);
    forward2DFieldDevice(it->second->datagradbot_g, it->second->datagradbot, Offset);
    forward2DFieldDevice(it->second->datagradtop_g, it->second->datagradtop, Offset);
    forward2DFieldDevice(it->second->datafluxbot_g, it->second->datafluxbot, Offset);
    forward2DFieldDevice(it->second->datafluxtop_g, it->second->datafluxtop, Offset);
    forward1DFieldDevice(it->second->datamean_g,    it->second->datamean, grid->kcells);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    forward3DFieldDevice(it->second->data_g,        it->second->data,        Offset);
    forward2DFieldDevice(it->second->databot_g,     it->second->databot,     Offset);
    forward2DFieldDevice(it->second->datatop_g,     it->second->datatop,     Offset);
    forward2DFieldDevice(it->second->datagradbot_g, it->second->datagradbot, Offset);
    forward2DFieldDevice(it->second->datagradtop_g, it->second->datagradtop, Offset);
    forward2DFieldDevice(it->second->datafluxbot_g, it->second->datafluxbot, Offset);
    forward2DFieldDevice(it->second->datafluxtop_g, it->second->datafluxtop, Offset);
    forward1DFieldDevice(it->second->datamean_g,    it->second->datamean, grid->kcells);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    forward3DFieldDevice(it->second->data_g, it->second->data, Offset);

  forward1DFieldDevice(rhoref_g,  rhoref , grid->kcells);
  forward1DFieldDevice(rhorefh_g, rhorefh, grid->kcells);

  //master->printMessage("Synchronized GPU with CPU (forward)\n");

  return 0;
}

int Fields::backwardDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    backward3DFieldDevice(it->second->data,        it->second->data_g,        Offset);
    backward2DFieldDevice(it->second->databot,     it->second->databot_g,     Offset);
    backward2DFieldDevice(it->second->datatop,     it->second->datatop_g,     Offset);
    backward2DFieldDevice(it->second->datagradbot, it->second->datagradbot_g, Offset);
    backward2DFieldDevice(it->second->datagradtop, it->second->datagradtop_g, Offset);
    backward2DFieldDevice(it->second->datafluxbot, it->second->datafluxbot_g, Offset);
    backward2DFieldDevice(it->second->datafluxtop, it->second->datafluxtop_g, Offset);
    backward1DFieldDevice(it->second->datamean,    it->second->datamean_g, grid->kcells);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    backward3DFieldDevice(it->second->data,        it->second->data_g,        Offset);
    backward2DFieldDevice(it->second->databot,     it->second->databot_g,     Offset);
    backward2DFieldDevice(it->second->datatop,     it->second->datatop_g,     Offset);
    backward2DFieldDevice(it->second->datagradbot, it->second->datagradbot_g, Offset);
    backward2DFieldDevice(it->second->datagradtop, it->second->datagradtop_g, Offset);
    backward2DFieldDevice(it->second->datafluxbot, it->second->datafluxbot_g, Offset);
    backward2DFieldDevice(it->second->datafluxtop, it->second->datafluxtop_g, Offset);
    backward1DFieldDevice(it->second->datamean,    it->second->datamean_g, grid->kcells);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    backward3DFieldDevice(it->second->data,        it->second->data_g,        Offset);

  backward1DFieldDevice(rhoref,  rhoref_g,  grid->kcells);
  backward1DFieldDevice(rhorefh, rhorefh_g, grid->kcells);

  //master->printMessage("Synchronized CPU with GPU (backward)\n");

  return 0;
}

void Fields::forward3DFieldDevice(double * field_g, double * field, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(&field_g[grid->memoffset], imemsizep,  field, imemsize, imemsize, grid->jcells*grid->kcells, hipMemcpyHostToDevice));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field_g, field, grid->ncells*sizeof(double), hipMemcpyHostToDevice));
}

void Fields::forward2DFieldDevice(double * field_g, double * field, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);
 
  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(&field_g[grid->memoffset], imemsizep,  field, imemsize, imemsize, grid->jcells,  hipMemcpyHostToDevice));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field_g, field, grid->ijcells*sizeof(double), hipMemcpyHostToDevice));
}

void Fields::forward1DFieldDevice(double * field_g, double * field, int ncells)
{
  cudaSafeCall(hipMemcpy(field_g, field, ncells*sizeof(double), hipMemcpyHostToDevice));
}

void Fields::backward3DFieldDevice(double * field, double * field_g, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(field, imemsize, &field_g[grid->memoffset], imemsizep, imemsize, grid->jcells*grid->kcells, hipMemcpyDeviceToHost));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field, field_g, grid->ncells*sizeof(double), hipMemcpyDeviceToHost));
}

void Fields::backward2DFieldDevice(double * field, double * field_g, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);
 
  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(field, imemsize, &field_g[grid->memoffset], imemsizep, imemsize, grid->jcells, hipMemcpyDeviceToHost));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field, field_g, grid->ijcells*sizeof(double), hipMemcpyDeviceToHost));
}

void Fields::backward1DFieldDevice(double * field, double * field_g, int ncells)
{
  cudaSafeCall(hipMemcpy(field, field_g, ncells*sizeof(double), hipMemcpyDeviceToHost));
}

int Fields::clearDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    cudaSafeCall(hipFree(it->second->data_g));
    cudaSafeCall(hipFree(it->second->databot_g));
    cudaSafeCall(hipFree(it->second->datatop_g));
    cudaSafeCall(hipFree(it->second->datagradbot_g));
    cudaSafeCall(hipFree(it->second->datagradtop_g));
    cudaSafeCall(hipFree(it->second->datafluxbot_g));
    cudaSafeCall(hipFree(it->second->datafluxtop_g));
    cudaSafeCall(hipFree(it->second->datamean_g));
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    cudaSafeCall(hipFree(it->second->data_g));
    cudaSafeCall(hipFree(it->second->databot_g));
    cudaSafeCall(hipFree(it->second->datatop_g));
    cudaSafeCall(hipFree(it->second->datagradbot_g));
    cudaSafeCall(hipFree(it->second->datagradtop_g));
    cudaSafeCall(hipFree(it->second->datafluxbot_g));
    cudaSafeCall(hipFree(it->second->datafluxtop_g));
    cudaSafeCall(hipFree(it->second->datamean_g));
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    cudaSafeCall(hipFree(it->second->data_g));
  }

  cudaSafeCall(hipFree(rhoref_g));
  cudaSafeCall(hipFree(rhorefh_g));

  return 0;
}

