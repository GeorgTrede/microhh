#include "fields.h"
#include "grid.h"
#include "master.h"

int cfields::prepareGPU()
{
  const int nmemsize = grid->ncellsp*sizeof(double);
  const int nmemsize2d = grid->ijcellsp*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMalloc(&it->second->data_g,        nmemsize);
    hipMalloc(&it->second->databot_g,     nmemsize2d);
    hipMalloc(&it->second->datatop_g,     nmemsize2d);
    hipMalloc(&it->second->datagradbot_g, nmemsize2d);
    hipMalloc(&it->second->datagradtop_g, nmemsize2d);
    hipMalloc(&it->second->datafluxbot_g, nmemsize2d);
    hipMalloc(&it->second->datafluxtop_g, nmemsize2d);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipMalloc(&it->second->data_g, nmemsize);
  }

  hipMalloc(&a["p"]->data_g, nmemsize);
  hipMalloc(&a["tmp1"]->data_g, nmemsize);
  hipMalloc(&a["tmp2"]->data_g, nmemsize);

  // copy all the data to the GPU
  forwardGPU();

  return 0;
}

int cfields::forwardGPU()
{
  const int jcells    = grid->jcells;
  const int jkcells   = grid->jcells * grid->kcells;
  const int imemsizep = grid->icellsp * sizeof(double);
  const int imemsize  = grid->icells  * sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep, it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->databot_g[grid->memoffset],     imemsizep, it->second->databot,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datatop_g[grid->memoffset],     imemsizep, it->second->datatop,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradbot_g[grid->memoffset], imemsizep, it->second->datagradbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradtop_g[grid->memoffset], imemsizep, it->second->datagradtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxbot_g[grid->memoffset], imemsizep, it->second->datafluxbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxtop_g[grid->memoffset], imemsizep, it->second->datafluxtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep, it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);

  hipMemcpy2D(&a["p"]->data_g[grid->memoffset],              imemsizep, a["p"]->data,            imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  hipMemcpy2D(&a["tmp1"]->data_g[grid->memoffset],           imemsizep, a["tmp1"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  hipMemcpy2D(&a["tmp2"]->data_g[grid->memoffset],           imemsizep, a["tmp2"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  

  //printf("--> forwardGPU\n");

  return 0;
}

int cfields::backwardGPU()
{
  const int jcells    = grid->jcells;
  const int jkcells   = grid->jcells * grid->kcells;
  const int imemsizep = grid->icellsp * sizeof(double);
  const int imemsize  = grid->icells  * sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->databot,     imemsize, &it->second->databot_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datatop,     imemsize, &it->second->datatop_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradbot, imemsize, &it->second->datagradbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradtop, imemsize, &it->second->datagradtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxbot, imemsize, &it->second->datafluxbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxtop, imemsize, &it->second->datafluxtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);

  hipMemcpy2D(a["p"]->data,              imemsize, &a["p"]->data_g[grid->memoffset],            imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  hipMemcpy2D(a["tmp1"]->data,           imemsize, &a["tmp1"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  hipMemcpy2D(a["tmp2"]->data,           imemsize, &a["tmp2"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  

  //printf("--> backwardGPU\n");

  return 0;
}

int cfields::clearGPU()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipFree(&it->second->data_g);
    hipFree(&it->second->databot_g);
    hipFree(&it->second->datatop_g);
    hipFree(&it->second->datagradbot_g);
    hipFree(&it->second->datagradtop_g);
    hipFree(&it->second->datafluxbot_g);
    hipFree(&it->second->datafluxtop_g);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipFree(&it->second->data_g);
  }

  hipFree(&a["p"]->data_g);
  hipFree(&a["tmp1"]->data_g);
  hipFree(&a["tmp2"]->data_g);

  return 0;
}

