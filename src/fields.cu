#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "fields.h"
#include "grid.h"
#include "master.h"
#include "boundary.h" // TMP BVS
#include "constants.h"
#include "tools.h"

// TODO use interp2 functions instead of manual interpolation
__global__ void fields_calcmom_2nd(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                                   double * __restrict__ mom, double * __restrict__ dz,
                                   int istart, int jstart, int kstart,
                                   int iend,   int jend,   int kend,
                                   int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 
  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    mom[ijk] = (0.5*(u[ijk]+u[ijk+ii]) + 0.5*(v[ijk]+v[ijk+jj]) + 0.5*(w[ijk]+w[ijk+kk]))*dz[k];
  }
}

__global__ void fields_calctke_2nd(double * __restrict__ u, double * __restrict__ v, double * __restrict__ w, 
                                   double * __restrict__ tke, double * __restrict__ dz,
                                   int istart, int jstart, int kstart,
                                   int iend,   int jend,   int kend,
                                   int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 
  int ii = 1;

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    tke[ijk] = ( 0.5*(pow(u[ijk],2)+pow(u[ijk+ii],2)) 
               + 0.5*(pow(v[ijk],2)+pow(v[ijk+jj],2)) 
               + 0.5*(pow(w[ijk],2)+pow(w[ijk+kk],2)))*dz[k];
  }
}

__global__ void fields_calcmass_2nd(double * __restrict__ s, double * __restrict__ mass, double * __restrict__ dz,
                                    int istart, int jstart, int kstart,
                                    int iend,   int jend,   int kend,
                                    int jj, int kk)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
  int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
  int k = blockIdx.z + kstart; 

  if(i < iend && j < jend && k < kend)
  {
    int ijk = i + j*jj + k*kk;
    mass[ijk] = s[ijk]*dz[k];
  }
}

#ifdef USECUDA
int Fields::exec()
{
  // calculate the means for the prognostic scalars
  if(calcprofs)
  {
    for(fieldmap::iterator it=sp.begin(); it!=sp.end(); ++it)
      grid->calcMean_g(it->second->datamean_g, &it->second->data_g[grid->memoffset], atmp["tmp1"]->data_g);
  }

  return 0;
}
#endif

#ifdef USECUDA
double Fields::checkmom()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  fields_calcmom_2nd<<<gridGPU, blockGPU>>>(&u->data_g[offs], &v->data_g[offs], &w->data_g[offs], 
                                            &atmp["tmp1"]->data_g[offs], grid->dz_g,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend,
                                            grid->icellsp, grid->ijcellsp);
  cudaCheckError();

  double mom = grid->getSum_g(&atmp["tmp1"]->data_g[offs], atmp["tmp2"]->data_g); 
  grid->getSum(&mom);
  mom /= (grid->itot*grid->jtot*grid->zsize);

  return mom;
}
#endif

#ifdef USECUDA
double Fields::checktke()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;

  fields_calctke_2nd<<<gridGPU, blockGPU>>>(&u->data_g[offs], &v->data_g[offs], &w->data_g[offs], 
                                            &atmp["tmp1"]->data_g[offs], grid->dz_g,
                                            grid->istart,  grid->jstart, grid->kstart,
                                            grid->iend,    grid->jend,   grid->kend,
                                            grid->icellsp, grid->ijcellsp);
  cudaCheckError();

  double tke = grid->getSum_g(&atmp["tmp1"]->data_g[offs], atmp["tmp2"]->data_g); 

  grid->getSum(&tke);
  tke /= (grid->itot*grid->jtot*grid->zsize);
  tke *= 0.5;

  return tke;
}
#endif

#ifdef USECUDA
double Fields::checkmass()
{
  const int blocki = cuda::blockSizeI;
  const int blockj = cuda::blockSizeJ;
  const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
  const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

  dim3 gridGPU (gridi, gridj, grid->kcells);
  dim3 blockGPU(blocki, blockj, 1);

  const int offs = grid->memoffset;
  double mass;

  // CvH for now, do the mass check on the first scalar... Do we want to change this?
  fieldmap::iterator itProg=sp.begin();
  if(sp.begin() != sp.end())
  {
    fields_calcmass_2nd<<<gridGPU, blockGPU>>>(&itProg->second->data_g[offs], &atmp["tmp1"]->data_g[offs], grid->dz_g,
                                               grid->istart,  grid->jstart, grid->kstart,
                                               grid->iend,    grid->jend,   grid->kend,
                                               grid->icellsp, grid->ijcellsp);
    cudaCheckError();

    mass = grid->getSum_g(&atmp["tmp1"]->data_g[offs], atmp["tmp2"]->data_g); 
    grid->getSum(&mass);
    mass /= (grid->itot*grid->jtot*grid->zsize);
  }
  else
    mass = 0; 

  return mass;
}
#endif

int Fields::prepareDevice()
{
  const int nmemsize   = grid->ncellsp*sizeof(double);
  const int nmemsize1d = grid->kcells*sizeof(double);

  // Prognostic fields
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
    it->second->initDevice();
 
  // Diagnostic fields 
  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
    it->second->initDevice();

  // Tendencies
  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    cudaSafeCall(hipMalloc(&it->second->data_g, nmemsize));

  // Temporary fields
  atmp["tmp1"]->initDevice();
  atmp["tmp2"]->initDevice();

  // Reference profiles
  cudaSafeCall(hipMalloc(&rhoref_g,  nmemsize1d));
  cudaSafeCall(hipMalloc(&rhorefh_g, nmemsize1d));

  // copy all the data to the GPU
  forwardDevice();

  return 0;
}

int Fields::forwardDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
    forwardField3dDevice(it->second);

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
    forwardField3dDevice(it->second);

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    forward3DFieldDevice(it->second->data_g, it->second->data, Offset);

  forwardField3dDevice(atmp["tmp1"]);
  forwardField3dDevice(atmp["tmp2"]);

  forward1DFieldDevice(rhoref_g,  rhoref , grid->kcells);
  forward1DFieldDevice(rhorefh_g, rhorefh, grid->kcells);

  //master->printMessage("Synchronized GPU with CPU (forward)\n");

  return 0;
}

int Fields::backwardDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
    backwardField3dDevice(it->second);

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
    backwardField3dDevice(it->second);

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    backward3DFieldDevice(it->second->data,        it->second->data_g,        Offset);

  backwardField3dDevice(atmp["tmp1"]);
  backwardField3dDevice(atmp["tmp2"]);

  backward1DFieldDevice(rhoref,  rhoref_g,  grid->kcells);
  backward1DFieldDevice(rhorefh, rhorefh_g, grid->kcells);

  //master->printMessage("Synchronized CPU with GPU (backward)\n");

  return 0;
}

int Fields::clearDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
    it->second->clearDevice();

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
    it->second->clearDevice();

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    cudaSafeCall(hipFree(it->second->data_g));

  atmp["tmp1"]->clearDevice();
  atmp["tmp2"]->clearDevice();
  
  cudaSafeCall(hipFree(rhoref_g));
  cudaSafeCall(hipFree(rhorefh_g));

  return 0;
}

/* BvS: it would make more sense to put this routine in field3d.cu, but how to solve this with the calls to fields.cu? */
void Fields::forwardField3dDevice(Field3d *fld)
{
  forward3DFieldDevice(fld->data_g,        fld->data,        Offset);
  forward2DFieldDevice(fld->databot_g,     fld->databot,     Offset);
  forward2DFieldDevice(fld->datatop_g,     fld->datatop,     Offset);
  forward2DFieldDevice(fld->datagradbot_g, fld->datagradbot, Offset);
  forward2DFieldDevice(fld->datagradtop_g, fld->datagradtop, Offset);
  forward2DFieldDevice(fld->datafluxbot_g, fld->datafluxbot, Offset);
  forward2DFieldDevice(fld->datafluxtop_g, fld->datafluxtop, Offset);
  forward1DFieldDevice(fld->datamean_g,    fld->datamean, grid->kcells);
}

/* BvS: it would make more sense to put this routine in field3d.cu, but how to solve this with the calls to fields.cu? */
void Fields::backwardField3dDevice(Field3d *fld)
{
  backward3DFieldDevice(fld->data,        fld->data_g,        Offset);
  backward2DFieldDevice(fld->databot,     fld->databot_g,     Offset);
  backward2DFieldDevice(fld->datatop,     fld->datatop_g,     Offset);
  backward2DFieldDevice(fld->datagradbot, fld->datagradbot_g, Offset);
  backward2DFieldDevice(fld->datagradtop, fld->datagradtop_g, Offset);
  backward2DFieldDevice(fld->datafluxbot, fld->datafluxbot_g, Offset);
  backward2DFieldDevice(fld->datafluxtop, fld->datafluxtop_g, Offset);
  backward1DFieldDevice(fld->datamean,    fld->datamean_g, grid->kcells);
}

void Fields::forward3DFieldDevice(double * field_g, double * field, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(&field_g[grid->memoffset], imemsizep,  field, imemsize, imemsize, grid->jcells*grid->kcells, hipMemcpyHostToDevice));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field_g, field, grid->ncells*sizeof(double), hipMemcpyHostToDevice));
}

void Fields::forward2DFieldDevice(double * field_g, double * field, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);
 
  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(&field_g[grid->memoffset], imemsizep,  field, imemsize, imemsize, grid->jcells,  hipMemcpyHostToDevice));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field_g, field, grid->ijcells*sizeof(double), hipMemcpyHostToDevice));
}

void Fields::forward1DFieldDevice(double * field_g, double * field, int ncells)
{
  cudaSafeCall(hipMemcpy(field_g, field, ncells*sizeof(double), hipMemcpyHostToDevice));
}

void Fields::backward3DFieldDevice(double * field, double * field_g, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(field, imemsize, &field_g[grid->memoffset], imemsizep, imemsize, grid->jcells*grid->kcells, hipMemcpyDeviceToHost));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field, field_g, grid->ncells*sizeof(double), hipMemcpyDeviceToHost));
}

void Fields::backward2DFieldDevice(double * field, double * field_g, OffsetType sw)
{
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);
 
  if(sw == Offset)
    cudaSafeCall(hipMemcpy2D(field, imemsize, &field_g[grid->memoffset], imemsizep, imemsize, grid->jcells, hipMemcpyDeviceToHost));
  else if(sw == NoOffset)
    cudaSafeCall(hipMemcpy(field, field_g, grid->ijcells*sizeof(double), hipMemcpyDeviceToHost));
}

void Fields::backward1DFieldDevice(double * field, double * field_g, int ncells)
{
  cudaSafeCall(hipMemcpy(field, field_g, ncells*sizeof(double), hipMemcpyDeviceToHost));
}
