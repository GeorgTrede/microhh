/*
 * MicroHH
 * Copyright (c) 2011-2014 Chiel van Heerwaarden
 * Copyright (c) 2011-2014 Thijs Heus
 * Copyright (c)      2014 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "fields.h"
#include "grid.h"
#include "master.h"
#include "boundary.h" // TMP BVS

#ifdef USECUDA
int cfields::exec()
{
  // calculate the means for the prognostic scalars
  if(calcprofs)
  {
    for(fieldmap::iterator it=sp.begin(); it!=sp.end(); ++it)
      grid->calcmean_g(it->second->datamean_g, &it->second->data_g[grid->memoffset], s["tmp1"]->data_g);
  }

  return 0;
}
#endif

int cfields::prepareDevice()
{
  const int nmemsize   = grid->ncellsp*sizeof(double);
  const int nmemsize1d = grid->kcells*sizeof(double);
  const int nmemsize2d = (grid->ijcellsp+grid->memoffset)*sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMalloc(&it->second->data_g,        nmemsize  );
    hipMalloc(&it->second->databot_g,     nmemsize2d);
    hipMalloc(&it->second->datatop_g,     nmemsize2d);
    hipMalloc(&it->second->datagradbot_g, nmemsize2d);
    hipMalloc(&it->second->datagradtop_g, nmemsize2d);
    hipMalloc(&it->second->datafluxbot_g, nmemsize2d);
    hipMalloc(&it->second->datafluxtop_g, nmemsize2d);
    hipMalloc(&it->second->datamean_g,    nmemsize1d);
  }

  // BvS: slightly wasteful, but make sure we have all fields...
  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipMalloc(&it->second->data_g,        nmemsize  );
    hipMalloc(&it->second->databot_g,     nmemsize2d);
    hipMalloc(&it->second->datatop_g,     nmemsize2d);
    hipMalloc(&it->second->datagradbot_g, nmemsize2d);
    hipMalloc(&it->second->datagradtop_g, nmemsize2d);
    hipMalloc(&it->second->datafluxbot_g, nmemsize2d);
    hipMalloc(&it->second->datafluxtop_g, nmemsize2d);
    hipMalloc(&it->second->datamean_g,    nmemsize1d);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipMalloc(&it->second->data_g, nmemsize);
  }

  hipMalloc(&rhoref_g,  nmemsize1d);
  hipMalloc(&rhorefh_g, nmemsize1d);

  // copy all the data to the GPU
  forwardDevice();

  return 0;
}

int cfields::forwardDevice()
{
  const int jcells     = grid->jcells;
  const int jkcells    = grid->jcells * grid->kcells;
  const int nmemsize1d = grid->kcells*sizeof(double);
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep,  it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->databot_g[grid->memoffset],     imemsizep,  it->second->databot,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datatop_g[grid->memoffset],     imemsizep,  it->second->datatop,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradbot_g[grid->memoffset], imemsizep,  it->second->datagradbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradtop_g[grid->memoffset], imemsizep,  it->second->datagradtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxbot_g[grid->memoffset], imemsizep,  it->second->datafluxbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxtop_g[grid->memoffset], imemsizep,  it->second->datafluxtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy(it->second->datamean_g, it->second->datamean, nmemsize1d, hipMemcpyHostToDevice);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep,  it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->databot_g[grid->memoffset],     imemsizep,  it->second->databot,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datatop_g[grid->memoffset],     imemsizep,  it->second->datatop,     imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradbot_g[grid->memoffset], imemsizep,  it->second->datagradbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datagradtop_g[grid->memoffset], imemsizep,  it->second->datagradtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxbot_g[grid->memoffset], imemsizep,  it->second->datafluxbot, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy2D(&it->second->datafluxtop_g[grid->memoffset], imemsizep,  it->second->datafluxtop, imemsize, imemsize, jcells,  hipMemcpyHostToDevice);
    hipMemcpy(it->second->datamean_g, it->second->datamean, nmemsize1d, hipMemcpyHostToDevice);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy2D(&it->second->data_g[grid->memoffset],        imemsizep, it->second->data,        imemsize, imemsize, jkcells, hipMemcpyHostToDevice);

  //hipMemcpy2D(&a["p"]->data_g[grid->memoffset],              imemsizep, a["p"]->data,            imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  //hipMemcpy2D(&a["tmp1"]->data_g[grid->memoffset],           imemsizep, a["tmp1"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  //hipMemcpy2D(&a["tmp2"]->data_g[grid->memoffset],           imemsizep, a["tmp2"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  
  //hipMemcpy2D(&a["tmp3"]->data_g[grid->memoffset],           imemsizep, a["tmp3"]->data,         imemsize, imemsize, jkcells, hipMemcpyHostToDevice);  

  hipMemcpy(rhoref_g,  rhoref,  nmemsize1d, hipMemcpyHostToDevice);
  hipMemcpy(rhorefh_g, rhorefh, nmemsize1d, hipMemcpyHostToDevice);

  //master->printMessage("Synchronized GPU with CPU (forward)\n");

  return 0;
}

int cfields::backwardDevice()
{
  const int jcells     = grid->jcells;
  const int jkcells    = grid->jcells * grid->kcells;
  const int nmemsize1d = grid->kcells*sizeof(double);
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->databot,     imemsize, &it->second->databot_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datatop,     imemsize, &it->second->datatop_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradbot, imemsize, &it->second->datagradbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradtop, imemsize, &it->second->datagradtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxbot, imemsize, &it->second->datafluxbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxtop, imemsize, &it->second->datafluxtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datamean, it->second->datamean_g, nmemsize1d, hipMemcpyDeviceToHost);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->databot,     imemsize, &it->second->databot_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datatop,     imemsize, &it->second->datatop_g[grid->memoffset],     imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradbot, imemsize, &it->second->datagradbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datagradtop, imemsize, &it->second->datagradtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxbot, imemsize, &it->second->datafluxbot_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy2D(it->second->datafluxtop, imemsize, &it->second->datafluxtop_g[grid->memoffset], imemsizep, imemsize, jcells,  hipMemcpyDeviceToHost);
    hipMemcpy(it->second->datamean, it->second->datamean_g, nmemsize1d, hipMemcpyDeviceToHost);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
    hipMemcpy2D(it->second->data,        imemsize, &it->second->data_g[grid->memoffset],        imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);

  //hipMemcpy2D(a["p"]->data,              imemsize, &a["p"]->data_g[grid->memoffset],            imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  //hipMemcpy2D(a["tmp1"]->data,           imemsize, &a["tmp1"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  //hipMemcpy2D(a["tmp2"]->data,           imemsize, &a["tmp2"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  
  //hipMemcpy2D(a["tmp3"]->data,           imemsize, &a["tmp3"]->data_g[grid->memoffset],         imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);  

  hipMemcpy(rhoref,  rhoref_g,  nmemsize1d, hipMemcpyDeviceToHost);
  hipMemcpy(rhorefh, rhorefh_g, nmemsize1d, hipMemcpyDeviceToHost);

  //master->printMessage("Synchronized CPU with GPU (backward)\n");

  return 0;
}

// TMP BvS
void cfields::D2H(double * hdata, double * ddata)
{
  const int jkcells    = grid->jcells * grid->kcells;
  const int imemsizep  = grid->icellsp * sizeof(double);
  const int imemsize   = grid->icells  * sizeof(double);

  hipMemcpy2D(hdata, imemsize, &ddata[grid->memoffset], imemsizep, imemsize, jkcells, hipMemcpyDeviceToHost);
}

// TMP BvS
void cfields::printSlice(double * data, int k)
{
  int ijk;
  for (int j=0; j<grid->jcells; ++j)
  {
    for (int i=0; i<grid->icells; ++i)
    {
      ijk = i + j*grid->icells + k*grid->ijcells;
      printf("% 9.4e ",data[ijk]);
    }  
    printf("\n");
  }
  printf("\n");
}

int cfields::clearDevice()
{
  for(fieldmap::const_iterator it=ap.begin(); it!=ap.end(); ++it)
  {
    hipFree(it->second->data_g);
    hipFree(it->second->databot_g);
    hipFree(it->second->datatop_g);
    hipFree(it->second->datagradbot_g);
    hipFree(it->second->datagradtop_g);
    hipFree(it->second->datafluxbot_g);
    hipFree(it->second->datafluxtop_g);
    hipFree(it->second->datamean_g);
  }

  for(fieldmap::const_iterator it=sd.begin(); it!=sd.end(); ++it)
  {
    hipFree(it->second->data_g);
    hipFree(it->second->databot_g);
    hipFree(it->second->datatop_g);
    hipFree(it->second->datagradbot_g);
    hipFree(it->second->datagradtop_g);
    hipFree(it->second->datafluxbot_g);
    hipFree(it->second->datafluxtop_g);
    hipFree(it->second->datamean_g);
  }

  for(fieldmap::const_iterator it=at.begin(); it!=at.end(); ++it)
  {
    hipFree(it->second->data_g);
  }

  //hipFree(&a["p"]->data_g);
  //hipFree(&a["tmp1"]->data_g);
  //hipFree(&a["tmp2"]->data_g);
  //hipFree(&a["tmp3"]->data_g);

  hipFree(rhoref_g);
  hipFree(rhorefh_g);

  return 0;
}

